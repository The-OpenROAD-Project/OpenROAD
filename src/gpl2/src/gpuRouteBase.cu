#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
// BSD 3-Clause License
//
// Copyright (c) 2023, Google LLC
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
///////////////////////////////////////////////////////////////////////////////

#include <algorithm>
#include <cmath>
#include <iostream>
#include <string>
#include <utility>

#include "gpuRouteBase.h"
#include "grt/GlobalRouter.h"
#include "odb/db.h"
#include "placerBase.h"
#include "utl/Logger.h"

using grt::GlobalRouter;
using std::make_pair;
using std::pair;
using std::sort;
using std::string;
using std::vector;

using utl::GPL2;

namespace gpl2 {

/////////////////////////////////////////////
// RouteBaseVars

RouteBaseVars::RouteBaseVars()
{
  reset();
}

void RouteBaseVars::reset()
{
  inflationRatioCoef = 2.5;
  maxInflationRatio = 2.5;
  maxDensity = 0.90;
  targetRC = 1.25;
  ignoreEdgeRatio = 0.8;
  minInflationRatio = 1.01;
  rcK1 = rcK2 = 1.0;
  rcK3 = rcK4 = 0.0;
  maxBloatIter = 1;
  maxInflationIter = 4;
}

/////////////////////////////////////////////
// GpuRouteBase

GpuRouteBase::GpuRouteBase()
    : rbVars_(), db_(nullptr), grouter_(nullptr), nbc_(nullptr), log_(nullptr)
{
}

GpuRouteBase::GpuRouteBase(RouteBaseVars rbVars,
                           odb::dbDatabase* db,
                           grt::GlobalRouter* grouter,
                           std::shared_ptr<PlacerBaseCommon> nbc,
                           std::vector<std::shared_ptr<PlacerBase>> nbVec,
                           utl::Logger* log)
    : GpuRouteBase()
{
  rbVars_ = rbVars;
  db_ = db;
  grouter_ = grouter;
  nbc_ = std::move(nbc);
  log_ = log;
  nbVec_ = std::move(nbVec);
}

GpuRouteBase::~GpuRouteBase()
{
}

}  // namespace gpl2
