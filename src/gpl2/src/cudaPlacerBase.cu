#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
// BSD 3-Clause License
//
// Copyright (c) 2023, Google LLC
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

#include <chrono>
#include <cmath>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <memory>
#include <numeric>

#include "odb/db.h"
#include "placerBase.h"
#include "placerObjects.h"
#include "poissonSolver.h"
#include "util.h"
#include "utl/Logger.h"
// basic vectors
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
// memory related
#include <thrust/copy.h>
#include <thrust/fill.h>
// algorithm related
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/replace.h>
#include <thrust/transform.h>

namespace gpl2 {

using utl::GPL2;

///////////////////////////////////////////////////////////////////////////////////
// PlacerBaseCommon
///////////////////////////////////////////////////////////////////////////////////

void PlacerBaseCommon::initCUDAKernel()
{
  // calculate the information on the host side
  hInstDCx_.resize(numPlaceInsts_);
  hInstDCy_.resize(numPlaceInsts_);
  int instIdx = 0;
  for (auto& inst : placeInsts_) {
    hInstDCx_[instIdx] = inst->cx();
    hInstDCy_[instIdx] = inst->cy();
    instIdx++;
  }

  // allocate the objects on host side
  dInstDCxPtr_ = setThrustVector<int>(numPlaceInsts_, dInstDCx_);
  dInstDCyPtr_ = setThrustVector<int>(numPlaceInsts_, dInstDCy_);

  // copy from host to device
  thrust::copy(hInstDCx_.begin(), hInstDCx_.end(), dInstDCx_.begin());
  thrust::copy(hInstDCy_.begin(), hInstDCy_.end(), dInstDCy_.begin());

  // allocate memory on device side
  dWLGradXPtr_ = setThrustVector<float>(numPlaceInsts_, dWLGradX_);
  dWLGradYPtr_ = setThrustVector<float>(numPlaceInsts_, dWLGradY_);

  // create the wlGradOp
  wlGradOp_ = new WirelengthOp(this);
}

void PlacerBaseCommon::freeCUDAKernel()
{
  dInstDCxPtr_ = nullptr;
  dInstDCyPtr_ = nullptr;

  dWLGradXPtr_ = nullptr;
  dWLGradYPtr_ = nullptr;
}

// Update the database information
void PlacerBaseCommon::updateDB()
{
  if (clusterFlag_ == true) {
    updateDBCluster();
    return;
  }

  thrust::copy(dInstDCx_.begin(), dInstDCx_.end(), hInstDCx_.begin());
  thrust::copy(dInstDCy_.begin(), dInstDCy_.end(), hInstDCy_.begin());

  int manufactureGird = db_->getTech()->getManufacturingGrid();

  for (auto inst : placeInsts_) {
    const int instId = inst->instId();
    inst->setCenterLocation(
        static_cast<int>(hInstDCx_[instId]) / manufactureGird * manufactureGird,
        static_cast<int>(hInstDCy_[instId]) / manufactureGird
            * manufactureGird);
    inst->dbSetLocation();
    inst->dbSetPlaced();
  }
}

void PlacerBaseCommon::updateDBCluster()
{
  thrust::copy(dInstDCx_.begin(), dInstDCx_.end(), hInstDCx_.begin());
  thrust::copy(dInstDCy_.begin(), dInstDCy_.end(), hInstDCy_.begin());

  int manufactureGird = db_->getTech()->getManufacturingGrid();
  odb::dbBlock* block = getBlock();
  // insts fill with real instances
  // update the clusters
  odb::dbSet<odb::dbInst> insts = block->getInsts();
  for (odb::dbInst* inst : insts) {
    auto type = inst->getMaster()->getType();
    if (!type.isCore() && !type.isBlock()) {
      continue;
    }
    const int clusterId
        = odb::dbIntProperty::find(inst, "cluster_id")->getValue();
    const int cx = hInstDCx_[clusterId];
    const int cy = hInstDCy_[clusterId];
    odb::dbBox* bbox = inst->getBBox();
    int width = bbox->getDX();
    int height = bbox->getDY();
    int lx = cx - width / 2;
    int ly = cy - height / 2;
    if (lx < die().coreLx()) {
      lx = die().coreLx();
    }

    if (ly < die().coreLy()) {
      ly = die().coreLy();
    }

    if (lx + width > die().coreUx()) {
      lx = die().coreUx() - width;
    }

    if (ly + height > die().coreUy()) {
      ly = die().coreUy() - height;
    }

    lx = lx / manufactureGird * manufactureGird;
    ly = ly / manufactureGird * manufactureGird;

    inst->setLocation(lx, ly);
    inst->setPlacementStatus(odb::dbPlacementStatus::PLACED);
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////
// PlacerBase
/////////////////////////////////////////////////////////////////////////////////////////////

void PlacerBase::initCUDAKernel()
{
  // calculate the information on the host side
  thrust::host_vector<int> hPlaceInstIds(numPlaceInsts_);
  int instIdx = 0;
  for (auto& inst : placeInsts_) {
    hPlaceInstIds[instIdx] = inst->instId();
    instIdx++;
  }

  // allocate the objects on host side
  dPlaceInstIdsPtr_ = setThrustVector<int>(numPlaceInsts_, dPlaceInstIds_);

  // copy from host to device
  thrust::copy(
      hPlaceInstIds.begin(), hPlaceInstIds.end(), dPlaceInstIds_.begin());

  thrust::host_vector<int> hInstDDx(numInsts_);
  thrust::host_vector<int> hInstDDy(numInsts_);
  thrust::host_vector<int> hInstDCx(numInsts_);
  thrust::host_vector<int> hInstDCy(numInsts_);
  thrust::host_vector<float> hWireLengthPrecondi(numInsts_);
  thrust::host_vector<float> hDensityPrecondi(numInsts_);

  // calculate the information on the host side
  instIdx = 0;
  for (auto& inst : insts_) {
    hInstDDx[instIdx] = inst->dDx();
    hInstDDy[instIdx] = inst->dDy();
    hInstDCx[instIdx] = inst->cx();
    hInstDCy[instIdx] = inst->cy();
    hWireLengthPrecondi[instIdx] = inst->wireLengthPreconditioner();
    hDensityPrecondi[instIdx] = inst->densityPreconditioner();
    instIdx++;
  }

  dInstDDxPtr_ = setThrustVector<int>(numInsts_, dInstDDx_);
  dInstDDyPtr_ = setThrustVector<int>(numInsts_, dInstDDy_);
  dInstDCxPtr_ = setThrustVector<int>(numInsts_, dInstDCx_);
  dInstDCyPtr_ = setThrustVector<int>(numInsts_, dInstDCy_);

  dWireLengthPrecondiPtr_
      = setThrustVector<float>(numInsts_, dWireLengthPrecondi_);
  dDensityPrecondiPtr_ = setThrustVector<float>(numInsts_, dDensityPrecondi_);

  thrust::copy(hInstDDx.begin(), hInstDDx.end(), dInstDDx_.begin());
  thrust::copy(hInstDDy.begin(), hInstDDy.end(), dInstDDy_.begin());
  thrust::copy(hInstDCx.begin(), hInstDCx.end(), dInstDCx_.begin());
  thrust::copy(hInstDCy.begin(), hInstDCy.end(), dInstDCy_.begin());
  thrust::copy(hWireLengthPrecondi.begin(),
               hWireLengthPrecondi.end(),
               dWireLengthPrecondi_.begin());
  thrust::copy(hDensityPrecondi.begin(),
               hDensityPrecondi.end(),
               dDensityPrecondi_.begin());

  dDensityGradXPtr_ = setThrustVector<float>(numInsts_, dDensityGradX_);
  dDensityGradYPtr_ = setThrustVector<float>(numInsts_, dDensityGradY_);

  dCurSLPCoordiPtr_ = setThrustVector<FloatPoint>(numInsts_, dCurSLPCoordi_);
  dCurSLPWireLengthGradXPtr_
      = setThrustVector<float>(numInsts_, dCurSLPWireLengthGradX_);
  dCurSLPWireLengthGradYPtr_
      = setThrustVector<float>(numInsts_, dCurSLPWireLengthGradY_);
  dCurSLPDensityGradXPtr_
      = setThrustVector<float>(numInsts_, dCurSLPDensityGradX_);
  dCurSLPDensityGradYPtr_
      = setThrustVector<float>(numInsts_, dCurSLPDensityGradY_);
  dCurSLPSumGradsPtr_
      = setThrustVector<FloatPoint>(numInsts_, dCurSLPSumGrads_);

  dPrevSLPCoordiPtr_ = setThrustVector<FloatPoint>(numInsts_, dPrevSLPCoordi_);
  dPrevSLPWireLengthGradXPtr_
      = setThrustVector<float>(numInsts_, dPrevSLPWireLengthGradX_);
  dPrevSLPWireLengthGradYPtr_
      = setThrustVector<float>(numInsts_, dPrevSLPWireLengthGradY_);
  dPrevSLPDensityGradXPtr_
      = setThrustVector<float>(numInsts_, dPrevSLPDensityGradX_);
  dPrevSLPDensityGradYPtr_
      = setThrustVector<float>(numInsts_, dPrevSLPDensityGradY_);
  dPrevSLPSumGradsPtr_
      = setThrustVector<FloatPoint>(numInsts_, dPrevSLPSumGrads_);

  dNextSLPCoordiPtr_ = setThrustVector<FloatPoint>(numInsts_, dNextSLPCoordi_);
  dNextSLPWireLengthGradXPtr_
      = setThrustVector<float>(numInsts_, dNextSLPWireLengthGradX_);
  dNextSLPWireLengthGradYPtr_
      = setThrustVector<float>(numInsts_, dNextSLPWireLengthGradY_);
  dNextSLPDensityGradXPtr_
      = setThrustVector<float>(numInsts_, dNextSLPDensityGradX_);
  dNextSLPDensityGradYPtr_
      = setThrustVector<float>(numInsts_, dNextSLPDensityGradY_);
  dNextSLPSumGradsPtr_
      = setThrustVector<FloatPoint>(numInsts_, dNextSLPSumGrads_);

  dCurCoordiPtr_ = setThrustVector<FloatPoint>(numInsts_, dCurCoordi_);
  dNextCoordiPtr_ = setThrustVector<FloatPoint>(numInsts_, dNextCoordi_);

  dSumGradsXPtr_ = setThrustVector<float>(numInsts_, dSumGradsX_);
  dSumGradsYPtr_ = setThrustVector<float>(numInsts_, dSumGradsY_);

  densityOp_ = new DensityOp(this);
}

void PlacerBase::freeCUDAKernel()
{
  densityOp_ = nullptr;
  dPlaceInstIdsPtr_ = nullptr;

  dDensityGradXPtr_ = nullptr;
  dDensityGradYPtr_ = nullptr;

  dCurSLPCoordiPtr_ = nullptr;
  dCurSLPSumGradsPtr_ = nullptr;

  dPrevSLPCoordiPtr_ = nullptr;
  dPrevSLPSumGradsPtr_ = nullptr;

  dNextSLPCoordiPtr_ = nullptr;
  dNextSLPSumGradsPtr_ = nullptr;

  dCurSLPCoordiPtr_ = nullptr;
  dCurSLPSumGradsPtr_ = nullptr;
}

// Make sure the instances are within the region
__device__ float getDensityCoordiLayoutInside(int instWidth,
                                              float cx,
                                              int coreLx,
                                              int coreUx)
{
  float adjVal = cx;
  if (cx - instWidth / 2 < coreLx) {
    adjVal = coreLx + instWidth / 2;
  }

  if (cx + instWidth / 2 > coreUx) {
    adjVal = coreUx - instWidth / 2;
  }

  return adjVal;
}

__global__ void updateDensityCoordiLayoutInsideKernel(const int numInsts,
                                                      const int coreLx,
                                                      const int coreLy,
                                                      const int coreUx,
                                                      const int coreUy,
                                                      const int* instDDx,
                                                      const int* instDDy,
                                                      int* instDCx,
                                                      int* instDCy)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numInsts) {
    instDCx[instIdx] = getDensityCoordiLayoutInside(
        instDDx[instIdx], instDCx[instIdx], coreLx, coreUx);
    instDCy[instIdx] = getDensityCoordiLayoutInside(
        instDDy[instIdx], instDCy[instIdx], coreLy, coreUy);
  }
}

__global__ void initDensityCoordiKernel(int numInsts,
                                        const int* instDCx,
                                        const int* instDCy,
                                        FloatPoint* dCurCoordiPtr,
                                        FloatPoint* dCurSLPCoordiPtr,
                                        FloatPoint* dPrevSLPCoordiPtr)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numInsts) {
    const FloatPoint loc(instDCx[instIdx], instDCy[instIdx]);
    dCurCoordiPtr[instIdx] = loc;
    dCurSLPCoordiPtr[instIdx] = loc;
    dPrevSLPCoordiPtr[instIdx] = loc;
  }
}

void PlacerBase::initDensity1()
{
  // update density coordinate for each instance
  int numThreads = 256;
  int numBlocks = (numInsts_ + numThreads - 1) / numThreads;
  updateDensityCoordiLayoutInsideKernel<<<numBlocks, numThreads>>>(
      numInsts_,
      bg_.lx(),
      bg_.ly(),
      bg_.ux(),
      bg_.uy(),
      dInstDDxPtr_,
      dInstDDyPtr_,
      dInstDCxPtr_,
      dInstDCyPtr_);

  // initialize the dCurSLPCoordiPtr_, dPrevSLPCoordiPtr_
  // and dCurCoordiPtr_
  initDensityCoordiKernel<<<numBlocks, numThreads>>>(numInsts_,
                                                     dInstDCxPtr_,
                                                     dInstDCyPtr_,
                                                     dCurCoordiPtr_,
                                                     dCurSLPCoordiPtr_,
                                                     dPrevSLPCoordiPtr_);

  // We need to sync up bewteen pb and pbCommon
  updateGCellDensityCenterLocation(dCurSLPCoordiPtr_);
  pbCommon_->updatePinLocation();
  // calculate the previous hpwl
  // update the location of instances within this region
  // while the instances in other regions will not change
  prevHpwl_ = pbCommon_->hpwl();

  // FFT update
  updateDensityForceBin();

  // update parameters
  baseWireLengthCoef_ = npVars_.initWireLengthCoef
                        / (static_cast<float>(binSizeX() + binSizeY()) * 0.5);

  sumOverflow_ = static_cast<float>(overflowArea())
                 / static_cast<float>(nesterovInstsArea());

  sumOverflowUnscaled_ = static_cast<float>(overflowAreaUnscaled())
                         / static_cast<float>(nesterovInstsArea());
}

// (a)  // (a) define the get distance method
// getDistance is only defined on the host side
struct getTupleDistanceFunctor
{
  __host__ __device__ float operator()(
      const thrust::tuple<FloatPoint, FloatPoint>& t)
  {
    const FloatPoint& a = thrust::get<0>(t);
    const FloatPoint& b = thrust::get<1>(t);
    float dist = 0.0f;
    dist += (a.x - b.x) * (a.x - b.x);
    dist += (a.y - b.y) * (a.y - b.y);
    return dist;
  }
};

__host__ float getDistance(const FloatPoint* a,
                           const FloatPoint* b,
                           const int numInsts)
{
  if (numInsts <= 0) {
    return 0.0;
  }

  thrust::device_ptr<FloatPoint> aBegin(const_cast<FloatPoint*>(a));
  thrust::device_ptr<FloatPoint> aEnd = aBegin + numInsts;

  thrust::device_ptr<FloatPoint> bBegin(const_cast<FloatPoint*>(b));
  thrust::device_ptr<FloatPoint> bEnd = bBegin + numInsts;

  float sumDistance = thrust::transform_reduce(
      thrust::make_zip_iterator(thrust::make_tuple(aBegin, bBegin)),
      thrust::make_zip_iterator(thrust::make_tuple(aEnd, bEnd)),
      getTupleDistanceFunctor(),
      0.0f,
      thrust::plus<float>());

  return std::sqrt(sumDistance / (2.0 * numInsts));
}

template <typename T>
struct myAbs
{
  __host__ __device__ double operator()(const T& x) const
  {
    if (x >= 0)
      return x;
    else
      return x * -1;
  }
};

__host__ float getAbsGradSum(const float* a, const int numInsts)
{
  thrust::device_ptr<float> aBegin(const_cast<float*>(a));
  thrust::device_ptr<float> aEnd = aBegin + numInsts;
  double sumAbs = thrust::transform_reduce(
      aBegin, aEnd, myAbs<float>(), 0.0, thrust::plus<double>());
  return sumAbs;
}

float PlacerBase::getStepLength(const FloatPoint* prevSLPCoordi,
                                const FloatPoint* prevSLPSumGrads,
                                const FloatPoint* curSLPCoordi,
                                const FloatPoint* curSLPSumGrads) const
{
  float coordiDistance = getDistance(prevSLPCoordi, curSLPCoordi, numInsts_);
  float gradDistance = getDistance(prevSLPSumGrads, curSLPSumGrads, numInsts_);
  return coordiDistance / gradDistance;
}

// Function: initDensity2
float PlacerBase::initDensity2()
{
  // the wirelength force on each instance is zero
  if (wireLengthGradSum_ == 0) {
    densityPenalty_ = npVars_.initDensityPenalty;
    updatePrevGradient();
  }

  if (wireLengthGradSum_ != 0) {
    densityPenalty_
        = (wireLengthGradSum_ / densityGradSum_) * npVars_.initDensityPenalty;
  }

  sumOverflow_ = static_cast<float>(overflowArea())
                 / static_cast<float>(nesterovInstsArea());

  sumOverflowUnscaled_ = static_cast<float>(overflowAreaUnscaled())
                         / static_cast<float>(nesterovInstsArea());

  stepLength_ = getStepLength(dPrevSLPCoordiPtr_,
                              dPrevSLPSumGradsPtr_,
                              dCurSLPCoordiPtr_,
                              dCurSLPSumGradsPtr_);

  return stepLength_;
}

__global__ void sumGradientKernel(const int numInsts,
                                  const float densityPenalty,
                                  const float minPrecondi,
                                  const float* wireLengthPrecondi,
                                  const float* densityPrecondi,
                                  const float* wireLengthGradientsX,
                                  const float* wireLengthGradientsY,
                                  const float* densityGradientsX,
                                  const float* densityGradientsY,
                                  FloatPoint* sumGrads)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numInsts) {
    sumGrads[instIdx].x = wireLengthGradientsX[instIdx]
                          + densityPenalty * densityGradientsX[instIdx];
    sumGrads[instIdx].y = wireLengthGradientsY[instIdx]
                          + densityPenalty * densityGradientsY[instIdx];
    FloatPoint sumPrecondi(
        wireLengthPrecondi[instIdx] + densityPenalty * densityPrecondi[instIdx],
        wireLengthPrecondi[instIdx]
            + densityPenalty * densityPrecondi[instIdx]);

    if (sumPrecondi.x < minPrecondi) {
      sumPrecondi.x = minPrecondi;
    }

    if (sumPrecondi.y < minPrecondi) {
      sumPrecondi.y = minPrecondi;
    }

    sumGrads[instIdx].x /= sumPrecondi.x;
    sumGrads[instIdx].y /= sumPrecondi.y;
  }
}

void PlacerBase::updatePrevGradient()
{
  updateGradients(dPrevSLPWireLengthGradXPtr_,
                  dPrevSLPWireLengthGradYPtr_,
                  dPrevSLPDensityGradXPtr_,
                  dPrevSLPDensityGradYPtr_,
                  dPrevSLPSumGradsPtr_);
}

void PlacerBase::updateCurGradient()
{
  updateGradients(dCurSLPWireLengthGradXPtr_,
                  dCurSLPWireLengthGradYPtr_,
                  dCurSLPDensityGradXPtr_,
                  dCurSLPDensityGradYPtr_,
                  dCurSLPSumGradsPtr_);
}

void PlacerBase::updateNextGradient()
{
  updateGradients(dNextSLPWireLengthGradXPtr_,
                  dNextSLPWireLengthGradYPtr_,
                  dNextSLPDensityGradXPtr_,
                  dNextSLPDensityGradYPtr_,
                  dNextSLPSumGradsPtr_);
}

void PlacerBase::updateGradients(float* wireLengthGradientsX,
                                 float* wireLengthGradientsY,
                                 float* densityGradientsX,
                                 float* densityGradientsY,
                                 FloatPoint* sumGrads)
{
  if (isConverged_) {
    return;
  }

  wireLengthGradSum_ = 0;
  densityGradSum_ = 0;

  // get the forces on each instance
  getWireLengthGradientWA(wireLengthGradientsX, wireLengthGradientsY);
  getDensityGradient(densityGradientsX, densityGradientsY);

  wireLengthGradSum_ += getAbsGradSum(wireLengthGradientsX, numInsts_);
  wireLengthGradSum_ += getAbsGradSum(wireLengthGradientsY, numInsts_);
  densityGradSum_ += getAbsGradSum(densityGradientsX, numInsts_);
  densityGradSum_ += getAbsGradSum(densityGradientsY, numInsts_);

  int numThreads = 256;
  int numBlocks = (numInsts_ + numThreads - 1) / numThreads;
  sumGradientKernel<<<numBlocks, numThreads>>>(numInsts_,
                                               densityPenalty_,
                                               npVars_.minPreconditioner,
                                               dWireLengthPrecondiPtr_,
                                               dDensityPrecondiPtr_,
                                               wireLengthGradientsX,
                                               wireLengthGradientsY,
                                               densityGradientsX,
                                               densityGradientsY,
                                               sumGrads);
}

// sync up the instances location based on the corrodinates
__global__ void updateGCellDensityCenterLocationKernel(
    const int numInsts,
    const FloatPoint* coordis,
    int* instDCx,
    int* instDCy)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numInsts) {
    instDCx[instIdx] = coordis[instIdx].x;
    instDCy[instIdx] = coordis[instIdx].y;
  }
}

// sync up the instances between pbCommon and current pb
__global__ void syncPlaceInstsCommonKernel(const int numPlaceInsts,
                                           const int* placeInstIds,
                                           const int* placeInstDCx,
                                           const int* placeInstDCy,
                                           int* instDCxCommon,
                                           int* instDCyCommon)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numPlaceInsts) {
    int instId = placeInstIds[instIdx];
    instDCxCommon[instId] = placeInstDCx[instIdx];
    instDCyCommon[instId] = placeInstDCy[instIdx];
  }
}

void PlacerBase::updateGCellDensityCenterLocation(const FloatPoint* coordis)
{
  const int numThreads = 256;
  const int numBlocks = (numInsts_ + numThreads - 1) / numThreads;
  const int numPlaceInstBlocks = (numPlaceInsts_ + numThreads - 1) / numThreads;

  updateGCellDensityCenterLocationKernel<<<numBlocks, numThreads>>>(
      numInsts_, coordis, dInstDCxPtr_, dInstDCyPtr_);

  syncPlaceInstsCommonKernel<<<numPlaceInstBlocks, numThreads>>>(
      numPlaceInsts_,
      dPlaceInstIdsPtr_,
      dInstDCxPtr_,
      dInstDCyPtr_,
      pbCommon_->dInstDCxPtr(),
      pbCommon_->dInstDCyPtr());

  densityOp_->updateGCellLocation(dInstDCxPtr_, dInstDCyPtr_);
}

__global__ void getWireLengthGradientWAKernel(const int numPlaceInsts,
                                              const int* dPlaceInstIdsPtr,
                                              const float* dWLGradXCommonPtr,
                                              const float* dWLGradYCommonPtr,
                                              float* dWireLengthGradXPtr,
                                              float* dWireLengthGradYPtr)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numPlaceInsts) {
    int instId = dPlaceInstIdsPtr[instIdx];
    dWireLengthGradXPtr[instIdx] = dWLGradXCommonPtr[instId];
    dWireLengthGradYPtr[instIdx] = dWLGradYCommonPtr[instId];
  }
}

void PlacerBase::getWireLengthGradientWA(float* wireLengthGradientsX,
                                         float* wireLengthGradientsY)
{
  int numThreads = 256;
  int numBlocks = (numPlaceInsts_ + numThreads - 1) / numThreads;

  getWireLengthGradientWAKernel<<<numBlocks, numThreads>>>(
      numPlaceInsts_,
      dPlaceInstIdsPtr_,
      pbCommon_->dWLGradXPtr(),
      pbCommon_->dWLGradYPtr(),
      wireLengthGradientsX,
      wireLengthGradientsY);
}

void PlacerBase::getDensityGradient(float* densityGradientsX,
                                    float* densityGradientsY)
{
  densityOp_->getDensityGradient(densityGradientsX, densityGradientsY);
}

// calculate the next state based on current state
__global__ void nesterovUpdateCooridnatesKernel(
    const int numInsts,
    const int coreLx,
    const int coreLy,
    const int coreUx,
    const int coreUy,
    const float stepLength,
    const float coeff,
    const int* instDDx,
    const int* instDDy,
    const FloatPoint* curCoordiPtr,
    const FloatPoint* curSLPCoordiPtr,
    const FloatPoint* curSLPSumGradsPtr,
    FloatPoint* nextCoordiPtr,
    FloatPoint* nextSLPCoordiPtr)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numInsts) {
    FloatPoint nextCoordi(
        curSLPCoordiPtr[instIdx].x + stepLength * curSLPSumGradsPtr[instIdx].x,
        curSLPCoordiPtr[instIdx].y + stepLength * curSLPSumGradsPtr[instIdx].y);

    FloatPoint nextSLPCoordi(
        nextCoordi.x + coeff * (nextCoordi.x - curCoordiPtr[instIdx].x),
        nextCoordi.y + coeff * (nextCoordi.y - curCoordiPtr[instIdx].y));

    // check the boundary
    nextCoordiPtr[instIdx]
        = FloatPoint(getDensityCoordiLayoutInside(
                         instDDx[instIdx], nextCoordi.x, coreLx, coreUx),
                     getDensityCoordiLayoutInside(
                         instDDy[instIdx], nextCoordi.y, coreLy, coreUy));

    nextSLPCoordiPtr[instIdx]
        = FloatPoint(getDensityCoordiLayoutInside(
                         instDDx[instIdx], nextSLPCoordi.x, coreLx, coreUx),
                     getDensityCoordiLayoutInside(
                         instDDy[instIdx], nextSLPCoordi.y, coreLy, coreUy));
  }
}

void PlacerBase::nesterovUpdateCoordinates(float coeff)
{
  if (isConverged_) {
    return;
  }

  int numThreads = 256;
  int numBlocks = (numInsts_ + numThreads - 1) / numThreads;

  nesterovUpdateCooridnatesKernel<<<numBlocks, numThreads>>>(
      numInsts_,
      bg_.lx(),
      bg_.ly(),
      bg_.ux(),
      bg_.uy(),
      stepLength_,
      coeff,
      dInstDDxPtr_,
      dInstDDyPtr_,
      dCurCoordiPtr_,
      dCurSLPCoordiPtr_,
      dCurSLPSumGradsPtr_,
      dNextCoordiPtr_,
      dNextSLPCoordiPtr_);

  // update density
  updateGCellDensityCenterLocation(dNextSLPCoordiPtr_);
  updateDensityForceBin();
}

__global__ void updateInitialPrevSLPCoordiKernel(
    const int numInsts,
    const int coreLx,
    const int coreLy,
    const int coreUx,
    const int coreUy,
    const int* instDDx,
    const int* instDDy,
    const float initialPrevCoordiUpdateCoef,
    const FloatPoint* dCurSLPCoordiPtr,
    const FloatPoint* dCurSLPSumGradsPtr,
    FloatPoint* dPrevSLPCoordiPtr)
{
  int instIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (instIdx < numInsts) {
    const float preCoordiX
        = dCurSLPCoordiPtr[instIdx].x
          - initialPrevCoordiUpdateCoef * dCurSLPSumGradsPtr[instIdx].x;
    const float preCoordiY
        = dCurSLPCoordiPtr[instIdx].y
          - initialPrevCoordiUpdateCoef * dCurSLPSumGradsPtr[instIdx].y;
    const FloatPoint newCoordi(
        getDensityCoordiLayoutInside(
            instDDx[instIdx], preCoordiX, coreLx, coreUx),
        getDensityCoordiLayoutInside(
            instDDy[instIdx], preCoordiY, coreLy, coreUy));
    dPrevSLPCoordiPtr[instIdx] = newCoordi;
  }
}

void PlacerBase::updateInitialPrevSLPCoordi()
{
  const int numThreads = 256;
  const int numBlocks = (numInsts_ + numThreads - 1) / numThreads;
  updateInitialPrevSLPCoordiKernel<<<numBlocks, numThreads>>>(
      numInsts_,
      bg_.lx(),
      bg_.ly(),
      bg_.ux(),
      bg_.uy(),
      dInstDDxPtr_,
      dInstDDyPtr_,
      npVars_.initialPrevCoordiUpdateCoef,
      dCurSLPCoordiPtr_,
      dCurSLPSumGradsPtr_,
      dPrevSLPCoordiPtr_);
}

}  // namespace gpl2
