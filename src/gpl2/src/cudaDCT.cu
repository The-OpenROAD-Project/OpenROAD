#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
// BSD 3-Clause License
//
// Copyright (c) 2023, Google LLC
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// The density force is calculated by solving the Poisson equation.
// It is originally developed by the graduate student Jaekyung Kim
// (jkim97@postech.ac.kr) at Pohang University of Science and Technology
// (POSTECH), then modified by our UCSD team. We thank Jaekyung Kim for his
// contribution.
//
//
///////////////////////////////////////////////////////////////////////////////

#include <hipfft/hipfft.h>

#include <cassert>

#include "cudaDCT.h"
#include "cudaUtil.h"

#define TPB 16

__global__ void dct2d_preprocess(const hipfftReal* x,
                                 hipfftReal* y,
                                 const int M,
                                 const int N,
                                 const int halfN)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;
  if (hid < M && wid < N) {
    int index;
    int cond = (((hid & 1) == 0) << 1) | ((wid & 1) == 0);
    switch (cond) {
      case 0:
        index = INDEX((M << 1) - (hid + 1), N - ((wid + 1) >> 1), halfN);
        break;
      case 1:
        index = INDEX((M << 1) - (hid + 1), (wid >> 1), halfN);
        break;
      case 2:
        index = INDEX(hid, N - ((wid + 1) >> 1), halfN);
        break;
      case 3:
        index = INDEX(hid, (wid >> 1), halfN);
        break;
      default:
        break;
    }
    y[index] = x[INDEX(hid, wid, N)];
  }
}

__global__ __launch_bounds__(TPB* TPB, 10) void dct2d_postprocess(
    const hipfftComplex* V,
    hipfftReal* y,
    const int M,
    const int N,
    const int halfM,
    const int halfN,
    const hipfftReal two_over_MN,
    const hipfftReal four_over_MN,
    const hipfftComplex* __restrict__ expkM,
    const hipfftComplex* __restrict__ expkN)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;
  if (hid < halfM && wid < halfN) {
    int cond = ((hid != 0) << 1) | (wid != 0);
    switch (cond) {
      case 0: {
        y[0] = V[0].x * four_over_MN;
        y[halfN] = RealPartOfMul(expkN[halfN], V[halfN]) * four_over_MN;

        y[INDEX(halfM, 0, N)]
            = expkM[halfM].x * V[INDEX(halfM, 0, halfN + 1)].x * four_over_MN;

        y[INDEX(halfM, halfN, N)]
            = expkM[halfM].x
              * RealPartOfMul(expkN[halfN], V[INDEX(halfM, halfN, halfN + 1)])
              * four_over_MN;
        break;
      }

      case 1: {
        hipfftComplex tmp;

        tmp = V[wid];
        y[wid] = RealPartOfMul(expkN[wid], tmp) * four_over_MN;
        y[N - wid] = -ImaginaryPartOfMul(expkN[wid], tmp) * four_over_MN;

        tmp = V[INDEX(halfM, wid, halfN + 1)];
        y[INDEX(halfM, wid, N)]
            = expkM[halfM].x * RealPartOfMul(expkN[wid], tmp) * four_over_MN;
        y[INDEX(halfM, N - wid, N)] = -expkM[halfM].x
                                      * ImaginaryPartOfMul(expkN[wid], tmp)
                                      * four_over_MN;
        break;
      }

      case 2: {
        hipfftComplex tmp1, tmp2, tmp_up, tmp_down;
        tmp1 = V[INDEX(hid, 0, halfN + 1)];
        tmp2 = V[INDEX(M - hid, 0, halfN + 1)];
        tmp_up.x = expkM[hid].x * (tmp1.x + tmp2.x)
                   + expkM[hid].y * (tmp2.y - tmp1.y);
        tmp_down.x = -expkM[hid].y * (tmp1.x + tmp2.x)
                     + expkM[hid].x * (tmp2.y - tmp1.y);
        y[INDEX(hid, 0, N)] = tmp_up.x * two_over_MN;
        y[INDEX(M - hid, 0, N)] = tmp_down.x * two_over_MN;

        tmp1 = complexAdd(V[INDEX(hid, halfN, halfN + 1)],
                          V[INDEX(M - hid, halfN, halfN + 1)]);
        tmp2 = complexSubtract(V[INDEX(hid, halfN, halfN + 1)],
                               V[INDEX(M - hid, halfN, halfN + 1)]);
        tmp_up.x = expkM[hid].x * tmp1.x - expkM[hid].y * tmp2.y;
        tmp_up.y = expkM[hid].x * tmp1.y + expkM[hid].y * tmp2.x;
        tmp_down.x = -expkM[hid].y * tmp1.x - expkM[hid].x * tmp2.y;
        tmp_down.y = -expkM[hid].y * tmp1.y + expkM[hid].x * tmp2.x;
        y[INDEX(hid, halfN, N)]
            = RealPartOfMul(expkN[halfN], tmp_up) * two_over_MN;
        y[INDEX(M - hid, halfN, N)]
            = RealPartOfMul(expkN[halfN], tmp_down) * two_over_MN;
        break;
      }

      case 3: {
        hipfftComplex tmp1, tmp2, tmp_up, tmp_down;
        tmp1 = complexAdd(V[INDEX(hid, wid, halfN + 1)],
                          V[INDEX(M - hid, wid, halfN + 1)]);
        tmp2 = complexSubtract(V[INDEX(hid, wid, halfN + 1)],
                               V[INDEX(M - hid, wid, halfN + 1)]);
        tmp_up.x = expkM[hid].x * tmp1.x - expkM[hid].y * tmp2.y;
        tmp_up.y = expkM[hid].x * tmp1.y + expkM[hid].y * tmp2.x;
        tmp_down.x = -expkM[hid].y * tmp1.x - expkM[hid].x * tmp2.y;
        tmp_down.y = -expkM[hid].y * tmp1.y + expkM[hid].x * tmp2.x;
        y[INDEX(hid, wid, N)] = RealPartOfMul(expkN[wid], tmp_up) * two_over_MN;
        y[INDEX(M - hid, wid, N)]
            = RealPartOfMul(expkN[wid], tmp_down) * two_over_MN;
        y[INDEX(hid, N - wid, N)]
            = -ImaginaryPartOfMul(expkN[wid], tmp_up) * two_over_MN;
        y[INDEX(M - hid, N - wid, N)]
            = -ImaginaryPartOfMul(expkN[wid], tmp_down) * two_over_MN;
        break;
      }

      default:
        assert(0);
        break;
    }
  }
}

void fft2D(hipfftReal* d_input, hipfftComplex* d_output, hipfftHandle& plan)
{
  hipfftExecR2C(plan, d_input, d_output);
  // hipDeviceSynchronize();
}

void dct_2d_fft(const int M,
                const int N,
                hipfftHandle& plan,
                const hipfftComplex* expkM,
                const hipfftComplex* expkN,
                const hipfftReal* input,
                hipfftReal* pre,
                hipfftComplex* fft,
                hipfftReal* post)
{
  if (!isPowerOf2(N) || !isPowerOf2(M)) {
    printf("Input length is not power of 2.\n");
    assert(0);
  }

  dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
  dim3 gridSize2((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
  dim3 blockSize(TPB, TPB, 1);

  dct2d_preprocess<<<gridSize, blockSize>>>(input, pre, M, N, N / 2);

  // hipDeviceSynchronize();

  fft2D(pre, fft, plan);

  dct2d_postprocess<<<gridSize2, blockSize>>>(fft,
                                              post,
                                              M,
                                              N,
                                              M / 2,
                                              N / 2,
                                              2.0 / (M * N),
                                              4.0 / (M * N),
                                              expkM,
                                              expkN);
  // cuda2eviceSynchronize();
}

////////////////////////////////////////////////////////////////////////////////////

__global__ void idct2d_postprocess(const hipfftReal* x,
                                   hipfftReal* y,
                                   const int M,
                                   const int N,
                                   const int halfN)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;
  if (hid < M && wid < N) {
    int cond = ((hid < M / 2) << 1) | (wid < N / 2);
    int index;
    switch (cond) {
      case 0:
        index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
        break;
      case 1:
        index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
        break;
      case 2:
        index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
        break;
      case 3:
        index = INDEX(hid << 1, wid << 1, N);
        break;
      default:
        assert(0);
        break;
    }
    y[index] = x[INDEX(hid, wid, N)];
  }
}

__global__ __launch_bounds__(TPB* TPB, 8) void idct2d_preprocess(
    const hipfftReal* input,
    hipfftComplex* output,
    const int M,
    const int N,
    const int halfM,
    const int halfN,
    const hipfftComplex* __restrict__ expkM,
    const hipfftComplex* __restrict__ expkN,
    const hipfftComplex* __restrict__ expkMN_1,
    const hipfftComplex* __restrict__ expkMN_2)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;
  if (hid < halfM && wid < halfN) {
    int cond = ((hid != 0) << 1) | (wid != 0);
    switch (cond) {
      case 0: {
        hipfftReal tmp1;
        hipfftComplex tmp_up;

        output[0].x = input[0];
        output[0].y = 0;

        tmp1 = input[halfN];
        tmp_up.x = tmp1;
        tmp_up.y = tmp1;
        output[halfN] = complexMulConj(expkN[halfN], tmp_up);

        tmp1 = input[INDEX(halfM, 0, N)];
        tmp_up.x = tmp1;
        tmp_up.y = tmp1;
        output[INDEX(halfM, 0, halfN + 1)]
            = complexMulConj(expkM[halfM], tmp_up);

        tmp1 = input[INDEX(halfM, halfN, N)];
        tmp_up.x = 0;
        tmp_up.y = 2 * tmp1;
        output[INDEX(halfM, halfN, halfN + 1)]
            = complexMulConj(expkMN_1[halfM + halfN], tmp_up);
        break;
      }

      case 1: {
        hipfftComplex tmp_up;
        tmp_up.x = input[wid];
        tmp_up.y = input[N - wid];
        output[wid] = complexMulConj(expkN[wid], tmp_up);

        hipfftReal tmp1 = input[INDEX(halfM, wid, N)];
        hipfftReal tmp2 = input[INDEX(halfM, N - wid, N)];
        tmp_up.x = tmp1 - tmp2;
        tmp_up.y = tmp1 + tmp2;
        output[INDEX(halfM, wid, halfN + 1)]
            = complexMulConj(expkMN_1[halfM + wid], tmp_up);
        break;
      }

      case 2: {
        hipfftReal tmp1, tmp3;
        hipfftComplex tmp_up, tmp_down;

        tmp1 = input[INDEX(hid, 0, N)];
        tmp3 = input[INDEX(M - hid, 0, N)];
        tmp_down.x = tmp3;
        tmp_down.y = tmp1;

        // two outputs are conjugate
        tmp_up = complexMul(expkM[M - hid], tmp_down);
        output[INDEX(hid, 0, halfN + 1)] = tmp_up;
        output[INDEX(M - hid, 0, halfN + 1)] = complexConj(tmp_up);

        tmp1 = input[INDEX(hid, halfN, N)];
        tmp3 = input[INDEX(M - hid, halfN, N)];
        tmp_up.x = tmp1 - tmp3;
        tmp_up.y = tmp3 + tmp1;
        tmp_down.x = tmp3 - tmp1;
        tmp_down.y = tmp1 + tmp3;

        output[INDEX(hid, halfN, halfN + 1)]
            = complexMulConj(expkMN_1[hid + halfN], tmp_up);
        output[INDEX(M - hid, halfN, halfN + 1)]
            = complexMulConj(expkMN_2[halfN - hid + (N - 1)], tmp_down);
        break;
      }

      case 3: {
        hipfftReal tmp1 = input[INDEX(hid, wid, N)];
        hipfftReal tmp2 = input[INDEX(hid, N - wid, N)];
        hipfftReal tmp3 = input[INDEX(M - hid, wid, N)];
        hipfftReal tmp4 = input[INDEX(M - hid, N - wid, N)];
        hipfftComplex tmp_up, tmp_down;
        tmp_up.x = tmp1 - tmp4;
        tmp_up.y = tmp3 + tmp2;
        tmp_down.x = tmp3 - tmp2;
        tmp_down.y = tmp1 + tmp4;

        output[INDEX(hid, wid, halfN + 1)]
            = complexMulConj(expkMN_1[hid + wid], tmp_up);
        output[INDEX(M - hid, wid, halfN + 1)]
            = complexMulConj(expkMN_2[wid - hid + (N - 1)], tmp_down);
        break;
      }

      default:
        assert(0);
        break;
    }
  }
}

void ifft2D(hipfftComplex* d_input, hipfftReal* d_output, hipfftHandle& plan)
{
  hipfftExecC2R(plan, d_input, d_output);
  // hipDeviceSynchronize();
}

void idct_2d_fft(const int M,
                 const int N,
                 hipfftHandle& plan,
                 const hipfftComplex* expkMForInverse,
                 const hipfftComplex* expkNForInverse,
                 const hipfftComplex* expkMN1,
                 const hipfftComplex* expkMN2,
                 const hipfftReal* input,
                 hipfftComplex* pre,
                 hipfftReal* ifft,
                 hipfftReal* post)
{
  if (!isPowerOf2(N) || !isPowerOf2(M)) {
    printf("Input length is not power of 2.\n");
    assert(0);
  }

  hipMemset(pre, 0, M * (N / 2 + 1) * sizeof(hipfftComplex));

  dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
  dim3 gridSize2((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
  dim3 blockSize(TPB, TPB, 1);

  idct2d_preprocess<<<gridSize2, blockSize>>>(input,
                                              pre,
                                              M,
                                              N,
                                              M / 2,
                                              N / 2,
                                              expkMForInverse,
                                              expkNForInverse,
                                              expkMN1,
                                              expkMN2);
  // hipDeviceSynchronize();

  ifft2D(pre, ifft, plan);

  idct2d_postprocess<<<gridSize, blockSize>>>(ifft, post, M, N, N / 2);

  // hipDeviceSynchronize();
}

__global__ void idct_idxst_preprocess(const hipfftReal* input,
                                      hipfftReal* output,
                                      const int M,
                                      const int N)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;

  if (hid < M && wid < N) {
    int idx_in = INDEX(M - hid, wid, N);
    int idx_out = INDEX(hid, wid, N);

    if (hid == 0)
      output[idx_out] = 0;
    else
      output[idx_out] = input[idx_in];
  }
}

__global__ void idct_idxst_postprocess(const hipfftReal* input,
                                       hipfftReal* output,
                                       const int M,
                                       const int N)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;

  if (hid < M && wid < N) {
    int idx = INDEX(hid, wid, N);

    if (hid % 2 == 0)
      output[idx] = +input[idx];
    else
      output[idx] = -input[idx];
  }
}

void idct_idxst(const int M,
                const int N,
                hipfftHandle& plan,
                const hipfftComplex* expkMForInverse,
                const hipfftComplex* expkNForInverse,
                const hipfftComplex* expkMN1,
                const hipfftComplex* expkMN2,
                const hipfftReal* input,
                hipfftReal* workSpaceReal1,
                hipfftComplex* workSpaceComplex,
                hipfftReal* workSpaceReal2,
                hipfftReal* workSpaceReal3,
                hipfftReal* output)
{
  if (!isPowerOf2(N) || !isPowerOf2(M)) {
    printf("Input length is not power of 2.\n");
    assert(0);
  }

  dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
  dim3 blockSize(TPB, TPB, 1);

  idct_idxst_preprocess<<<gridSize, blockSize>>>(input, workSpaceReal1, M, N);

  // hipDeviceSynchronize();

  idct_2d_fft(M,
              N,
              plan,
              expkMForInverse,
              expkNForInverse,
              expkMN1,
              expkMN2,
              workSpaceReal1,
              workSpaceComplex,
              workSpaceReal2,
              workSpaceReal3);

  idct_idxst_postprocess<<<gridSize, blockSize>>>(workSpaceReal3, output, M, N);

  // hipDeviceSynchronize();
}

__global__ void idxst_idct_preprocess(const hipfftReal* input,
                                      hipfftReal* output,
                                      const int M,
                                      const int N)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;

  if (hid < M && wid < N) {
    int idx_in = INDEX(hid, N - wid, N);
    int idx_out = INDEX(hid, wid, N);

    if (wid == 0)
      output[idx_out] = 0;
    else
      output[idx_out] = input[idx_in];
  }
}

__global__ void idxst_idct_postprocess(const hipfftReal* input,
                                       hipfftReal* output,
                                       const int M,
                                       const int N)
{
  const int wid = blockDim.x * blockIdx.x + threadIdx.x;
  const int hid = blockDim.y * blockIdx.y + threadIdx.y;

  if (hid < M && wid < N) {
    int idx = INDEX(hid, wid, N);

    if (wid % 2 == 0)
      output[idx] = +input[idx];
    else
      output[idx] = -input[idx];
  }
}

void idxst_idct(const int M,
                const int N,
                hipfftHandle& plan,
                const hipfftComplex* expkMForInverse,
                const hipfftComplex* expkNForInverse,
                const hipfftComplex* expkMN1,
                const hipfftComplex* expkMN2,
                const hipfftReal* input,
                hipfftReal* workSpaceReal1,
                hipfftComplex* workSpaceComplex,
                hipfftReal* workSpaceReal2,
                hipfftReal* workSpaceReal3,
                hipfftReal* output)
{
  if (!isPowerOf2(N) || !isPowerOf2(M)) {
    printf("Input length is not power of 2.\n");
    assert(0);
  }

  dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
  dim3 blockSize(TPB, TPB, 1);

  idxst_idct_preprocess<<<gridSize, blockSize>>>(input, workSpaceReal1, M, N);

  // hipDeviceSynchronize();

  idct_2d_fft(M,
              N,
              plan,
              expkMForInverse,
              expkNForInverse,
              expkMN1,
              expkMN2,
              workSpaceReal1,
              workSpaceComplex,
              workSpaceReal2,
              workSpaceReal3);

  idxst_idct_postprocess<<<gridSize, blockSize>>>(workSpaceReal3, output, M, N);

  // hipDeviceSynchronize();
}
