#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
// BSD 3-Clause License
//
// Copyright (c) 2023, Google LLC
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
///////////////////////////////////////////////////////////////////////////////

#include <hipfft/hipfft.h>
#include <math_functions.h>  // Include CUDA math functions header
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <algorithm>
#include <cassert>
#include <cfloat>
#include <climits>
#include <cmath>
#include <ctime>
#include <fstream>
#include <iostream>
#include <memory>
#include <random>
#include <vector>

#include "placerBase.h"
#include "placerObjects.h"

namespace gpl2 {

/////////////////////////////////////////////////////////
// Class WirelengthOp
void WirelengthOp::initCUDAKernel()
{
  // Initialize related information
  std::vector<int> hInstPinIdx;
  thrust::host_vector<int> hInstPinPos(numInsts_ + 1);
  thrust::host_vector<int> hPinInstId(numPins_);

  std::vector<int> hNetPinIdx;
  thrust::host_vector<int> hNetPinPos(numNets_ + 1);
  thrust::host_vector<int> hPinNetId(numPins_);

  thrust::host_vector<float> hNetWeight(numNets_);
  thrust::host_vector<float> hNetVirtualWeight(numNets_);

  int pinIdx = 0;
  for (auto pin : pbc_->pins()) {
    hPinInstId[pinIdx] = pin->instId();
    hPinNetId[pinIdx] = pin->netId();
    pinIdx++;
  }

  int instIdx = 0;
  hInstPinPos[0] = 0;
  for (auto& inst : pbc_->insts()) {
    for (auto& pin : inst->pins()) {
      hInstPinIdx.push_back(pin->pinId());
    }
    hInstPinPos[instIdx + 1] = hInstPinPos[instIdx] + inst->numPins();
    instIdx++;
  }

  int netIdx = 0;
  hNetPinPos[0] = 0;
  for (auto& net : pbc_->nets()) {
    for (auto& pin : net->pins()) {
      hNetPinIdx.push_back(pin->pinId());
    }

    hNetWeight[netIdx] = net->weight();
    hNetVirtualWeight[netIdx] = net->virtualWeight();
    hNetPinPos[netIdx + 1] = hNetPinPos[netIdx] + net->numPins();
    netIdx++;
  }

  // Allocate memory on the device side
  dInstPinIdxPtr_ = setThrustVector<int>(hInstPinIdx.size(), dInstPinIdx_);
  dInstPinPosPtr_ = setThrustVector<int>(numInsts_ + 1, dInstPinPos_);
  dPinInstIdPtr_ = setThrustVector<int>(numPins_, dPinInstId_);

  dNetPinIdxPtr_ = setThrustVector<int>(hNetPinIdx.size(), dNetPinIdx_);
  dNetWeightPtr_ = setThrustVector<float>(numNets_, dNetWeight_);
  dNetVirtualWeightPtr_ = setThrustVector<float>(numNets_, dNetVirtualWeight_);
  dNetPinPosPtr_ = setThrustVector<int>(numNets_ + 1, dNetPinPos_);
  dPinNetIdPtr_ = setThrustVector<int>(numPins_, dPinNetId_);

  // copy from host to device
  thrust::copy(hInstPinIdx.begin(), hInstPinIdx.end(), dInstPinIdx_.begin());
  thrust::copy(hInstPinPos.begin(), hInstPinPos.end(), dInstPinPos_.begin());
  thrust::copy(hPinInstId.begin(), hPinInstId.end(), dPinInstId_.begin());

  thrust::copy(hNetWeight.begin(), hNetWeight.end(), dNetWeight_.begin());
  thrust::copy(hNetVirtualWeight.begin(),
               hNetVirtualWeight.end(),
               dNetVirtualWeight_.begin());

  thrust::copy(hNetPinIdx.begin(), hNetPinIdx.end(), dNetPinIdx_.begin());
  thrust::copy(hNetPinPos.begin(), hNetPinPos.end(), dNetPinPos_.begin());
  thrust::copy(hPinNetId.begin(), hPinNetId.end(), dPinNetId_.begin());

  // Check the pin information
  thrust::host_vector<int> hPinX(numPins_);
  thrust::host_vector<int> hPinY(numPins_);
  thrust::host_vector<int> hPinOffsetX(numPins_);
  thrust::host_vector<int> hPinOffsetY(numPins_);

  // This is for fixed instances
  for (auto& pin : pbc_->pins()) {
    const int pinId = pin->pinId();
    hPinX[pinId] = pin->cx();
    hPinY[pinId] = pin->cy();
    hPinOffsetX[pinId] = pin->offsetCx();
    hPinOffsetY[pinId] = pin->offsetCy();
  }

  // allocate memory on the device side
  dPinXPtr_ = setThrustVector<int>(numPins_, dPinX_);
  dPinYPtr_ = setThrustVector<int>(numPins_, dPinY_);
  dPinOffsetXPtr_ = setThrustVector<int>(numPins_, dPinOffsetX_);
  dPinOffsetYPtr_ = setThrustVector<int>(numPins_, dPinOffsetY_);
  dPinGradXPtr_ = setThrustVector<float>(numPins_, dPinGradX_);
  dPinGradYPtr_ = setThrustVector<float>(numPins_, dPinGradY_);

  dPinAPosXPtr_ = setThrustVector<float>(numPins_, dPinAPosX_);
  dPinANegXPtr_ = setThrustVector<float>(numPins_, dPinANegX_);
  dPinAPosYPtr_ = setThrustVector<float>(numPins_, dPinAPosY_);
  dPinANegYPtr_ = setThrustVector<float>(numPins_, dPinANegY_);
  dNetBPosXPtr_ = setThrustVector<float>(numNets_, dNetBPosX_);
  dNetBNegXPtr_ = setThrustVector<float>(numNets_, dNetBNegX_);
  dNetBPosYPtr_ = setThrustVector<float>(numNets_, dNetBPosY_);
  dNetBNegYPtr_ = setThrustVector<float>(numNets_, dNetBNegY_);
  dNetCPosXPtr_ = setThrustVector<float>(numNets_, dNetCPosX_);
  dNetCNegXPtr_ = setThrustVector<float>(numNets_, dNetCNegX_);
  dNetCPosYPtr_ = setThrustVector<float>(numNets_, dNetCPosY_);
  dNetCNegYPtr_ = setThrustVector<float>(numNets_, dNetCNegY_);

  dNetLxPtr_ = setThrustVector<int>(numNets_, dNetLx_);
  dNetLyPtr_ = setThrustVector<int>(numNets_, dNetLy_);
  dNetUxPtr_ = setThrustVector<int>(numNets_, dNetUx_);
  dNetUyPtr_ = setThrustVector<int>(numNets_, dNetUy_);
  dNetWidthPtr_ = setThrustVector<int>(numNets_, dNetWidth_);
  dNetHeightPtr_ = setThrustVector<int>(numNets_, dNetHeight_);

  // copy from host to device
  thrust::copy(hPinX.begin(), hPinX.end(), dPinX_.begin());
  thrust::copy(hPinY.begin(), hPinY.end(), dPinY_.begin());
  thrust::copy(hPinOffsetX.begin(), hPinOffsetX.end(), dPinOffsetX_.begin());
  thrust::copy(hPinOffsetY.begin(), hPinOffsetY.end(), dPinOffsetY_.begin());
}

void WirelengthOp::freeCUDAKernel()
{
  numInsts_ = 0;
  numPins_ = 0;
  numNets_ = 0;
  numPlaceInsts_ = 0;

  pbc_ = nullptr;
  logger_ = nullptr;

  dInstPinIdxPtr_ = nullptr;
  dInstPinPosPtr_ = nullptr;
  dPinInstIdPtr_ = nullptr;

  dNetPinIdxPtr_ = nullptr;
  dNetPinPosPtr_ = nullptr;
  dPinNetIdPtr_ = nullptr;

  dPinXPtr_ = nullptr;
  dPinYPtr_ = nullptr;
  dPinOffsetXPtr_ = nullptr;
  dPinOffsetYPtr_ = nullptr;
  dPinGradXPtr_ = nullptr;
  dPinGradYPtr_ = nullptr;
  dPinAPosXPtr_ = nullptr;
  dPinANegXPtr_ = nullptr;
  dPinAPosYPtr_ = nullptr;
  dPinANegYPtr_ = nullptr;

  dNetWidthPtr_ = nullptr;
  dNetHeightPtr_ = nullptr;
  dNetLxPtr_ = nullptr;
  dNetLyPtr_ = nullptr;
  dNetUxPtr_ = nullptr;
  dNetUyPtr_ = nullptr;
  dNetWeightPtr_ = nullptr;
  dNetVirtualWeightPtr_ = nullptr;
  dNetBPosXPtr_ = nullptr;
  dNetBNegXPtr_ = nullptr;
  dNetBPosYPtr_ = nullptr;
  dNetBNegYPtr_ = nullptr;
  dNetCPosXPtr_ = nullptr;
  dNetCNegXPtr_ = nullptr;
  dNetCPosYPtr_ = nullptr;
  dNetCNegYPtr_ = nullptr;
}

// All other operations only for placeable Instances
__global__ void updatePinLocationKernel(const int numPlaceInsts,
                                        const int* dInstPinIdxPtr,
                                        const int* dInstPinPosPtr,
                                        const int* dPinOffsetXPtr,
                                        const int* dPinOffsetYPtr,
                                        const int* instDCx,
                                        const int* instDCy,
                                        int* dPinXPtr,
                                        int* dPinYPtr)
{
  const int instId = blockIdx.x * blockDim.x + threadIdx.x;
  if (instId < numPlaceInsts) {
    const int pinStart = dInstPinPosPtr[instId];
    const int pinEnd = dInstPinPosPtr[instId + 1];
    const float instDCxVal = instDCx[instId];
    const float instDCyVal = instDCy[instId];
    for (int pinId = pinStart; pinId < pinEnd; ++pinId) {
      const int pinIdx = dInstPinIdxPtr[pinId];
      dPinXPtr[pinIdx] = instDCxVal + dPinOffsetXPtr[pinIdx];
      dPinYPtr[pinIdx] = instDCyVal + dPinOffsetYPtr[pinIdx];
    }
  }
}

__global__ void updateNetBBoxKernel(int numNets,
                                    const int* dNetPinIdxPtr,
                                    const int* dNetPinPosPtr,
                                    const int* dPinXPtr,
                                    const int* dPinYPtr,
                                    int* dNetLxPtr,
                                    int* dNetLyPtr,
                                    int* dNetUxPtr,
                                    int* dNetUyPtr,
                                    int* dNetWidthPtr,
                                    int* dNetHeightPtr)
{
  const int netId = blockIdx.x * blockDim.x + threadIdx.x;
  if (netId < numNets) {
    const int pinStart = dNetPinPosPtr[netId];
    const int pinEnd = dNetPinPosPtr[netId + 1];
    int netLx = INT_MAX;
    int netLy = INT_MAX;
    int netUx = 0;
    int netUy = 0;
    for (int pinId = pinStart; pinId < pinEnd; ++pinId) {
      const int pinIdx = dNetPinIdxPtr[pinId];
      const int pinX = dPinXPtr[pinIdx];
      const int pinY = dPinYPtr[pinIdx];
      netLx = min(netLx, pinX);
      netLy = min(netLy, pinY);
      netUx = max(netUx, pinX);
      netUy = max(netUy, pinY);
    }

    if (netLx > netUx || netLy > netUy) {
      netLx = 0;
      netUx = 0;
      netLy = 0;
      netUy = 0;
    }

    dNetLxPtr[netId] = netLx;
    dNetLyPtr[netId] = netLy;
    dNetUxPtr[netId] = netUx;
    dNetUyPtr[netId] = netUy;
    dNetWidthPtr[netId] = netUx - netLx;
    dNetHeightPtr[netId] = netUy - netLy;
  }
}

void WirelengthOp::updatePinLocation(const int* instDCx, const int* instDCy)
{
  int numThreads = 256;
  int numBlocks = (numInsts_ + numThreads - 1) / numThreads;
  updatePinLocationKernel<<<numBlocks, numThreads>>>(numPlaceInsts_,
                                                     dInstPinIdxPtr_,
                                                     dInstPinPosPtr_,
                                                     dPinOffsetXPtr_,
                                                     dPinOffsetYPtr_,
                                                     instDCx,
                                                     instDCy,
                                                     dPinXPtr_,
                                                     dPinYPtr_);

  int numNetThreads = 256;
  int numNetBlocks = (numNets_ + numNetThreads - 1) / numThreads;
  updateNetBBoxKernel<<<numNetBlocks, numThreads>>>(numNets_,
                                                    dNetPinIdxPtr_,
                                                    dNetPinPosPtr_,
                                                    dPinXPtr_,
                                                    dPinYPtr_,
                                                    dNetLxPtr_,
                                                    dNetLyPtr_,
                                                    dNetUxPtr_,
                                                    dNetUyPtr_,
                                                    dNetWidthPtr_,
                                                    dNetHeightPtr_);
}

struct TypeConvertor
{
  __host__ __device__ int64_t operator()(const int& x) const
  {
    return static_cast<int64_t>(x);
  }
};

int64_t WirelengthOp::computeHPWL()
{
  int64_t hpwl = 0;
  hpwl = thrust::transform_reduce(dNetWidth_.begin(),
                                  dNetWidth_.end(),
                                  TypeConvertor(),
                                  hpwl,
                                  thrust::plus<int64_t>());

  hpwl = thrust::transform_reduce(dNetHeight_.begin(),
                                  dNetHeight_.end(),
                                  TypeConvertor(),
                                  hpwl,
                                  thrust::plus<int64_t>());
  return hpwl;
}

struct WeightHPWLFunctor
{
  float virtualWeightFactor_;

  WeightHPWLFunctor(float virtualWeightFactor)
      : virtualWeightFactor_(virtualWeightFactor)
  {
  }

  __host__ __device__ int64_t
  operator()(const thrust::tuple<int, int, float, float>& t) const
  {
    const int width = thrust::get<0>(t);
    const int height = thrust::get<1>(t);
    const float weight = thrust::get<2>(t);
    const float virtualWeight = thrust::get<3>(t);
    const float sumWeight = weight + virtualWeight * virtualWeightFactor_;
    return static_cast<int64_t>(sumWeight * (width + height));
  }
};

int64_t WirelengthOp::computeWeightedHPWL(float virtualWeightFactor)
{
  int64_t hpwl = 0;
  hpwl = thrust::transform_reduce(
      thrust::make_zip_iterator(thrust::make_tuple(dNetWidth_.begin(),
                                                   dNetHeight_.begin(),
                                                   dNetWeight_.begin(),
                                                   dNetVirtualWeight_.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(dNetWidth_.end(),
                                                   dNetHeight_.end(),
                                                   dNetWeight_.end(),
                                                   dNetVirtualWeight_.end())),
      WeightHPWLFunctor(virtualWeightFactor),
      hpwl,
      thrust::plus<int64_t>());

  return hpwl;
}

// Compute aPos and aNeg
__global__ void computeAPosNegKernel(const int numPins,
                                     const float wlCoeffX,
                                     const float wlCoeffY,
                                     const int* dPinXPtr,
                                     const int* dPinYPtr,
                                     const int* dPinNetIdPtr,
                                     const int* dNetLxPtr,
                                     const int* dNetLyPtr,
                                     const int* dNetUxPtr,
                                     const int* dNetUyPtr,
                                     float* dPinAPosXPtr,
                                     float* dPinANegXPtr,
                                     float* dPinAPosYPtr,
                                     float* dPinANegYPtr)
{
  const unsigned int pinId = blockIdx.x * blockDim.x + threadIdx.x;
  if (pinId < numPins) {
    const int netId = dPinNetIdPtr[pinId];
    dPinAPosXPtr[pinId] = expf(wlCoeffX * (dPinXPtr[pinId] - dNetUxPtr[netId]));
    dPinANegXPtr[pinId]
        = expf(-1.0 * wlCoeffX * (dPinXPtr[pinId] - dNetLxPtr[netId]));
    dPinAPosYPtr[pinId] = expf(wlCoeffY * (dPinYPtr[pinId] - dNetUyPtr[netId]));
    dPinANegYPtr[pinId]
        = expf(-1.0 * wlCoeffY * (dPinYPtr[pinId] - dNetLyPtr[netId]));
  }
}

__global__ void computeBCPosNegKernel(int numNets,
                                      const int* __restrict__ dNetPinPosPtr,
                                      const int* __restrict__ dNetPinIdxPtr,
                                      const int* __restrict__ dPinXPtr,
                                      const int* __restrict__ dPinYPtr,
                                      const float* __restrict__ dPinAPosXPtr,
                                      const float* __restrict__ dPinANegXPtr,
                                      const float* __restrict__ dPinAPosYPtr,
                                      const float* __restrict__ dPinANegYPtr,
                                      float* dNetBPosXPtr,
                                      float* dNetBNegXPtr,
                                      float* dNetBPosYPtr,
                                      float* dNetBNegYPtr,
                                      float* dNetCPosXPtr,
                                      float* dNetCNegXPtr,
                                      float* dNetCPosYPtr,
                                      float* dNetCNegYPtr)
{
  const unsigned int netId = blockIdx.x * blockDim.x + threadIdx.x;
  if (netId < numNets) {
    const int pinStart = dNetPinPosPtr[netId];
    const int pinEnd = dNetPinPosPtr[netId + 1];
    float bPosX = 0.0;
    float bNegX = 0.0;
    float bPosY = 0.0;
    float bNegY = 0.0;

    float cPosX = 0.0;
    float cNegX = 0.0;
    float cPosY = 0.0;
    float cNegY = 0.0;

    for (int pinId = pinStart; pinId < pinEnd; ++pinId) {
      const int pinIdx = dNetPinIdxPtr[pinId];
      bPosX += dPinAPosXPtr[pinIdx];
      bNegX += dPinANegXPtr[pinIdx];
      bPosY += dPinAPosYPtr[pinIdx];
      bNegY += dPinANegYPtr[pinIdx];

      cPosX += dPinXPtr[pinIdx] * dPinAPosXPtr[pinIdx];
      cNegX += dPinXPtr[pinIdx] * dPinANegXPtr[pinIdx];
      cPosY += dPinYPtr[pinIdx] * dPinAPosYPtr[pinIdx];
      cNegY += dPinYPtr[pinIdx] * dPinANegYPtr[pinIdx];
    }

    dNetBPosXPtr[netId] = bPosX;
    dNetBNegXPtr[netId] = bNegX;
    dNetBPosYPtr[netId] = bPosY;
    dNetBNegYPtr[netId] = bNegY;

    dNetCPosXPtr[netId] = cPosX;
    dNetCNegXPtr[netId] = cNegX;
    dNetCPosYPtr[netId] = cPosY;
    dNetCNegYPtr[netId] = cNegY;
  }
}

__global__ void computePinWAGradKernel(const int numPins,
                                       const float wlCoeffX,
                                       const float wlCoeffY,
                                       const int* __restrict__ dPinNetIdPtr,
                                       const int* __restrict__ dNetPinPosPtr,
                                       const int* __restrict__ dPinXPtr,
                                       const int* __restrict__ dPinYPtr,
                                       const float* __restrict__ dPinAPosXPtr,
                                       const float* __restrict__ dPinANegXPtr,
                                       const float* __restrict__ dPinAPosYPtr,
                                       const float* __restrict__ dPinANegYPtr,
                                       const float* __restrict__ dNetBPosXPtr,
                                       const float* __restrict__ dNetBNegXPtr,
                                       const float* __restrict__ dNetBPosYPtr,
                                       const float* __restrict__ dNetBNegYPtr,
                                       const float* __restrict__ dNetCPosXPtr,
                                       const float* __restrict__ dNetCNegXPtr,
                                       const float* __restrict__ dNetCPosYPtr,
                                       const float* __restrict__ dNetCNegYPtr,
                                       float* dPinGradXPtr,
                                       float* dPinGradYPtr)
{
  int pinIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (pinIdx < numPins) {
    const int netId = dPinNetIdPtr[pinIdx];
    const int netSize = dNetPinPosPtr[netId + 1] - dNetPinPosPtr[netId];

    // TODO:  if we need to remove high-fanout nets,
    // we can remove it here

    float netBNegX2 = dNetBNegXPtr[netId] * dNetBNegXPtr[netId];
    float netBPosX2 = dNetBPosXPtr[netId] * dNetBPosXPtr[netId];
    float netBNegY2 = dNetBNegYPtr[netId] * dNetBNegYPtr[netId];
    float netBPosY2 = dNetBPosYPtr[netId] * dNetBPosYPtr[netId];

    float pinXWlCoeffX = dPinXPtr[pinIdx] * wlCoeffX;
    float pinYWlCoeffY = dPinYPtr[pinIdx] * wlCoeffY;

    dPinGradXPtr[pinIdx] = ((1.0f - pinXWlCoeffX) * dNetBNegXPtr[netId]
                            + wlCoeffX * dNetCNegXPtr[netId])
                               * dPinANegXPtr[pinIdx] / netBNegX2
                           - ((1.0f + pinXWlCoeffX) * dNetBPosXPtr[netId]
                              - wlCoeffX * dNetCPosXPtr[netId])
                                 * dPinAPosXPtr[pinIdx] / netBPosX2;

    dPinGradYPtr[pinIdx] = ((1.0f - pinYWlCoeffY) * dNetBNegYPtr[netId]
                            + wlCoeffY * dNetCNegYPtr[netId])
                               * dPinANegYPtr[pinIdx] / netBNegY2
                           - ((1.0f + pinYWlCoeffY) * dNetBPosYPtr[netId]
                              - wlCoeffY * dNetCPosYPtr[netId])
                                 * dPinAPosYPtr[pinIdx] / netBPosY2;
  }
}

// define the kernel for updating wirelength force
// on each instance
__global__ void computeWirelengthGradientWAKernel(
    const int numPlaceInsts,
    const float virtualWeightFactor,
    const int* dPinNetIdPtr,
    const float* dNetWeightPtr,
    const float* dNetVirtualWeightPtr,
    const int* dInstPinIdxPtr,
    const int* dInstPinPosPtr,
    const float* dPinGradX,
    const float* dPinGradY,
    float* wirelengthForceX,
    float* wirelengthForceY)
{
  const int instId = blockIdx.x * blockDim.x + threadIdx.x;
  if (instId < numPlaceInsts) {
    const int pinStart = dInstPinPosPtr[instId];
    const int pinEnd = dInstPinPosPtr[instId + 1];
    float wlGradX = 0.0;
    float wlGradY = 0.0;
    for (int pinId = pinStart; pinId < pinEnd; ++pinId) {
      const int pinIdx = dInstPinIdxPtr[pinId];
      const int netId = dPinNetIdPtr[pinIdx];
      const float weight = dNetWeightPtr[netId]
                           + dNetVirtualWeightPtr[netId] * virtualWeightFactor;
      wlGradX += dPinGradX[pinIdx] * weight;
      wlGradY += dPinGradY[pinIdx] * weight;
    }

    wirelengthForceX[instId] = wlGradX;
    wirelengthForceY[instId] = wlGradY;
  }
}

void WirelengthOp::computeWireLengthForce(const float wlCoeffX,
                                          const float wlCoeffY,
                                          const float virtualWeightFactor,
                                          float* wirelengthForceX,
                                          float* wirelengthForceY)
{
  int numThreads = 256;
  int numNetBlocks = (numNets_ + numThreads - 1) / numThreads;
  int numInstBlocks = (numPlaceInsts_ + numThreads - 1) / numThreads;
  int numPinBlocks = (numPins_ + numThreads - 1) / numThreads;

  computeAPosNegKernel<<<numPinBlocks, numThreads>>>(numPins_,
                                                     wlCoeffX,
                                                     wlCoeffY,
                                                     dPinXPtr_,
                                                     dPinYPtr_,
                                                     dPinNetIdPtr_,
                                                     dNetLxPtr_,
                                                     dNetLyPtr_,
                                                     dNetUxPtr_,
                                                     dNetUyPtr_,
                                                     dPinAPosXPtr_,
                                                     dPinANegXPtr_,
                                                     dPinAPosYPtr_,
                                                     dPinANegYPtr_);

  computeBCPosNegKernel<<<numNetBlocks, numThreads>>>(numNets_,
                                                      dNetPinPosPtr_,
                                                      dNetPinIdxPtr_,
                                                      dPinXPtr_,
                                                      dPinYPtr_,
                                                      dPinAPosXPtr_,
                                                      dPinANegXPtr_,
                                                      dPinAPosYPtr_,
                                                      dPinANegYPtr_,
                                                      dNetBPosXPtr_,
                                                      dNetBNegXPtr_,
                                                      dNetBPosYPtr_,
                                                      dNetBNegYPtr_,
                                                      dNetCPosXPtr_,
                                                      dNetCNegXPtr_,
                                                      dNetCPosYPtr_,
                                                      dNetCNegYPtr_);

  computePinWAGradKernel<<<numPinBlocks, numThreads>>>(numPins_,
                                                       wlCoeffX,
                                                       wlCoeffY,
                                                       dPinNetIdPtr_,
                                                       dNetPinPosPtr_,
                                                       dPinXPtr_,
                                                       dPinYPtr_,
                                                       dPinAPosXPtr_,
                                                       dPinANegXPtr_,
                                                       dPinAPosYPtr_,
                                                       dPinANegYPtr_,
                                                       dNetBPosXPtr_,
                                                       dNetBNegXPtr_,
                                                       dNetBPosYPtr_,
                                                       dNetBNegYPtr_,
                                                       dNetCPosXPtr_,
                                                       dNetCNegXPtr_,
                                                       dNetCPosYPtr_,
                                                       dNetCNegYPtr_,
                                                       dPinGradXPtr_,
                                                       dPinGradYPtr_);

  // get the force on each instance
  computeWirelengthGradientWAKernel<<<numInstBlocks, numThreads>>>(
      numPlaceInsts_,
      virtualWeightFactor,
      dPinNetIdPtr_,
      dNetWeightPtr_,
      dNetVirtualWeightPtr_,
      dInstPinIdxPtr_,
      dInstPinPosPtr_,
      dPinGradXPtr_,
      dPinGradYPtr_,
      wirelengthForceX,
      wirelengthForceY);
}

}  // namespace gpl2
