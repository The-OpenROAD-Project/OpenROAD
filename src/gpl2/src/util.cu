///////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2018-2020, The Regents of the University of California
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <cmath>
#include <iostream>
#include <memory>
#include <numeric>

#include "util.h"
// basic vectors
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
// memory related
#include <thrust/copy.h>
#include <thrust/fill.h>
// algorithm related
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/replace.h>
#include <thrust/transform.h>

namespace gpl2 {

// utilities function
void freeCUDA(void* cuda_pointer)
{
  hipError_t err = hipFree(cuda_pointer);
  if (err != hipSuccess) {
    std::cerr << "Failed to free the pointer (error code ";
    std::cerr << hipGetErrorString(err) << ")!\n";
  }
}

void getLastCUDAErr()
{
  // Check for any errors launching the kernel
  hipError_t cudaerr = hipGetLastError();
  if (cudaerr != hipSuccess) {
    std::cerr << "CUDA failed with error: " << hipGetErrorString(cudaerr);
  }
}

}  // namespace gpl2
