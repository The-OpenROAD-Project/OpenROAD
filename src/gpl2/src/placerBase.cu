#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2018-2023, The Regents of the University of California
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
///////////////////////////////////////////////////////////////////////////////

#include <odb/db.h>
#include <stdio.h>

#include <chrono>
#include <cmath>
#include <iostream>
#include <memory>
#include <numeric>
#include <sstream>
#include <unordered_set>

#include "db_sta/dbNetwork.hh"
#include "placerBase.h"
#include "placerObjects.h"
#include "poissonSolver.h"
#include "sta/Liberty.hh"
#include "util.h"
#include "utl/Logger.h"

namespace gpl2 {

using namespace std;
using utl::GPL2;

#define REPLACE_SQRT2 1.414213562373095048801L

///////////////////////////////////////////////////////////////////////////////////
// PlacerBaseVars
///////////////////////////////////////////////////////////////////////////////////

PlacerBaseVars::PlacerBaseVars()
{
  reset();
}

void PlacerBaseVars::reset()
{
  padLeft = padRight = 0;
  skipIoMode = false;
  row_limit = 6;
}

/////////////////////////////////////////////////////////////////////////////////////
// NesterovBaseVars
/////////////////////////////////////////////////////////////////////////////////////

NesterovBaseVars::NesterovBaseVars()
{
  reset();
}

void NesterovBaseVars::reset()
{
  targetDensity = 1.0;
  binCntX = binCntY = 0;
  minWireLengthForceBar = -300;
  isSetBinCnt = false;
  useUniformTargetDensity = true;
}

/////////////////////////////////////////////////////////////////////////////////////
// NesterovPlaceVars
/////////////////////////////////////////////////////////////////////////////////////

NesterovPlaceVars::NesterovPlaceVars()
{
  reset();
}

// default variables for NesterovPlaceVars
void NesterovPlaceVars::reset()
{
  maxNesterovIter = 5000;
  maxBackTrack = 10;
  initDensityPenalty = 0.0008;
  initWireLengthCoef = 0.25;
  targetOverflow = 0.1;
  minPhiCoef = 0.95;
  maxPhiCoef = 1.03;
  minPreconditioner = 1.0;
  initialPrevCoordiUpdateCoef = 100;
  referenceHpwl = 446000000;
  routabilityCheckOverflow = 0.20;
  timingDrivenMode = true;
  routabilityDrivenMode = true;
}

/////////////////////////////////////////////////////////////////////////////////////
// PlacerBaseCommon
/////////////////////////////////////////////////////////////////////////////////////

PlacerBaseCommon::PlacerBaseCommon()
    : network_(nullptr),
      db_(nullptr),
      log_(nullptr),
      pbVars_(),
      wlGradOp_(nullptr),
      die_(),
      siteSizeX_(0),
      siteSizeY_(0),
      numInsts_(0),
      haloWidth_(0),
      virtualIter_(0),
      clusterFlag_(false),
      numPlaceInsts_(0),
      numFixedInsts_(0),
      numDummyInsts_(0),
      placeInstsArea_(0),
      nonPlaceInstsArea_(0),
      macroInstsArea_(0),
      stdCellInstsArea_(0),
      virtualWeightFactor_(0.0),
      dInstDCxPtr_(nullptr),
      dInstDCyPtr_(nullptr),
      dWLGradXPtr_(nullptr),
      dWLGradYPtr_(nullptr)
{
}

PlacerBaseCommon::PlacerBaseCommon(sta::dbNetwork* network,
                                   odb::dbDatabase* db,
                                   PlacerBaseVars pbVars,
                                   utl::Logger* log,
                                   float haloWidth,
                                   int virtualIter,
                                   int numHops,
                                   float bloatFactor,
                                   bool clusterFlag,
                                   bool dataflowFlag,
                                   bool datapathFlag,
                                   bool clusterConstraintFlag)
    : PlacerBaseCommon()
{
  network_ = network;
  db_ = db;
  log_ = log;
  pbVars_ = pbVars;
  block_ = db_->getChip()->getBlock();

  clusterFlag_ = clusterFlag;
  dataflowFlag_ = dataflowFlag;
  datapathFlag_ = datapathFlag;
  clusterConstraintFlag_ = clusterConstraintFlag;

  virtualIter_ = virtualIter;
  numHops_ = numHops;
  bloatFactor_ = bloatFactor;

  std::cout << "[INFO] clusterFlag = " << clusterFlag_ << std::endl;
  std::cout << "[INFO] dataflowFlag = " << dataflowFlag_ << std::endl;
  std::cout << "[INFO] datapathFlag = " << datapathFlag_ << std::endl;
  std::cout << "[INFO] clusterConstraintFlag = " << clusterConstraintFlag_
            << std::endl;
  std::cout << "[INFO] virtualIter = " << virtualIter_ << std::endl;
  std::cout << "[INFO] numHops = " << numHops_ << std::endl;
  std::cout << "[INFO] haloWidth = " << haloWidth << std::endl;
  std::cout << "[INFO] bloatFactor = " << bloatFactor << std::endl;

  float dbuPerMicron
      = static_cast<float>(db_->getTech()->getDbUnitsPerMicron());
  haloWidth_ = haloWidth * dbuPerMicron;

  createDataFlow();

  if (clusterFlag_ == true) {
    initClusterNetlist();  // for cluster placement
  } else {
    init();  // for stamdard cell placement
  }
}

PlacerBaseCommon::~PlacerBaseCommon()
{
  reset();
}

void PlacerBaseCommon::evaluateHPWL()
{
  odb::dbBlock* block = db_->getChip()->getBlock();
  odb::dbSet<odb::dbNet> nets = block->getNets();
  unsigned long int hpwl = 0;
  for (odb::dbNet* net : nets) {
    odb::dbSigType netType = net->getSigType();
    // escape nets with VDD/VSS/reset nets
    if (netType == odb::dbSigType::SIGNAL || netType == odb::dbSigType::CLOCK) {
      int minX = std::numeric_limits<int>::max();
      int minY = std::numeric_limits<int>::max();
      int maxX = std::numeric_limits<int>::min();
      int maxY = std::numeric_limits<int>::min();

      for (odb::dbITerm* iTerm : net->getITerms()) {
        int offsetLx = std::numeric_limits<int>::max();
        int offsetLy = std::numeric_limits<int>::max();
        int offsetUx = std::numeric_limits<int>::min();
        int offsetUy = std::numeric_limits<int>::min();

        int offsetCx_ = 0;
        int offsetCy_ = 0;

        for (odb::dbMPin* mPin : iTerm->getMTerm()->getMPins()) {
          for (odb::dbBox* box : mPin->getGeometry()) {
            offsetLx = std::min(box->xMin(), offsetLx);
            offsetLy = std::min(box->yMin(), offsetLy);
            offsetUx = std::max(box->xMax(), offsetUx);
            offsetUy = std::max(box->yMax(), offsetUy);
          }
        }

        int lx = iTerm->getInst()->getBBox()->xMin();
        int ly = iTerm->getInst()->getBBox()->yMin();

        int instCenterX = iTerm->getInst()->getMaster()->getWidth() / 2;
        int instCenterY = iTerm->getInst()->getMaster()->getHeight() / 2;

        // Pin SHAPE is NOT FOUND;
        // (may happen on OpenDB bug case)
        if (offsetLx == INT_MAX || offsetLy == INT_MAX || offsetUx == INT_MIN
            || offsetUy == INT_MIN) {
          // offset is center of instances
          offsetCx_ = offsetCy_ = 0;
        } else {
          // offset is Pin BBoxs' center, so
          // subtract the Origin coordinates (e.g. instCenterX, instCenterY)
          //
          // Transform coordinates
          // from (origin: 0,0)
          // to (origin: instCenterX, instCenterY)
          //
          offsetCx_ = (offsetLx + offsetUx) / 2 - instCenterX;
          offsetCy_ = (offsetLy + offsetUy) / 2 - instCenterY;
        }

        int cx = lx + instCenterX + offsetCx_;
        int cy = ly + instCenterY + offsetCy_;

        minX = std::min(minX, cx);
        minY = std::min(minY, cy);
        maxX = std::max(maxX, cx);
        maxY = std::max(maxY, cy);
      }

      for (auto bTerm : net->getBTerms()) {
        int lx = std::numeric_limits<int>::max();
        int ly = std::numeric_limits<int>::max();
        int ux = std::numeric_limits<int>::min();
        int uy = std::numeric_limits<int>::min();

        for (odb::dbBPin* bPin : bTerm->getBPins()) {
          odb::Rect bbox = bPin->getBBox();
          lx = std::min(bbox.xMin(), lx);
          ly = std::min(bbox.yMin(), ly);
          ux = std::max(bbox.xMax(), ux);
          uy = std::max(bbox.yMax(), uy);
        }

        int cx = (lx + ux) / 2;
        int cy = (ly + uy) / 2;

        minX = std::min(minX, cx);
        minY = std::min(minY, cy);
        maxX = std::max(maxX, cx);
        maxY = std::max(maxY, cy);
      }
      hpwl += (maxX - minX) + (maxY - minY);
    }
  }
  std::cout << "[Total HPWL] HPWL = " << hpwl << std::endl;
}

// In this mode, we assume there is no fixed instances
// This is for mixed-size placement
void PlacerBaseCommon::initClusterNetlist()
{
  odb::dbBlock* block = db_->getChip()->getBlock();
  // get the site information
  odb::dbSite* site = nullptr;
  for (auto* row : block->getRows()) {
    if (row->getSite()->getClass() != odb::dbSiteClass::PAD) {
      site = row->getSite();
      break;
    }
  }

  // siteSize update
  siteSizeX_ = site->getWidth();
  siteSizeY_ = site->getHeight();

  // get the die information
  odb::Rect coreRect = block->getCoreArea();
  odb::Rect dieRect = block->getDieArea();
  die_ = Die(dieRect, coreRect);

  // Reorder the cluster id
  int numClusters = 0;
  std::map<int, int> clusterIdReorderMap;

  // insts fill with real instances
  // update the clusters
  odb::dbSet<odb::dbInst> insts = block->getInsts();
  for (odb::dbInst* inst : insts) {
    auto type = inst->getMaster()->getType();
    if (!type.isCore() && !type.isBlock()) {
      continue;
    }
    // check if the instance has a cluster id attribute
    auto clusterIdProp = odb::dbIntProperty::find(inst, "cluster_id");
    if (clusterIdProp != nullptr) {
      const int clusterId = clusterIdProp->getValue();
      if (clusterIdReorderMap.find(clusterId) == clusterIdReorderMap.end()) {
        clusterIdReorderMap[clusterId] = numClusters++;
      }
      odb::dbIntProperty::find(inst, "cluster_id")
          ->setValue(clusterIdReorderMap[clusterId]);
    }
  }

  std::cout << "[INFO] Number of clusters = " << numClusters << std::endl;
  std::vector<int64_t> clustersArea(numClusters, 0);
  for (odb::dbInst* inst : insts) {
    auto type = inst->getMaster()->getType();
    if (!type.isCore() && !type.isBlock()) {
      continue;
    }
    odb::dbBox* bbox = inst->getBBox();
    int haloWidth = 0;
    if (inst->getMaster()->isBlock()) {
      haloWidth = haloWidth_;
    }
    const int64_t instArea
        = static_cast<int64_t>(bbox->getDX() + haloWidth * 2)
          * static_cast<int64_t>(bbox->getDY() + haloWidth * 2);
    // check if the instance has a cluster id attribute
    auto clusterIdProp = odb::dbIntProperty::find(inst, "cluster_id");
    if (clusterIdProp != nullptr) {
      const int clusterId = clusterIdProp->getValue();
      clustersArea[clusterId] += instArea;
    }
  }

  // We need to bloat the area of clusters to speed the clusters
  int64_t coreArea = die_.coreArea();
  int64_t sumArea = std::accumulate(
      clustersArea.begin(), clustersArea.end(), static_cast<int64_t>(0));

  // In the default node (bloatFactor_ == 1.0), we bloat the clusters to fill
  // the entire core area
  std::cout << "[INFO] bloatFactor = " << bloatFactor_ << std::endl;
  float bloatFactor = bloatFactor_;
  if (bloatFactor >= 0.99) {
    bloatFactor = static_cast<float>(coreArea) / static_cast<float>(sumArea);
    std::cout << "[INFO] Reset the bloatFactor to " << bloatFactor << std::endl;
  }

  for (auto& area : clustersArea) {
    area = area * bloatFactor;
  }

  // create fake instances
  numInsts_ = numClusters;
  numPlaceInsts_ = numClusters;
  numFixedInsts_ = numInsts_ - numPlaceInsts_;
  // allocate the objects on host side
  instStor_.resize(numInsts_);
  // different to original RePlAce codes,
  // we assign inst_id property to each inst
  // So we do not use inst_map_
  int placeInstId = -1;
  int fixedInstId = -1;
  const float cluster_ratio
      = 1.0;  // We assume the aspect ratio of the cluster is 1.0
  for (int clusterId = 0; clusterId < numClusters; clusterId++) {
    // create fake instances
    placeInstId++;
    const int64_t height = std::sqrt(clustersArea[clusterId] * cluster_ratio);
    const int64_t width = clustersArea[clusterId] / height;
    Instance myInst(0, 0, width, height, false);  // just a small instance
    myInst.setMacro();
    myInst.setInstId(placeInstId);
    instStor_[placeInstId] = myInst;
    const int64_t instArea = myInst.area();
    if (myInst.isMacro()) {
      macroInstsArea_ += instArea;
    } else {
      stdCellInstsArea_ += instArea;
    }
    placeInstsArea_ += instArea;
  }

  // create pointers on for the host objects
  for (auto& inst : instStor_) {
    insts_.push_back(&inst);
    if (!inst.isFixed()) {
      placeInsts_.push_back(&inst);
    }
  }

  // store the connections between clusters
  std::map<int, std::map<int, int>> adjMatrix;
  // nets fill
  odb::dbSet<odb::dbNet> nets = block->getNets();
  // TODO: I obersve that if the reserve size is not large enough, the program
  // will crash
  netStor_.reserve(nets.size() * 10);  // Here we do not use resize
  pinStor_.reserve(nets.size() * 10);  // average degree is around 3
  int netId = -1;
  int pinId = -1;
  for (odb::dbNet* net : nets) {
    odb::dbSigType netType = net->getSigType();
    odb::dbIntProperty::create(net, "netId", -1);

    // escape nets with VDD/VSS/reset nets
    if (netType == odb::dbSigType::SIGNAL || netType == odb::dbSigType::CLOCK) {
      // check number of clusters connected to this net
      std::set<int> iTermClusters;
      // check all the instance pins connected to this net
      for (odb::dbITerm* iTerm : net->getITerms()) {
        // map the pin to its inst
        const int clusterId
            = odb::dbIntProperty::find(iTerm->getInst(), "cluster_id")
                  ->getValue();
        iTermClusters.insert(clusterId);
      }

      if (iTermClusters.size() + net->getBTerms().size() <= 1) {
        continue;
      }

      // ignore high-fanout nets to avoid divergence
      if (iTermClusters.size() > 100) {
        continue;
      }

      // create fake pins for clusters
      netId++;
      odb::dbIntProperty::find(net, "netId")->setValue(netId);
      Net myNet(netId);
      myNet.setWeight(1.0);
      myNet.setVirtualWeight(0.0);
      netStor_.push_back(myNet);

      // create fake pins for clusters
      // check all the instance pins connected to this net
      for (auto& clusterId : iTermClusters) {
        pinId++;
        Pin myPin(pinId);
        myPin.setNet(&netStor_.back());
        myPin.setInstance(&instStor_[clusterId]);
        pinStor_.push_back(myPin);
      }

      for (odb::dbBTerm* bTerm : net->getBTerms()) {
        pinId++;
        odb::dbIntProperty::create(bTerm, "pinId", pinId);
        Pin myPin(bTerm, log_);
        // link the pin with the net
        myPin.setNet(&netStor_.back());
        pinStor_.push_back(myPin);
      }
    }
  }

  std::map<int, odb::dbBTerm*> btermClusterMap;
  int btermClusterId = numClusters;
  for (odb::dbBTerm* bterm : block->getBTerms()) {
    btermClusterMap[btermClusterId++] = bterm;
  }

  for (int clusterId = 0; clusterId < adjMatrix_.size(); clusterId++) {
    for (auto& adj : adjMatrix_[clusterId]) {
      if (clusterId >= numClusters) {
        // check if pinId exists
        if (odb::dbIntProperty::find(btermClusterMap[clusterId], "pinId")
            == nullptr) {
          continue;
        }
      }

      if (adj.first >= numClusters) {
        // check if pinId exists
        if (odb::dbIntProperty::find(btermClusterMap[adj.first], "pinId")
            == nullptr) {
          continue;
        }
      }

      netId++;
      Net myNet(netId);
      myNet.setWeight(adj.second);
      myNet.setVirtualWeight(0.0);
      netStor_.push_back(myNet);

      if (clusterId >= numClusters) {
        pinId++;
        Pin myPin(btermClusterMap[clusterId], log_);
        // link the pin with the net
        myPin.setNet(&netStor_.back());
        pinStor_.push_back(myPin);
      } else {
        pinId++;
        Pin myPin(pinId);
        myPin.setNet(&netStor_.back());
        myPin.setInstance(&instStor_[clusterId]);
        pinStor_.push_back(myPin);
      }

      if (adj.first >= numClusters) {
        pinId++;
        Pin myPin(btermClusterMap[adj.first], log_);
        // link the pin with the net
        myPin.setNet(&netStor_.back());
        pinStor_.push_back(myPin);
      } else {
        pinId++;
        Pin myPin(pinId);
        myPin.setNet(&netStor_.back());
        myPin.setInstance(&instStor_[adj.first]);
        pinStor_.push_back(myPin);
      }
    }
  }

  for (auto& net : netStor_) {
    nets_.push_back(&net);
  }

  for (auto& pin : pinStor_) {
    pins_.push_back(&pin);
    if (pin.net() != nullptr) {
      pin.net()->addPin(&pin);
    }
    if (pin.isITerm()) {
      pin.instance()->addPin(&pin);
    }
  }

  // Initialize the virtual weight related variables
  initVirtualWeightFactor_ = 0.00;
  virtualWeightFactor_ = 0.00;

  // print the statistics
  printInfo();
}

void splitString(std::string& inputString)
{
  if (inputString.back() != '_') {
    return;
  }

  // Create a stringstream from the input string
  std::istringstream ss(inputString);
  // Create a vector to store the split parts
  std::vector<std::string> parts;
  std::string part;
  while (std::getline(ss, part, '_')) {
    // Add each part to the vector
    parts.push_back(part);
  }

  if (parts.size() == 1) {
    return;
  }

  inputString.erase(inputString.length() - parts.back().length() - 2);
}

size_t customHash(const std::string& input)
{
  size_t hash = 0;
  for (char ch : input) {
    // Multiply the current hash value by a prime number
    // and add the character's ASCII value
    hash = hash * 11 + static_cast<size_t>(ch);
  }
  return hash;
}

bool checkSDFF(std::string input)
{
  if (input.at(0) == 'S') {
    return true;
  }

  return false;
}

void addVirtualConnection(std::map<int, float>& adjList,
                          const int clusterId,
                          const int seqVertexId,
                          const int level,
                          const int numHops,
                          const int bitWidth,
                          std::map<int, odb::dbBTerm*>& ioPinVertex,
                          std::map<int, odb::dbInst*>& instVertex,
                          const std::vector<Vertex>& seqVertices)
{
  std::set<int> sinkClusters;
  std::set<int>
      sinkSeqVertexIds;  // We need to further traverse the sequential graph
  for (auto& sink : seqVertices[seqVertexId].sinks) {
    if (sink.first >= ioPinVertex.size()) {
      // check if we should stop here
      if (odb::dbIntProperty::find(instVertex[sink.first], "cluster_id")
          == nullptr) {
        continue;
      }

      const int sinkClusterId
          = odb::dbIntProperty::find(instVertex[sink.first], "cluster_id")
                ->getValue();
      auto prop = odb::dbIntProperty::find(instVertex[sink.first], "dVertexId");
      if (sinkClusterId != clusterId && prop != nullptr) {
        sinkClusters.insert(sinkClusterId);
      } else {
        auto seqProp
            = odb::dbIntProperty::find(instVertex[sink.first], "seqVertexId");
        if (seqProp != nullptr) {
          const int sinkSeqVertexId = seqProp->getValue();
          sinkSeqVertexIds.insert(sinkSeqVertexId);
        } else {
          std::cout << "instName = " << instVertex[sink.first]->getName()
                    << std::endl;
        }
      }
    } else {
      auto prop
          = odb::dbIntProperty::find(ioPinVertex[sink.first], "dVertexId");
      if (prop != nullptr) {
        const int sinkClusterId
            = odb::dbIntProperty::find(ioPinVertex[sink.first], "cluster_id")
                  ->getValue();
        sinkClusters.insert(sinkClusterId);
      }
    }
  }

  for (auto& sinkClusterId : sinkClusters) {
    if (adjList.find(sinkClusterId) == adjList.end()) {
      adjList[sinkClusterId] = bitWidth / std::pow(2.0, level);
    } else {
      adjList[sinkClusterId] += bitWidth / std::pow(2.0, level);
    }
  }

  if (level < numHops) {
    for (auto& sink : sinkSeqVertexIds) {
      addVirtualConnection(adjList,
                           clusterId,
                           sink,
                           level + 1,
                           numHops,
                           bitWidth,
                           ioPinVertex,
                           instVertex,
                           seqVertices);
    }
  }
}

void addDataflowEdge(DVertex& dVertex,
                     const int seqVertexId,
                     const int maxDist,
                     std::map<int, odb::dbBTerm*>& ioPinVertex,
                     std::map<int, odb::dbInst*>& instVertex,
                     const std::vector<Vertex>& seqVertices)
{
  for (auto& sink : seqVertices[seqVertexId].sinks) {
    const float weight
        = static_cast<float>(sink.second) / static_cast<float>(maxDist);
    int sinkDVertexId = -1;
    if (sink.first >= ioPinVertex.size()) {
      auto prop = odb::dbIntProperty::find(instVertex[sink.first], "dVertexId");
      if (prop == nullptr) {
        std::cout << "instName = " << instVertex[sink.first]->getName()
                  << std::endl;
        continue;
      } else {
        sinkDVertexId = prop->getValue();
      }
    } else {
      auto prop
          = odb::dbIntProperty::find(ioPinVertex[sink.first], "dVertexId");
      if (prop == nullptr) {
        std::cout << "ioName = " << ioPinVertex[sink.first]->getName()
                  << std::endl;
        continue;
      } else {
        sinkDVertexId = prop->getValue();
      }
    }
    dVertex.addSink(sinkDVertexId, weight);
  }
}

// Create Dataflow Information
// model each std cell instance, IO pin and macro pin as vertices
void PlacerBaseCommon::createDataFlow()
{
  // if (datapathFlag_ == false && dataflowFlag_ == false) {
  //  return;
  //}

  std::map<int64_t, int> instMap;
  std::map<int64_t, int> multiFFMap;
  int numMacros = 0;

  // assign vertex_id property of each instance
  for (auto inst : block_->getInsts()) {
    odb::dbMaster* master = inst->getMaster();
    // check if the instance is a Pad, Cover or a block
    // We ignore nets connecting Pads, Covers
    // for blocks, we iterate over the block pins
    if (master->isPad() || master->isCover()) {
      continue;
    }

    if (master->isBlock()) {
      numMacros++;
      continue;
    }

    const sta::LibertyCell* libertyCell = network_->libertyCell(inst);
    if (libertyCell == nullptr) {
      continue;
    }

    if (!libertyCell->hasSequentials()) {
      continue;  // ignore combinational cell
    }

    const std::string masterName = master->getName();
    if (checkSDFF(masterName)) {
      continue;  // ignore scan FF
    }

    std::string instName = inst->getName();
    splitString(instName);

    const size_t instHash = customHash(instName);
    if (instMap.find(instHash) == instMap.end()) {
      instMap[instHash] = 1;
    } else {
      instMap[instHash] += 1;
    }
  }

  int maxFFBits = 0;
  int64_t hashId = 0;
  for (auto& inst : instMap) {
    if (inst.second >= busLimit_) {
      multiFFMap[inst.first] = inst.second;
      if (maxFFBits < inst.second) {
        hashId = inst.first;
        maxFFBits = inst.second;
      }
    }
  }

  if (maxFFBits == 0) {
    log_->report("[INFO] This is no multi-bit Flip Flops detected");
    dataflowVertices_.clear();
    return;
  }

  // create the data flow vertices
  dataflowVertices_.resize(multiFFMap.size() + numMacros
                           + block_->getBTerms().size());
  int dVertexId = 0;
  std::map<int64_t, int> multiFFDVertexIdMap;
  for (auto& inst : multiFFMap) {
    dataflowVertices_[dVertexId] = DVertex(dVertexId);
    multiFFDVertexIdMap[inst.first]
        = dVertexId;  // map from hash id to dVertexId
    dVertexId++;
  }

  for (auto inst : block_->getInsts()) {
    odb::dbMaster* master = inst->getMaster();
    // check if the instance is a Pad, Cover or a block
    // We ignore nets connecting Pads, Covers
    // for blocks, we iterate over the block pins
    if (master->isPad() || master->isCover()) {
      continue;
    }

    if (master->isBlock()) {
      dataflowVertices_[dVertexId] = DVertex(dVertexId, inst);
      auto prop = odb::dbIntProperty::find(inst, "dVertexId");
      if (prop == nullptr) {
        odb::dbIntProperty::create(inst, "dVertexId", dVertexId);
      }
      dVertexId++;
      continue;
    }

    const sta::LibertyCell* libertyCell = network_->libertyCell(inst);
    if (libertyCell == nullptr) {
      continue;
    }

    if (!libertyCell->hasSequentials()) {
      continue;  // ignore combinational cell
    }

    const std::string masterName = master->getName();
    if (checkSDFF(masterName)) {
      continue;  // ignore scan FF
    }

    std::string instName = inst->getName();
    splitString(instName);

    const size_t instHash = customHash(instName);
    if (multiFFMap.find(instHash) == multiFFMap.end()) {
      continue;
    }

    const int instDVertexId = multiFFDVertexIdMap[instHash];
    dataflowVertices_[instDVertexId].addInst(inst);
    auto prop = odb::dbIntProperty::find(inst, "dVertexId");
    if (prop == nullptr) {
      odb::dbIntProperty::create(inst, "dVertexId", instDVertexId);
    }
  }

  for (auto bterm : block_->getBTerms()) {
    dataflowVertices_[dVertexId] = DVertex(dVertexId, bterm);
    auto prop = odb::dbIntProperty::find(bterm, "dVertexId");
    if (prop == nullptr) {
      odb::dbIntProperty::create(bterm, "dVertexId", dVertexId);
    }
    dVertexId++;
  }

  // create sequential graph
  std::map<int, odb::dbBTerm*> ioPinVertex;
  std::map<int, odb::dbInst*> instVertex;
  std::vector<Vertex> seqVertices;
  // create the original netlist
  std::vector<std::vector<int>> vertices;
  std::vector<std::vector<int>> sinkHyperedges;  // dircted hypergraph

  createSeqGraph(
      ioPinVertex, instVertex, seqVertices, vertices, sinkHyperedges);

  // for debug, print the statistics
  // print the average fannots
  int64_t totalFanouts = 0;
  int maxDist = 0;
  for (auto& vertex : seqVertices) {
    totalFanouts += vertex.sinks.size();
    for (auto& sink : vertex.sinks) {
      maxDist = max(maxDist, sink.second);
    }
  }

  // Reorder the cluster id
  int numClusters = 0;
  std::map<int, int> clusterIdReorderMap;

  // insts fill with real instances
  // update the clusters
  odb::dbSet<odb::dbInst> insts = block_->getInsts();
  for (odb::dbInst* inst : insts) {
    auto type = inst->getMaster()->getType();
    if (!type.isCore() && !type.isBlock()) {
      continue;
    }
    // check if the instance has a cluster id attribute
    auto clusterIdProp = odb::dbIntProperty::find(inst, "cluster_id");
    if (clusterIdProp != nullptr) {
      const int clusterId = clusterIdProp->getValue();
      if (clusterIdReorderMap.find(clusterId) == clusterIdReorderMap.end()) {
        clusterIdReorderMap[clusterId] = numClusters++;
      }
      odb::dbIntProperty::find(inst, "cluster_id")
          ->setValue(clusterIdReorderMap[clusterId]);
    }
  }

  // create clusterId on each IO pin
  // clear the property
  for (odb::dbBTerm* bterm : block_->getBTerms()) {
    odb::dbIntProperty* prop = odb::dbIntProperty::find(bterm, "cluster_id");
    if (prop
        != nullptr) {  // Ensure the property exists before trying to delete it
      odb::dbIntProperty::find(bterm, "cluster_id")->setValue(numClusters++);
    } else {
      odb::dbIntProperty::create(bterm, "cluster_id", numClusters++);
    }
  }

  if (numClusters > 0 && dataflowFlag_ == true) {
    // We need to determine the connections between clusters
    // We have number of clusters from 0, ..., numClusters_ - 1 (really clusters
    // + IOs)
    adjMatrix_.resize(numClusters);
    // A breath-first search at G_{seq} starts simultaneously from all
    // components of block i traversing only outgoing edges through glue logic.
    // When a component of block j is reached, the bitwidth of its predecessor
    // in the path is added to the bin corresponding to the number of flops
    // stage create the virtaul nodes for each multiFF instance
    for (auto& dVertex : dataflowVertices_) {
      if (dVertex.isBTerm()) {
        const int seqVertexId
            = odb::dbIntProperty::find(dVertex.getBTerm(), "seqVertexId")
                  ->getValue();
        const int clusterId
            = odb::dbIntProperty::find(dVertex.getBTerm(), "cluster_id")
                  ->getValue();
        addVirtualConnection(adjMatrix_[clusterId],
                             clusterId,
                             seqVertexId,
                             0,
                             numHops_,
                             1,
                             ioPinVertex,
                             instVertex,
                             seqVertices);
      } else if (dVertex.isMacro()) {
        const int seqVertexId
            = odb::dbIntProperty::find(dVertex.getMacro(), "seqVertexId")
                  ->getValue();
        const int clusterId
            = odb::dbIntProperty::find(dVertex.getMacro(), "cluster_id")
                  ->getValue();
        int bitWidth = 0;
        for (odb::dbITerm* pin : dVertex.getMacro()->getITerms()) {
          if (pin->getSigType() != odb::dbSigType::SIGNAL
              || pin->getIoType() != odb::dbIoType::OUTPUT) {
            continue;
          }
          bitWidth++;
        }
        addVirtualConnection(adjMatrix_[clusterId],
                             clusterId,
                             seqVertexId,
                             0,
                             numHops_,
                             bitWidth,
                             ioPinVertex,
                             instVertex,
                             seqVertices);
      } else {
        for (auto& inst : dVertex.getFFs()) {
          const int seqVertexId
              = odb::dbIntProperty::find(inst, "seqVertexId")->getValue();
          const int clusterId
              = odb::dbIntProperty::find(inst, "cluster_id")->getValue();
          addVirtualConnection(adjMatrix_[clusterId],
                               clusterId,
                               seqVertexId,
                               0,
                               numHops_,
                               1,
                               ioPinVertex,
                               instVertex,
                               seqVertices);
        }
      }
    }
  }

  if (datapathFlag_ == true) {
    // create the virtaul nodes for each multiFF instance
    for (auto& dVertex : dataflowVertices_) {
      if (dVertex.isBTerm()) {
        const int seqVertexId
            = odb::dbIntProperty::find(dVertex.getBTerm(), "seqVertexId")
                  ->getValue();
        addDataflowEdge(dVertex,
                        seqVertexId,
                        maxDist,
                        ioPinVertex,
                        instVertex,
                        seqVertices);
      } else if (dVertex.isMacro()) {
        const int seqVertexId
            = odb::dbIntProperty::find(dVertex.getMacro(), "seqVertexId")
                  ->getValue();
        addDataflowEdge(dVertex,
                        seqVertexId,
                        maxDist,
                        ioPinVertex,
                        instVertex,
                        seqVertices);
      } else {
        for (auto& inst : dVertex.getFFs()) {
          const int seqVertexId
              = odb::dbIntProperty::find(inst, "seqVertexId")->getValue();
          addDataflowEdge(dVertex,
                          seqVertexId,
                          maxDist,
                          ioPinVertex,
                          instVertex,
                          seqVertices);
        }
      }
    }
  }
}

// Extract the sequence graph from the original netlist
// sequence graph consists of FFs, macros and IOs, we do not consider SDFFs
// Edges between sequential components are inferred by
// analyzing their transitive fanin/fanout in the original netlist

void buildSeqGraphFromVertex(
    Vertex& vertex,
    const int vertexId,
    int step,
    const std::vector<std::vector<int>>& vertices,
    const std::vector<std::vector<int>>& sinkHyperedges,
    const std::vector<bool>& stopFlagVec,
    std::unordered_set<int>& visited)
{
  if (stopFlagVec[vertexId] == true && vertex.src != vertexId) {
    vertex.addSink(vertexId, step);
    return;
  }

  visited.insert(vertexId);
  for (auto hyperedgeId : vertices[vertexId]) {
    for (auto& sink : sinkHyperedges[hyperedgeId]) {
      if (visited.find(sink) != visited.end()) {
        continue;  // This sink has been visited
      }
      buildSeqGraphFromVertex(vertex,
                              sink,
                              step + 1,
                              vertices,
                              sinkHyperedges,
                              stopFlagVec,
                              visited);
    }
  }
}

void PlacerBaseCommon::clearPinProperty()
{
  for (odb::dbBTerm* bterm : block_->getBTerms()) {
    odb::dbProperty::destroyProperties(bterm);
  }
}

void PlacerBaseCommon::clearInstProperty()
{
  for (auto inst : block_->getInsts()) {
    odb::dbProperty::destroyProperties(inst);
  }
}

void PlacerBaseCommon::createSeqGraph(
    std::map<int, odb::dbBTerm*>& ioPinVertex,
    std::map<int, odb::dbInst*>& instVertex,
    std::vector<Vertex>& seqVertices,
    // create the original netlist, directed hypergraph
    std::vector<std::vector<int>>& vertices,
    std::vector<std::vector<int>>& sinkHyperedges)
{
  std::vector<bool> stopFlagVec;  // stop at IO pins, FFs and macros
  int vertexId = 0;
  // assign vertexId property of each Bterm
  // All boundary terms are marked as sequential stopping points
  for (odb::dbBTerm* term : block_->getBTerms()) {
    auto prop1 = odb::dbIntProperty::find(term, "vertexId");
    if (prop1 == nullptr) {
      odb::dbIntProperty::create(term, "vertexId", vertexId);
    }
    ioPinVertex[vertexId] = term;
    vertexId++;
    stopFlagVec.push_back(true);
    auto prop2 = odb::dbIntProperty::find(term, "seqVertexId");
    if (prop2 == nullptr) {
      odb::dbIntProperty::create(term, "seqVertexId", seqVertices.size());
    }
    seqVertices.emplace_back(vertexId, true);  // true means it is an IO pin
  }

  // assign vertexId property of each instance
  for (auto inst : block_->getInsts()) {
    odb::dbMaster* master = inst->getMaster();
    // check if the instance is a Pad, Cover or a block
    // We ignore nets connecting Pads, Covers
    // for blocks, we iterate over the block pins
    if (master->isPad() || master->isCover()) {
      continue;
    }

    const sta::LibertyCell* libertyCell = network_->libertyCell(inst);
    if (libertyCell == nullptr) {
      continue;
    }

    // mark sequential instances
    if (master->isBlock()
        || (libertyCell->hasSequentials() && !checkSDFF(master->getName()))) {
      auto prop1 = odb::dbIntProperty::find(inst, "vertexId");
      if (prop1 == nullptr) {
        odb::dbIntProperty::create(inst, "vertexId", vertexId);
      }
      auto prop2 = odb::dbIntProperty::find(inst, "seqVertexId");
      if (prop2 == nullptr) {
        odb::dbIntProperty::create(inst, "seqVertexId", seqVertices.size());
      }
      instVertex[vertexId] = inst;
      stopFlagVec.push_back(true);  // Sequential cells
      seqVertices.emplace_back(vertexId,
                               false);  // false means it is not an IO pin
      vertexId++;
    } else if (!libertyCell->hasSequentials()) {
      auto prop1 = odb::dbIntProperty::find(inst, "vertexId");
      if (prop1 == nullptr) {
        odb::dbIntProperty::create(inst, "vertexId", vertexId);
      }
      instVertex[vertexId] = inst;
      stopFlagVec.push_back(false);  // Comb cells
      vertexId++;
    } else {
      auto prop1 = odb::dbIntProperty::find(inst, "vertexId");
      if (prop1 == nullptr) {
        odb::dbIntProperty::create(inst, "vertexId", -1);
      }
    }
  }

  // create the original netlist
  vertices.resize(stopFlagVec.size());
  // traverse the netlist
  for (odb::dbNet* net : block_->getNets()) {
    // ignore all the power net
    if (net->getSigType().isSupply()
        || net->getITerms().size() >= largeNetThreshold_) {
      continue;
    }

    int driverId = -1;      // driver vertex id
    std::set<int> loadsId;  // load vertex id
    bool padSDFFFlag = false;

    // check the connected instances
    for (odb::dbITerm* iterm : net->getITerms()) {
      odb::dbInst* inst = iterm->getInst();
      odb::dbMaster* master = inst->getMaster();
      // check if the instance is a Pad, Cover or empty block (such as marker)
      // We ignore nets connecting Pads, Covers, or markers
      if (master->isPad() || master->isCover()) {
        padSDFFFlag = true;
        break;
      }

      const int vertexId
          = odb::dbIntProperty::find(inst, "vertexId")->getValue();
      if (vertexId == -1) {
        continue;
      }

      if (iterm->getIoType() == odb::dbIoType::OUTPUT) {
        driverId = vertexId;
      } else {
        loadsId.insert(vertexId);
      }
    }

    if (padSDFFFlag) {
      continue;  // the nets with Pads should be ignored
    }

    // check the connected IO pins  of the net
    for (odb::dbBTerm* bterm : net->getBTerms()) {
      const int vertexId
          = odb::dbIntProperty::find(bterm, "vertexId")->getValue();
      if (bterm->getIoType() == odb::dbIoType::INPUT) {
        driverId = vertexId;
      } else {
        loadsId.insert(vertexId);
      }
    }

    //
    // Skip high fanout nets or nets that do not have valid driver or loads
    //
    if (driverId < 0 || loadsId.size() < 1
        || loadsId.size() > largeNetThreshold_) {
      continue;
    }

    // Create the hyperedge
    std::vector<int> hyperedge;
    for (auto& load : loadsId) {
      if (load != driverId) {
        hyperedge.push_back(load);
      }
    }

    vertices[driverId].push_back(sinkHyperedges.size());
    sinkHyperedges.push_back(hyperedge);
  }  // end net traversal

  std::unordered_set<int> visited;
  // Build the sequence graph
  for (auto& vertex : seqVertices) {
    visited.clear();
    buildSeqGraphFromVertex(
        vertex, vertex.src, 0, vertices, sinkHyperedges, stopFlagVec, visited);
  }
  visited.clear();
}

// ---------------------------------------------------------------------------------------------------------------------
// for standard cell placement
// ---------------------------------------------------------------------------------------------------------------------

void PlacerBaseCommon::init()
{
  odb::dbBlock* block = db_->getChip()->getBlock();
  // get the site information
  odb::dbSite* site = nullptr;
  for (auto* row : block->getRows()) {
    if (row->getSite()->getClass() != odb::dbSiteClass::PAD) {
      site = row->getSite();
      break;
    }
  }

  // siteSize update
  siteSizeX_ = site->getWidth();
  siteSizeY_ = site->getHeight();

  // get the die information
  odb::Rect coreRect = block->getCoreArea();
  odb::Rect dieRect = block->getDieArea();
  die_ = Die(dieRect, coreRect);

  // cluster constraint
  // Reorder the cluster id
  int numClusters = 0;
  numPlaceInsts_ = 0;
  numInsts_ = 0;
  // insts fill with real instances
  odb::dbSet<odb::dbInst> insts = block->getInsts();
  for (odb::dbInst* inst : insts) {
    auto type = inst->getMaster()->getType();
    if (!type.isCore() && !type.isBlock()) {
      continue;
    }
    numInsts_++;
    if (!isFixedOdbInst(inst)) {
      // call the utility function to check if the instance is a fixed instance
      numPlaceInsts_++;
    }

    // check if the instance has a cluster id attribute
    auto clusterIdProp = odb::dbIntProperty::find(inst, "cluster_id");
    if (clusterIdProp != nullptr) {
      const int clusterId = clusterIdProp->getValue();
      numClusters = max(numClusters, clusterId + 1);
    }
  }

  if (clusterConstraintFlag_ == false) {
    numClusters = 0;
  }

  // datapath constraint
  // Number of multi-bit FFs
  int numMultiFFs = 0;
  for (auto& vertex : dataflowVertices_) {
    if (vertex.getFFs().size() > 1) {
      numMultiFFs++;
    }
  }

  if (datapathFlag_ == false) {
    numMultiFFs = 0;
    dataflowVertices_.clear();
  }

  std::map<int, std::pair<int, int>> clusterLoc;
  const int numRealPlaceInsts = numPlaceInsts_;
  numInsts_ += numClusters + numMultiFFs;
  numPlaceInsts_ += numClusters + numMultiFFs;
  numFixedInsts_ = numInsts_ - numPlaceInsts_;

  // allocate the objects on host side
  instStor_.resize(numInsts_);
  dbInstStor_.resize(numInsts_);
  // different to original RePlAce codes,
  // we assign inst_id property to each inst
  // So we do not use inst_map_
  int placeInstId = -1;
  int fixedInstId = -1;
  for (odb::dbInst* inst : insts) {
    // check the instance id
    odb::dbIntProperty::create(inst, "instId", -1);
    auto type = inst->getMaster()->getType();
    if (!type.isCore() && !type.isBlock()) {
      continue;
    }
    int instId = -1;
    // check if the instance is a fixed instance
    if (isFixedOdbInst(inst)) {
      fixedInstId++;
      instId = numPlaceInsts_ + fixedInstId;
    } else {
      placeInstId++;
      instId = placeInstId;
    }
    odb::dbIntProperty::find(inst, "instId")->setValue(instId);
    // create the GpuInstance
    Instance myInst(inst,
                    pbVars_.padLeft * siteSizeX_,
                    pbVars_.padRight * siteSizeX_,
                    siteSizeY_,
                    pbVars_.row_limit,
                    log_);
    if (type.isBlock()) {
      myInst.setHaloWidth(haloWidth_);
    }
    // Fixed instaces need to be snapped outwards to the nearest site
    // boundary.  A partially overlapped site is unusable and this
    // is the simplest way to ensure it is counted as fully used.
    if (myInst.isFixed()) {
      myInst.snapOutward(coreRect.ll(), siteSizeX_, siteSizeY_);
    }
    dbInstStor_[instId] = inst;
    instStor_[instId] = myInst;

    // for clustered netlist
    auto prop = odb::dbIntProperty::find(inst, "cluster_id");
    if (prop != nullptr) {
      const int clusterId = prop->getValue();
      if (clusterLoc.find(clusterId) == clusterLoc.end()) {
        clusterLoc[clusterId] = std::make_pair(myInst.cx(), myInst.cy());
      }
    }

    const int64_t instArea = myInst.area();
    if (myInst.isMacro()) {
      macroInstsArea_ += instArea;
    } else {
      stdCellInstsArea_ += instArea;
    }

    if (myInst.isFixed()) {
      nonPlaceInstsArea_ += instArea;
    } else {
      placeInstsArea_ += instArea;
    }
  }

  // for clustered netlist
  // create virtual instances for each cluster
  // 2729, 2800 is the size of a dummy instance
  for (int clusterId = 0; clusterId < numClusters; clusterId++) {
    placeInstId++;
    auto& loc = clusterLoc[clusterId];
    Instance myInst(
        loc.first, loc.second, 2729, 2800, false);  // just a small instance
    dbInstStor_[placeInstId] = nullptr;
    instStor_[placeInstId] = myInst;
  }

  // for multi-bit FFs
  for (auto& vertex : dataflowVertices_) {
    if (vertex.getFFs().size() > 1) {
      placeInstId++;
      int64_t cx = 0;
      int64_t cy = 0;
      for (auto& inst : vertex.getFFs()) {
        int lx = 0;
        int ly = 0;
        inst->getLocation(lx, ly);
        lx += floor(inst->getBBox()->getDX() / 2);
        ly += floor(inst->getBBox()->getDY() / 2);
        cx += lx;
        cy += ly;
      }
      cx = static_cast<float>(cx) / vertex.getFFs().size();
      cy = static_cast<float>(cy) / vertex.getFFs().size();
      Instance myInst(cx, cy, 2729, 2800, false);  // just a small instance
      vertex.instId = placeInstId;
      dbInstStor_[placeInstId] = nullptr;
      instStor_[placeInstId] = myInst;
    }
  }

  // create pointers on for the host objects
  for (auto& inst : instStor_) {
    insts_.push_back(&inst);
    if (!inst.isFixed()) {
      placeInsts_.push_back(&inst);
    }
  }

  // nets fill
  odb::dbSet<odb::dbNet> nets = block->getNets();
  // TODO: I obersve that if the reserve size is not large enough, the program
  // will crash
  netStor_.reserve(nets.size() * 100);  // Here we do not use resize
  dbNetStor_.reserve(nets.size() * 100);
  pinStor_.reserve(nets.size() * 100);  // average degree is around 3
  dbPinStor_.reserve(nets.size() * 100);
  int netId = -1;
  int pinId = -1;
  int ignoreNet = 0;
  for (odb::dbNet* net : nets) {
    odb::dbSigType netType = net->getSigType();
    odb::dbIntProperty* prop = odb::dbIntProperty::find(net, "netId");
    if (prop
        != nullptr) {  // Ensure the property exists before trying to delete it
      odb::dbProperty::destroy(prop);
    }
    odb::dbIntProperty::create(net, "netId", -1);

    // escape nets with VDD/VSS/reset nets
    if (netType == odb::dbSigType::SIGNAL || netType == odb::dbSigType::CLOCK) {
      const int num_fanouts = net->getITerms().size() + net->getBTerms().size();
      // We can enable this to further improve the runtime
      if (num_fanouts <= 1 || num_fanouts > 10000000) {
        ignoreNet++;
        continue;
      }

      netId++;
      dbNetStor_.push_back(net);
      odb::dbIntProperty::find(net, "netId")->setValue(netId);

      Net myNet(net);
      netStor_.push_back(myNet);

      // check all the instance pins connected to this net
      for (odb::dbITerm* iTerm : net->getITerms()) {
        pinId++;
        odb::dbIntProperty::create(iTerm, "pinId", pinId);
        Pin myPin(iTerm, log_);
        // link the pin with the net
        myPin.setNet(&netStor_.back());
        // map the pin to its inst
        int instId
            = odb::dbIntProperty::find(iTerm->getInst(), "instId")->getValue();
        myPin.setInstance(&instStor_[instId]);
        pinStor_.push_back(myPin);
      }

      for (odb::dbBTerm* bTerm : net->getBTerms()) {
        pinId++;
        odb::dbIntProperty* prop = odb::dbIntProperty::find(bTerm, "pinId");
        if (prop != nullptr) {  // Ensure the property exists before trying to
                                // delete it
          odb::dbProperty::destroy(prop);
        }
        odb::dbIntProperty::create(bTerm, "pinId", pinId);
        Pin myPin(bTerm, log_);
        // link the pin with the net
        myPin.setNet(&netStor_.back());
        pinStor_.push_back(myPin);
      }
    }
  }

  // create cluster constraints
  if (numClusters > 0) {
    // create virtual nets within each cluster
    for (odb::dbInst* inst : insts) {
      const int instId = odb::dbIntProperty::find(inst, "instId")->getValue();
      if (instId == -1) {
        continue;
      }
      const int clusterId
          = odb::dbIntProperty::find(inst, "cluster_id")->getValue();
      // create a virtual net
      // create a virtual pin for each instance and a cluster
      netId++;
      Net myNet(netId);
      myNet.setVirtualWeight(1.0);
      netStor_.push_back(myNet);

      // create a virtual pin for each instance
      pinId++;
      Pin myPin(pinId);
      myPin.setNet(&netStor_.back());
      myPin.setInstance(&instStor_[instId]);
      pinStor_.push_back(myPin);

      // create a virtual pin for each cluster
      pinId++;
      Pin myPin2(pinId);
      myPin2.setNet(&netStor_.back());
      myPin2.setInstance(&instStor_[numRealPlaceInsts + clusterId]);
      pinStor_.push_back(myPin2);
    }
  }

  // create the datapath constraints
  int FF_limit = 2;
  float FF_weight = 1.0 / 2980;  // default value exp(-8)

  if (numMultiFFs > 0) {
    for (auto& vertex : dataflowVertices_) {
      if (vertex.getFFs().size() >= FF_limit) {
        int numSinks = 0;
        for (auto& sink : vertex.sinks) {
          for (auto& inst : dataflowVertices_[sink.first].getFFs()) {
            const int instId
                = odb::dbIntProperty::find(inst, "instId")->getValue();
            if (instId == -1) {
              continue;
            }
            numSinks++;
          }

          if (dataflowVertices_[sink.first].isBTerm()) {
            auto prop1 = odb::dbIntProperty::find(
                dataflowVertices_[sink.first].getBTerm(), "pinId");
            if (prop1 == nullptr) {  // Ensure the property exists before trying
                                     // to delete it
              continue;
            }
            numSinks++;
          }
        }

        if (numSinks == 0) {
          continue;
        }

        int numFFs = vertex.getFFs().size();
        // create a virtual net
        // create a virtual pin for each instance and a cluster
        netId++;
        Net myNet(netId);
        myNet.setVirtualWeight(1.0 / (numFFs - 1) * FF_weight);
        netStor_.push_back(myNet);

        // create a virtual pin for each instance
        pinId++;
        Pin myPin(pinId);
        myPin.setNet(&netStor_.back());
        myPin.setInstance(&instStor_[vertex.instId]);
        pinStor_.push_back(myPin);

        // create a virtual pin for each cluster
        for (auto& inst : vertex.getFFs()) {
          const int instId
              = odb::dbIntProperty::find(inst, "instId")->getValue();
          if (instId == -1) {
            continue;
          }

          pinId++;
          Pin myPin2(pinId);
          myPin2.setNet(&netStor_.back());
          myPin2.setInstance(&instStor_[instId]);
          pinStor_.push_back(myPin2);
        }
      }
    }
  }

  for (auto& net : netStor_) {
    nets_.push_back(&net);
  }

  for (auto& pin : pinStor_) {
    pins_.push_back(&pin);
    if (pin.net() != nullptr) {
      pin.net()->addPin(&pin);
    }
    if (pin.isITerm()) {
      pin.instance()->addPin(&pin);
    }
  }

  initVirtualWeightFactor_ = std::exp(virtualIter_);
  virtualWeightFactor_ = std::exp(virtualIter_);

  log_->report("[INFO] initVirtualWeightFactor = "
               + std::to_string(initVirtualWeightFactor_));
  log_->report("[INFO] virtualWeightFactor = "
               + std::to_string(virtualWeightFactor_));

  // print the statistics
  printInfo();
}

void PlacerBaseCommon::updateVirtualWeightFactor(int iter)
{
  virtualWeightFactor_ = initVirtualWeightFactor_ / std::exp(iter);
}

void PlacerBaseCommon::reset()
{
  db_ = nullptr;
  pbVars_.reset();

  // we need to free the memory on host and device
  // clear the vectors
  instStor_.clear();
  pinStor_.clear();
  netStor_.clear();

  dbInstStor_.clear();
  dbPinStor_.clear();
  dbNetStor_.clear();

  placeInsts_.clear();
  nets_.clear();
  insts_.clear();
  pins_.clear();

  numInsts_ = 0;
  numPlaceInsts_ = 0;
  numFixedInsts_ = 0;
  numDummyInsts_ = 0;

  placeInstsArea_ = 0;
  nonPlaceInstsArea_ = 0;
  macroInstsArea_ = 0;
  stdCellInstsArea_ = 0;

  clearPinProperty();

  if (clusterFlag_ == false) {
    clearInstProperty();
  }

  freeCUDAKernel();
}

// basic information
void PlacerBaseCommon::printInfo() const
{
  std::string msg;
  float dbuPerMicron
      = static_cast<float>(db_->getTech()->getDbUnitsPerMicron());

  msg = "NumInstances: " + std::to_string(numInsts_);
  log_->report(msg);

  msg = "NumPlaceInstances: " + std::to_string(numPlaceInsts_);
  log_->report(msg);

  msg = "NumFixedInstances: " + std::to_string(numFixedInsts_);
  log_->report(msg);

  msg = "NumDummyInstances: " + std::to_string(numDummyInsts_);
  log_->report(msg);

  msg = "DieAreaLxLy: "
        + floatToStringWithPrecision(
            static_cast<float>(die_.dieLx()) / dbuPerMicron, 2);
  msg += " "
         + floatToStringWithPrecision(
             static_cast<float>(die_.dieLy()) / dbuPerMicron, 2);
  log_->report(msg);

  msg = "DieAreaUxUy: "
        + floatToStringWithPrecision(
            static_cast<float>(die_.dieUx()) / dbuPerMicron, 2);
  msg += " "
         + floatToStringWithPrecision(
             static_cast<float>(die_.dieUy()) / dbuPerMicron, 2);
  log_->report(msg);

  msg = "CoreAreaLxLy: "
        + floatToStringWithPrecision(
            static_cast<float>(die_.coreLx()) / dbuPerMicron, 2);
  msg += " "
         + floatToStringWithPrecision(
             static_cast<float>(die_.coreLy()) / dbuPerMicron, 2);
  log_->report(msg);

  msg = "CoreAreaUxUy: "
        + floatToStringWithPrecision(
            static_cast<float>(die_.coreUx()) / dbuPerMicron, 2);
  msg += " "
         + floatToStringWithPrecision(
             static_cast<float>(die_.coreUy()) / dbuPerMicron, 2);
  log_->report(msg);

  const int64_t coreArea = die_.coreArea();
  float util = static_cast<double>(stdCellInstsArea_ + macroInstsArea_)
               / static_cast<double>(coreArea) * 100;
  msg = "CoreArea: "
        + floatToStringWithPrecision(
            static_cast<float>(coreArea) / dbuPerMicron / dbuPerMicron, 2);
  log_->report(msg);

  msg = "NonPlaceInstsArea: "
        + floatToStringWithPrecision(static_cast<float>(nonPlaceInstsArea_)
                                         / dbuPerMicron / dbuPerMicron,
                                     2);
  log_->report(msg);

  msg = "PlaceInstsArea: "
        + floatToStringWithPrecision(
            static_cast<float>(placeInstsArea_) / dbuPerMicron / dbuPerMicron,
            2);
  log_->report(msg);

  msg = "StdInstsArea: "
        + floatToStringWithPrecision(
            static_cast<float>(stdCellInstsArea_) / dbuPerMicron / dbuPerMicron,
            2);
  log_->report(msg);

  msg = "MacroInstsArea: "
        + floatToStringWithPrecision(
            static_cast<float>(macroInstsArea_) / dbuPerMicron / dbuPerMicron,
            2);
  log_->report(msg);

  msg = "Util(%): " + floatToStringWithPrecision(util, 2);
  log_->report(msg);

  if (util >= 100.1) {
    log_->error(GPL2, 301, "Utilization exceeds 100%.");
  }
}

int64_t PlacerBaseCommon::hpwl() const
{
  if (wlGradOp_ != nullptr) {
    // return wlGradOp_->computeHPWL();
    return wlGradOp_->computeWeightedHPWL(virtualWeightFactor_);
  } else {
    return 0;
  }
}

void PlacerBaseCommon::updatePinLocation()
{
  if (wlGradOp_ != nullptr) {
    wlGradOp_->updatePinLocation(dInstDCxPtr_, dInstDCyPtr_);
  }
}

// calculate the wirelength gradient
// we encapsulate the GPU acceleration into the WireLengthGradientOp class
void PlacerBaseCommon::updateWireLengthForce(const float wlCoeffX,
                                             const float wlCoeffY)
{
  if (wlGradOp_ != nullptr) {
    wlGradOp_->computeWireLengthForce(
        wlCoeffX, wlCoeffY, virtualWeightFactor_, dWLGradXPtr_, dWLGradYPtr_);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Class PlacerBase
////////////////////////////////////////////////////////////////////////////////

PlacerBase::PlacerBase()
    : db_(nullptr),
      log_(nullptr),
      pbCommon_(nullptr),
      group_(nullptr),
      densityOp_(nullptr),
      bg_(),
      die_(),
      siteSizeX_(0),
      siteSizeY_(0),
      nbVars_(),
      npVars_(),
      fillerDx_(0),
      fillerDy_(0),
      whiteSpaceArea_(0),
      movableArea_(0),
      totalFillerArea_(0),
      placeInstsArea_(0),
      nonPlaceInstsArea_(0),
      macroInstsArea_(0),
      stdInstsArea_(0),
      numInsts_(0),
      numNonPlaceInsts_(0),
      numPlaceInsts_(0),
      numFixedInsts_(0),
      numDummyInsts_(0),
      numFillerInsts_(0),
      dInstDDxPtr_(nullptr),
      dInstDDyPtr_(nullptr),
      dInstDCxPtr_(nullptr),
      dInstDCyPtr_(nullptr),
      dWireLengthPrecondiPtr_(nullptr),
      dDensityPrecondiPtr_(nullptr),
      sumPhi_(0.0),
      targetDensity_(0.0),
      uniformTargetDensity_(0.0),
      densityPenalty_(0.0),
      baseWireLengthCoef_(0.0),
      sumOverflow_(0.0),
      sumOverflowUnscaled_(0.0),
      prevHpwl_(0),
      isDiverged_(false),
      isMaxPhiCoefChanged_(false),
      minSumOverflow_(1e30),
      hpwlWithMinSumOverflow_(1e30),
      iter_(0),
      isConverged_(false),
      stepLength_(0.0),
      wireLengthGradSum_(0.0),
      densityGradSum_(0.0),
      dDensityGradXPtr_(nullptr),
      dDensityGradYPtr_(nullptr),
      dWireLengthGradXPtr_(nullptr),
      dWireLengthGradYPtr_(nullptr),
      dCurSLPCoordiPtr_(nullptr),
      dCurSLPWireLengthGradXPtr_(nullptr),
      dCurSLPWireLengthGradYPtr_(nullptr),
      dCurSLPDensityGradXPtr_(nullptr),
      dCurSLPDensityGradYPtr_(nullptr),
      dCurSLPSumGradsPtr_(nullptr),
      dPrevSLPCoordiPtr_(nullptr),
      dPrevSLPWireLengthGradXPtr_(nullptr),
      dPrevSLPWireLengthGradYPtr_(nullptr),
      dPrevSLPDensityGradXPtr_(nullptr),
      dPrevSLPDensityGradYPtr_(nullptr),
      dPrevSLPSumGradsPtr_(nullptr),
      dNextSLPCoordiPtr_(nullptr),
      dNextSLPWireLengthGradXPtr_(nullptr),
      dNextSLPWireLengthGradYPtr_(nullptr),
      dNextSLPDensityGradXPtr_(nullptr),
      dNextSLPDensityGradYPtr_(nullptr),
      dNextSLPSumGradsPtr_(nullptr),
      dCurCoordiPtr_(nullptr),
      dNextCoordiPtr_(nullptr)
{
}

// Constructor
PlacerBase::PlacerBase(NesterovBaseVars nbVars,
                       odb::dbDatabase* db,
                       std::shared_ptr<PlacerBaseCommon> pbCommon,
                       utl::Logger* log,
                       odb::dbGroup* group)
    : PlacerBase()
{
  nbVars_ = nbVars;
  db_ = db;
  log_ = log;
  pbCommon_ = std::move(pbCommon);
  group_ = group;
  init();
}

PlacerBase::~PlacerBase()
{
  reset();
}

void PlacerBase::reset()
{
  freeCUDAKernel();
}

void PlacerBase::init()
{
  // set a fixed seed
  srand(42);
  // get the die information
  die_ = pbCommon_->die();
  // siteSize update
  siteSizeX_ = pbCommon_->siteSizeX();
  siteSizeY_ = pbCommon_->siteSizeY();
  // Here we call the objects on the host side
  for (auto& inst : pbCommon_->insts()) {
    if (!inst->isInstance()) {
      continue;
    }

    // check whether the instance is in the group
    if (inst->dbInst() && inst->dbInst()->getGroup() != group_) {
      continue;
    }
    if (inst->isFixed() && isCoreAreaOverlap(die_, inst)) {
      // Check whether fixed instance is
      // within the corearea
      // outside of corearea is none of RePlAce's business
      nonPlaceInsts_.push_back(inst);
      nonPlaceInstsArea_ += getOverlapWithCoreArea(die_, inst);
    } else {
      placeInsts_.push_back(inst);
      insts_.push_back(inst);
      int64_t instArea = inst->area();
      placeInstsArea_ += instArea;
      // macro cells should be macroInstsArea_
      if (inst->isMacro()) {
        macroInstsArea_ += instArea;
      } else {
        stdInstsArea_ += instArea;
      }
    }
  }

  // create the dummy instances
  // insts fill with fake instances (fragmented row or blockage)
  initInstsForUnusableSites();
  for (auto& inst : dummyInsts_) {
    nonPlaceInstsArea_ += inst.area();
    nonPlaceInsts_.push_back(&inst);
  }

  // update gFillerCells
  initFillerGCells();
  for (auto& inst : fillerInsts_) {
    insts_.push_back(&inst);
  }

  // get the statistics
  numPlaceInsts_ = placeInsts_.size();
  numDummyInsts_ = dummyInsts_.size();
  numFillerInsts_ = fillerInsts_.size();
  // Place insts can be moved, we identify them as insts
  numInsts_ = numPlaceInsts_ + numFillerInsts_;
  // fixed instances and dummy instances cannot be moved, we identify
  // them as nonPlaceInsts
  numNonPlaceInsts_ = nonPlaceInsts_.size();

  log_->report("[INFO] Number of instances: " + std::to_string(numInsts_));
  log_->report("[INFO] Number of place instances: "
               + std::to_string(numPlaceInsts_));

  // initialize bin grid structure
  if (nbVars_.isSetBinCnt) {
    bg_.setBinCnt(nbVars_.binCntX, nbVars_.binCntY);
  }
  bg_.setPlacerBase(this);
  bg_.setLogger(log_);
  bg_.setCorePoints(&die());
  bg_.setTargetDensity(targetDensity_);
  bg_.initBins();

  // update densitySize and densityScale in each gCell
  updateDensitySize();
}

std::pair<int, int> getMinMaxIdx(int ll,
                                 int uu,
                                 int coreLL,
                                 int siteSize,
                                 int minIdx,
                                 int maxIdx)
{
  int lowerIdx = (ll - coreLL) / siteSize;
  int upperIdx = (fastModulo((uu - coreLL), siteSize) == 0)
                     ? (uu - coreLL) / siteSize
                     : (uu - coreLL) / siteSize + 1;
  return std::make_pair(std::max(minIdx, lowerIdx), std::min(maxIdx, upperIdx));
}

// Use dummy instance to fill unusable sites.  Sites are unusable
// due to fragmented rows or placement blockages.
void PlacerBase::initInstsForUnusableSites()
{
  odb::dbSet<odb::dbRow> rows = db_->getChip()->getBlock()->getRows();
  odb::dbSet<odb::dbPowerDomain> pds
      = db_->getChip()->getBlock()->getPowerDomains();

  int64_t siteCountX = (die_.coreUx() - die_.coreLx()) / siteSizeX_;
  int64_t siteCountY = (die_.coreUy() - die_.coreLy()) / siteSizeY_;

  enum PlaceInfo
  {
    Empty,     // This site cannot be used
    Row,       // This site can be used
    FixedInst  // This site is blocked by the fixed instance
  };

  //
  // Initialize siteGrid as empty
  //
  std::vector<PlaceInfo> siteGrid(siteCountX * siteCountY, PlaceInfo::Empty);
  // check if this belongs to a group
  // if there is a group, only mark the sites that belong to the group as Row
  // if there is no group, then mark all as Row, and then for each power domain,
  // mark the sites that belong to the power domain as Empty
  if (group_ != nullptr) {
    for (auto boundary : group_->getRegion()->getBoundaries()) {
      odb::Rect rect = boundary->getBox();
      std::pair<int, int> pairX = getMinMaxIdx(
          rect.xMin(), rect.xMax(), die_.coreLx(), siteSizeX_, 0, siteCountX);
      std::pair<int, int> pairY = getMinMaxIdx(
          rect.yMin(), rect.yMax(), die_.coreLy(), siteSizeY_, 0, siteCountY);
      for (int i = pairX.first; i < pairX.second; i++) {
        for (int j = pairY.first; j < pairY.second; j++) {
          siteGrid[j * siteCountX + i] = Row;
        }
      }
    }
  } else {
    // fill in rows' bbox
    for (odb::dbRow* row : rows) {
      odb::Rect rect = row->getBBox();
      std::pair<int, int> pairX = getMinMaxIdx(
          rect.xMin(), rect.xMax(), die_.coreLx(), siteSizeX_, 0, siteCountX);
      std::pair<int, int> pairY = getMinMaxIdx(
          rect.yMin(), rect.yMax(), die_.coreLy(), siteSizeY_, 0, siteCountY);
      for (int i = pairX.first; i < pairX.second; i++) {
        for (int j = pairY.first; j < pairY.second; j++) {
          siteGrid[j * siteCountX + i] = Row;
        }
      }
    }
  }

  // Mark blockage areas as empty so that their sites will be blocked.
  for (odb::dbBlockage* blockage : db_->getChip()->getBlock()->getBlockages()) {
    odb::dbInst* inst = blockage->getInstance();
    if (inst && !inst->isFixed()) {
      continue;
    }
    odb::dbBox* bbox = blockage->getBBox();
    std::pair<int, int> pairX = getMinMaxIdx(
        bbox->xMin(), bbox->xMax(), die_.coreLx(), siteSizeX_, 0, siteCountX);
    std::pair<int, int> pairY = getMinMaxIdx(
        bbox->yMin(), bbox->yMax(), die_.coreLy(), siteSizeY_, 0, siteCountY);

    // The placement density may be partial blockage
    // TODO : handle the partial blockage
    for (int j = pairY.first; j < pairY.second; j++) {
      for (int i = pairX.first; i < pairX.second; i++) {
        siteGrid[j * siteCountX + i] = Empty;
      }
    }
  }

  // fill fixed instances' bbox
  for (auto& inst : pbCommon_->insts()) {
    if (!inst->isFixed()) {
      continue;
    }
    std::pair<int, int> pairX = getMinMaxIdx(
        inst->lx(), inst->ux(), die_.coreLx(), siteSizeX_, 0, siteCountX);
    std::pair<int, int> pairY = getMinMaxIdx(
        inst->ly(), inst->uy(), die_.coreLy(), siteSizeY_, 0, siteCountY);

    for (int i = pairX.first; i < pairX.second; i++) {
      for (int j = pairY.first; j < pairY.second; j++) {
        siteGrid[j * siteCountX + i] = FixedInst;
      }
    }
  }

  // In the case of top level power domain i.e no group,
  // mark all other power domains as empty
  if (group_ == nullptr) {
    for (odb::dbPowerDomain* pd : pds) {
      if (pd->getGroup() != nullptr) {
        for (auto boundary : pd->getGroup()->getRegion()->getBoundaries()) {
          odb::Rect rect = boundary->getBox();
          std::pair<int, int> pairX = getMinMaxIdx(rect.xMin(),
                                                   rect.xMax(),
                                                   die_.coreLx(),
                                                   siteSizeX_,
                                                   0,
                                                   siteCountX);

          std::pair<int, int> pairY = getMinMaxIdx(rect.yMin(),
                                                   rect.yMax(),
                                                   die_.coreLy(),
                                                   siteSizeY_,
                                                   0,
                                                   siteCountY);

          for (int i = pairX.first; i < pairX.second; i++) {
            for (int j = pairY.first; j < pairY.second; j++) {
              siteGrid[j * siteCountX + i] = Empty;
            }
          }
        }
      }
    }
  }

  //
  // Search the "Empty" coordinates on site-grid
  // --> These sites need to be dummyInstance
  //
  for (int j = 0; j < siteCountY; j++) {
    for (int i = 0; i < siteCountX; i++) {
      // if empty spot found
      if (siteGrid[j * siteCountX + i] == Empty) {
        int startX = i;
        // find end points
        while (i < siteCountX && siteGrid[j * siteCountX + i] == Empty) {
          i++;
        }
        int endX = i;
        Instance myInst(die_.coreLx() + siteSizeX_ * startX,
                        die_.coreLy() + siteSizeY_ * j,
                        siteSizeX_ * (endX - startX),
                        siteSizeY_,
                        true);  // dummy instances
        dummyInsts_.push_back(myInst);
      }
    }
  }
}

// Note that filler cells can be moved around
// create the filler cells
void PlacerBase::initFillerGCells()
{
  // extract average dx/dy in range (10%, 90%)
  // Following codes are operated on the host side
  std::vector<int> dxStor;
  std::vector<int> dyStor;

  dxStor.reserve(pbCommon_->numPlaceInsts());
  dyStor.reserve(pbCommon_->numPlaceInsts());
  for (auto& placeInst : pbCommon_->placeInsts()) {
    dxStor.push_back(placeInst->dx());
    dyStor.push_back(placeInst->dy());
  }

  // sort
  std::sort(dxStor.begin(), dxStor.end());
  std::sort(dyStor.begin(), dyStor.end());

  // average from (10 - 90%)
  int64_t dxSum = 0;
  int64_t dySum = 0;

  int minIdx = dxStor.size() * 0.05;
  int maxIdx = dyStor.size() * 0.95;

  // when #instances are too small,
  // extracts average values in whole ranges
  if (minIdx == maxIdx) {
    minIdx = 0;
    maxIdx = dxStor.size();
  }

  for (int i = minIdx; i < maxIdx; i++) {
    dxSum += dxStor[i];
    dySum += dyStor[i];
  }

  // the avgDx and avgDy will be used as filler cells'
  // width and height
  fillerDx_ = static_cast<int>(dxSum / (maxIdx - minIdx));
  fillerDy_ = static_cast<int>(dySum / (maxIdx - minIdx));

  int64_t coreArea = die_.coreArea();
  whiteSpaceArea_ = coreArea - static_cast<int64_t>(nonPlaceInstsArea_);

  // targetDensity initialize
  if (nbVars_.useUniformTargetDensity) {
    // calculate the default uniform target density
    targetDensity_ = static_cast<float>(stdInstsArea_)
                         / static_cast<float>(whiteSpaceArea_ - macroInstsArea_)
                     + 0.01;
  } else {
    targetDensity_ = nbVars_.targetDensity;
  }

  // TODO density screening
  movableArea_ = whiteSpaceArea_ * targetDensity_;
  totalFillerArea_ = movableArea_ - nesterovInstsArea();
  uniformTargetDensity_ = static_cast<float>(nesterovInstsArea())
                          / static_cast<float>(whiteSpaceArea_);

  if (totalFillerArea_ < 0) {
    uniformTargetDensity_ = ceilf(uniformTargetDensity_ * 100) / 100;
    log_->error(GPL2,
                302,
                "Use a higher -density or "
                "re-floorplan with a larger core area.\n"
                "Given target density: {:.2f}\n"
                "Suggested target density: {:.2f}",
                targetDensity_,
                uniformTargetDensity_);
  }

  // calculate the number of filler cells
  // This may have some overflow issue
  numFillerInsts_ = static_cast<int>(
      totalFillerArea_ / (static_cast<int64_t>(fillerDx_ * fillerDy_)));
  if (numFillerInsts_ < 0) {
    numFillerInsts_ = 0;
  }

  if (pbCommon_->getClusterFlag() == true) {
    numFillerInsts_ = 0;
  }

  log_->report("totalFillerArea =  " + std::to_string(totalFillerArea_));
  log_->report("NumFillerCells = " + std::to_string(numFillerInsts_));
  log_->report("FillerCellSize = " + std::to_string(fillerDx_) + " , "
               + std::to_string(fillerDy_));

  //
  // mt19937 supports huge range of random values.
  // rand()'s RAND_MAX is only 32767.
  //
  mt19937 randVal(0);
  for (int i = 0; i < numFillerInsts_; i++) {
    // instability problem between g++ and clang++!
    auto randX = randVal();
    auto randY = randVal();
    int cx = randX % pbCommon_->die().coreDx() + pbCommon_->die().coreLx();
    int cy = randY % pbCommon_->die().coreDy() + pbCommon_->die().coreLy();
    // place filler cells on random coordi and
    // set size as avgDx and avgDy
    Instance myGCell(cx, cy, fillerDx_, fillerDy_, false);  // filler instance
    fillerInsts_.push_back(myGCell);
  }
}

// update densitySize and densityScale in each gCell
// We do not need to convert this into GPU kernel
// This is not the bottleneck currently
void PlacerBase::updateDensitySize()
{
  int instId = 0;
  for (auto& inst : insts_) {
    float scaleX = 0, scaleY = 0;
    float densitySizeX = 0, densitySizeY = 0;
    if (inst->dx() < REPLACE_SQRT2 * bg_.binSizeX()) {
      scaleX = static_cast<float>(inst->dx())
               / static_cast<float>(REPLACE_SQRT2 * bg_.binSizeX());
      densitySizeX = REPLACE_SQRT2 * static_cast<float>(bg_.binSizeX());
    } else {
      scaleX = 1.0;
      densitySizeX = inst->dx();
    }

    if (inst->dy() < REPLACE_SQRT2 * bg_.binSizeY()) {
      scaleY = static_cast<float>(inst->dy())
               / static_cast<float>(REPLACE_SQRT2 * bg_.binSizeY());
      densitySizeY = REPLACE_SQRT2 * static_cast<float>(bg_.binSizeY());
    } else {
      scaleY = 1.0;
      densitySizeY = inst->dy();
    }
    inst->setDensitySize(densitySizeX, densitySizeY, scaleX * scaleY);
  }
}

// update the step length
bool PlacerBase::nesterovUpdateStepLength()
{
  if (isConverged_) {
    return true;
  }

  float newStepLength = getStepLength(dCurSLPCoordiPtr_,
                                      dCurSLPSumGradsPtr_,
                                      dNextSLPCoordiPtr_,
                                      dNextSLPSumGradsPtr_);

  if (isnan(newStepLength) || isinf(newStepLength)) {
    isDiverged_ = true;
    divergeMsg_ = "RePlAce diverged at newStepLength.";
    divergeCode_ = 305;
    return false;
  }

  if (newStepLength > stepLength_ * 0.95) {
    stepLength_ = newStepLength;
    return false;
  }

  if (newStepLength < 0.01) {
    stepLength_ = 0.01;
    return false;
  }

  stepLength_ = newStepLength;

  return true;
}

// NestrovePlace related functions
void PlacerBase::updateDensityCenterCur()
{
  updateGCellDensityCenterLocation(dCurCoordiPtr_);
}

void PlacerBase::updateDensityCenterCurSLP()
{
  updateGCellDensityCenterLocation(dCurSLPCoordiPtr_);
}

void PlacerBase::updateDensityCenterPrevSLP()
{
  updateGCellDensityCenterLocation(dPrevSLPCoordiPtr_);
}

void PlacerBase::updateDensityCenterNextSLP()
{
  updateGCellDensityCenterLocation(dNextSLPCoordiPtr_);
}

void PlacerBase::updateDensityForceBin()
{
  densityOp_->updateDensityForceBin();
}

// dynamic adjustment for better convergence with large designs
void PlacerBase::nesterovAdjustPhi()
{
  if (isConverged_) {
    return;
  }

  // dynamic adjustment for
  // better convergence with
  // large designs
  if (!isMaxPhiCoefChanged_ && sumOverflowUnscaled_ < 0.35f) {
    isMaxPhiCoefChanged_ = true;
    npVars_.maxPhiCoef *= 0.99;
  }
}

float PlacerBase::getPhiCoef(float scaledDiffHpwl) const
{
  float retCoef = (scaledDiffHpwl < 0)
                      ? npVars_.maxPhiCoef
                      : npVars_.maxPhiCoef
                            * pow(npVars_.maxPhiCoef, scaledDiffHpwl * -1.0);
  retCoef = std::max(npVars_.minPhiCoef, retCoef);
  return retCoef;
}

bool PlacerBase::checkConvergence()
{
  if (isConverged_) {
    return true;
  }

  if (numPlaceInsts_ <= 10000) {
    npVars_.targetOverflow = 0.2;
  }

  if (sumOverflowUnscaled_ <= npVars_.targetOverflow) {
    if (group_) {
      std::string msg = "[NesterovSolve] PowerDomain ";
      msg += group_->getName();
      msg += " finished with Overflow: ";
      msg += std::to_string(sumOverflowUnscaled_);
      log_->report(msg);
    } else {
      std::string msg = "[NesterovSolve] Finished with Overflow: ";
      msg += std::to_string(sumOverflowUnscaled_);
      log_->report(msg);
    }

    isConverged_ = true;
    return true;
  }
  return false;
}

float PlacerBase::overflowArea() const
{
  return densityOp_->sumOverflow();
}

// exchange the states:  prev -> current, current -> next
// update the parameters
void PlacerBase::updateNextIter(int iter)
{
  if (isConverged_) {
    return;
  }

  // Previous <= Current
  std::swap(dCurSLPCoordiPtr_, dPrevSLPCoordiPtr_);
  std::swap(dCurSLPSumGradsPtr_, dPrevSLPSumGradsPtr_);
  std::swap(dCurSLPWireLengthGradXPtr_, dPrevSLPWireLengthGradXPtr_);
  std::swap(dCurSLPWireLengthGradYPtr_, dPrevSLPWireLengthGradYPtr_);
  std::swap(dCurSLPDensityGradXPtr_, dPrevSLPDensityGradXPtr_);
  std::swap(dCurSLPDensityGradYPtr_, dPrevSLPDensityGradYPtr_);

  // Current <= Next
  std::swap(dCurSLPCoordiPtr_, dNextSLPCoordiPtr_);
  std::swap(dCurSLPSumGradsPtr_, dNextSLPSumGradsPtr_);
  std::swap(dCurSLPWireLengthGradXPtr_, dNextSLPWireLengthGradXPtr_);
  std::swap(dCurSLPWireLengthGradYPtr_, dNextSLPWireLengthGradYPtr_);
  std::swap(dCurSLPDensityGradXPtr_, dNextSLPDensityGradXPtr_);
  std::swap(dCurSLPDensityGradYPtr_, dNextSLPDensityGradYPtr_);

  std::swap(dCurCoordiPtr_, dNextCoordiPtr_);

  // In a macro dominated design like mock-array-big you may be placing
  // very few std cells in a sea of fixed macros. The overflow denominator
  // may be quite small and prevent convergence. This is mostly due to
  // our limited ability to move instances off macros cleanly.
  // As that improves this should no longer be needed.
  const float fractionOfMaxIters
      = static_cast<float>(iter) / npVars_.maxNesterovIter;
  const float overflowDenominator
      = std::max(static_cast<float>(nesterovInstsArea()),
                 fractionOfMaxIters * nonPlaceInstsArea() * 0.05f);

  sumOverflow_ = overflowArea() / overflowDenominator;
  sumOverflowUnscaled_ = overflowAreaUnscaled() / overflowDenominator;

  int64_t hpwl = pbCommon_->hpwl();
  float phiCoef = getPhiCoef(static_cast<float>(hpwl - prevHpwl_)
                             / npVars_.referenceHpwl);

  prevHpwl_ = hpwl;
  // TODO:  use autotuner to autotune this parameter for better tradeoff between
  // overflow and wirelength
  // densityPenalty_ *= phiCoef * 1.01;
  densityPenalty_ *= phiCoef * 0.99;

  if (iter == 0 || (iter + 1) % 10 == 0) {
    std::string msg = "[NesterovSolve] Iter: " + std::to_string(iter + 1) + " ";
    msg += "overflow: " + std::to_string(sumOverflowUnscaled_) + " ";
    msg += "HPWL: " + std::to_string(prevHpwl_) + " ";
    msg += "densityPenalty: " + std::to_string(double(densityPenalty_));
    log_->report(msg);
  }

  if (iter > 50 && minSumOverflow_ > sumOverflowUnscaled_) {
    minSumOverflow_ = sumOverflowUnscaled_;
    hpwlWithMinSumOverflow_ = prevHpwl_;
  }
}

////////////////////////////////////////////////////////////////////////////
// class BinGrid
////////////////////////////////////////////////////////////////////////////
BinGrid::BinGrid()
    : log_(nullptr),
      pb_(nullptr),
      numBins_(0),
      lx_(0),
      ly_(0),
      ux_(0),
      uy_(0),
      binCntX_(0),
      binCntY_(0),
      binSizeX_(0),
      binSizeY_(0),
      targetDensity_(0),
      isSetBinCnt_(0)
{
}

BinGrid::BinGrid(Die* die) : BinGrid()
{
  setCorePoints(die);
}

BinGrid::~BinGrid()
{
  log_ = nullptr;
  pb_ = nullptr;
  binStor_.clear();
  bins_.clear();
  numBins_ = 0;
  binCntX_ = binCntY_ = 0;
  binSizeX_ = binSizeY_ = 0;
  isSetBinCnt_ = 0;
}

void BinGrid::setCorePoints(const Die* die)
{
  lx_ = die->coreLx();
  ly_ = die->coreLy();
  ux_ = die->coreUx();
  uy_ = die->coreUy();
}

void BinGrid::setBinCnt(int binCntX, int binCntY)
{
  isSetBinCnt_ = true;
  binCntX_ = binCntX;
  binCntY_ = binCntY;
}

static unsigned int roundDownToPowerOfTwo(unsigned int x)
{
  x |= (x >> 1);
  x |= (x >> 2);
  x |= (x >> 4);
  x |= (x >> 8);
  x |= (x >> 16);
  return x ^ (x >> 1);
}

void BinGrid::initBins()
{
  int64_t totalBinArea
      = static_cast<int64_t>(ux_ - lx_) * static_cast<int64_t>(uy_ - ly_);
  int64_t averagePlaceInstArea = pb_->placeInstsArea() / pb_->numPlaceInsts();

  int64_t idealBinArea
      = std::round(static_cast<float>(averagePlaceInstArea) / targetDensity_);

  int idealBinCnt = totalBinArea / idealBinArea;
  if (idealBinCnt < 4) {  // the smallest we allow is 2x2 bins
    idealBinCnt = 4;
  }

  if (!isSetBinCnt_) {
    // Consider the apect ratio of the block when computing the number
    // of bins so that the bins remain relatively square.
    const unsigned int width = ux_ - lx_;
    const unsigned int height = uy_ - ly_;
    const int ratio = roundDownToPowerOfTwo(std::max(width, height)
                                            / std::min(width, height));
    int foundBinCnt = 2;
    // find binCnt: 2, 4, 8, 16, 32, 64, ...
    // s.t. #bins(binCnt) <= idealBinCnt <= #bins(binCnt*2).
    for (foundBinCnt = 2; foundBinCnt <= 1024; foundBinCnt *= 2) {
      if ((foundBinCnt == 2
           || foundBinCnt * (foundBinCnt * ratio) <= idealBinCnt)
          && 4 * foundBinCnt * (foundBinCnt * ratio) > idealBinCnt) {
        break;
      }
    }

    if (width > height) {
      binCntX_ = foundBinCnt * ratio;
      binCntY_ = foundBinCnt;
    } else {
      binCntX_ = foundBinCnt;
      binCntY_ = foundBinCnt * ratio;
    }
  }

  binSizeX_ = ceil(static_cast<float>((ux_ - lx_)) / binCntX_);
  binSizeY_ = ceil(static_cast<float>((uy_ - ly_)) / binCntY_);
  // create bins_ on host and device
  numBins_ = binCntX_ * binCntY_;

  log_->report("[INFO] binSizeX = " + std::to_string(binSizeX_));
  log_->report("[INFO] binSizeY = " + std::to_string(binSizeY_));
  log_->report("[INFO] binCntX = " + std::to_string(binCntX_));
  log_->report("[INFO] binCntY = " + std::to_string(binCntY_));
  log_->report("[INFO] numBins = " + std::to_string(numBins_));
  log_->report("[INFO] targetDensity = " + std::to_string(targetDensity_));

  // create bins_ on host
  binStor_.reserve(numBins_);
  for (int idxY = 0; idxY < binCntY_; ++idxY) {
    for (int idxX = 0; idxX < binCntX_; ++idxX) {
      const int x = lx_ + idxX * binSizeX_;
      const int y = ly_ + idxY * binSizeY_;
      const int sizeX = std::min(ux_ - x, binSizeX_);
      const int sizeY = std::min(uy_ - y, binSizeY_);
      binStor_.emplace_back(
          idxX, idxY, x, y, x + sizeX, y + sizeY, targetDensity_);
    }
  }

  for (auto& bin : binStor_) {
    bins_.push_back(&bin);
  }

  updateBinsNonPlaceArea();
}

static int64_t getOverlapArea(const Bin* bin,
                              const Instance* inst,
                              int dbu_per_micron)
{
  int rectLx = max(bin->lx(), inst->lx()), rectLy = max(bin->ly(), inst->ly()),
      rectUx = min(bin->ux(), inst->ux()), rectUy = min(bin->uy(), inst->uy());

  if (rectLx >= rectUx || rectLy >= rectUy) {
    return 0;
  }

  if (inst->isMacro()) {
    const float meanX = (inst->cx() - inst->lx()) / (float) dbu_per_micron;
    const float meanY = (inst->cy() - inst->ly()) / (float) dbu_per_micron;

    // For the bivariate normal distribution, we are using
    // the shifted means of X and Y.
    // Sigma is used as the mean/4 for both dimensions
    const biNormalParameters i
        = {meanX,
           meanY,
           meanX / 4,
           meanY / 4,
           (rectLx - inst->lx()) / (float) dbu_per_micron,
           (rectLy - inst->ly()) / (float) dbu_per_micron,
           (rectUx - inst->lx()) / (float) dbu_per_micron,
           (rectUy - inst->ly()) / (float) dbu_per_micron};

    const float original = static_cast<float>(rectUx - rectLx)
                           * static_cast<float>(rectUy - rectLy);
    const float scaled = calculateBiVariateNormalCDF(i)
                         * static_cast<float>(inst->ux() - inst->lx())
                         * static_cast<float>(inst->uy() - inst->ly());

    // For heavily dense regions towards the center of the macro,
    // we are using an upper limit of 1.15*(overlap) between the macro
    // and the bin.
    if (scaled >= original) {
      return min<float>(scaled, original * 1.15);
    }
    // If the scaled value is smaller than the actual overlap
    // then use the original overlap value instead.
    // This is implemented to prevent cells from being placed
    // at the outer sides of the macro.
    else {
      return original;
    }
  } else {
    return static_cast<float>(rectUx - rectLx)
           * static_cast<float>(rectUy - rectLy);
  }
}

void BinGrid::updateBinsNonPlaceArea()
{
  for (auto bin : bins_) {
    bin->setNonPlaceArea(0);
  }

  for (auto& inst : pb_->nonPlaceInsts()) {
    std::pair<int, int> pairX = getMinMaxIdxX(inst);
    std::pair<int, int> pairY = getMinMaxIdxY(inst);
    for (int i = pairX.first; i < pairX.second; i++) {
      for (int j = pairY.first; j < pairY.second; j++) {
        Bin* bin = bins_[j * binCntX_ + i];
        // Note that nonPlaceArea should have scale-down with
        // target density.
        // See MS-replace paper
        //
        bin->addNonPlaceArea(
            getOverlapArea(
                bin,
                inst,
                pb_->db()->getChip()->getBlock()->getDbUnitsPerMicron())
            * bin->targetDensity());
      }
    }
  }
}

std::pair<int, int> BinGrid::getMinMaxIdxX(const Instance* inst) const
{
  int lowerIdx = (inst->lx() - lx()) / binSizeX_;
  int upperIdx = (fastModulo((inst->ux() - lx()), binSizeX_) == 0)
                     ? (inst->ux() - lx()) / binSizeX_
                     : (inst->ux() - lx()) / binSizeX_ + 1;

  return std::make_pair(std::max(lowerIdx, 0), std::min(upperIdx, binCntX_));
}

std::pair<int, int> BinGrid::getMinMaxIdxY(const Instance* inst) const
{
  int lowerIdx = (inst->ly() - ly()) / binSizeY_;
  int upperIdx = (fastModulo((inst->uy() - ly()), binSizeY_) == 0)
                     ? (inst->uy() - ly()) / binSizeY_
                     : (inst->uy() - ly()) / binSizeY_ + 1;

  return std::make_pair(std::max(lowerIdx, 0), std::min(upperIdx, binCntY_));
}

}  // namespace gpl2
