#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
// BSD 3-Clause License
//
// Copyright (c) 2023, Google LLC
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// The density force is calculated by solving the Poisson equation.
// It is originally developed by the graduate student Jaekyung Kim
// (jkim97@postech.ac.kr) at Pohang University of Science and Technology
// (POSTECH), then modified by our UCSD team. We thank Jaekyung Kim for his
// contribution.
//
//
///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>

#include <cassert>
#include <cmath>
#include <fstream>
#include <iostream>
#include <memory>

#include "poissonSolver.h"
#include "util.h"

namespace gpl2 {

__global__ void precomputeExpk(hipfftComplex* expkM,
                               hipfftComplex* expkN,
                               const int M,
                               const int N)
{
  const int tID = blockDim.x * blockIdx.x + threadIdx.x;

  if (tID <= M / 2) {
    int hID = tID;
    hipfftComplex W_h_4M = make_float2(__cosf((float) PI * hID / (2 * M)),
                                      -__sinf((float) PI * hID / (M * 2)));
    expkM[hID] = W_h_4M;
  }
  if (tID <= N / 2) {
    int wid = tID;
    hipfftComplex W_w_4N = make_float2(__cosf((float) PI * wid / (2 * N)),
                                      -__sinf((float) PI * wid / (N * 2)));
    expkN[wid] = W_w_4N;
  }
}

__global__ void precomputeExpkForInverse(hipfftComplex* expkM,
                                         hipfftComplex* expkN,
                                         hipfftComplex* expkMN_1,
                                         hipfftComplex* expkMN_2,
                                         const int M,
                                         const int N)
{
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < M) {
    int hid = tid;
    hipfftComplex W_h_4M = make_float2(__cosf((float) PI * hid / (2 * M)),
                                      -__sinf((float) PI * hid / (M * 2)));
    expkM[hid] = W_h_4M;
    // expkMN_1
    hipfftComplex W_h_4M_offset
        = make_float2(__cosf((float) PI * (hid + M) / (2 * M)),
                      -__sinf((float) PI * (hid + M) / (M * 2)));
    expkMN_1[hid] = W_h_4M;
    expkMN_1[hid + M] = W_h_4M_offset;

    // expkMN_2
    W_h_4M = make_float2(-__sinf((float) PI * (hid - (N - 1)) / (M * 2)),
                         -__cosf((float) PI * (hid - (N - 1)) / (2 * M)));

    W_h_4M_offset
        = make_float2(-__sinf((float) PI * (hid - (N - 1) + M) / (M * 2)),
                      -__cosf((float) PI * (hid - (N - 1) + M) / (2 * M)));
    expkMN_2[hid] = W_h_4M;
    expkMN_2[hid + M] = W_h_4M_offset;
  }
  if (tid <= N / 2) {
    int wid = tid;
    hipfftComplex W_w_4N = make_float2(__cosf((float) PI * wid / (2 * N)),
                                      -__sinf((float) PI * wid / (N * 2)));
    expkN[wid] = W_w_4N;
  }
}

__global__ void divideByWSquare(const int binCntX,
                                const int binCntY,
                                const int binSizeX,
                                const int binSizeY,
                                hipfftReal* input)
{
  const int wID = blockDim.x * blockIdx.x + threadIdx.x;
  const int hID = blockDim.y * blockIdx.y + threadIdx.y;

  if (wID < binCntX && hID < binCntY) {
    int binID = wID + hID * binCntX;

    if (hID == 0 && wID == 0)
      input[binID] = 0.0;
    else {
      float denom1 = (2.0 * float(FFT_PI) * wID) / binCntX;
      float denom2
          = (2.0 * float(FFT_PI) * hID) / binCntY * binSizeY / binSizeX;

      input[binID] /= (denom1 * denom1 + denom2 * denom2);
    }
  }
}

__global__ void multiplyW(const int binCntX,
                          const int binCntY,
                          const int binSizeX,
                          const int binSizeY,
                          const hipfftReal* auv,
                          hipfftReal* inputForX,
                          hipfftReal* inputForY)
{
  const int wID = blockDim.x * blockIdx.x + threadIdx.x;
  const int hID = blockDim.y * blockIdx.y + threadIdx.y;

  if (wID < binCntX && hID < binCntY) {
    int binID = wID + hID * binCntX;

    float w_u = (2.0 * float(FFT_PI) * wID) / binCntX;
    float w_v = (2.0 * float(FFT_PI) * hID) / binCntY * binSizeY / binSizeX;

    inputForX[binID] = w_u * auv[binID];
    inputForY[binID] = w_v * auv[binID];
  }
}

void PoissonSolver::solvePoissonPotential(const float* binDensity,
                                          float* potential)
{
  int numThread = 16;

  dim3 gridSize((binCntX_ + numThread - 1) / numThread,
                (binCntY_ + numThread - 1) / numThread,
                1);

  dim3 blockSize(numThread, numThread, 1);

  // Step #1. Compute Coefficient (a_uv)
  dct_2d_fft(binCntY_,
             binCntX_,
             plan_,
             d_expkM_,
             d_expkN_,
             binDensity,
             d_workSpaceReal1_,
             d_workSpaceComplex_,
             d_auv_);

  // Step #2. Divide by (w_u^2 + w_v^2)
  divideByWSquare<<<gridSize, blockSize>>>(
      binCntX_, binCntY_, binSizeX_, binSizeY_, d_auv_);

  // Step #3. Compute Potential
  idct_2d_fft(binCntY_,
              binCntX_,
              planInverse_,
              d_expkMForInverse_,
              d_expkNForInverse_,
              d_expkMN1_,
              d_expkMN2_,
              d_auv_,
              d_workSpaceComplex_,
              d_workSpaceReal1_,
              potential);
}

void PoissonSolver::solvePoisson(const float* binDensity,
                                 float* potential,
                                 float* electroForceX,
                                 float* electroForceY)
{
  int numThread = 16;

  dim3 gridSize((binCntX_ + numThread - 1) / numThread,
                (binCntY_ + numThread - 1) / numThread,
                1);

  dim3 blockSize(numThread, numThread, 1);

  // Step #1. Compute Coefficient (a_uv)
  dct_2d_fft(binCntY_,
             binCntX_,
             plan_,
             d_expkM_,
             d_expkN_,
             binDensity,
             d_workSpaceReal1_,
             d_workSpaceComplex_,
             d_auv_);

  // Step #2. Divide by (w_u^2 + w_v^2)
  divideByWSquare<<<gridSize, blockSize>>>(
      binCntX_, binCntY_, binSizeX_, binSizeY_, d_auv_);

  // Step #3. Compute Potential
  idct_2d_fft(binCntY_,
              binCntX_,
              planInverse_,
              d_expkMForInverse_,
              d_expkNForInverse_,
              d_expkMN1_,
              d_expkMN2_,
              d_auv_,
              d_workSpaceComplex_,
              d_workSpaceReal1_,
              potential);

  // Step #4. Multiply w_u , w_v
  multiplyW<<<gridSize, blockSize>>>(binCntX_,
                                     binCntY_,
                                     binSizeX_,
                                     binSizeY_,
                                     d_auv_,
                                     d_inputForX_,
                                     d_inputForY_);

  // Step #5. Compute ElectroForceX
  idxst_idct(binCntY_,
             binCntX_,
             planInverse_,
             d_expkMForInverse_,
             d_expkNForInverse_,
             d_expkMN1_,
             d_expkMN2_,
             d_inputForX_,
             d_workSpaceReal1_,
             d_workSpaceComplex_,
             d_workSpaceReal2_,
             d_workSpaceReal3_,
             electroForceX);

  // Step #6. Compute ElectroForceY
  idct_idxst(binCntY_,
             binCntX_,
             planInverse_,
             d_expkMForInverse_,
             d_expkNForInverse_,
             d_expkMN1_,
             d_expkMN2_,
             d_inputForY_,
             d_workSpaceReal1_,
             d_workSpaceComplex_,
             d_workSpaceReal2_,
             d_workSpaceReal3_,
             electroForceY);

  hipDeviceSynchronize();
}

void PoissonSolver::initCUDAKernel()
{
  CUDA_CHECK(hipMalloc((void**) &d_binDensity_,
                        binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(
      hipMalloc((void**) &d_auv_, binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**) &d_potential_,
                        binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(
      hipMalloc((void**) &d_efX_, binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(
      hipMalloc((void**) &d_efY_, binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**) &d_workSpaceReal1_,
                        binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**) &d_workSpaceReal2_,
                        binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**) &d_workSpaceReal3_,
                        binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**) &d_workSpaceComplex_,
                        (binCntX_ / 2 + 1) * binCntY_ * sizeof(hipfftComplex)));

  // expk
  // For DCT2D
  CUDA_CHECK(hipMalloc((void**) &d_expkM_,
                        (binCntY_ / 2 + 1) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void**) &d_expkN_,
                        (binCntX_ / 2 + 1) * sizeof(hipfftComplex)));

  // For IDCT2D & IDXST_IDCT & IDCT_IDXST
  CUDA_CHECK(hipMalloc((void**) &d_expkMForInverse_,
                        (binCntY_) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void**) &d_expkNForInverse_,
                        (binCntX_ / 2 + 1) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void**) &d_expkMN1_,
                        (binCntX_ + binCntY_) * sizeof(hipfftComplex)));

  CUDA_CHECK(hipMalloc((void**) &d_expkMN2_,
                        (binCntX_ + binCntY_) * sizeof(hipfftComplex)));

  // For Input For IDXST_IDCT & IDCT_IDXST
  CUDA_CHECK(hipMalloc((void**) &d_inputForX_,
                        binCntX_ * binCntY_ * sizeof(hipfftReal)));

  CUDA_CHECK(hipMalloc((void**) &d_inputForY_,
                        binCntX_ * binCntY_ * sizeof(hipfftReal)));

  int numThread = 1024;
  int numBin = std::max(binCntX_, binCntY_);
  int numBlock = (numBin - 1 + numThread) / numThread;

  precomputeExpk<<<numBlock, numThread>>>(
      d_expkM_, d_expkN_, binCntY_, binCntX_);

  precomputeExpkForInverse<<<numBlock, numThread>>>(d_expkMForInverse_,
                                                    d_expkNForInverse_,
                                                    d_expkMN1_,
                                                    d_expkMN2_,
                                                    binCntY_,
                                                    binCntX_);

  hipfftPlan2d(&plan_, binCntY_, binCntX_, HIPFFT_R2C);
  hipfftPlan2d(&planInverse_, binCntY_, binCntX_, HIPFFT_C2R);
}

void PoissonSolver::freeCUDAKernel()
{
  CUDA_CHECK(hipFree(d_binDensity_));
  CUDA_CHECK(hipFree(d_auv_));
  CUDA_CHECK(hipFree(d_potential_));

  CUDA_CHECK(hipFree(d_efX_));
  CUDA_CHECK(hipFree(d_efY_));

  CUDA_CHECK(hipFree(d_workSpaceReal1_));
  CUDA_CHECK(hipFree(d_workSpaceReal2_));
  CUDA_CHECK(hipFree(d_workSpaceReal3_));

  CUDA_CHECK(hipFree(d_workSpaceComplex_));

  CUDA_CHECK(hipFree(d_expkN_));
  CUDA_CHECK(hipFree(d_expkM_));

  CUDA_CHECK(hipFree(d_expkNForInverse_));
  CUDA_CHECK(hipFree(d_expkMForInverse_));

  CUDA_CHECK(hipFree(d_expkMN1_));
  CUDA_CHECK(hipFree(d_expkMN2_));

  CUDA_CHECK(hipFree(d_inputForX_));
  CUDA_CHECK(hipFree(d_inputForY_));

  hipfftDestroy(plan_);
  hipfftDestroy(planInverse_);
}

}  // namespace gpl2
