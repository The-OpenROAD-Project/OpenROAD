#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////
//
// BSD 3-Clause License
//
// Copyright (c) 2019, The Regents of the University of California
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
///////////////////////////////////////////////////////////////////////////////

#include <cusp/precond/diagonal.h>
#include <cusp/blas/blas.h>
#include <cusp/krylov/bicgstab.h>

#include "gpuSolver.h"

namespace gpl {
using namespace std;
using utl::GPL;

void GpuSolver::cudaerror(hipError_t code)
{
  if (code != hipSuccess) {
    log_->error(GPL,
                1,
                "[CUDA ERROR] {} at line {} in file {} \n",
                hipGetErrorString(code),
                __LINE__,
                __FILE__);
  }
}
void GpuSolver::cusparseerror(hipsparseStatus_t code)
{
  if (code != HIPSPARSE_STATUS_SUCCESS) {
    log_->error(GPL,
                1,
                "[CUSPARSE ERROR] {} at line {} in file {}\n",
                hipsparseGetErrorString(code),
                __LINE__,
                __FILE__);
  }
}

void GpuSolver::cusolvererror(hipsolverStatus_t code)
{
  if (code != HIPSOLVER_STATUS_SUCCESS) {
    log_->error(GPL,
                1,
                "[CUSOLVER ERROR] {} at line {} in file {}\n",
                hipGetErrorString(*(hipError_t*) &code),
                __LINE__,
                __FILE__);
  }
}

GpuSolver::GpuSolver(SMatrix& placeInstForceMatrix,
                     Eigen::VectorXf& fixedInstForceVec,
                     utl::Logger* logger)
{
  // {cooRowIndex_, cooColIndex_, cooVal_} are the host vectors used to store
  // the sparse format of placeInstForceMatrix.
  nnz_ = placeInstForceMatrix.nonZeros();
  vector<int> cooRowIndex, cooColIndex;
  vector<float> cooVal;
  cooRowIndex.reserve(nnz_);
  cooColIndex.reserve(nnz_);
  cooVal.reserve(nnz_);

  for (int row = 0; row < placeInstForceMatrix.outerSize(); row++) {
    for (typename Eigen::SparseMatrix<float, Eigen::RowMajor>::InnerIterator it(
             placeInstForceMatrix, row);
         it;
         ++it) {
      cooRowIndex.push_back(it.row());
      cooColIndex.push_back(it.col());
      cooVal.push_back(it.value());
    }
  }

  m_ = fixedInstForceVec.size();
  nnz_ = cooVal.size();
  log_ = logger;
  d_cooRowIndex_.resize(nnz_);
  d_cooColIndex_.resize(nnz_);
  d_cooVal_.resize(nnz_);
  d_fixedInstForceVec_.resize(m_);
  d_instLocVec_.resize(m_);

  // Copy the COO formatted triplets to device
  thrust::copy(cooRowIndex.begin(), cooRowIndex.end(), d_cooRowIndex_.begin());
  thrust::copy(cooColIndex.begin(), cooColIndex.end(), d_cooColIndex_.begin());
  thrust::copy(cooVal.begin(), cooVal.end(), d_cooVal_.begin());
  thrust::copy(&fixedInstForceVec[0],
               &fixedInstForceVec[m_ - 1],
               d_fixedInstForceVec_.begin());

  // Set raw pointers to point to the triplets in the device
  r_cooRowIndex_ = thrust::raw_pointer_cast(d_cooRowIndex_.data());
  r_cooColIndex_ = thrust::raw_pointer_cast(d_cooColIndex_.data());
  r_cooVal_ = thrust::raw_pointer_cast(d_cooVal_.data());
  r_fixedInstForceVec_ = thrust::raw_pointer_cast(d_fixedInstForceVec_.data());
  r_instLocVec_ = thrust::raw_pointer_cast(d_instLocVec_.data());
}

void GpuSolver::cusolverCal(Eigen::VectorXf& instLocVec)
{
  // Updated CUDA solver using CUSP library
  thrust::device_ptr<int> p_rowInd
      = thrust::device_pointer_cast(r_cooRowIndex_);
  thrust::device_ptr<int> p_colInd
      = thrust::device_pointer_cast(r_cooColIndex_);
  thrust::device_ptr<float> p_val = thrust::device_pointer_cast(r_cooVal_);
  thrust::device_ptr<float> d_fixedInstForceVec_
      = thrust::device_pointer_cast(r_fixedInstForceVec_);
  thrust::device_ptr<float> p_instLocVec_ = thrust::device_pointer_cast(r_instLocVec_);

  // use array1d_view to wrap the individual arrays
  typedef typename cusp::array1d_view<thrust::device_ptr<int>>
      DeviceIndexArrayView;
  typedef typename cusp::array1d_view<thrust::device_ptr<float>>
      DeviceValueArrayView;
  DeviceIndexArrayView row_indices(p_rowInd, p_rowInd + nnz_);
  DeviceIndexArrayView column_indices(p_colInd, p_colInd + nnz_);
  DeviceValueArrayView values(p_val, p_val + nnz_);
  DeviceValueArrayView d_x(p_instLocVec_, p_instLocVec_ + m_);
  DeviceValueArrayView d_b(d_fixedInstForceVec_, d_fixedInstForceVec_ + m_);

  // combine the three array1d_views into a coo_matrix_view
  typedef cusp::coo_matrix_view<DeviceIndexArrayView,
                                DeviceIndexArrayView,
                                DeviceValueArrayView>
      DeviceView;

  // construct a coo_matrix_view from the array1d_views
  DeviceView d_A(m_, m_, nnz_, row_indices, column_indices, values);

  // set stopping criteria.
  int iteration_limit = 100;
  float relative_tolerance = 1e-15;
  bool verbose = false;  // Decide if the CUDA solver prints the iteration
                         // details or not.
  cusp::monitor<float> monitor_(
      d_b, iteration_limit, relative_tolerance, verbose);

  // setup preconditioner
  cusp::precond::diagonal<float, cusp::device_memory> d_M(d_A);

  // solve the linear system A * x = b with the BICGSTAB method
  cusp::krylov::bicgstab(d_A, d_x, d_b, monitor_, d_M);

  // Sync and Copy data to host
  cudaerror(hipMemcpy(instLocVec.data(),
                       r_instLocVec_,
                       sizeof(float) * m_,
                       hipMemcpyDeviceToHost));

  // Calculate  AX = A * X - B
  cusp::coo_matrix<int, float, cusp::device_memory> A(d_A);
  cusp::array1d<float, cusp::device_memory> X(d_x);
  cusp::array1d<float, cusp::device_memory> B(d_b);
  cusp::array1d<float, cusp::device_memory> AX(m_);
  cusp::multiply(A, X, AX);
  cusp::blas::axpy(B, AX, -1);

  // Calculate L1 norm of the residual vector.
  error_ = cusp::blas::nrm1(AX) / cusp::blas::nrm1(B);
}

float GpuSolver::error()
{
  return (error_ > 0) ? error_ : -error_;
}

GpuSolver::~GpuSolver()
{
}
}  // namespace gpl
