#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <string>
#include <functional>

#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"
#include "stt/SteinerTreeBuilder.h"
#include "gr/FlexGRCMap.h"
#include "gr/FlexGR.h"
#include "frBaseTypes.h"
#include "frDesign.h"
#include "gr/FlexGR_util.h"


namespace drt {


__device__
void addRawDemandSegment__device(
  uint64_t* d_cmap, 
  int xDim, int yDim, int numLayers,
  int parentX, int parentY, int childX, int childY, 
  int childLayerNum)
{


}



__global__
void layerAssignNodeCompute__kernel(  
  NodeStruct* d_nodes,
  unsigned* d_bestLayerCombs,
  unsigned* d_bestLayerCosts,
  uint64_t* d_costMap,
  bool* d_layerDir,
  int xDim, int yDim, int numLayers,
  int nodeStartIdx, int nodeEndIdx, int depth,
  unsigned VIACOST_DEVICE,
  unsigned VIA_ACCESS_LAYERNUM_DEVICE,
  unsigned BLOCKCOST_DEVICE,
  unsigned MARKERCOST_DEVICE)
{
  int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  int nodeId = tIdx + nodeStartIdx;
  if (nodeId >= nodeEndIdx) {
    return;
  }

  NodeStruct& node = d_nodes[nodeId];
  if (node.level != depth) {
    return;
  }

  int numChild = node.childCnt;
  int numComb = 1;
  for (int i = 0; i < numChild; i++) numComb *= numLayers;
  
  printf("Node %d, numComb: %d\n", nodeId, numComb);

  // iterate over all combinations and get the combination with lowest overall cost  
  for (int layerNum = 0; layerNum < numLayers; layerNum++) {
    unsigned currLayerBestCost = UINT_MAX;
    unsigned currLayerBestComb = 0;
    
    // cache node-specific parameters outside the combination loop
    int minPinLayer = node.minLayerNum;
    int maxPinLayer = node.maxLayerNum;
    int parentIdx = node.parentIdx;
    int curLocX = node.x;
    int curLocY = node.y;

    // Iterate over each combination
    for (unsigned comb = 0; comb < numComb; comb++) {
      unsigned currComb = comb; // current combination index
      unsigned downStreamCost = 0;
      unsigned downstreamViaCost = 0;
      int downstreamMinLayerNum = INT_MAX;
      int downstreamMaxLayerNum = INT_MIN;
       
      // Iterate through all children for the current combination
      for (int i = 0; node.childCnt; i++) {
        int childIdx = node.children[i];
        int childLayerNum = currComb % numLayers;  // determine child layer
        downstreamMinLayerNum = min(downstreamMinLayerNum, childLayerNum);
        downstreamMaxLayerNum = max(downstreamMaxLayerNum, childLayerNum);
        currComb /= numLayers; // update combination index for next child
        // Accumulate downstream cost for this child
        downStreamCost += d_bestLayerCosts[childIdx * numLayers + childLayerNum];
      }
      
      downstreamMinLayerNum = min(downstreamMinLayerNum, minPinLayer);
      downstreamMaxLayerNum = max(downstreamMaxLayerNum, maxPinLayer);
      // Compute the number of vias (tune the via cost as needed)
      const unsigned numVias = max(layerNum, maxPinLayer) - min(layerNum, minPinLayer);

      // Compute upstream edge congestion cost
      unsigned congestionCost = 0;
      if (layerNum <= (VIA_ACCESS_LAYERNUM_DEVICE / 2 - 1)) { // Pin layer routing adjustment
        congestionCost += VIACOST_DEVICE * 8;
      }

      // If the node has a parent, compute congestion cost along the connecting edge
      if (parentIdx != -1) {
        int parentX = d_nodes[parentIdx].x;
        int parentY = d_nodes[parentIdx].y;

        if (curLocX == parentX) { // vertical segment
          congestionCost += 100;
        } else if (curLocY == parentY) { // horizontal segment
          congestionCost += 100;
        } else {
          printf("Node %d: current node and parent node are not aligned collinearly\n", nodeId);
        }
      }

      unsigned currLayerCost = downStreamCost + downstreamViaCost + congestionCost;
      if (currLayerCost < currLayerBestCost) {
        currLayerBestCost = currLayerCost;
        currLayerBestComb = comb;
      }
    } // end of combination loop
    
    d_bestLayerCosts[nodeId * numLayers + layerNum] = currLayerBestCost;
    d_bestLayerCombs[nodeId * numLayers + layerNum] = currLayerBestComb;
  } // end of layer loop     
}


// Update the layer assignment for each net
// from top (parent) to bottom (child)
__global__
void layerAssignNodeCommit_kernel(
  NodeStruct* d_nodes,
  const unsigned* d_bestLayerCombs,
  const unsigned* d_bestLayerCosts,
  int xDim, int yDim, int numLayers,
  int nodeStartIdx, int nodeEndIdx,
  int depth)
{
  int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  int nodeId = tIdx + nodeStartIdx;
  if (nodeId >= nodeEndIdx) {
    return;
  }

  NodeStruct& node = d_nodes[nodeId];
  if (node.level != depth) {
    return;
  }

  if (node.parentIdx == -1) { // root node
    frLayerNum minCostLayerNum = 0;
    unsigned minCost = UINT_MAX;
    for (frLayerNum layerNum = 0; layerNum < numLayers; layerNum++) {
      if (d_bestLayerCosts[nodeId * numLayers + layerNum]  < minCost) {
        minCostLayerNum = layerNum;
        minCost = d_bestLayerCosts[nodeId * numLayers + layerNum];
      }
    }

    node.layerNum = minCostLayerNum;
  }

  // Update the layer assignment for each child
  int currLayerNum = node.layerNum;
  int comb = d_bestLayerCombs[nodeId * numLayers + currLayerNum];
  // Iterate through all children for the current combination
  for (int i = 0; node.childCnt; i++) {
    int childIdx = node.children[i];
    d_nodes[childIdx].layerNum = comb % numLayers; // determine child layer
    comb /= numLayers; // update combination index for next child
  }
}
    

// update the congestion map
// We use atomicAdd to update the congestion map
// update all the children segment
__global__
void layerAssignSegmentCommit__kernel(
  NodeStruct* d_nodes,
  uint64_t* d_cmap,
  bool* d_layerDir,
  int xDim, int yDim, int numLayers,
  int nodeStartIdx, int nodeEndIdx, 
  int depth)
{  
  int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  int nodeId = tIdx + nodeStartIdx;
  if (nodeId >= nodeEndIdx) {
    return;
  }

  NodeStruct& node = d_nodes[nodeId];
  if (node.level != depth) {
    return;
  }

  // update the demand for the current node and its children
  int parentX = node.x;
  int parentY = node.y;
  int numChild = node.childCnt; 
  for (int i = 0; node.childCnt; i++) {
    int childIdx = node.children[i];
    NodeStruct& childNode = d_nodes[childIdx];
    int childLayerNum = childNode.layerNum;
    int childX = childNode.x;
    int childY = childNode.y;
    // To be implemented 
    // addRawDemand__device(d_cmap, xDim, yDim, numLayers, parentX, parentY, childX, childY, childLayerNum);
  }
}



// Perform the node level parallelization for the update of the GR layer assignment
void FlexGR::layerAssign_node_compute_CUDA(
  std::vector<unsigned>& bestLayerCosts,
  std::vector<unsigned>& bestLayerCombs,
  std::vector<int>& netBatchNodePtr,
  std::vector<int>& netBatchMaxDepth,
  std::vector<NodeStruct>& nodes)
{    
  int xDim, yDim, zDim;
  cmap_->getDim(xDim, yDim, zDim);
  auto& h_costMap = cmap_->getBits();
  bool* layerDir = new bool[zDim];  // 0 for horizontal, 1 for vertical
  for (int layerNum = 0; layerNum < zDim; layerNum++) {
    auto dir = design_->getTech()->getLayer((layerNum + 1) * 2)->getDir();
    layerDir[layerNum] = (dir == dbTechLayerDir::HORIZONTAL) ? 0 : 1;
  }

  for (int layerNum = 0; layerNum < zDim; layerNum++) {
    std::cout << "Layer " << layerNum << " is " << (layerDir[layerNum] ? "vertical" : "horizontal") << std::endl;
  }

  unsigned VIACOST_DEVICE = router_cfg_->VIACOST;
  unsigned VIA_ACCESS_LAYERNUM_DEVICE = router_cfg_->VIA_ACCESS_LAYERNUM;
  unsigned BLOCKCOST_DEVICE = router_cfg_->BLOCKCOST;
  unsigned MARKERCOST_DEVICE = router_cfg_->MARKERCOST;

  std::cout << "[INFO] VIACOST_DEVICE: " << VIACOST_DEVICE << std::endl;
  std::cout << "[INFO] VIA_ACCESS_LAYERNUM_DEVICE: " << VIA_ACCESS_LAYERNUM_DEVICE << std::endl;
  std::cout << "[INFO] BLOCKCOST_DEVICE: " << BLOCKCOST_DEVICE << std::endl;
  std::cout << "[INFO] MARKERCOST_DEVICE: " << MARKERCOST_DEVICE << std::endl;

  // Allocate memory for the device
  unsigned* d_bestLayerCosts;
  unsigned* d_bestLayerCombs;
  NodeStruct* d_nodes;
  uint64_t* d_costMap;
  bool* d_layerDir;

  hipMalloc(&d_bestLayerCosts, bestLayerCosts.size() * sizeof(unsigned));
  hipMalloc(&d_bestLayerCombs, bestLayerCombs.size() * sizeof(unsigned));
  hipMalloc(&d_nodes, nodes.size() * sizeof(NodeStruct));
  hipMalloc(&d_costMap, h_costMap.size() * sizeof(uint64_t));
  hipMalloc(&d_layerDir, zDim * sizeof(bool));

  hipMemcpy(d_bestLayerCosts, bestLayerCosts.data(), bestLayerCosts.size() * sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(d_bestLayerCombs, bestLayerCombs.data(), bestLayerCombs.size() * sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(d_nodes, nodes.data(), nodes.size() * sizeof(NodeStruct), hipMemcpyHostToDevice);
  hipMemcpy(d_costMap, h_costMap.data(), h_costMap.size() * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_layerDir, layerDir, zDim * sizeof(bool), hipMemcpyHostToDevice);


  int numBatch = netBatchMaxDepth.size();
  // Launch the kernel
  for (int batchId = 0; batchId < numBatch; batchId++) {
    int maxDepth = netBatchMaxDepth[batchId];
    int nodeStartIdx = netBatchNodePtr[batchId];
    int nodeEndIdx = netBatchNodePtr[batchId + 1];
    int numNodes = nodeEndIdx - nodeStartIdx;
    int numThreads = 256;
    int numBlocks = (numNodes + numThreads - 1) / numThreads;
    
    // node-level compute
    for (int depth = maxDepth; depth >= 0; depth--) { 
      layerAssignNodeCompute__kernel<<<numBlocks, numThreads>>>(
        d_nodes,
        d_bestLayerCombs, 
        d_bestLayerCosts,
        d_costMap,
        d_layerDir,
        xDim, yDim, zDim, 
        nodeStartIdx, nodeEndIdx, depth,
        VIACOST_DEVICE, 
        VIA_ACCESS_LAYERNUM_DEVICE,
        BLOCKCOST_DEVICE, 
        MARKERCOST_DEVICE);
    }
    
    hipDeviceSynchronize();  // Wait for the kernel to finish
    cudaCheckError();
  }

  // Copy the results back to the host
  hipMemcpy(bestLayerCosts.data(), d_bestLayerCosts, bestLayerCosts.size() * sizeof(unsigned), hipMemcpyDeviceToHost);
  hipMemcpy(bestLayerCombs.data(), d_bestLayerCombs, bestLayerCombs.size() * sizeof(unsigned), hipMemcpyDeviceToHost);

  // Free the memory
  hipFree(d_bestLayerCosts);
  hipFree(d_bestLayerCombs);
  hipFree(d_nodes);
  hipFree(d_costMap);
  hipFree(d_layerDir);

  delete[] layerDir;
}

} // namespace drt
