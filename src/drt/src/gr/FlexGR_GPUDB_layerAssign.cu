#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR_GPUDB.h"

#include <omp.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <string>

#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"

namespace drt {


/*
__global__ void layerAssignNodeCompute__kernel(  
  NodeStruct* d_nodes,
  unsigned* d_bestLayerCombs,
  unsigned* d_bestLayerCosts,
  int nodeStartIdx, int nodeEndIdx, int depth,
  int xDim, int yDim, int numLayers,
  unsigned VIACOST, unsigned VIA_ACCESS_LAYERNUM,
  unsigned LA_PIN_LAYER_COST_FACTOR,
  int maxChild)
{
  extern __shared__ unsigned s_data[];
  int nodeId = nodeStartIdx + blockIdx.x;
  if (nodeId >= nodeEndIdx) return;

  // Shared memory pointers setup
  unsigned* s_childCosts = s_data;
  unsigned* s_fixedCostPerLayer = s_data + maxChild * numLayers;
  unsigned* s_reductionCosts = s_fixedCostPerLayer + numLayers;
  unsigned* s_reductionCombs = s_reductionCosts + blockDim.x;

  // Load node info to shared memory
  __shared__ int s_childCnt, s_minPinLayer, s_maxPinLayer, s_parentIdx, s_curLocX, s_curLocY;
  if (threadIdx.x == 0) {
    NodeStruct node = d_nodes[nodeId];
    s_childCnt = node.childCnt;
    s_minPinLayer = node.minLayerNum;
    s_maxPinLayer = node.maxLayerNum;
    s_parentIdx = node.parentIdx;
    s_curLocX = node.x;
    s_curLocY = node.y;
    
    // Handle invalid nodes immediately
    if (node.level != depth || s_childCnt > maxChild) {
      s_childCnt = -1; // Mark as invalid
    }
  }
  __syncthreads();

  // Skip invalid nodes
  if (s_childCnt == -1) return;
  if (s_childCnt == 0) {  // Leaf node special case
    if (threadIdx.x == 0) {
      for (int layerNum = 0; layerNum < numLayers; layerNum++) {
        unsigned numVias = max(layerNum, s_maxPinLayer) - min(layerNum, s_minPinLayer);
        unsigned viaCost = VIACOST * numVias;
        unsigned congestionCost = 0;
        if (layerNum <= (VIA_ACCESS_LAYERNUM / 2 - 1)) {
          congestionCost += VIACOST * LA_PIN_LAYER_COST_FACTOR;
        }
        if (s_parentIdx != -1) {
          congestionCost += 100; // Simplified alignment cost
        }
        d_bestLayerCosts[nodeId * numLayers + layerNum] = viaCost + congestionCost;
        d_bestLayerCombs[nodeId * numLayers + layerNum] = 0;
      }
    }
    return;
  }

  // Load child costs to shared memory
  if (threadIdx.x == 0) {
    NodeStruct node = d_nodes[nodeId];
    for (int i = 0; i < s_childCnt; i++) {
      int childIdx = node.children[i];
      for (int l = 0; l < numLayers; l++) {
        s_childCosts[i * numLayers + l] = d_bestLayerCosts[childIdx * numLayers + l];
      }
    }
  }
  __syncthreads();

  // Precompute fixed costs per layer
  if (threadIdx.x < numLayers) {
    int layerNum = threadIdx.x;
    unsigned numVias = max(layerNum, s_maxPinLayer) - min(layerNum, s_minPinLayer);
    unsigned viaCost = VIACOST * numVias;
    unsigned congestionCost = 0;
    
    if (layerNum <= (VIA_ACCESS_LAYERNUM / 2 - 1)) {
      congestionCost += VIACOST * LA_PIN_LAYER_COST_FACTOR;
    }
    if (s_parentIdx != -1) {
      congestionCost += 100; // Simplified alignment cost
    }
    s_fixedCostPerLayer[layerNum] = viaCost + congestionCost;
  }
  __syncthreads();

  // Process each layer combination
  for (int layerNum = 0; layerNum < numLayers; layerNum++) {
    unsigned fixedCost = s_fixedCostPerLayer[layerNum];
    unsigned long long numComb = 1;
    for (int i = 0; i < s_childCnt; i++) {
      numComb *= numLayers;
    }

    // Parallelize combination evaluation
    unsigned long long combPerThread = (numComb + blockDim.x - 1) / blockDim.x;
    unsigned long long start = threadIdx.x * combPerThread;
    unsigned long long end = min(start + combPerThread, numComb);

    unsigned myMinCost = UINT_MAX;
    unsigned myBestComb = 0;

    // Evaluate combinations in parallel
    for (unsigned long long comb = start; comb < end; comb++) {
      unsigned temp = comb;
      unsigned cost = 0;
      for (int i = 0; i < s_childCnt; i++) {
        int layer_i = temp % numLayers;
        temp /= numLayers;
        cost += s_childCosts[i * numLayers + layer_i];
      }
      if (cost < myMinCost) {
        myMinCost = cost;
        myBestComb = comb;
      }
    }

    // Parallel reduction for best combination
    s_reductionCosts[threadIdx.x] = myMinCost;
    s_reductionCombs[threadIdx.x] = myBestComb;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
      if (threadIdx.x < stride) {
        if (s_reductionCosts[threadIdx.x + stride] < s_reductionCosts[threadIdx.x]) {
          s_reductionCosts[threadIdx.x] = s_reductionCosts[threadIdx.x + stride];
          s_reductionCombs[threadIdx.x] = s_reductionCombs[threadIdx.x + stride];
        }
      }
      __syncthreads();
    }

    // Store results
    if (threadIdx.x == 0) {
      d_bestLayerCosts[nodeId * numLayers + layerNum] = s_reductionCosts[0] + fixedCost;
      d_bestLayerCombs[nodeId * numLayers + layerNum] = s_reductionCombs[0];
    }
    __syncthreads();
  }
}
*/








__global__
void layerAssignNodeCompute__kernel(  
  NodeStruct* d_nodes,
  unsigned* d_bestLayerCombs,
  unsigned* d_bestLayerCosts,
  int nodeStartIdx, int nodeEndIdx, int depth,
  uint64_t* d_cmap, bool* d_layerDir,
  int xDim, int yDim, int numLayers,
  unsigned VIACOST, unsigned VIA_ACCESS_LAYERNUM,
  unsigned BLOCKCOST, unsigned MARKERCOST,
  unsigned LA_PIN_LAYER_COST_FACTOR)
{
  int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nodeId = tIdx + nodeStartIdx;
  if (nodeId >= nodeEndIdx) { return;  } // Out of bounds 

  // Copy to register
  NodeStruct node = d_nodes[nodeId];
  if (node.level != depth) { return; }

  int numChild = node.childCnt;
  int numComb = 1;
  #pragma unroll
  for (int i = 0; i < 4; i++) // Unroll if max 4 children
    if (i < numChild) numComb *= numLayers;
  
  // cache node-specific parameters outside the combination loop
  const int minPinLayer = node.minLayerNum;
  const int maxPinLayer = node.maxLayerNum;
  const int curLocX = node.x;
  const int curLocY = node.y;
  int parentIdx = node.parentIdx;
  // Preload children into local array (max 4)
  int children[4];  
  #pragma unroll
  for (int i = 0; i < 4; i++)
    children[i] = node.children[i];

  // iterate over all combinations and get the combination with lowest overall cost  
  for (int layerNum = 0; layerNum < numLayers; layerNum++) {
    unsigned currLayerBestCost = UINT_MAX;
    unsigned currLayerBestComb = 0;
    // Iterate over each combination
    for (unsigned comb = 0; comb < numComb; comb++) {
      unsigned currComb = comb; // current combination index
      unsigned downStreamCost = 0;
      int downstreamMinLayerNum = INT_MAX;
      int downstreamMaxLayerNum = INT_MIN;
      // Iterate through all children for the current combination
      // For each child, determine its layer in this combination and add cost
      #pragma unroll
      for (int i = 0; i < 4; i++) {
        if (i < numChild) {
          int childIdx = children[i];
          int childLayerNum = currComb % numLayers;  // determine child layer
          downstreamMinLayerNum = min(downstreamMinLayerNum, childLayerNum);
          downstreamMaxLayerNum = max(downstreamMaxLayerNum, childLayerNum);
          currComb /= numLayers; // update combination index for next child
          // Accumulate downstream cost for this child
          downStreamCost += d_bestLayerCosts[childIdx * numLayers + childLayerNum];
        }
      }
      
      downstreamMinLayerNum = min(downstreamMinLayerNum, minPinLayer);
      downstreamMaxLayerNum = max(downstreamMaxLayerNum, maxPinLayer);
      const unsigned downstreamViaCost =
        (max(layerNum, maxPinLayer) - min(layerNum, minPinLayer)) * VIACOST;

      // Compute upstream edge congestion cost
      // Pin layer routing penalty
      unsigned congestionCost = 0;
      if (layerNum <= (VIA_ACCESS_LAYERNUM / 2 - 1)) { // Pin layer routing adjustment
        congestionCost += VIACOST * LA_PIN_LAYER_COST_FACTOR;
      }

      // If the node has a parent, compute congestion cost along the connecting edge
      if (parentIdx != -1) {
        int parentX = d_nodes[parentIdx].x;
        int parentY = d_nodes[parentIdx].y;
        if (curLocX == parentX) { // vertical segment
          bool isLayerBlocked = d_layerDir[layerNum] == false; // false means horizontal
          congestionCost += getSegmentCostV__device(
            d_cmap, curLocX, min(curLocY, parentY), max(curLocY, parentY), layerNum,
            xDim, yDim, numLayers, isLayerBlocked, BLOCKCOST, MARKERCOST);
        } else if (curLocY == parentY) { // horizontal segment
          bool isLayerBlocked = d_layerDir[layerNum] == true; // true means vertical
          congestionCost += getSegmentCostH__device(
            d_cmap, min(curLocX, parentX), max(curLocX, parentX), curLocY, layerNum,
            xDim, yDim, numLayers, isLayerBlocked, BLOCKCOST, MARKERCOST);
        } else {
          printf("LANodeCompute Error Node %d: current node and parent node are not aligned collinearly\n", nodeId);
        }
      }

      unsigned currLayerCost = downStreamCost + downstreamViaCost + congestionCost;
      if (currLayerCost < currLayerBestCost) {
        currLayerBestCost = currLayerCost;
        currLayerBestComb = comb;
      }
    } // end of combination loop
    
    d_bestLayerCosts[nodeId * numLayers + layerNum] = currLayerBestCost;
    d_bestLayerCombs[nodeId * numLayers + layerNum] = currLayerBestComb;
  } // end of layer loop     
}



__global__
void layerAssignNodeCommit__kernel(  
  NodeStruct* d_nodes,
  unsigned* d_bestLayerCombs,
  unsigned* d_bestLayerCosts,
  int nodeStartIdx, int nodeEndIdx, int depth,
  uint64_t* d_cmap, int xDim, int yDim, int numLayers)
{
  int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nodeId = tIdx + nodeStartIdx;
  if (nodeId >= nodeEndIdx) { return;  } // Out of bounds 

  NodeStruct& node = d_nodes[nodeId];
  if (node.level != depth) { return; }
  
  if (node.level == 0) { // root node
    unsigned minCost = UINT_MAX;
    int bestLayerNum = -1;
    for (int l = 0; l < numLayers; l++) {
      unsigned cost = d_bestLayerCosts[nodeId * numLayers + l];
      if (cost < minCost) {
        minCost = cost;
        bestLayerNum = l;
      }
    }
    node.layerNum = bestLayerNum;
  }

  // Decode the child's layer from bestComb
  int numChild = node.childCnt;
  unsigned comb = d_bestLayerCombs[nodeId * numLayers + node.layerNum];
  for (int i = 0; i < numChild; i++) { 
    int myLayer = comb % numLayers;
    comb /= numLayers;
    int childId = node.children[i];
    d_nodes[childId].layerNum = myLayer; // Assign layer to child node  
    // update cmap
    int childX = d_nodes[childId].x;
    int childY = d_nodes[childId].y;
    int parentX = node.x;
    int parentY = node.y;
    if (childX == parentX) { // vertical segment
      addSegmentV__device(d_cmap, min(childY, parentY), max(childY, parentY), childX, myLayer,
        xDim, yDim, numLayers);
    } else if (childY == parentY) { // horizontal segment
      addSegmentH__device(d_cmap, min(childX, parentX), max(childX, parentX), childY, myLayer,
        xDim, yDim, numLayers);
    } else {
      printf("LANodeCommit Error Node %d: current node and parent node are not aligned collinearly\n", nodeId);
    }
  }
}


// To do: large-net with highest depth should be done in CPU mode
// instead of GPU mode
// It seems that we do not the batches and netBatchPtr in this function
void FlexGRGPUDB::layerAssign_CUDA(
  std::vector<frNet*>& sortedNets,
  std::vector<int>& batches,
  std::vector<int>& netBatchPtr,
  std::vector<NodeStruct>& nodes,
  std::vector<int>& netBatchMaxDepth,
  std::vector<int>& netBatchNodePtr)
{
  int totNumNodes = nodes.size();

  NodeStruct* d_nodes;
  unsigned* d_bestLayerCombs;
  unsigned* d_bestLayerCosts;

  hipMalloc((void**)&d_nodes, nodes.size() * sizeof(NodeStruct));
  hipMalloc((void**)&d_bestLayerCombs, totNumNodes * sizeof(unsigned));
  hipMalloc((void**)&d_bestLayerCosts, totNumNodes * sizeof(unsigned));

  // Copy the data from the host to the device
  hipMemcpy(d_nodes, nodes.data(), nodes.size() * sizeof(NodeStruct), hipMemcpyHostToDevice);
  hipMemset(d_bestLayerCombs, 0, totNumNodes * sizeof(unsigned));
  hipMemset(d_bestLayerCosts, UINT_MAX, totNumNodes * sizeof(unsigned));

  // sync CMap
  syncCMapHostToDevice();

  cudaCheckError();

  if (debugMode_) {
    for (auto& node : nodes) {
      // check if the node is aligned with its children 
      for (int i = 0; i < node.childCnt; i++) {
        auto& childNode = nodes[node.children[i]];
        if (node.x != childNode.x && node.y != childNode.y) {    
          std::cout << "[ERROR] FlexGRGPUDB::layerAssign_CUDA: "
                    << "Node " << node.nodeIdx
                    << " is not aligned with its child node "
                    << childNode.nodeIdx << ".\n";
          std::cout << "Number of children: " << static_cast<int>(node.childCnt) << "\n";
          // print all the child nodes
          std::cout << "Child nodes: ";
          for (int j = 0; j < node.childCnt; j++) {
            auto& child = nodes[node.children[j]];
            std::cout << child.nodeIdx << " (x: " << child.x
                      << ", y: " << child.y << ") ";
          }
          std::cout << "\n";
          std::cout << "node.x = " << node.x
                    << ", node.y = " << node.y
                    << ", childNode.x = " << childNode.x
                    << ", childNode.y = " << childNode.y << "\n";
          if (childNode.parentIdx == -1) {
            std::cout << "Child node is a root node, no parent to check.\n";
          } else {
            // Print parent node information
            auto parentNode = nodes[childNode.parentIdx];
            std::cout << "Parent node: "
                      << parentNode.nodeIdx << ", "
                      << "x = " << parentNode.x
                      << ", y = " << parentNode.y << "\n";
          }
          std::cout << "Exiting due to misalignment.\n";
          exit(1); 
        }
      }
      
      
      if (node.parentIdx == -1) {
        continue; // Skip root nodes
      }

      auto& parentNode = nodes[node.parentIdx];
      if (node.x != parentNode.x && node.y != parentNode.y) {
        std::cout << "[ERROR] FlexGRGPUDB::layerAssign_CUDA: "
                  << "Node " << node.nodeIdx
                  << " is not aligned with its parent node "
                  << parentNode.nodeIdx << ".\n";
        std::cout << "node.x = " << node.x
                  << ", node.y = " << node.y
                  << ", parentNode.x = " << parentNode.x
                  << ", parentNode.y = " << parentNode.y << "\n";
        exit(1); 
      }
    }
  }


  // Node-level layer assignment kernel
  int numBatches = netBatchMaxDepth.size();
  // launch the kernel with one thread per node
  for (int batchId = 0; batchId < numBatches; batchId++) {
    int maxDepth = netBatchMaxDepth[batchId];
    int nodeStartIdx = netBatchNodePtr[batchId];
    int nodeEndIdx = netBatchNodePtr[batchId + 1];
    int numNodes = nodeEndIdx - nodeStartIdx;
    int numThreads = 256;
    int numBlocks = (numNodes + numThreads - 1) / numThreads;
   
    if (debugMode_) {
      std::string msg = std::string("[INFO] ")
                      + std::string("FlexGRGPUDB::layerAssign_CUDA: ")
                      + "batchId: " + std::to_string(batchId) + ", "
                      + "maxDepth: " + std::to_string(maxDepth) + ", "
                      + "nodeStartIdx: " + std::to_string(nodeStartIdx) + ", "
                      + "nodeEndIdx: " + std::to_string(nodeEndIdx) + ", "
                      + "numNodes: " + std::to_string(numNodes) + ", "
                      + "numBlocks: " + std::to_string(numBlocks) + ", "
                      + "numThreads: " + std::to_string(numThreads);
      logger_->report(msg);
    }    

    // node-level compute
    for (int depth = maxDepth; depth >= 0; depth--) { 
      layerAssignNodeCompute__kernel<<<numBlocks, numThreads>>>(
        d_nodes, d_bestLayerCombs, d_bestLayerCosts,
        nodeStartIdx, nodeEndIdx, depth,
        d_cmap_bits_3D, d_layerDir, xDim, yDim, zDim, 
        router_cfg_->VIACOST,
        router_cfg_->VIA_ACCESS_LAYERNUM,
        router_cfg_->BLOCKCOST,
        router_cfg_->MARKERCOST,
        LA_PIN_LAYER_COST_FACTOR);
    }

    hipDeviceSynchronize();  // Wait for the kernel to finish
    cudaCheckError();
   
    for (int depth = 0; depth < maxDepth; depth++) {
      layerAssignNodeCommit__kernel<<<numBlocks, numThreads>>>(
        d_nodes, d_bestLayerCombs, d_bestLayerCosts,
        nodeStartIdx, nodeEndIdx, depth, 
        d_cmap_bits_3D, xDim, yDim, zDim);
    }
  
    hipDeviceSynchronize();  // Wait for the kernel to finish
    cudaCheckError();
  }

  cudaCheckError();  

  // copy the solution from the device to the host
  hipMemcpy(nodes.data(), d_nodes, nodes.size() * sizeof(NodeStruct), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_nodes);
  hipFree(d_bestLayerCombs);
  hipFree(d_bestLayerCosts);
  cudaCheckError();
}





}  // namespace drt
