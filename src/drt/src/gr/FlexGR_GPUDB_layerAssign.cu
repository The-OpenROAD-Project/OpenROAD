#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR_GPUDB.h"

#include <omp.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <string>

#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"

namespace drt {

__global__
void layerAssignNodeCompute__kernel(  
  NodeStruct* d_nodes,
  unsigned* d_bestLayerCombs,
  unsigned* d_bestLayerCosts,
  int nodeStartIdx, int nodeEndIdx, int depth,
  uint64_t* d_cmap, bool* d_layerDir,
  int xDim, int yDim, int numLayers,
  unsigned VIACOST, unsigned VIA_ACCESS_LAYERNUM,
  unsigned BLOCKCOST, unsigned MARKERCOST,
  unsigned LA_PIN_LAYER_COST_FACTOR)
{
  int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nodeId = tIdx + nodeStartIdx;
  if (nodeId >= nodeEndIdx) { return;  } // Out of bounds 

  // Copy to register
  NodeStruct node = d_nodes[nodeId];
  if (node.level != depth) { return; }

  int numChild = node.childCnt;
  int numComb = 1;
  #pragma unroll
  for (int i = 0; i < 4; i++) // Unroll if max 4 children
    if (i < numChild) numComb *= numLayers;
  
  // cache node-specific parameters outside the combination loop
  const int minPinLayer = node.minLayerNum;
  const int maxPinLayer = node.maxLayerNum;
  const int curLocX = node.x;
  const int curLocY = node.y;
  int parentIdx = node.parentIdx;
  // Preload children into local array (max 4)
  int children[4];  
  #pragma unroll
  for (int i = 0; i < 4; i++)
    children[i] = node.children[i];

  // iterate over all combinations and get the combination with lowest overall cost  
  for (int layerNum = 0; layerNum < numLayers; layerNum++) {
    unsigned currLayerBestCost = UINT_MAX;
    unsigned currLayerBestComb = 0;
    // Iterate over each combination
    for (unsigned comb = 0; comb < numComb; comb++) {
      unsigned currComb = comb; // current combination index
      unsigned downStreamCost = 0;
      int downstreamMinLayerNum = INT_MAX;
      int downstreamMaxLayerNum = INT_MIN;
      // Iterate through all children for the current combination
      // For each child, determine its layer in this combination and add cost
      #pragma unroll
      for (int i = 0; i < 4; i++) {
        if (i < numChild) {
          int childIdx = children[i];
          int childLayerNum = currComb % numLayers;  // determine child layer
          downstreamMinLayerNum = min(downstreamMinLayerNum, childLayerNum);
          downstreamMaxLayerNum = max(downstreamMaxLayerNum, childLayerNum);
          currComb /= numLayers; // update combination index for next child
          // Accumulate downstream cost for this child
          downStreamCost += d_bestLayerCosts[childIdx * numLayers + childLayerNum];
        }
      }
      
      downstreamMinLayerNum = min(downstreamMinLayerNum, minPinLayer);
      downstreamMaxLayerNum = max(downstreamMaxLayerNum, maxPinLayer);
      const unsigned downstreamViaCost =
        (max(layerNum, maxPinLayer) - min(layerNum, minPinLayer)) * VIACOST;

      // Compute upstream edge congestion cost
      // Pin layer routing penalty
      unsigned congestionCost = 0;
      if (layerNum <= (VIA_ACCESS_LAYERNUM / 2 - 1)) { // Pin layer routing adjustment
        congestionCost += VIACOST * LA_PIN_LAYER_COST_FACTOR;
      }

      // If the node has a parent, compute congestion cost along the connecting edge
      if (parentIdx != -1) {
        int parentX = d_nodes[parentIdx].x;
        int parentY = d_nodes[parentIdx].y;

        if (curLocX == parentX) { // vertical segment
          congestionCost += 100;
        } else if (curLocY == parentY) { // horizontal segment
          congestionCost += 100;
        } else {
          printf("Node %d: current node and parent node are not aligned collinearly\n", nodeId);
        }
      }

      unsigned currLayerCost = downStreamCost + downstreamViaCost + congestionCost;
      if (currLayerCost < currLayerBestCost) {
        currLayerBestCost = currLayerCost;
        currLayerBestComb = comb;
      }
    } // end of combination loop
    
    d_bestLayerCosts[nodeId * numLayers + layerNum] = currLayerBestCost;
    d_bestLayerCombs[nodeId * numLayers + layerNum] = currLayerBestComb;
  } // end of layer loop     
}



// To do: large-net with highest depth should be done in CPU mode
// instead of GPU mode

void FlexGRGPUDB::layerAssign_CUDA(
  std::vector<frNet*>& sortedNets,
  std::vector<std::vector<int> >& batches,
  std::vector<NodeStruct>& nodes,
  std::vector<int>& netBatchMaxDepth,
  std::vector<int>& netBatchNodePtr)
{
  int totNumNodes = nodes.size();

  NodeStruct* d_nodes;
  unsigned* d_bestLayerCombs;
  unsigned* d_bestLayerCosts;

  hipMalloc((void**)&d_nodes, nodes.size() * sizeof(NodeStruct));
  hipMalloc((void**)&d_bestLayerCombs, totNumNodes * sizeof(unsigned));
  hipMalloc((void**)&d_bestLayerCosts, totNumNodes * sizeof(unsigned));

  // Copy the data from the host to the device
  hipMemcpy(d_nodes, nodes.data(), nodes.size() * sizeof(NodeStruct), hipMemcpyHostToDevice);
  hipMemset(d_bestLayerCombs, 0, totNumNodes * sizeof(unsigned));
  hipMemset(d_bestLayerCosts, UINT_MAX, totNumNodes * sizeof(unsigned));

  // sync CMap
  syncCMapHostToDevice();

  cudaCheckError();

  // Node-level layer assignment kernel
  int numBatches = netBatchMaxDepth.size();
  // launch the kernel with one thread per node
  for (int batchId = 0; batchId < numBatches; batchId++) {
    int maxDepth = netBatchMaxDepth[batchId];
    int nodeStartIdx = netBatchNodePtr[batchId];
    int nodeEndIdx = netBatchNodePtr[batchId + 1];
    int numNodes = nodeEndIdx - nodeStartIdx;
    int numThreads = 256;
    int numBlocks = (numNodes + numThreads - 1) / numThreads;
   
    if (debugMode_) {
      std::string msg = std::string("[INFO] ")
                      + std::string("FlexGRGPUDB::layerAssign_CUDA: ")
                      + "batchId: " + std::to_string(batchId) + ", "
                      + "maxDepth: " + std::to_string(maxDepth) + ", "
                      + "nodeStartIdx: " + std::to_string(nodeStartIdx) + ", "
                      + "nodeEndIdx: " + std::to_string(nodeEndIdx) + ", "
                      + "numNodes: " + std::to_string(numNodes) + ", "
                      + "numBlocks: " + std::to_string(numBlocks) + ", "
                      + "numThreads: " + std::to_string(numThreads);
      logger_->report(msg);
    }    

    // node-level compute
    for (int depth = maxDepth; depth >= 0; depth--) { 
      layerAssignNodeCompute__kernel<<<numBlocks, numThreads>>>(
        d_nodes, d_bestLayerCombs, d_bestLayerCosts,
        nodeStartIdx, nodeEndIdx, depth,
        d_cmap_bits_3D, d_layerDir, xDim, yDim, zDim, 
        router_cfg_->VIACOST,
        router_cfg_->VIA_ACCESS_LAYERNUM,
        router_cfg_->BLOCKCOST,
        router_cfg_->MARKERCOST,
        LA_PIN_LAYER_COST_FACTOR);
    }
    
    hipDeviceSynchronize();  // Wait for the kernel to finish
    cudaCheckError();
  }


  cudaCheckError();  

  // copy the solution from the device to the host
  hipMemcpy(nodes.data(), d_nodes, nodes.size() * sizeof(NodeStruct), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_nodes);
  hipFree(d_bestLayerCombs);
  hipFree(d_bestLayerCosts);
  cudaCheckError();
}





}  // namespace drt
