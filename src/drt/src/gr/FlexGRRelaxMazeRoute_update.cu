#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2024, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <string>
#include <functional>

#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"
#include "FlexGR_util.h"
#include "gr/FlexGR.h"
#include "stt/SteinerTreeBuilder.h"
#include "gr/FlexGR_util_update.h"
#include "gr/FlexGRCMap.h"
#include "gr/FlexGR_util_update.h"

namespace drt {

// Device function for obtaining the cost for each grid
__device__
float getGridCost2D_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  int xIdx,
  int yIdx,
  frDirEnum dir,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  int zIdx = 0;
  int zDim = 1; // 2D Map
  float cost = 0;
  auto supply = getRawSupply_device(d_cmap, xDim, yDim, zDim, xIdx, yIdx, zIdx, dir) * congestionThresh;
  auto demand = getRawDemand_device(d_cmap, xDim, yDim, zDim, xIdx, yIdx, zIdx, dir);
  
  // congestion cost
  float exp_val = exp(min(10.0f, static_cast<float>(supply) - demand));  
  // Calculate the factor safely
  float factor = 8.0f / (1.0f + exp_val); 
  // Compute congestion cost with safety limits
  float congCost = demand * (1.0f + factor) / (supply + 1.0f);
  cost += congCost;

  // blockage cost
  if (hasBlock_device(d_cmap, xDim, yDim, zDim, xIdx, yIdx, zIdx, dir)) {
    cost += BLOCKCOST_DEVICE * 100;
  }
   
  // overflow cost
  if (demand >= supply) {
    cost += 128;
  }

  // history cost
  cost += 4 * congCost * getHistoryCost_device(d_cmap, xDim, yDim, zDim, xIdx, yIdx, zIdx);    
  return cost;
}


// Device function for computing segment cost
__device__
float compute_segment_cost_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  int startIdx,
  int endIdx,
  int fixedIdx, 
  frDirEnum dir,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  float cost = 0;
  int minIdx = min(startIdx, endIdx);
  int maxIdx = max(startIdx, endIdx);
  for (int idx = minIdx; idx <= maxIdx; idx++) {
    int xIdx = (dir == frDirEnum::E) ? idx : fixedIdx;
    int yIdx = (dir == frDirEnum::E) ? fixedIdx : idx;
    cost += getGridCost2D_device(d_cmap, xDim, yDim, xIdx, yIdx, dir, congestionThresh, BLOCKCOST_DEVICE);
  }
  return cost;
}


// define the device function for I-Shape kernel
__device__
float compute_I_shape_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  IntPair srcLoc,
  IntPair dstLoc,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE) 
{
  if (srcLoc.x() == dstLoc.x() && srcLoc.y() == dstLoc.y()) {
    return 0.0;
  } 
  
  if (srcLoc.x() == dstLoc.x()) { // vertical
    return compute_segment_cost_device(
      d_cmap, xDim, yDim, srcLoc.y(), dstLoc.y(), srcLoc.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);
  } 
  
  if (srcLoc.y() == dstLoc.y()) { // horizontal
    return compute_segment_cost_device(
      d_cmap, xDim, yDim, srcLoc.x(), dstLoc.x(), srcLoc.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);
  } 

  return 0;
}


// define the device function for L-Shape kernel
__device__ 
float compute_L_Shape_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  IntPair srcLoc,
  IntPair dstLoc,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  if (srcLoc.x() == dstLoc.x() || srcLoc.y() == dstLoc.y()) {
    return compute_I_shape_device(d_cmap, xDim, yDim, srcLoc, dstLoc, congestionThresh, BLOCKCOST_DEVICE);
  }

  // Check the two possible L-Shape
  IntPair corner1 = {srcLoc.x(), dstLoc.y()};
  IntPair corner2 = {dstLoc.x(), srcLoc.y()};

  float cost1 = compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.y(), corner1.y(), srcLoc.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE) 
    + compute_segment_cost_device(d_cmap, xDim, yDim, corner1.x(), dstLoc.x(), corner1.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);  

  float cost2 = compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.x(), corner2.x(), srcLoc.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE) 
    + compute_segment_cost_device(d_cmap, xDim, yDim, corner2.y(), dstLoc.y(), corner2.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);
  
  return std::min(cost1, cost2);
}


__device__
float compute_Z_Shape_util_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  IntPair srcLoc,
  IntPair dstLoc,
  IntPair corner1, 
  IntPair corner2,
  float  congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  // I-Shape
  if (srcLoc.x() == dstLoc.x() || srcLoc.y() == dstLoc.y()) {
    return compute_I_shape_device(d_cmap, xDim, yDim, srcLoc, dstLoc, congestionThresh, BLOCKCOST_DEVICE);
  }

  // L-Shape
  if (corner1.x() == corner2.x() && corner1.y() == corner2.y()) {
    if (corner1.x() == srcLoc.x()) { // vertical first
      float cost = 0.0;
      cost += compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.y(), corner1.y(), srcLoc.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);
      cost += compute_segment_cost_device(d_cmap, xDim, yDim, corner1.x(), dstLoc.x(), dstLoc.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);
      return cost;
    } else { // horizontal first
      float cost = 0.0;
      cost += compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.x(), corner1.x(), srcLoc.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);
      cost += compute_segment_cost_device(d_cmap, xDim, yDim, corner1.y(), dstLoc.y(), dstLoc.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);
      return cost;
    }
  }

  if (srcLoc.x() == corner1.x()) { // vertical first
    float cost = 0.0;
    cost += compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.y(), corner1.y(), srcLoc.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);
    cost += compute_segment_cost_device(d_cmap, xDim, yDim, corner1.x(), corner2.x(), corner1.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);
    cost += compute_segment_cost_device(d_cmap, xDim, yDim, corner2.y(), dstLoc.y(), dstLoc.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);
    return cost;
  } else { // horizontal first
    float cost = 0.0;
    cost += compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.x(), corner1.x(), srcLoc.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);
    cost += compute_segment_cost_device(d_cmap, xDim, yDim, corner1.y(), corner2.y(), corner1.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);
    cost += compute_segment_cost_device(d_cmap, xDim, yDim, corner2.x(), dstLoc.x(), dstLoc.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);
    return cost;
  }
 
  printf("Error_A !!!\n");
  return FLT_MAX;
}




__device__
IntPair compute_L_shape_corner_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  IntPair srcLoc,
  IntPair dstLoc,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  if (srcLoc.x() == dstLoc.x() || srcLoc.y() == dstLoc.y()) {
    return IntPair(-1, -1);
  }

  // Check the two possible L-Shape
  IntPair corner1 = {srcLoc.x(), dstLoc.y()};
  IntPair corner2 = {dstLoc.x(), srcLoc.y()};

  float cost1 = compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.y(), corner1.y(), srcLoc.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE) 
    + compute_segment_cost_device(d_cmap, xDim, yDim, corner1.x(), dstLoc.x(), corner1.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE);  

  float cost2 = compute_segment_cost_device(d_cmap, xDim, yDim, srcLoc.x(), corner2.x(), srcLoc.y(), frDirEnum::E, congestionThresh, BLOCKCOST_DEVICE) 
    + compute_segment_cost_device(d_cmap, xDim, yDim, corner2.y(), dstLoc.y(), corner2.x(), frDirEnum::N, congestionThresh, BLOCKCOST_DEVICE);

  //printf("cost1 = %f, cost2 = %f\n, corner1 = (%d, %d), corner2 = (%d, %d), srcLoc = (%d, %d), dstLoc = (%d, %d)\n", 
  //  cost1, cost2, corner1.x(), corner1.y(), corner2.x(), corner2.y(), srcLoc.x(), srcLoc.y(), dstLoc.x(), dstLoc.y());

  if (cost1 < cost2) {
    return corner1;
  } else {
    return corner2;
  }
}


// Device function for commit the segment
__device__
void commit_segment_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  IntPair srcLoc,
  IntPair dstLoc)
{
  int zIdx = 0;
  int zDim = 1; // 2D Map
  int minX = min(srcLoc.start, dstLoc.start);
  int minY = min(srcLoc.end, dstLoc.end);
  int maxX = max(srcLoc.start, dstLoc.start);
  int maxY = max(srcLoc.end, dstLoc.end);
  
  if (minY == maxY) { // horizontal segment
    for (int xIdx = minX; xIdx < minY; xIdx++) {
      addRawDemand_device(d_cmap, xDim, yDim, zDim, xIdx, minY, zIdx, frDirEnum::E);
      addRawDemand_device(d_cmap, xDim, yDim, zDim, xIdx + 1, minY, zIdx, frDirEnum::E);
    }
  } else if (minX == maxX) { // vertical segment
    for (int yIdx = minY; yIdx < maxY; yIdx++) {
      addRawDemand_device(d_cmap, xDim, yDim, zDim, minX, yIdx, zIdx, frDirEnum::N);
      addRawDemand_device(d_cmap, xDim, yDim, zDim, minX, yIdx + 1, zIdx, frDirEnum::N);
    }
  } else {
    printf("Error ! current node and parent node are are not aligned collinearly in commit_segment_device\n");
  }
}



// Define unions for atomic operations
union FloatUInt {
  float f = FLT_MAX;
  unsigned int ui;
};

union CostComb {
  unsigned long long int uint64;
  struct {
    unsigned int costBits;
    unsigned int combIdx;
  } data;
};

// Atomic function to update best cost and combination index
__device__
void atomicMinCostComb(unsigned long long int* address, float cost, unsigned int combIdx) {
  CostComb old, assumed, desired;

  // Convert cost to its bit representation
  FloatUInt costBits;
  costBits.f = cost;
  desired.data.costBits = costBits.ui;
  desired.data.combIdx = combIdx;
  old.uint64 = *address;
  while (true) {
    assumed.uint64 = old.uint64;
    // Extract the assumed cost
    FloatUInt assumedCostBits;
    assumedCostBits.ui = assumed.data.costBits;
    float assumedCost = assumedCostBits.f;
    // If the current best cost is less than or equal, no need to update
    if (assumedCost <= cost) {
      break;
    }

    // Attempt to update atomically
    old.uint64 = atomicCAS(address, assumed.uint64, desired.uint64);
    // If the value hasn't changed, update was successful
    if (old.uint64 == assumed.uint64) {
      break;
    }
  }
}


// Define the device function for general Z-shape pattern routing
__device__
float compute_Z_Shape_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  IntPair srcLoc,
  IntPair dstLoc,
  float  congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  if (srcLoc.x() == dstLoc.x() || srcLoc.y() == dstLoc.y()) {
    return compute_I_shape_device(d_cmap, xDim, yDim, srcLoc, dstLoc, congestionThresh, BLOCKCOST_DEVICE);
  }
   
  float cost = FLT_MAX;
  IntPair corner1, corner2;
  int lx = min(srcLoc.x(), dstLoc.x());
  int ux = max(srcLoc.x(), dstLoc.x());
  int ly = min(srcLoc.y(), dstLoc.y());
  int uy = max(srcLoc.y(), dstLoc.y());

  // horizontal first
  for (int x = lx + 1; x <= ux; x++) {
    corner1 = {x, srcLoc.y()};
    corner2 = {x, dstLoc.y()};
    float tempCost = compute_Z_Shape_util_device(
      d_cmap, xDim, yDim, srcLoc, dstLoc, corner1, corner2, congestionThresh, BLOCKCOST_DEVICE);
    if (tempCost < cost) {
      cost = tempCost;
    } 
  }

  // vertical first
  for (int y = ly + 1; y <= uy; y++) {
    corner1 = {srcLoc.x(), y};
    corner2 = {dstLoc.x(), y};
    float tempCost = compute_Z_Shape_util_device(
      d_cmap, xDim, yDim, srcLoc, dstLoc, corner1, corner2, congestionThresh, BLOCKCOST_DEVICE);
    if (tempCost < cost) {
      cost = tempCost;
    } 
  }

  if (cost == FLT_MAX) {
    printf("Error_B !!! Cost is FLT_MAX !!!\n");
  }

  return cost;
}



__device__
IntPair compute_Z_shape_corner_device(
  uint64_t* d_cmap,
  int xDim,
  int yDim,
  IntPair srcLoc,
  IntPair dstLoc,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  if (srcLoc.x() == dstLoc.x() || srcLoc.y() == dstLoc.y()) {
    return IntPair(-1, -1);
  }

  float bestCost = FLT_MAX;
  IntPair bestCorner = IntPair(-1, -1);
  
  IntPair corner1, corner2;
  int lx = min(srcLoc.x(), dstLoc.x());
  int ux = max(srcLoc.x(), dstLoc.x());
  int ly = min(srcLoc.y(), dstLoc.y());
  int uy = max(srcLoc.y(), dstLoc.y());

  // horizontal first
  for (int x = lx + 1; x <= ux; x++) {
    corner1 = {x, srcLoc.y()};
    corner2 = {x, dstLoc.y()};
    float cost = compute_Z_Shape_util_device(
      d_cmap, xDim, yDim, srcLoc, dstLoc, corner1, corner2, congestionThresh, BLOCKCOST_DEVICE);
    if (cost < bestCost) {
      bestCost = cost;
      bestCorner = corner1;
    } 
  }

  // vertical first
  for (int y = ly + 1; y <= uy; y++) {
    corner1 = {srcLoc.x(), y};
    corner2 = {dstLoc.x(), y};
    float cost = compute_Z_Shape_util_device(
      d_cmap, xDim, yDim, srcLoc, dstLoc, corner1, corner2, congestionThresh, BLOCKCOST_DEVICE);
    if (cost < bestCost) {
      bestCost = cost;
      bestCorner = corner1;
    } 
  }

  return bestCorner;
}


__global__ 
void steiner_loc_process_combinations_kernel(
    float* d_nodeLocBestCost,
    unsigned* d_nodeLocBestComb,
    uint64_t* d_cmap,
    const IntPair* d_nodeLoc,
    const IntPair* d_nodeLocPtr,
    const int* d_nodeEdgeIdx,
    const int* d_nodeLevel,
    int nodeId,
    int depth,
    int nodeLocStartIdx,
    int numLocs,
    int numComb,
    int childStartIdx,
    int childEndIdx,
    int maxNumLocs,
    int xDim,
    int yDim,
    float congestionThresh,
    unsigned BLOCKCOST_DEVICE)
{
    extern __shared__ char sharedMemory[];
    IntPair* sharedLoc = (IntPair*)sharedMemory;
    // Ensure proper alignment for unsigned long long int
    size_t offset = sizeof(IntPair);
    size_t padding = (8 - (offset % 8)) % 8;
    unsigned long long int* sharedBestCostComb = (unsigned long long int*)(sharedMemory + offset + padding);
    unsigned long long int& bestCostComb = sharedBestCostComb[0];

    int locIdx = blockIdx.x; // Parent location index
    // Calculate thread's unique index
    int threadId = threadIdx.x + threadIdx.y * blockDim.x;
    int totalThreads = blockDim.x * blockDim.y;
    bool active = true;
    if (locIdx >= numLocs) { active = false; }

    // Load parent location into shared memory
    int nodeLocIdx = nodeLocStartIdx + locIdx;
    if (threadId == 0) {
      sharedLoc[0] = d_nodeLoc[nodeLocIdx];
      FloatUInt costBits;
      costBits.f = FLT_MAX;
      bestCostComb = ((unsigned long long int)(UINT_MAX) << 32) | costBits.ui;
    }
    __syncthreads();



    // Process combinations in a loop
    for (int combIdx = threadId; combIdx < numComb; combIdx += totalThreads) {
      if (!active || combIdx >= numComb) { continue; }
      const auto& parentLoc = sharedLoc[0];
      int numChildren = childEndIdx - childStartIdx;
      unsigned currComb = combIdx;
      float cost = 0.0f;
      bool valid = true;
      int validError = -1;

      // Compute cost for this combination
      for (int childIdx = childStartIdx; childIdx < childEndIdx; childIdx++) {
        int childNodeIdx = d_nodeEdgeIdx[childIdx];    
        int numChildLocs = d_nodeLocPtr[childNodeIdx].end - d_nodeLocPtr[childNodeIdx].start;
        if (numChildLocs <= 0) {
          valid = false;
          validError = 1;
          break;
        }
 
        int locOffset = currComb % maxNumLocs;
        currComb /= maxNumLocs;

        int childLocIdx = d_nodeLocPtr[childNodeIdx].start + locOffset;
        // Bounds check (optional now, but can keep for safety)
        if (childLocIdx >= d_nodeLocPtr[childNodeIdx].end) {
          valid = false;
          validError = 2;
          break;
        }
      
        float childCost = d_nodeLocBestCost[childLocIdx];
        if (childCost == FLT_MAX) {
          validError = 3;
          // check the child information
          for (int idx = d_nodeLocPtr[childNodeIdx].start; idx < d_nodeLocPtr[childNodeIdx].end; idx++) {
            printf("childLocIdx = %d, childCost = %f, childNodeIdx = %d, childDepth = %d, locIdx = %d, nodeId = %d, depth = %d, childLoc.x = %d, childLoc.y = %d\n", 
              idx, d_nodeLocBestCost[idx], childNodeIdx, d_nodeLevel[childNodeIdx], locIdx, nodeId, depth,
              d_nodeLoc[idx].x(), d_nodeLoc[idx].y());
          }
          printf("Child cost is FLT_MAX !!! locIdx = %d, nodeId = %d, depth = %d, comIdx = %d\n", 
            locIdx, nodeId, depth, combIdx);
          valid = false;
          break;
        } else {
          cost += childCost;
        }

        //float L_cost = compute_L_Shape_device(
        //  d_cmap, xDim, yDim, parentLoc, d_nodeLoc[childLocIdx], congestionThresh, BLOCKCOST_DEVICE);
        
        float L_cost = compute_Z_Shape_device(
            d_cmap, xDim, yDim, parentLoc, d_nodeLoc[childLocIdx], congestionThresh, BLOCKCOST_DEVICE);

        if (L_cost == FLT_MAX) {
          validError = 4;
          printf("L_cost is FLT_MAX !!! locIdx = %d, nodeId = %d, depth = %d, comIdx = %d\n", 
            locIdx, nodeId, depth, combIdx);
          valid = false;
          break;
        } else {
          cost += L_cost;
        }
      }

      //if (nodeId == 8613 || nodeId == 7495) {
      //  printf("nodeId = %d, valid = %d, validError = %d, cost = %f, locIdx = %d, CombIdx = %d, numComb = %d, numChildren = %d, childStartIdx = %d, childEndIdx = %d\n", 
      //    nodeId, valid, validError,
      //    cost, locIdx, combIdx, numComb, numChildren, childStartIdx, childEndIdx);
      //} 

      if (valid) {
        atomicMinCostComb(&bestCostComb, cost, combIdx);
        if (cost == FLT_MAX) {
          printf("Cost is FLT_MAX !!! locIdx = %d, nodeId = %d, depth = %d, comIdx = %d, numComb = %d\n", 
            locIdx, nodeId, depth, combIdx, numComb);
        }
      }          
    }
    __syncthreads();

    // Write final results back to global memory
    if (threadId == 0) {
      // Extract cost and combIdx from bestCostComb
      CostComb result;
      result.uint64 = bestCostComb;
      FloatUInt costBits;
      costBits.ui = result.data.costBits;
      float bestCost = costBits.f;
      unsigned int bestComb = result.data.combIdx;
      d_nodeLocBestCost[nodeLocIdx] = bestCost;
      d_nodeLocBestComb[nodeLocIdx] = bestComb;
    }

    __syncthreads();

    if (d_nodeLocBestCost[nodeLocIdx] == FLT_MAX) {
      printf("BestCost is FLT_MAX !!! locIdx = %d, bestComb = %d, depth = %d, nodeId = %d, numComb = %d\n", 
        locIdx, d_nodeLocBestComb[nodeLocIdx], depth, nodeId, numComb);
    }
}


/*
__global__
void steiner_loc_process_combinations_kernel(
  float* d_nodeLocBestCost,
  unsigned* d_nodeLocBestComb,
  uint64_t* d_cmap,
  const IntPair* d_nodeLoc,
  const IntPair* d_nodeLocPtr,
  const int* d_nodeEdgeIdx,
  const int* d_nodeLevel,
  int nodeId, // just for debugging
  int depth,
  int nodeLocStartIdx,
  int numLocs,
  int numComb,
  int childStartIdx,
  int childEndIdx,
  int maxNumLocs,
  int xDim,
  int yDim,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  extern __shared__ char sharedMemory[];
  IntPair* sharedLoc = (IntPair*)sharedMemory;
  size_t offset = sizeof(IntPair);
  size_t padding = (8 - (offset % 8)) % 8;
  unsigned long long int* sharedBestCostComb = (unsigned long long int*)(sharedMemory + offset + padding);
  unsigned long long int& bestCostComb = sharedBestCostComb[0];

  int locIdx = blockIdx.x; // Parent location index
  int totalThreadsPerBlock = blockDim.x * blockDim.y;
  int combIdx = threadIdx.x + threadIdx.y * blockDim.x; // Combination index
  bool active = true;
  if (locIdx >= numLocs || combIdx >= numComb) { active = false; }

  // Load parent location into shared memory (1 location per block)
  int nodeLocIdx = nodeLocStartIdx + locIdx;
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    sharedLoc[0] = d_nodeLoc[nodeLocIdx];
    FloatUInt costBits;
    costBits.f = FLT_MAX;
    bestCostComb = ((unsigned long long int)(UINT_MAX) << 32) | costBits.ui;
  }  
  __syncthreads();

  if (active) {
    const auto& parentLoc = sharedLoc[0];
    int numChildren = childEndIdx - childStartIdx;
    unsigned currComb = combIdx;
    float cost = 0.0f;
    bool valid = true;

    // Compute cost for this combination
    for (int childIdx = childStartIdx; childIdx < childEndIdx; childIdx++) {
      int childNodeIdx = d_nodeEdgeIdx[childIdx];    
      int numChildLocs = d_nodeLocPtr[childNodeIdx].end - d_nodeLocPtr[childNodeIdx].start;
      if (numChildLocs <= 0) {
        valid = false;
        break;
      }
 
      int locOffset = currComb % maxNumLocs;
      currComb /= maxNumLocs;
    
      int childLocIdx = d_nodeLocPtr[childNodeIdx].start + locOffset;
      // Bounds check (optional now, but can keep for safety)
      if (childLocIdx >= d_nodeLocPtr[childNodeIdx].end) {
        valid = false;
        break;
      }
      
      float childCost = d_nodeLocBestCost[childLocIdx];
      if (childCost == FLT_MAX) {
        // check the child information
        for (int idx = d_nodeLocPtr[childNodeIdx].start; idx < d_nodeLocPtr[childNodeIdx].end; idx++) {
          printf("childLocIdx = %d, childCost = %f, childNodeIdx = %d, childDepth = %d, locIdx = %d, nodeId = %d, depth = %d, childLoc.x = %d, childLoc.y = %d\n", 
            idx, d_nodeLocBestCost[idx], childNodeIdx, d_nodeLevel[childNodeIdx], locIdx, nodeId, depth,
            d_nodeLoc[idx].x(), d_nodeLoc[idx].y());
        }
        printf("Child cost is FLT_MAX !!! locIdx = %d, nodeId = %d, depth = %d, comIdx = %d\n", 
          locIdx, nodeId, depth, combIdx);
        valid = false;
        break;
      } else {
        cost += childCost;
      }

      float L_cost = compute_L_Shape_device(
        d_cmap, xDim, yDim, parentLoc, d_nodeLoc[childLocIdx], congestionThresh, BLOCKCOST_DEVICE);

      if (L_cost == FLT_MAX) {
        printf("L_cost is FLT_MAX !!! locIdx = %d, nodeId = %d, depth = %d, comIdx = %d\n", 
          locIdx, nodeId, depth, combIdx);
        valid = false;
        break;
      } else {
        cost += L_cost;
      }
    }
  
    if (valid) {
      atomicMinCostComb(&bestCostComb, cost, combIdx);
    }  
  }

  __syncthreads();
  // Write final results back to global memory
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    // Extract cost and combIdx from bestCostComb
    CostComb result;
    result.uint64 = bestCostComb;
    FloatUInt costBits;
    costBits.ui = result.data.costBits;
    float bestCost = costBits.f;
    unsigned int bestComb = result.data.combIdx;
    d_nodeLocBestCost[nodeLocIdx] = bestCost;
    d_nodeLocBestComb[nodeLocIdx] = bestComb;
   
    //if (bestCost == FLT_MAX) {
    //  printf("BestCost is FLT_MAX !!! locIdx = %d, bestComb = %d, depth = %d, nodeId = %d\n", 
    //    locIdx, bestComb, depth, nodeId);
    //}
  }

  __syncthreads();

  if (d_nodeLocBestCost[nodeLocIdx] == FLT_MAX) {
    printf("BestCost is FLT_MAX !!! locIdx = %d, bestComb = %d, depth = %d, nodeId = %d\n", 
      locIdx, d_nodeLocBestComb[nodeLocIdx], depth, nodeId);
  }
}
*/

__global__ 
void steiner_node_compute_update_kernel(
  float* d_nodeLocBestCost,
  unsigned* d_nodeLocBestComb,
  uint64_t* d_cmap, // congestion map
  const int* d_netBatch,
  const int* d_nodeCntPtr, // store the connection of the net
  const int* d_nodeLevel,
  const IntPair* d_nodeLoc,
  const IntPair* d_nodeLocPtr,
  const int* d_nodeEdgeIdx, // store the relationship between the nodes
  const IntPair* d_nodeEdgePtr,
  int xDim,
  int yDim,
  int maxNumNodes,
  int maxNumLocs, // maximum number of locations for each node
  int batchStartIdx,
  int batchEndIdx,
  int depth,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  //int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  int threadId = threadIdx.x + threadIdx.y * blockDim.x;
  int tIdx = blockIdx.x * (blockDim.x * blockDim.y) + threadId;
  //int tIdx = threadIdx.x + threadIdx.y * blockDim.x;
  int netIdx = tIdx / maxNumNodes + batchStartIdx;
  if (netIdx >= batchEndIdx)  { return; }

  int netId = d_netBatch[netIdx];
  int nodeId = tIdx % maxNumNodes + d_nodeCntPtr[netId];  
  if (nodeId >= d_nodeCntPtr[netId + 1]) { return; }

  int nodeLevel = d_nodeLevel[nodeId];
  if (nodeLevel != depth) { return; }
  
  int childStartIdx = d_nodeEdgePtr[nodeId].start;
  int childEndIdx = d_nodeEdgePtr[nodeId].end; 

  int nodeLocStartIdx = d_nodeLocPtr[nodeId].start;
  int nodeLocEndIdx = d_nodeLocPtr[nodeId].end;
  int numLocs = nodeLocEndIdx - nodeLocStartIdx;  
  int numChildren = childEndIdx - childStartIdx;

  if (numChildren == 0) {
    d_nodeLocBestCost[nodeLocStartIdx] = 0.0f;
    d_nodeLocBestComb[nodeLocStartIdx] = 0;
    return;
  }

  int numComb = 1;
  for (int i = 0; i < numChildren; i++) {
    numComb *= maxNumLocs;
  }

  //dim3 blockDim(256); // Threads per block
  //dim3 gridDim(numLocs, (numComb + blockDim.x - 1) / blockDim.x);
  //dim3 blockDim(32, (numComb + 32 - 1) / 32);
  int blockDimY = min(32, (numComb + 32 - 1) / 32);
  dim3 blockDim(32, blockDimY);
  dim3 gridDim(numLocs, 1);
 
  //printf("numLocs = %d, numComb = %d  "
  //       "blockDim = %d  %d  "
  //       "gridDim = %d  %d\n", 
  //       numLocs, numComb, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
  
  size_t offset = sizeof(IntPair);
  size_t padding = (8 - (offset % 8)) % 8;
  size_t sharedMemSize = offset + padding + sizeof(unsigned long long int);
  // Launch the kernel for loc level computation
  steiner_loc_process_combinations_kernel<<<gridDim, blockDim, sharedMemSize>>>(
    d_nodeLocBestCost,
    d_nodeLocBestComb,
    d_cmap,
    d_nodeLoc,
    d_nodeLocPtr,
    d_nodeEdgeIdx,
    d_nodeLevel,
    nodeId,
    depth,
    nodeLocStartIdx,
    numLocs,
    numComb,
    childStartIdx,
    childEndIdx,
    maxNumLocs,
    xDim,
    yDim,
    congestionThresh,
    BLOCKCOST_DEVICE);

  // hipDeviceSynchronize();
  // Check for errors after kernel launch
  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    printf("Error in kernel launch (steiner_node_compute_update_kernel (test)): %s\n", hipGetErrorString(cudaStatus));
  }
}


// Determine the optimal location for the each steiner node
// from level 0 to the last level
__global__ 
void steiner_node_commit_update_kernel(
  float* d_nodeLocBestCost,
  unsigned* d_nodeLocBestComb,
  int* d_bestOptimalLoc, // store the loc id
  const int* d_netBatch,
  const int* d_nodeCntPtr, // store the connection of the net
  const int* d_nodeLevel,
  const IntPair* d_nodeLoc,
  const IntPair* d_nodeLocPtr,
  const int* d_nodeEdgeIdx, // store the relationship between the nodes
  const IntPair* d_nodeEdgePtr,
  const int* d_nodeParentIdx,
  int maxNumNodes,
  int maxNumLocs,
  int batchStartIdx,
  int batchEndIdx,
  int depth)
{
  //int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  //int tIdx = threadIdx.x + threadIdx.y * blockDim.x;
  int threadId = threadIdx.x + threadIdx.y * blockDim.x;
  int tIdx = blockIdx.x * (blockDim.x * blockDim.y) + threadId;
  int netIdx = tIdx / maxNumNodes + batchStartIdx;

  // Check if the thread is out of bound
  if (netIdx >= batchEndIdx) return;

  int netId = d_netBatch[netIdx];
  int nodeId = tIdx % maxNumNodes + d_nodeCntPtr[netId];  
  if (nodeId >= d_nodeCntPtr[netId + 1]) return;

  int nodeLevel = d_nodeLevel[nodeId];
  if (nodeLevel != depth) return;

  int nodeLocStartIdx = d_nodeLocPtr[nodeId].start;
  int nodeLocEndIdx = d_nodeLocPtr[nodeId].end;
  int numLocs = nodeLocEndIdx - nodeLocStartIdx;

  int parentIdx = d_nodeParentIdx[nodeId];
  if (parentIdx == -1) { // root node
    int bestLocId = -1;
    float bestCost = FLT_MAX;
    for (int idx = nodeLocStartIdx; idx < nodeLocEndIdx; idx++) {
      if (d_nodeLocBestCost[idx] < bestCost) {
        bestCost = d_nodeLocBestCost[idx];
        bestLocId = idx;
      }
    }
    d_bestOptimalLoc[nodeId] = bestLocId;
  }  
  
  // update the location of the child nodes
  int childStartIdx = d_nodeEdgePtr[nodeId].start;
  int childEndIdx = d_nodeEdgePtr[nodeId].end; 
  int comb = d_nodeLocBestComb[d_bestOptimalLoc[nodeId]];
  //if (comb > 0) { 
  //  printf("nodeId = %d, bestLocId = %d, comb = %d\n", nodeId, d_bestOptimalLoc[nodeId], comb);
  //}

  for (int childIdx = childStartIdx; childIdx < childEndIdx; childIdx++) {
    int childNodeIdx = d_nodeEdgeIdx[childIdx];
    d_bestOptimalLoc[childNodeIdx] = comb % maxNumLocs + d_nodeLocPtr[childNodeIdx].start;
    comb /= maxNumLocs;
  }

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    printf("Error in kernel launch (steiner_node_commit_update_kernel): %s\n", hipGetErrorString(cudaStatus));
  }
}
    

// Compute the corner location for the nodes and commit the segment
// From level 0 to the last level
__global__ 
void corner_node_update_kernel(
  IntPair* d_cornerLoc,
  int* d_bestOptimalLoc, // store the loc id
  uint64_t* d_cmap, // congestion map
  const int* d_netBatch,
  const int* d_nodeCntPtr, // store the connection of the net
  const int* d_nodeLevel,
  const IntPair* d_nodeLoc,
  const IntPair* d_nodeLocPtr,
  const int* d_nodeEdgeIdx, // store the relationship between the nodes
  const IntPair* d_nodeEdgePtr,
  int xDim,
  int yDim,
  int maxNumNodes,
  int batchStartIdx,
  int batchEndIdx,
  int depth,
  float congestionThresh,
  unsigned BLOCKCOST_DEVICE)
{
  //int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  //int tIdx = threadIdx.x + threadIdx.y * blockDim.x;
  int threadId = threadIdx.x + threadIdx.y * blockDim.x;
  int tIdx = blockIdx.x * (blockDim.x * blockDim.y) + threadId;
  int netIdx = tIdx / maxNumNodes + batchStartIdx;

  // Check if the thread is out of bound
  if (netIdx >= batchEndIdx) return;
  int netId = d_netBatch[netIdx];
  
  int nodeId = tIdx % maxNumNodes + d_nodeCntPtr[netId];  
  if (nodeId >= d_nodeCntPtr[netId + 1]) return;

  int nodeLevel = d_nodeLevel[nodeId];
  if (nodeLevel != depth) return;
  
  int childStartIdx = d_nodeEdgePtr[nodeId].start;
  int childEndIdx = d_nodeEdgePtr[nodeId].end; 

  if (childEndIdx == childStartIdx) { // leaf node
    return;
  }

  auto& parentLoc = d_nodeLoc[d_bestOptimalLoc[nodeId]];
  // Determine the corner location for each child node
  for (int childIdx = childStartIdx; childIdx < childEndIdx; childIdx++) {
    int childNodeIdx = d_nodeEdgeIdx[childIdx];
    auto& childLoc = d_nodeLoc[d_bestOptimalLoc[childNodeIdx]];
    d_cornerLoc[childNodeIdx] = compute_L_shape_corner_device(d_cmap,
      xDim, yDim, parentLoc, childLoc, congestionThresh, BLOCKCOST_DEVICE);
    // Commit the segment
    if (d_cornerLoc[childNodeIdx].x() == -1 || d_cornerLoc[childNodeIdx].y() == -1) {
      commit_segment_device(d_cmap, xDim, yDim, parentLoc, childLoc);
    } else {
      if (childLoc.x() == d_cornerLoc[childNodeIdx].x() && 
          childLoc.y() == d_cornerLoc[childNodeIdx].y()) {
        printf("Error ! The corner location is the same as the child location\n");
      } else if (parentLoc.x() == d_cornerLoc[childNodeIdx].x() && 
                 parentLoc.y() == d_cornerLoc[childNodeIdx].y()) {
        printf("Error ! The corner location is the same as the parent location\n");
      }
      
      commit_segment_device(d_cmap, xDim, yDim, parentLoc, d_cornerLoc[childNodeIdx]);
      commit_segment_device(d_cmap, xDim, yDim, d_cornerLoc[childNodeIdx], childLoc);
    }
  }

  // Check for errors after kernel launch
  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    printf("Error in kernel launch (corner_node_update_kernel): %s\n", hipGetErrorString(cudaStatus));
  }
}

// Use DAG-Based Approach for updating the Steiner Tree 
// Basic idea: We do not explicitly change the structure of the Steiner Tree.
// We assume the location of the Steiner nodes are not fixed.
// Then use greedy approach to determine the location of the Steiner nodes.
// Let's work on the 2D Grid Grpah
// Here we only need to consider the nets with congestion
void FlexGR::RRR_SteinerTreeShift(std::vector<frNet*>& nets2RR)
{
  logger_->report("[INFO][FlexGR] Start Steiner Tree Shift...");
  int gridXSize = xDim_;
  int gridYSize = yDim_;
  for (auto& net : nets2RR) {
    initGR_ripup_update(net);
  }

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  printf("Number of GPUs available: %d\n", deviceCount);
  hipSetDevice(1); // Switch to GPU 1

  // Function for converting 2D index to 1d index  
  auto getIdx = [&](int x, int y) { return y * gridXSize + x; };
  std::vector<uint64_t> congestion_map;
  congestion_map.resize(gridXSize * gridYSize);
  for (int xIdx = 0; xIdx < gridXSize; xIdx++) {
    for (int yIdx = 0; yIdx < gridYSize; yIdx++) {
      congestion_map[getIdx(xIdx, yIdx)] = cmap2D_->getOriginalBits(xIdx, yIdx, 0);
    }
  }  

  std::cout << "test a" << std::endl;
  removeLoop(nets2RR);
  checkValidNet_update();

  //removeLoop(nets2RR);
  // Step 1:  Perform overlap detection to generate net batch
  std::vector<std::vector<int> > netBatch;
  batchGeneration_update(nets2RR, netBatch);  

  // Step 2: node levelization
  int totalNumNodes = 0;
  int maxNumNodes = 0; // Maximum number of nodes for each net
  int maxNumChildren = 0; // Maximum number of children for each node 
  int maxNumLocs = 0; // Maximum number of locations for each node
  std::vector<int> nodeCntPtrVec; // store the connection of the net
  std::vector<int> nodeLevel; // store the depth of the node
  std::vector<IntPair> locVec; // store the location of the node
  std::vector<IntPair> nodeLocPtr;
  std::vector<int> nodeParentIdx;  // store the parent and children relationship
  std::vector<int> nodeEdgeIdx;
  std::vector<IntPair> nodeEdgePtr;
  std::vector<int> netBatchMaxDepth;

  initGR_node_levelization_update(
    nets2RR, 
    netBatch, 
    gridXSize, gridYSize, 
    totalNumNodes, maxNumNodes, maxNumChildren, maxNumLocs,
    nodeCntPtrVec, 
    nodeLevel, 
    locVec, nodeLocPtr,
    nodeParentIdx, nodeEdgeIdx, nodeEdgePtr, 
    netBatchMaxDepth);

  // Define the block cost
  unsigned BLOCKCOST_DEVICE = BLOCKCOST;
  std::vector<float> nodeLocBestCost(locVec.size(), std::numeric_limits<float>::max());
  std::vector<unsigned> nodeLocBestComb(locVec.size(), std::numeric_limits<unsigned>::max());
  
  // translate the netBatch into 1D array
  std::vector<int> netBatch_1D;
  netBatch_1D.reserve(nets2RR.size());
  std::vector<int> netBatchPtr;
  netBatchPtr.reserve(netBatch.size() + 1);
  for (auto& netVec : netBatch) {
    netBatchPtr.push_back(netBatch_1D.size());
    netBatch_1D.insert(netBatch_1D.end(), 
                       std::make_move_iterator(netVec.begin()),
                       std::make_move_iterator(netVec.end()));
  }
  netBatchPtr.push_back(netBatch_1D.size());
  std::vector<int> bestOptimalLoc(totalNumNodes, -1);
  std::vector<IntPair> cornerLoc(totalNumNodes, {-1, -1}); // We use the L-Shape as the initial connection

  int* d_netBatch;
  uint64_t* d_congestion_map;  
  int* d_nodeCntPtr;  
  int* d_nodeLevel;
  IntPair* d_nodeLoc;
  IntPair* d_nodeLocPtr;
  int* d_nodeParentIdx;
  int* d_nodeEdgeIdx;
  IntPair* d_nodeEdgePtr;
  int* d_netBatchMaxDepth;
  float* d_nodeLocBestCost;
  unsigned* d_nodeLocBestComb;
  int* d_bestOptimalLoc;
  IntPair* d_cornerLoc;
  
  hipMalloc(&d_netBatch, netBatch_1D.size() * sizeof(int));
  hipMalloc(&d_congestion_map, congestion_map.size() * sizeof(uint64_t));
  hipMalloc(&d_nodeCntPtr, nodeCntPtrVec.size() * sizeof(int));
  hipMalloc(&d_nodeLevel, nodeLevel.size() * sizeof(int));
  hipMalloc(&d_nodeLoc, locVec.size() * sizeof(IntPair));
  hipMalloc(&d_nodeLocPtr, nodeLocPtr.size() * sizeof(IntPair));
  hipMalloc(&d_nodeParentIdx, nodeParentIdx.size() * sizeof(int));
  hipMalloc(&d_nodeEdgeIdx, nodeEdgeIdx.size() * sizeof(int));
  hipMalloc(&d_nodeEdgePtr, nodeEdgePtr.size() * sizeof(IntPair));
  hipMalloc(&d_netBatchMaxDepth, netBatchMaxDepth.size() * sizeof(int));
  hipMalloc(&d_nodeLocBestCost, nodeLocBestCost.size() * sizeof(float));
  hipMalloc(&d_nodeLocBestComb, nodeLocBestComb.size() * sizeof(unsigned));
  hipMalloc(&d_bestOptimalLoc, bestOptimalLoc.size() * sizeof(int));
  hipMalloc(&d_cornerLoc, cornerLoc.size() * sizeof(IntPair));

  hipMemcpy(d_netBatch, netBatch_1D.data(), netBatch_1D.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_congestion_map, congestion_map.data(), congestion_map.size() * sizeof(uint64_t), hipMemcpyHostToDevice);  
  hipMemcpy(d_nodeCntPtr, nodeCntPtrVec.data(), nodeCntPtrVec.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeLevel, nodeLevel.data(), nodeLevel.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeLoc, locVec.data(), locVec.size() * sizeof(IntPair), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeLocPtr, nodeLocPtr.data(), nodeLocPtr.size() * sizeof(IntPair), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeParentIdx, nodeParentIdx.data(), nodeParentIdx.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeEdgeIdx, nodeEdgeIdx.data(), nodeEdgeIdx.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeEdgePtr, nodeEdgePtr.data(), nodeEdgePtr.size() * sizeof(IntPair), hipMemcpyHostToDevice);
  hipMemcpy(d_netBatchMaxDepth, netBatchMaxDepth.data(), netBatchMaxDepth.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeLocBestCost, nodeLocBestCost.data(), nodeLocBestCost.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_nodeLocBestComb, nodeLocBestComb.data(), nodeLocBestComb.size() * sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(d_bestOptimalLoc, bestOptimalLoc.data(), bestOptimalLoc.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_cornerLoc, cornerLoc.data(), cornerLoc.size() * sizeof(IntPair), hipMemcpyHostToDevice);
 
  logger_->report("[INFO][FlexGR] Done CUDA memory initialization...\n");
  
  // Launch the kernel
  int numBatch = netBatch.size();
  int maxDepth = 0;
  std::cout << "Number of net batches: " << numBatch << std::endl;
  std::cout << "Launching the kernel..." << std::endl;

  for (int batchId = 0; batchId < numBatch; batchId++) {
    maxDepth = netBatchMaxDepth[batchId];
    //std::cout << "Batch id : " << batchId << "  " << "Max depth: " << maxDepth << std::endl;
    // Launch the kernel
    int startIdx = netBatchPtr[batchId];
    int endIdx = netBatchPtr[batchId + 1];
    int numNets = endIdx - startIdx;
    int numNodes = numNets * maxNumNodes; // total number of threads needed
    int numThreads = 64;
    int numBlocks = (numNodes + numThreads - 1) / numThreads;

    /* 
    int numSubBatches = (numThreads * numBlocks) / 1024 + 1;
    numSubBatches = std::min(numSubBatches, numNets > 0 ? numNets : 1); // Ensure numSubBatches <= numNets
    int subBatchSize = (numNets + numSubBatches - 1) / numSubBatches;
    subBatchSize = std::max(subBatchSize, 1); // Ensure subBatchSize >= 1

    int subBatchStartIdx = 0;
    int subBatchEndIdx = 0;
    for (int subBatchId = 0; subBatchId < numSubBatches; subBatchId++) {
      subBatchStartIdx = subBatchEndIdx;
      subBatchEndIdx = std::min(subBatchStartIdx + subBatchSize, numNets);
      int subBatchNumNets = subBatchEndIdx - subBatchStartIdx;
    
      if (subBatchNumNets == 0) {
        continue; // Skip empty sub-batches
      }
    
      int subBatchNumNodes = subBatchNumNets * maxNumNodes;
      int subBatchNumThreads = 64;
      int subBatchNumBlocks = (subBatchNumNodes + subBatchNumThreads - 1) / subBatchNumThreads;
      subBatchNumBlocks = std::max(subBatchNumBlocks, 1); // Ensure at l

      std::cout << "subBatchNumNets = " << subBatchNumNets << "  "
                << "subBatchNumNodes = " << subBatchNumNodes << "  "
                << "subBatchNumThreads = " << subBatchNumThreads << "  "
                << "subBatchNumBlocks = " << subBatchNumBlocks << std::endl;
     
      std::cout << "subBatchStartIdx = " << subBatchStartIdx + startIdx << "  "
                << "subBatchEndIdx = " << subBatchEndIdx  + startIdx << std::endl;

      int blockDimX = 32;
      int blockDimY = 2; // 32 * 2 = 64 threads per block
      dim3 blockDim(blockDimX, blockDimY);
                
      // Calculate total threads per block
      int threadsPerBlock = blockDimX * blockDimY; // Should match subBatchNumThreads (64)
      // Calculate grid dimensions
      int totalThreadsNeeded = subBatchNumNodes;
      int blocksNeeded = (totalThreadsNeeded + threadsPerBlock - 1) / threadsPerBlock;
      std::cout << "totalThreadsNeeded = " << totalThreadsNeeded << "  "
                << "threadsPerBlock = " << threadsPerBlock << "  "
                << "blocksNeeded = " << blocksNeeded << std::endl;      
      dim3 gridDim(blocksNeeded, 1);

      // node-level compute
      for (int depth = maxDepth; depth >= 0; depth--) {
        //std::cout << "depth = " << depth << std::endl;
        //std::cout << "depth = " << depth << "  "
        //          << "numNodes = " << numNodes << "  "
        //          << "numThreads = " << numThreads << "  "
        //          << "numBlocks = " << numBlocks << std::endl;

        steiner_node_compute_update_kernel<<<gridDim, blockDim>>>(
          d_nodeLocBestCost,
          d_nodeLocBestComb,
          d_congestion_map,
          d_netBatch,
          d_nodeCntPtr,
          d_nodeLevel,
          d_nodeLoc,
          d_nodeLocPtr,
          d_nodeEdgeIdx,
          d_nodeEdgePtr,
          gridXSize,
          gridYSize,
          maxNumNodes,
          maxNumLocs,
          startIdx + subBatchStartIdx,
          startIdx + subBatchEndIdx,
          depth,
          congestionThresh_,
          BLOCKCOST_DEVICE);
        //hipDeviceSynchronize();
      }

      //hipDeviceSynchronize();

      // add other kernels functions
      // perform the node-level commit
      for (int depth = 0; depth <= maxDepth; depth++) {
        steiner_node_commit_update_kernel<<<gridDim, blockDim>>>(
          d_nodeLocBestCost,
          d_nodeLocBestComb,
          d_bestOptimalLoc,
          d_netBatch,
          d_nodeCntPtr,
          d_nodeLevel,
          d_nodeLoc,
          d_nodeLocPtr,
          d_nodeEdgeIdx,
          d_nodeEdgePtr,
          d_nodeParentIdx,
          maxNumNodes,
          maxNumLocs,
          startIdx + subBatchStartIdx,
          startIdx + subBatchEndIdx,
          depth);
      }

      // update the corner location
      for (int depth = 0; depth <= maxDepth; depth++) {
        corner_node_update_kernel<<<gridDim, blockDim>>>(
          d_cornerLoc,
          d_bestOptimalLoc,
          d_congestion_map,
          d_netBatch,
          d_nodeCntPtr,
          d_nodeLevel,
          d_nodeLoc,
          d_nodeLocPtr,
          d_nodeEdgeIdx,
          d_nodeEdgePtr,
          gridXSize,
          gridYSize,
          maxNumNodes,
          startIdx + subBatchStartIdx,
          startIdx + subBatchEndIdx,
          depth,
          congestionThresh_,
          BLOCKCOST_DEVICE);
      }
    }
    */


  
    // node-level compute
    for (int depth = maxDepth; depth >= 0; depth--) {
      //std::cout << "depth = " << depth << std::endl;
      /*
      std::cout << "depth = " << depth << "  "
                << "numNodes = " << numNodes << "  "
                << "numThreads = " << numThreads << "  "
                << "numBlocks = " << numBlocks << std::endl;
      */
      steiner_node_compute_update_kernel<<<numBlocks, numThreads>>>(
        d_nodeLocBestCost,
        d_nodeLocBestComb,
        d_congestion_map,
        d_netBatch,
        d_nodeCntPtr,
        d_nodeLevel,
        d_nodeLoc,
        d_nodeLocPtr,
        d_nodeEdgeIdx,
        d_nodeEdgePtr,
        gridXSize,
        gridYSize,
        maxNumNodes,
        maxNumLocs,
        startIdx,
        endIdx,
        depth,
        congestionThresh_,
        BLOCKCOST_DEVICE);
      //hipDeviceSynchronize();
    }

    //hipDeviceSynchronize();

    // for debugging,  check the cost and combination
    //hipMemcpy(nodeLocBestCost.data(), d_nodeLocBestCost, nodeLocBestCost.size() * sizeof(float), hipMemcpyDeviceToHost);
    //for (int i = 0; i < 10; i++) {
    //  std::cout << "i = " << i << "cost: " << nodeLocBestCost[i] << std::endl;
    //}

    //std::cout << "finish node compute update kernel" << std::endl;

    // perform the node-level commit
    for (int depth = 0; depth <= maxDepth; depth++) {
      steiner_node_commit_update_kernel<<<numBlocks, numThreads>>>(
        d_nodeLocBestCost,
        d_nodeLocBestComb,
        d_bestOptimalLoc,
        d_netBatch,
        d_nodeCntPtr,
        d_nodeLevel,
        d_nodeLoc,
        d_nodeLocPtr,
        d_nodeEdgeIdx,
        d_nodeEdgePtr,
        d_nodeParentIdx,
        maxNumNodes,
        maxNumLocs,
        startIdx,
        endIdx,
        depth);
    }
  
    
    // for debugging,  check the cost and combination
    //hipMemcpy(bestOptimalLoc.data(), d_bestOptimalLoc, bestOptimalLoc.size() * sizeof(int), hipMemcpyDeviceToHost);
    //for (int i = 0; i < 10; i++) {
    //  std::cout << "i = " << i << "bestOptimalLoc: " << bestOptimalLoc[i] << std::endl;
    //}

    //std::cout << "finish node commit update kernel" << std::endl;

    for (int depth = 0; depth <= maxDepth; depth++) {
      corner_node_update_kernel<<<numBlocks, numThreads>>>(
        d_cornerLoc,
        d_bestOptimalLoc,
        d_congestion_map,
        d_netBatch,
        d_nodeCntPtr,
        d_nodeLevel,
        d_nodeLoc,
        d_nodeLocPtr,
        d_nodeEdgeIdx,
        d_nodeEdgePtr,
        gridXSize,
        gridYSize,
        maxNumNodes,
        startIdx,
        endIdx,
        depth,
        congestionThresh_,
        BLOCKCOST_DEVICE);
    }   
  }


  // for debugging,  check the cost and combination
  hipMemcpy(nodeLocBestCost.data(), d_nodeLocBestCost, nodeLocBestCost.size() * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(nodeLocBestComb.data(), d_nodeLocBestComb, nodeLocBestComb.size() * sizeof(unsigned), hipMemcpyDeviceToHost);
  hipMemcpy(bestOptimalLoc.data(), d_bestOptimalLoc, bestOptimalLoc.size() * sizeof(int), hipMemcpyDeviceToHost); 
  hipMemcpy(cornerLoc.data(), d_cornerLoc, cornerLoc.size() * sizeof(IntPair), hipMemcpyDeviceToHost);
  
  for (int i = 0; i < nodeLocBestCost.size(); i++) {
    if (nodeLocBestCost[i] == FLT_MAX) {
      std::cout << "Error ! nodeLocBestCost is FLT_MAX !!!" << "  "
                << "Bestcomb = " << nodeLocBestComb[i] << "  "
                << "i = " << i << std::endl;
    }
  }
  
  std::cout << "Finish the kernel launch" << std::endl;


  // update the location for the steiner node 
  std::function<void(frNode*, frNet*, int)> traverse_net_dfs_lambda = 
    [&](frNode* currNode, frNet* net, int baseIdx) {
    if (currNode->getType() != frNodeTypeEnum::frcSteiner) {
      return;
    }


    //if (currNode->isDontMove()) {
    //  
    //}

    int currNodeIdx = currNode->getIntProp();    
    Point loc(locVec[bestOptimalLoc[currNodeIdx]].x(), locVec[bestOptimalLoc[currNodeIdx]].y());
    Point realLoc = design_->getTopBlock()->getGCellCenter(loc);
     
    if (currNode->isDontMove()) {
      Point newLoc = currNode->getLoc();  
      Point gcellLoc = design_->getTopBlock()->getGCellIdx(newLoc);
      if (gcellLoc.x() != loc.x() || gcellLoc.y() != loc.y()) {
        std::cout << "Error ! The location is not the same as the optimal location\n";
        exit(1);
      }
    } else {
      currNode->setLoc(realLoc);
    }

    //currNode->setLoc(realLoc);
    // currNode->setLoc(design_->getTopBlock()->getGCellCenter(loc));
    Point newLoc = design_->getTopBlock()->getGCellIdx(currNode->getLoc());
    

    /*
    // check the parent loc
    if (currNode->getParent() != nullptr && currNode->getParent()->getType() == frNodeTypeEnum::frcSteiner) {
      auto parentLoc = currNode->getParent()->getLoc();
      auto parentGcellLoc = design_->getTopBlock()->getGCellIdx(parentLoc);
      if (parentGcellLoc.x() == newLoc.x() && parentGcellLoc.y() == newLoc.y()) {
        std::cout << "Error ! The parent loc is the same as the child loc\n";
      }
    }
    */

    
    // Traverse the children
    for (auto child : currNode->getChildren()) {
      traverse_net_dfs_lambda(child, net, baseIdx);
    } 
       
    // if (gcellLoc.x() != loc.x() || gcellLoc.y() != loc.y()) {
    //   std::cout << "Previous location: " << gcellLoc.x() << " " << gcellLoc.y() << "  "
    //            << "New location: " << loc.x() << " " << loc.y() << "  "
    //            << "node loc : " << newLoc.x() << " " << newLoc.y() << " "
    //            << "real loc : " << realLoc.x() << " " << realLoc.y() << " "
    //            << "nodeId=" << currNodeIdx << std::endl;
    //}
  };    

  auto printNetInfo = [&](frNet* net) {
    for (auto& node : net->getNodes()) {
      if (node->getType() != frNodeTypeEnum::frcSteiner) {
        continue;
      }

      auto parent = node->getParent();
      if (parent == nullptr || parent->getType() != frNodeTypeEnum::frcSteiner) {
        continue;
      }
  
      Point parentLoc = design_->getTopBlock()->getGCellIdx(parent->getLoc());
      Point nodeLoc = design_->getTopBlock()->getGCellIdx(node->getLoc());

      if (parentLoc.x() == nodeLoc.x() && parentLoc.y() == nodeLoc.y()) {
        std::cout << "netId = " << net->getId() << " "
                  << "nodeId = " << node->getIntProp() << " "
                  << "parentId = " << parent->getIntProp() << " "
                  << "Parent loc: " << parentLoc.x() << " " << parentLoc.y() << " "
                  << "Node loc: " << nodeLoc.x() << " " << nodeLoc.y() << " "
                  << "Error: two steiner nodes are at the same location\n";
        exit(1);
      }
    }
  };


  std::function<void(frNet*)> traverse_net_add_node_dfs_lambda = [&](frNet* net) {
    std::vector<frNode*> nodes;
    for (auto& node : net->getNodes()) {
      if (node->getType() != frNodeTypeEnum::frcSteiner) {
        continue;
      }

      auto parent = node->getParent();
      if (parent == nullptr || parent->getType() != frNodeTypeEnum::frcSteiner) {
        continue;
      }
      
      nodes.push_back(node.get());
    }
   
    /*
    // Print the node
    for (auto& node : nodes) {
      auto parentLoc = design_->getTopBlock()->getGCellIdx(node->getParent()->getLoc());
      auto childLoc = design_->getTopBlock()->getGCellIdx(node->getLoc());
      std::cout << "nodeId = " << node->getIntProp() << "  "
                << "Parent Node id: " << node->getParent()->getIntProp() << "  "
                << "Parent loc: " << parentLoc.x() << " " << parentLoc.y() << "  "
                << "Child loc: " << childLoc.x() << " " << childLoc.y() << std::endl;
    }
    */


    for (auto& node : nodes) {
      int nodeIdx = node->getIntProp();
      if (nodeIdx >= cornerLoc.size()) {
        std::cout << "Error ! nodeIdx is out of bound "
                  << "nodeIdx = " << nodeIdx << " corNocLoc.size() = " << cornerLoc.size() << std::endl;
        continue;
      }
      
      auto corner = cornerLoc[nodeIdx];
      if (corner.x() == -1 || corner.y() == -1) {
        continue;
      }
      
      Point parentLoc = design_->getTopBlock()->getGCellIdx(node->getParent()->getLoc());
      Point childLoc = design_->getTopBlock()->getGCellIdx(node->getLoc());


      /*
      std::cout << "New netId = " << net->getId() << "  "
                << "nodeId = " << node->getIntProp() << "  "
                << "Parent Node id: " << node->getParent()->getIntProp() << "  "
                << "parentLoc = " << parentLoc.x() << " " << parentLoc.y() << "  "
                << "childLoc = " << childLoc.x() << " " << childLoc.y() << "  "
                << "cornerLoc = " << corner.x() << " " << corner.y() << std::endl;
      */

      if (((parentLoc.x() != corner.x() && parentLoc.y() != corner.y()) ||  
         (childLoc.x() != corner.x() && childLoc.y() != corner.y()))) {
        std::cout << "Node id : " << nodeIdx << " "
                  << "Parent Node id: " << node->getParent()->getIntProp() << " "
                  << "Parent loc: " << parentLoc.x() << " " << parentLoc.y() << " "
                  << " Child loc: " << childLoc.x() << " " << childLoc.y() 
                  << " Corner loc: " << corner.x() << " " << corner.y() << std::endl;
      }

      if ((parentLoc.x() == corner.x() && parentLoc.y() == corner.y()) 
        || (childLoc.x() == corner.x() && childLoc.y() == corner.y())) {
        std::cout << "Node id : " << nodeIdx << " Parent loc: " << parentLoc.x() << " " << parentLoc.y() 
                  << " Child loc: " << childLoc.x() << " " << childLoc.y() 
                  << " Corner loc: " << corner.x() << " " << corner.y() << std::endl;
      }

      Point loc = design_->getTopBlock()->getGCellCenter(Point(corner.x(), corner.y()));            
      createCornerNode2D_update(net, node, node->getParent(), Point(corner.x(), corner.y()));

      /*
      // check the location
      Point loc1 = design_->getTopBlock()->getGCellIdx(node->getParent()->getLoc());
      Point loc0 = design_->getTopBlock()->getGCellIdx(node->getLoc());
      Point loc2 = design_->getTopBlock()->getGCellIdx(node->getParent()->getParent()->getLoc());
      // detect the non-colinear corner
      if ((loc0.x() != loc1.x() && loc0.y() != loc1.y()) ||
          (loc1.x() != loc2.x() && loc1.y() != loc2.y())) {
        std::cout << "Node id : " << nodeIdx << " Parent loc: " << parentLoc.x() << " " << parentLoc.y() 
                  << " Child loc: " << childLoc.x() << " " << childLoc.y() 
                  << " Corner loc: " << corner.x() << " " << corner.y() << std::endl;
        
        std::cout << "Error ! The corner is not colinear" << " "
                  << "loc0 = " << loc0.x() << " " << loc0.y() << " "
                  << "loc1 = " << loc1.x() << " " << loc1.y() << " "
                  << "loc2 = " << loc2.x() << " " << loc2.y() << std::endl;
      }

      if (loc0.x() == loc1.x() && loc0.y() == loc1.y()) {
        std::cout << "Error ! The corner location is the same as the parent location" << std::endl;
        std::cout << "Node id : " << nodeIdx << " Parent loc: " << parentLoc.x() << " " << parentLoc.y() 
                  << " Child loc: " << childLoc.x() << " " << childLoc.y() 
                  << " Corner loc: " << corner.x() << " " << corner.y() << std::endl;
        
        std::cout << "Error ! The corner is not colinear" << " "
                  << "loc0 = " << loc0.x() << " " << loc0.y() << " "
                  << "loc1 = " << loc1.x() << " " << loc1.y() << " "
                  << "loc2 = " << loc2.x() << " " << loc2.y() << std::endl; 
      }

      if (loc1.x() == loc2.x() && loc1.y() == loc2.y()) {
        std::cout << "Error ! The corner location is the same as the parent location" << std::endl;
        std::cout << "Node id : " << nodeIdx << " Parent loc: " << parentLoc.x() << " " << parentLoc.y() 
                  << " Child loc: " << childLoc.x() << " " << childLoc.y() 
                  << " Corner loc: " << corner.x() << " " << corner.y() << std::endl;
        
        std::cout << "Error ! The corner is not colinear" << " "
                  << "loc0 = " << loc0.x() << " " << loc0.y() << " "
                  << "loc1 = " << loc1.x() << " " << loc1.y() << " "
                  << "loc2 = " << loc2.x() << " " << loc2.y() << std::endl; 
      }
      */
    }

    for (auto& node : net->getNodes()) {
      if (node->getType() != frNodeTypeEnum::frcSteiner) {
        continue;
      }

      auto parent = node->getParent();
      if (parent == nullptr || parent->getType() != frNodeTypeEnum::frcSteiner) {
        continue;
      }
  
      Point parentLoc = design_->getTopBlock()->getGCellIdx(parent->getLoc());
      Point nodeLoc = design_->getTopBlock()->getGCellIdx(node->getLoc());

      if (parentLoc.x() == nodeLoc.x() && parentLoc.y() == nodeLoc.y()) {
        std::cout << "netId = " << net->getId() << " "
                  << "nodeId = " << node->getIntProp() << " "
                  << "parentId = " << parent->getIntProp() << " "
                  << "Parent loc: " << parentLoc.x() << " " << parentLoc.y() << " "
                  << "Node loc: " << nodeLoc.x() << " " << nodeLoc.y() << " "
                  << "Error: two steiner nodes are at the same location\n";
        exit(1);
      }
    }
  };

  int batchId = 0;
  int baseIdx = 0;
  // traverse the list in a DFS manner
  for (auto& batch : netBatch) {
    for (auto netId : batch) {
      auto& net = nets2RR[netId];
      baseIdx = nodeCntPtrVec[netId];
      //std::cout << "NetId = " << net->getId() << std::endl;
      //std::cout << "before  traverse_net_dfs_lambda " << std::endl;
      //printNetInfo(net);
      traverse_net_dfs_lambda(net->getRootGCellNode(), net, baseIdx);
      //std::cout << "before traverse_net_add_node_dfs_lambda " << std::endl;
      //printNetInfo(net);
      //traverse_net_add_node_dfs_lambda(net);
      //std::cout << "after traverse_net_add_node_dfs_lambda " << std::endl;
      //printNetInfo(net);
      //initGR_updateCongestion2D_net_update(net);
    }
  } 

  removeLoop(nets2RR);
  
  // traverse the list in a DFS manner
  for (auto& batch : netBatch) {
    for (auto netId : batch) {
      auto& net = nets2RR[netId];
      baseIdx = nodeCntPtrVec[netId];
      //std::cout << "NetId = " << net->getId() << std::endl;
      //std::cout << "before  traverse_net_dfs_lambda " << std::endl;
      //printNetInfo(net);
      //traverse_net_dfs_lambda(net->getRootGCellNode(), net, baseIdx);
      //std::cout << "before traverse_net_add_node_dfs_lambda " << std::endl;
      //printNetInfo(net);
      traverse_net_add_node_dfs_lambda(net);
      //std::cout << "after traverse_net_add_node_dfs_lambda " << std::endl;
      //printNetInfo(net);
      initGR_updateCongestion2D_net_update(net);
    }
  } 
  
  std::cout << "finish location update" << std::endl;
  checkValidNet_update();
  // exit(1);

  // Remove the CUDA memory
  hipFree(d_netBatch);
  hipFree(d_congestion_map);
  hipFree(d_nodeCntPtr);
  hipFree(d_nodeLevel);
  hipFree(d_nodeLoc);
  hipFree(d_nodeLocPtr);
  hipFree(d_nodeParentIdx);
  hipFree(d_nodeEdgeIdx);
  hipFree(d_nodeEdgePtr);
  hipFree(d_netBatchMaxDepth);
  hipFree(d_nodeLocBestCost);
  hipFree(d_nodeLocBestComb);
  hipFree(d_bestOptimalLoc); 
  hipFree(d_cornerLoc);
}

 
 
} // namespace drt

