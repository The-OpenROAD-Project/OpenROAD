#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2024, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR_util.h"
#include <iostream>
#include <sys/resource.h>

namespace drt {


__device__
int getIdx__device(int x, int y, int z, int xDim, int yDim, int zDim)
{
  if (z < 0 || z >= zDim) {
    return -1;
  }
  return z * xDim * yDim + y * xDim + x;    
}


__device__
bool getBit__device(uint64_t* d_cmap, unsigned idx, unsigned pos)
{
  return (d_cmap[idx] >> pos) & 1;
}


__device__
unsigned getBits__device(uint64_t* d_cmap, unsigned idx, unsigned pos, unsigned length)
{
  auto tmp = d_cmap[idx] & (((1ull << length) - 1) << pos);
  return tmp >> pos;
}



// Non-atomic version (for reference only)
__device__
void setBits_naive__device(uint64_t* d_cmap, unsigned idx, unsigned pos, unsigned length, unsigned val)
{
  d_cmap[idx] &= ~(((1ull << length) - 1) << pos);  // clear related bits to 0
  d_cmap[idx] |= ((uint64_t) val & ((1ull << length) - 1))
              << pos;  // only get last length bits of val
}



__device__
void setBits__device(uint64_t* d_cmap, unsigned idx, unsigned pos, unsigned length, unsigned val)
{
  uint64_t* address = &d_cmap[idx];
  const uint64_t clear_mask = ((1ull << length) - 1) << pos; // clear related bits to 0
  const uint64_t set_val = (static_cast<uint64_t>(val) & ((1ull << length) - 1)) << pos; // only get last length bits of val
  uint64_t old = *address;
  uint64_t new_val;
  do {
    old = *address;
    new_val = (old & ~clear_mask) | set_val;
  } while (atomicCAS(reinterpret_cast<unsigned long long int*>(address),
           static_cast<unsigned long long int>(old),
           static_cast<unsigned long long int>(new_val)) != old);
}



// Non-atomic version (for reference only)
__device__
void addToBits_naive__device(uint64_t* d_cmap, unsigned idx, unsigned pos, unsigned length, unsigned val)
{
  auto tmp = getBits__device(d_cmap, idx, pos, length) + val;
  tmp = (tmp > (1u << length)) ? (1u << length) : tmp;
  setBits_naive__device(d_cmap, idx, pos, length, tmp);
}


__device__
void addToBits__device(uint64_t* d_cmap, unsigned idx, unsigned pos, unsigned length, unsigned val)
{
  uint64_t* address = &d_cmap[idx];
  const uint64_t clear_mask = ((1ull << length) - 1) << pos;
  const uint64_t val_mask = (1ull << length) - 1;
  uint64_t old, new_val;
  old = *address;
  do {
    old = *address;
    uint64_t current_val = (old >> pos) & val_mask;
    current_val += val;
    // Clamp to original behavior 
    current_val = min(current_val, static_cast<uint64_t>(1ull << length));
    new_val = (old & ~clear_mask) | ((current_val & val_mask) << pos);
  } while (atomicCAS(reinterpret_cast<unsigned long long int*>(address),
           static_cast<unsigned long long int>(old),
           static_cast<unsigned long long int>(new_val)) != old);
}



__device__
bool hasBlock__device(uint64_t* d_cmap, 
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir)
{
  bool sol = false;
  auto idx = getIdx__device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      sol = getBit__device(d_cmap, idx, 3);
      break;
    case frDirEnum::N:
      sol = getBit__device(d_cmap, idx, 2);
      break;
    case frDirEnum::U:;
      break;
    default:;
  }  
  return sol;
}



__device__
unsigned getRawSupply__device(uint64_t* d_cmap, 
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir)
{
  unsigned supply = 0;
  auto idx = getIdx__device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      supply = getBits__device(d_cmap, idx, 24, CMAPSUPPLYSIZE);
      break;
    case frDirEnum::N:
      supply = getBits__device(d_cmap, idx, 16, CMAPSUPPLYSIZE);
      break;
    case frDirEnum::U:;
      break;
    default:;
  }
  return supply << CMAPFRACSIZE;
}


__device__
unsigned getRawDemand__device(uint64_t* d_cmap,
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir)
{
  unsigned demand = 0;
  auto idx = getIdx__device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      demand = getBits__device(d_cmap, idx, 48, CMAPDEMANDSIZE);
      break;
    case frDirEnum::N:
      demand = getBits__device(d_cmap, idx, 32, CMAPDEMANDSIZE);
      break;
    case frDirEnum::U:;
      break;
    default:;
  }

  return demand;
}


__device__
void addRawDemand__device(
  uint64_t* d_cmap,
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir, unsigned delta)
{
  int idx = getIdx__device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      addToBits__device(d_cmap, idx, 48, CMAPDEMANDSIZE, delta);
      break;
    case frDirEnum::N:
      addToBits__device(d_cmap, idx, 32, CMAPDEMANDSIZE, delta);
      break;
    case frDirEnum::U:
      break;
    default:;
  }
}


} // namespace drt

