#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2024, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR_util.h"
#include <iostream>
#include <sys/resource.h>

namespace drt {

constexpr int GRGRIDGRAPHHISTCOSTSIZE = 8;
constexpr int GRSUPPLYSIZE = 8;
constexpr int GRDEMANDSIZE = 16;
constexpr int GRFRACSIZE = 1;


__host__ __device__
void correct(frMIdx& x, frMIdx& y, frMIdx& z, frDirEnum& dir) 
{
  switch (dir) {
    case frDirEnum::W:
      x--;
      dir = frDirEnum::E;
      break;
    case frDirEnum::S:
      y--;
      dir = frDirEnum::N;
      break;
    case frDirEnum::D:
      z--;
      dir = frDirEnum::U;
      break;
    default:;
  }
}


__host__ __device__
void correctU(frMIdx& x, frMIdx& y, frMIdx& z, frDirEnum& dir) 
{
  switch (dir) {
    case frDirEnum::D:
      z--;
      dir = frDirEnum::U;
      break;
    default:;
  }
}


__host__ __device__
void reverse(frMIdx& x, frMIdx& y, frMIdx& z, frDirEnum& dir)
{
  switch (dir) {
    case frDirEnum::E:
      x++;
      dir = frDirEnum::W;
      break;
    case frDirEnum::S:
      y--;
      dir = frDirEnum::N;
      break;
    case frDirEnum::W:
      x--;
      dir = frDirEnum::E;
      break;
    case frDirEnum::N:
      y++;
      dir = frDirEnum::S;
      break;
    case frDirEnum::U:
      z++;
      dir = frDirEnum::D;
      break;
    case frDirEnum::D:
      z--;
      dir = frDirEnum::U;
      break;
    default:;
  }
}



__host__ __device__
bool addEdge(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir) 
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }
  
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      setBit(bits, idx, 0);
      break;
    case frDirEnum::N:
      setBit(bits, idx, 1);
      break;
    case frDirEnum::U:
      setBit(bits, idx, 2);
      break;
    default:
      return false;
  }  
  
  return true; 
}


__host__ __device__
bool removeEdge(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir) 
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }
  
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      resetBit(bits, idx, 0);
      break;
    case frDirEnum::N:
      resetBit(bits, idx, 1);
      break;
    case frDirEnum::U:
      resetBit(bits, idx, 2);
      break;
    default:
      return false;
  }  
  
  return true; 
}


__host__ __device__
bool setBlock(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir) 
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }
  
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      setBit(bits, idx, 3);
      break;
    case frDirEnum::N:
      setBit(bits, idx, 4);
      break;
    case frDirEnum::U:
      setBit(bits, idx, 5);
      break;
    default:
      return false;
  }  
  
  return true; 
}



__host__ __device__
bool resetBlock(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir) 
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }
  
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      resetBit(bits, idx, 3);
      break;
    case frDirEnum::N:
      resetBit(bits, idx, 4);
      break;
    case frDirEnum::U:
      resetBit(bits, idx, 5);
      break;
    default:
      return false;
  }  
  
  return true; 
}


__host__ __device__
void setHistoryCost(uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, unsigned histCostIn)
{
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  setBits(bits, idx, 8, GRGRIDGRAPHHISTCOSTSIZE, histCostIn);
}


__host__ __device__
void addHistoryCost(uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, unsigned in)
{
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);  
  addToBits(bits, idx, 8, GRGRIDGRAPHHISTCOSTSIZE, in);
}


__host__ __device__
void decayHistoryCost(uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z)
{
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);  
  subToBits(bits, idx, 8, GRGRIDGRAPHHISTCOSTSIZE, 1);
}


__host__ __device__
void decayHistoryCost(uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, double d)
{
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);  
  int currCost = (getBits(bits, idx, 8, GRGRIDGRAPHHISTCOSTSIZE));
  currCost *= d;
  currCost = std::max(0, currCost);
  setBits(bits, idx, 8, GRGRIDGRAPHHISTCOSTSIZE, currCost);
}



// E == H; N == V; currently no U / D
__host__ __device__
void setSupply(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, unsigned supplyIn)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      setBits(bits, idx, 16, GRSUPPLYSIZE, supplyIn);
      break;
    case frDirEnum::N:
      setBits(bits, idx, 24, GRSUPPLYSIZE, supplyIn);
      break;
    default:
      return;
  }
}


// E == H; N == V; currently no U / D
__host__ __device__
void setDemand(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, unsigned demandIn)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      return setBits(
        bits, idx, 48 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE, demandIn);
    case frDirEnum::N:
      return setBits(
        bits, idx, 32 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE, demandIn);
    default:
      return;
  }
}


// E == H; N == V; currently no U / D
__host__ __device__
void setRawDemand(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, unsigned rawDemandIn)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      return setBits(bits, idx, 48, GRDEMANDSIZE, rawDemandIn);
    case frDirEnum::N:
      return setBits(bits, idx, 32, GRDEMANDSIZE, rawDemandIn);
    default:
      return;
  }
}


// E == H; N == V; currently no U / D
__host__ __device__
void addDemand(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, unsigned delta)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      addToBits(bits, idx, 48 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE, delta);
      break;
    case frDirEnum::N:
      addToBits(bits, idx, 32 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE, delta);
      break;
    default:
      return;
  }
}



// E == H; N == V; currently no U / D
__host__ __device__ 
void addRawDemand(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, unsigned delta)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      addToBits(bits, idx, 48, GRDEMANDSIZE, delta);
      break;
    case frDirEnum::N:
      addToBits(bits, idx, 32, GRDEMANDSIZE, delta);
      break;
    default:
      return;
  }
}


// E == H; N == V; currently no U / D
__host__ __device__
void subDemand(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, unsigned delta)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      subToBits(bits, idx, 48 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE, delta);
      break;
    case frDirEnum::N:
      subToBits(bits, idx, 32 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE, delta);
      break;
    default:
      return;
  }
}

// E == H; N == V; currently no U / D
__host__ __device__
void subRawDemand(uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, unsigned delta)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      subToBits(bits, idx, 48, GRDEMANDSIZE, delta);
      break;
    case frDirEnum::N:
      subToBits(bits, idx, 32, GRDEMANDSIZE, delta);
      break;
    default:
      return;
  }
} 


// E == H; N == V; currently no U / D
__host__ __device__
unsigned getSupply(const uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir, bool isRaw)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return 0;
  }
    
  unsigned supply = 0;
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      supply = getBits(bits, idx, 24, GRSUPPLYSIZE);
      break;
    case frDirEnum::N:
      supply = getBits(bits, idx, 16, GRSUPPLYSIZE);
      break;
    default:
      return 0;
  }
  
  // Return the raw or processed supply value based on the isRaw flag
  return isRaw ? (supply << GRFRACSIZE) : supply;
}


__host__ __device__
unsigned getRawSupply(const uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir)
{
  return getSupply(bits, zDir, xDim, yDim, zDim, x, y, z, dir, true);
}


// E == H; N == V; currently no U / D
__host__ __device__
unsigned getDemand(const uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return 0;
  }
    
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      return getBits(bits, idx, 48 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE);
    case frDirEnum::N:
      return getBits(bits, idx, 32 + GRFRACSIZE, GRDEMANDSIZE - GRFRACSIZE);
    default:
      return 0;
  }
}


// E == H; N == V; currently no U / D
__host__ __device__
unsigned getRawDemand(const uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return 0;
  }
    
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      return getBits(bits, idx, 48, GRDEMANDSIZE);
    case frDirEnum::N:
      return getBits(bits, idx, 32, GRDEMANDSIZE);
    default:
      return 0;
  }
}


__host__ __device__
bool hasBlock(const uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }
  
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      return getBit(bits, idx, 3);
    case frDirEnum::N:
      return getBit(bits, idx, 4);
    case frDirEnum::U:
      return getBit(bits, idx, 5);
    default:
      return false;
  }
}


__host__ __device__
bool hasHistoryCost(const uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z)
{
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  return getBits(bits, idx, 8, GRGRIDGRAPHHISTCOSTSIZE);
}


__host__ __device__
bool hasEdge(const uint64_t* bits,
  const bool* zDir, int xDim, int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }
  
  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  switch (dir) {
    case frDirEnum::E:
      return getBit(bits, idx, 0);
    case frDirEnum::N:
      return getBit(bits, idx, 1);
    case frDirEnum::U:
      return getBit(bits, idx, 2);
    default:
      return false;
  }  
}


__host__ __device__
frCoord getEdgeLength(
  const frCoord* xCoords, const frCoord* yCoords, const frCoord* zHeights,
  const bool* zDirs, int xDim , int yDim, int zDim,
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir)
{
  correct(x, y, z, dir);
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return false;
  }

  switch (dir) {
    case frDirEnum::E:
      return xCoords[x + 1] - xCoords[x];
    case frDirEnum::N:
      return yCoords[y + 1] - yCoords[y];
    case frDirEnum::U:
      return zHeights[z + 1] - zHeights[z];
    default:
      return 0;
  }
}




__host__ __device__
bool hasCongCost(const uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z, frDirEnum dir)
{
  return (getRawDemand(bits, zDir, xDim, yDim, zDim, x, y, z, dir) > 
          getRawSupply(bits, zDir, xDim, yDim, zDim, x, y, z, dir));
}


__host__ __device__
unsigned getHistoryCost(const uint64_t* bits, 
  const bool* zDir, int xDim, int yDim, int zDim, 
  frMIdx x, frMIdx y, frMIdx z)
{
  if (!isValid(x, y, z, xDim, yDim, zDim)) {
    return 0;
  }

  auto idx = getIdx(x, y, z, xDim, yDim, zDir);
  return getBits(bits, idx, 8, GRGRIDGRAPHHISTCOSTSIZE);
}


} // namespace drt

