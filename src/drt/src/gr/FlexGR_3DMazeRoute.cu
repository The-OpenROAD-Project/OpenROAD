#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR.h"
#include <omp.h> 
#include <cmath>
#include <fstream>
#include <iostream>
 
#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <string>
#include <stdint.h> // For fixed-width integers
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <map>
#include <queue>
#include <fstream>
#include <iostream>
#include <iterator>
#include <memory>
#include <set>
#include <sstream>
#include <string>
#include <vector>
#include <numeric>
#include <algorithm>

// We always work on the entire grid 
// The grid system is always in terms of global coordinates
// So for each net, we need to translate the local index system into global system

namespace drt {


namespace cg = cooperative_groups;



enum Directions3D {
  DIR_NORTH_3D   = 0,
  DIR_RIGHT_3D   = 1,
  DIR_SOUTH_3D   = 2,
  DIR_LEFT_3D    = 3,
  DIR_UP_3D      = 4,
  DIR_DOWN_3D    = 5,
  DIR_NONE_3D    = 255
};


__host__ __device__ 
void initNodeData3D(NodeData3D& nd) {
  nd.forward_h_cost = INF32;
  nd.forward_g_cost = INF32;
  nd.backward_h_cost = INF32;
  nd.backward_g_cost = INF32;
  nd.forward_h_cost_prev = INF32;
  nd.forward_g_cost_prev = INF32;
  nd.backward_h_cost_prev = INF32;
  nd.backward_g_cost_prev = INF32;
  nd.forward_direction = DIR_NONE_3D;
  nd.backward_direction = DIR_NONE_3D;
  nd.forward_direction_prev = DIR_NONE_3D;
  nd.backward_direction_prev = DIR_NONE_3D;
  nd.golden_parent_x = -1;
  nd.golden_parent_y = -1;
  nd.golden_parent_z = -1;
  nd.flags.src_flag = 0;
  nd.flags.dst_flag = 0;
  nd.flags.forward_update_flag = 0;
  nd.flags.backward_update_flag = 0;
  nd.flags.forward_visited_flag = 0;
  nd.flags.backward_visited_flag = 0;
  nd.flags.forward_visited_flag_prev = 0;
  nd.flags.backward_visited_flag_prev = 0;
}


__device__  
uint8_t computeParentDirection3D(int d) {
  switch(d) {
    case 0: return DIR_NORTH_3D;
    case 1: return DIR_RIGHT_3D;
    case 2: return DIR_SOUTH_3D;
    case 3: return DIR_LEFT_3D;
    case 4: return DIR_UP_3D;
    case 5: return DIR_DOWN_3D;
    default: return DIR_NONE_3D;
  }
}


// Invert direction for backtracking
__device__  
uint8_t invertDirection3D(uint8_t d) {
  switch(d) {
    case DIR_NORTH_3D:    return DIR_SOUTH_3D;
    case DIR_SOUTH_3D:    return DIR_NORTH_3D;
    case DIR_LEFT_3D:     return DIR_RIGHT_3D;
    case DIR_RIGHT_3D:    return DIR_LEFT_3D;
    case DIR_UP_3D:       return DIR_DOWN_3D;
    case DIR_DOWN_3D:     return DIR_UP_3D;
    default:           return DIR_NONE_3D;
  }
}


__device__ __host__  
int3 idxToLoc_3D(int idx, int xDim, int yDim) {
  int z = idx / (xDim * yDim);
  int temp = idx % (xDim * yDim);
  int y = temp / xDim;
  int x = temp % xDim;
  return make_int3(x,y,z);
}


__device__ __host__  
int locToIdx_3D(int x, int y, int z, int xDim, int yDim) {
  return z * xDim * yDim + y * xDim + x;
}


__host__ __device__
unsigned getEdgeLength3D(
  const int* xCoords, 
  const int* yCoords, 
  const int* zHeights,
  int x, int y, int z, Directions3D dir)
{
  switch (dir) {
    case Directions3D::DIR_RIGHT_3D:
      return xCoords[x + 1] - xCoords[x];
    case Directions3D::DIR_NORTH_3D:
      return yCoords[y + 1] - yCoords[y];
    case Directions3D::DIR_UP_3D:
      return zHeights[z + 1] - zHeights[z];
    default:
      return 0;
  }
}


// We do not consider bending cost in this version
__host__ __device__
uint32_t getEdgeCost3D(
  const uint64_t* d_costMap,
  const int* d_xCoords,
  const int* d_yCoords,
  const int* d_zHeights,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST, 
  int idx, int x, int y, int z,
  Directions3D dir)
{
  return getEdgeLength3D(d_xCoords, d_yCoords, d_zHeights, x, y, z, dir);
}



__device__ 
uint32_t getNeighorCost3D(
  const uint64_t* d_costMap,
  const int* d_xCoords,
  const int* d_yCoords,
  const int* d_zHeights,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST,
  int idx, int x, int y, int z,
  int nbrIdx, int nx, int ny, int nz)
{
  uint32_t newG = 0;
  if (nx == x && ny == y - 1 && nz == z) {
    newG += getEdgeCost3D(d_costMap,     
      d_xCoords, d_yCoords, d_zHeights,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      nbrIdx, nx, ny, nz, Directions3D::DIR_NORTH_3D);
  } else if (nx == x && ny == y + 1 && nz == z) {
    newG += getEdgeCost3D(d_costMap, 
      d_xCoords, d_yCoords, d_zHeights,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      idx, x, y, z, Directions3D::DIR_NORTH_3D);
  } else if (nx == x - 1 && ny == y && nz == z) {
    newG += getEdgeCost3D(d_costMap, 
      d_xCoords, d_yCoords, d_zHeights,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      nbrIdx, nx, ny, nz, Directions3D::DIR_RIGHT_3D);
  } else if (nx == x + 1 && ny == y && nz == z) {
    newG += getEdgeCost3D(d_costMap, 
      d_xCoords, d_yCoords, d_zHeights,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      idx, x, y, z, Directions3D::DIR_RIGHT_3D);
  } else if (nx == x && ny == y && nz == z - 1) {
    newG += getEdgeCost3D(d_costMap, 
      d_xCoords, d_yCoords, d_zHeights,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      nbrIdx, nx, ny, nz, Directions3D::DIR_UP_3D);
  } else if (nx == x && ny == y && nz == z + 1) {
    newG += getEdgeCost3D(d_costMap, 
      d_xCoords, d_yCoords, d_zHeights,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      idx, x, y, z, Directions3D::DIR_UP_3D);
  }

  return newG;
}



__device__
void initNodeData3D__device(
  NodeData3D* d_nodes,
  int* d_pins, int pinIterStart, int pinIter,  // Pin related variables
  int LLX, int LLY, int URX, int URY, // Bounding box
  int xDim, int yDim, int zDim) 
{ 
  int total = (URX - LLX + 1) * (URY - LLY + 1) * zDim;
  int xDimTemp = URX - LLX + 1;
  int yDimTemp = URY - LLY + 1;
  int tid = threadIdx.x;
  int stride = blockDim.x;
  
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int3 local = idxToLoc_3D(localIdx, xDimTemp, yDimTemp);
    int x = local.x + LLX;
    int y = local.y + LLY;
    int z = local.z;
    int idx = locToIdx_3D(x, y, z, xDim, yDim);
  
    // The experimental results show that the heuristic cost is not needed
    d_nodes[idx].forward_h_cost = 0;
    d_nodes[idx].backward_h_cost = 0;

    if (d_nodes[idx].flags.src_flag) {
      d_nodes[idx].forward_g_cost = 0;
      d_nodes[idx].forward_g_cost_prev = 0;
      d_nodes[idx].flags.forward_visited_flag = true;
      d_nodes[idx].flags.forward_visited_flag_prev = true;
    } else {
      d_nodes[idx].forward_g_cost = INF32;
      d_nodes[idx].forward_g_cost_prev = INF32;
      d_nodes[idx].flags.forward_visited_flag = false;
      d_nodes[idx].flags.forward_visited_flag_prev = false;
    }

    if (d_nodes[idx].flags.dst_flag) {
      d_nodes[idx].backward_g_cost = 0;
      d_nodes[idx].backward_g_cost_prev = 0;
      d_nodes[idx].flags.backward_visited_flag = true;
      d_nodes[idx].flags.backward_visited_flag_prev = true;
    } else {
      d_nodes[idx].backward_g_cost = INF32;
      d_nodes[idx].backward_g_cost_prev = INF32;
      d_nodes[idx].flags.backward_visited_flag = false;
      d_nodes[idx].flags.backward_visited_flag_prev = false;
    }

    d_nodes[idx].forward_direction = DIR_NONE_3D;
    d_nodes[idx].backward_direction = DIR_NONE_3D;
    d_nodes[idx].forward_direction_prev = DIR_NONE_3D;
    d_nodes[idx].backward_direction_prev = DIR_NONE_3D;
    d_nodes[idx].flags.forward_update_flag = false;
    d_nodes[idx].flags.backward_update_flag = false;
    d_nodes[idx].flags.forward_visited_flag = false;
    d_nodes[idx].flags.backward_visited_flag = false;
    d_nodes[idx].flags.forward_visited_flag_prev = false;
    d_nodes[idx].flags.backward_visited_flag_prev = false;
  } 
}



// Define the device function for the biwaveBellmanFord_2D_v4__device
__device__
void runBiBellmanFord3D__device(
  int netId,
  NodeData3D* d_nodes,
  uint64_t* d_costMap, 
  const int* __restrict__ d_dX, 
  const int* __restrict__ d_dY,
  const int* __restrict__ d_dZ,
  const int* __restrict__ d_xCoords,
  const int* __restrict__ d_yCoords,
  const int* __restrict__ d_zHeights,
  int LLX, int LLY, int URX, int URY,
  int xDim, int yDim, int zDim, 
  int maxIters,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST)
{
  // To handle the case where maxIters is too small 
  if (URX - LLX == 0) {
    maxIters = maxIters < 2 * (URY - LLY) * zDim ? 2 * (URY - LLY) * zDim : maxIters;
  }

  if (URY - LLY == 0) {
    maxIters = maxIters < 2 * (URX - LLX) * zDim ? 2 * (URX - LLX) * zDim : maxIters;
  }
    
  // Each device function is handled by a single block
  int total = (URX - LLX + 1) * (URY - LLY + 1) * zDim;
  int tid = threadIdx.x;
  int stride = blockDim.x; 
  int xDimTemp = URX - LLX + 1;
  int yDimTemp = URY - LLY + 1;  

  // Define the shared memory for d_dx and d_dy
  __shared__ int s_dX[6];
  __shared__ int s_dY[6];
  __shared__ int s_dZ[6];
  __shared__ volatile int s_doneFlag;
  __shared__ volatile int s_minCost;
  //__shared__ volatile int s_meetId;
  __shared__ unsigned long long s_meet;
  __shared__ volatile int tracebackError;   // 0: no error; 1: error detected
  __shared__ int s_minForwardCost;
  __shared__ int s_minBackwardCost;
  
  // Load the d_dX and d_dY into shared memory
  if (tid < 6) {
    s_dX[tid] = d_dX[tid];
    s_dY[tid] = d_dY[tid];
    s_dZ[tid] = d_dZ[tid];
    if (tid == 0) {
      s_doneFlag = 0;
      s_minCost = 0x7FFFFFFF;
      s_meet = 0xFFFFFFFFFFFFFFFFULL;
      tracebackError = 0;
      s_minForwardCost = 0x7FFFFFFF;
      s_minBackwardCost = 0x7FFFFFFF;
    }
  }
  __syncthreads();

  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int3 local = idxToLoc_3D(localIdx, xDimTemp, yDimTemp);
    int x = local.x + LLX;
    int y = local.y + LLY;
    int z = local.z;
    int idx = locToIdx_3D(x, y, z, xDim, yDim);
    if (d_nodes[idx].flags.dst_flag) {
      atomicExch((int*)&s_doneFlag, 1);
      break; // Exit early if any destination node is found.
    }
  }
  __syncthreads();

  if (!s_doneFlag) {
    return;
  } 
  
  if (tid == 0) {
    s_doneFlag = 0;
  }

  __syncthreads();

  // We'll do up to maxIters or until no changes / front-meet
  for (int iter = 0; iter < maxIters && (s_doneFlag == 0); iter++)
  {
    bool localFrontsMeet = false;

    ////////////////////////////////////////////////////////////////////////////
    // (1) Forward & backward relaxation phase
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int3 local = idxToLoc_3D(localIdx, xDimTemp, yDimTemp);
      int x = local.x + LLX;
      int y = local.y + LLY;
      int z = local.z;
      int idx = locToIdx_3D(x, y, z, xDim, yDim);
      NodeData3D &nd = d_nodes[idx];

      // Forward relaxation:
      // Skip if src_flag is set.
      if (!nd.flags.src_flag) {
        uint32_t bestCost = nd.forward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 6; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          int nz = z + s_dZ[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY || nz < 0 || nz >= zDim) {
            continue;  // out of bounds
          }
          int nbrIdx = locToIdx_3D(nx, ny, nz, xDim, yDim);
          uint32_t neighborCost = d_nodes[nbrIdx].forward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          uint32_t newG = neighborCost +
            getNeighorCost3D(d_costMap, d_xCoords, d_yCoords, d_zHeights,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, z, nbrIdx, nx, ny, nz);
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) { // Found an improvement
          nd.forward_g_cost = bestCost;
          nd.forward_direction = computeParentDirection3D(bestD);
          nd.flags.forward_update_flag = 1;
        }
      } // end forward

      // Backward relaxation:
      // newCost = min over neighbors of (neighbor.backward_cost + edgeWeight).
      // Skip if dst_flag is set.
      if (!nd.flags.dst_flag) {
        uint32_t bestCost = nd.backward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 6; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          int nz = z + s_dZ[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY || nz < 0 || nz >= zDim) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_3D(nx, ny, nz, xDim, yDim);
          uint32_t neighborCost = d_nodes[nbrIdx].backward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          uint32_t newG = neighborCost +
            getNeighorCost3D(d_costMap, 
              d_xCoords, d_yCoords, d_zHeights,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, z, nbrIdx, nx, ny, nz);
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) {
          nd.backward_g_cost = bestCost;
          nd.backward_direction = computeParentDirection3D(bestD);
          nd.flags.backward_update_flag = 1;
        }
      } // end backward
    } // end for each node (relaxation)
    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // (2) Commit updated costs (double-buffering technique)
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int3 local = idxToLoc_3D(localIdx, xDimTemp, yDimTemp);
      int x = local.x + LLX;
      int y = local.y + LLY;
      int z = local.z;
      int idx = locToIdx_3D(x, y, z, xDim, yDim);
      NodeData3D &nd = d_nodes[idx];
      if (nd.flags.forward_update_flag) {
        nd.flags.forward_update_flag = false;
        nd.forward_g_cost_prev = nd.forward_g_cost;
        atomicMin(&s_minForwardCost, nd.forward_g_cost);
      }
      if (nd.flags.backward_update_flag) {
        nd.flags.backward_update_flag = false;
        nd.backward_g_cost_prev = nd.backward_g_cost;
        atomicMin(&s_minBackwardCost, nd.backward_g_cost);
      }
      // Save previous visited flags for later comparisons.
      nd.flags.forward_visited_flag_prev = nd.flags.forward_visited_flag;
      nd.flags.backward_visited_flag_prev = nd.flags.backward_visited_flag;
    }
    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // (3) Mark nodes using parallel reduction to select the minimum cost nodes.
    //     For the forward side, we find the node with the minimum forward_g_cost
    //     (among nodes not already visited) and mark its forward_visited_flag.
    //     Similarly for the backward side.    
    // Alternatively, one could scan all nodes to check for any node that has both flags true.
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int3 local = idxToLoc_3D(localIdx, xDimTemp, yDimTemp);
      int x = local.x + LLX;
      int y = local.y + LLY;
      int z = local.z;
      int idx = locToIdx_3D(x, y, z, xDim, yDim);
      NodeData3D &nd = d_nodes[idx];
      if (nd.forward_g_cost_prev <= s_minForwardCost) {
        nd.flags.forward_visited_flag = true;
      }

      if (nd.backward_g_cost_prev <= s_minBackwardCost) {
        nd.flags.backward_visited_flag = true;
      }

      // Check if the forward and backward fronts meet.
      if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
        localFrontsMeet = true;
      }
    }

    __syncthreads();

    if (localFrontsMeet) {
      atomicExch((int*)&s_doneFlag, 1);
    }
    
    if (tid == 0) {
      s_minForwardCost = 0x7FFFFFFF;
      s_minBackwardCost = 0x7FFFFFFF;
    }
    
    __syncthreads();

  } // end for (iter)

  __syncthreads();

  // Ensure all threads know the doneFlag
  bool converged = (s_doneFlag == 1);
  if (!converged) {
    if (tid == 0) {
      printf("Error! biwaveBellmanFord3D__device did not converge. doneFlag = false netId = %d\n", netId);
    }
    __syncthreads();
    return;
  } 
  __syncthreads();  

  // Iterate over your domain. Assume tid and stride are defined appropriately.
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int3 local = idxToLoc_3D(localIdx, xDimTemp, yDimTemp);
    int x = local.x + LLX;
    int y = local.y + LLY;
    int z = local.z;
    int idx = locToIdx_3D(x, y, z, xDim, yDim);
    auto& nd = d_nodes[idx];
    // Only consider nodes visited from both directions.
    if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
      // Assume each node has an integer cost value.
      int cost = nd.forward_g_cost + nd.backward_g_cost;
      // Pack the cost and idx into a 64-bit value.
      // Lower cost (in high bits) gives a lower overall value.
      unsigned long long candidate = (((unsigned long long)cost) << 32) | ((unsigned int)idx);
      atomicMin(&s_meet, candidate);
    }
  }
  __syncthreads();


  // After the loop, the meetId is stored in the lower 32 bits of s_meet.
  int s_meetId = (int)(s_meet & 0xFFFFFFFF);

  // Check if s_meetId is valid.
  if (s_meetId == 0x7FFFFFFF) {
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      printf("Error! biwaveBellmanFord3D__device did not converge. meetId = 0x7FFFFFFF, netId = %d\n", netId);
    }
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      atomicExch((int*)&tracebackError, 1);
    }
  }
  __syncthreads();
  
  // Only threads 0 and 1 perform the traceback if no error occurred.
  if (tracebackError == 0) {
    // ----- Forward Traceback (Thread 0) -----
    if (threadIdx.x == 0) {
      int tempIter = 0;      
      // Update the meetId accordingly to remove redundant path
      while (d_nodes[s_meetId].forward_direction == d_nodes[s_meetId].backward_direction && tempIter < total) {
        if (d_nodes[s_meetId].forward_direction == DIR_NONE_3D) {
          printf("Warning: forward_direction == DIR_NONE_3D netId = %d s_meetId = %d\n", netId, s_meetId);
          break;
        }
        
        int3 xyz = idxToLoc_3D(s_meetId, xDim, yDim);
        auto direction = d_nodes[s_meetId].forward_direction;
        int nx = xyz.x + s_dX[direction];
        int ny = xyz.y + s_dY[direction];
        int nz = xyz.z + s_dZ[direction];
        s_meetId = locToIdx_3D(nx, ny, nz, xDim, yDim);
        tempIter++;
      }

      if (tempIter >= total) {
        printf("Warning: reduce iteration exceeded maximum iterations. netId = %d\n", netId);
      }

      // forward trace
      int forwardCurId = s_meetId;
      int forwardIteration = 0;
      while (!d_nodes[forwardCurId].flags.src_flag && forwardIteration < total) {
        uint8_t fwdDir = d_nodes[forwardCurId].forward_direction;
        int3 xyz = idxToLoc_3D(forwardCurId, xDim, yDim);
        int nx = xyz.x + s_dX[fwdDir];
        int ny = xyz.y + s_dY[fwdDir];
        int nz = xyz.z + s_dZ[fwdDir];
        if (nx < LLX || nx > URX || ny < LLY || ny > URY || nz < 0 || nz >= zDim) {
          break;
        }
        d_nodes[forwardCurId].golden_parent_x = nx;
        d_nodes[forwardCurId].golden_parent_y = ny;
        d_nodes[forwardCurId].golden_parent_z = nz;
        d_nodes[forwardCurId].flags.src_flag = 1;
        forwardCurId = locToIdx_3D(nx, ny, nz, xDim, yDim);
        forwardIteration++;
      }
      
      if (forwardIteration >= total) {
        printf("Warning: Forward traceback exceeded maximum iterations. netId = %d\n", netId);
      }
   
      // backward trace
      int backwardCurId = s_meetId;
      int backwardIteration = 0;
      if (d_nodes[backwardCurId].flags.dst_flag == 1) {
        d_nodes[backwardCurId].flags.dst_flag = 0; // Reset dst flag.
        d_nodes[backwardCurId].flags.src_flag = 1;
      } else {
        while (!d_nodes[backwardCurId].flags.dst_flag && backwardIteration < total) {
          int3 xyz = idxToLoc_3D(backwardCurId, xDim, yDim);
          uint8_t backwardDir = d_nodes[backwardCurId].backward_direction;
          int nx = xyz.x + s_dX[backwardDir];
          int ny = xyz.y + s_dY[backwardDir];
          int nz = xyz.z + s_dZ[backwardDir];      
          if (nx < LLX || nx > URX || ny < LLY || ny > URY || nz < 0 || nz >= zDim) {
            break;
          }
          int nextId = locToIdx_3D(nx, ny, nz, xDim, yDim);       
          d_nodes[nextId].flags.src_flag = 1;
          d_nodes[nextId].golden_parent_x = xyz.x;
          d_nodes[nextId].golden_parent_y = xyz.y;
          d_nodes[nextId].golden_parent_z = xyz.z;
          backwardCurId = nextId;
          backwardIteration++;
        }
        
        d_nodes[backwardCurId].flags.dst_flag = 0;
        if (backwardIteration >= total) {
          printf("Warning: Backward traceback exceeded maximum iterations. netId = %d\n", netId);
        }
      }
    }
  }
  __syncthreads();
}


__device__ 
void biwaveBellmanFord3D__device(
  int netId,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData3D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_dZ,
  const int* d_xCoords,
  const int* d_yCoords,
  const int* d_zHeights,
  int maxIters,
  int xDim,
  int yDim,
  int zDim,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST, 
  int HISTCOST)
{
  // for this net
  int pinIdxStart = d_netPtr[netId];
  int pinIdxEnd = d_netPtr[netId + 1];
  int numPins = pinIdxEnd - pinIdxStart;
  Rect2D_CUDA netBBox = d_netBBoxVec[netId];
  int LLX = netBBox.xMin;
  int LLY = netBBox.yMin;
  int URX = netBBox.xMax;
  int URY = netBBox.yMax;

  for (int pinIter = 1; pinIter < numPins; pinIter++) {
    // Initialize the node data
    initNodeData3D__device(
      d_nodes,
      d_pins, pinIdxStart, pinIter, 
      LLX, LLY, URX, URY, 
      xDim, yDim, zDim);

    __syncthreads(); // Synchronize all threads in the block

    // Run the Bellman Ford algorithm
    runBiBellmanFord3D__device(
      netId, 
      d_nodes, d_costMap, 
      d_dX, d_dY, d_dZ,
      d_xCoords, d_yCoords, d_zHeights,
      LLX, LLY, URX, URY, 
      xDim, yDim, zDim, maxIters,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST);  

    __syncthreads(); // Synchronize all threads in the block
  }
}


__global__ 
void biwaveBellmanFord3D__kernel(
  int netStartId,
  int netEndId,
  int* d_netBatchIdx,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData3D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_dZ,
  const int* d_xCoords,
  const int* d_yCoords,
  const int* d_zHeights,
  int maxIters,
  int xDim,
  int yDim,
  int zDim,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST, 
  int HISTCOST)
{
  // Each net is handled by a single block
  for (int netId = netStartId + blockIdx.x; netId < netEndId; netId += gridDim.x) {
    biwaveBellmanFord3D__device(
      netId,
      d_netPtr,
      d_netBBoxVec,
      d_pins,
      d_costMap,
      d_nodes + d_netBatchIdx[netId] * xDim * yDim * zDim,
      d_dX,
      d_dY,
      d_dZ,
      d_xCoords,
      d_yCoords,
      d_zHeights,
      maxIters,
      xDim,
      yDim,
      zDim,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST, 
      HISTCOST);
  }
}



__global__ 
void initBatchNodeData3D__kernel(
  NodeData3D* d_nodes,
  int numNodes)
{
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numNodes; idx += gridDim.x * blockDim.x) {
    initNodeData3D(d_nodes[idx]);
  }
}

__global__
void initParent3D__kernel(
  Point3D_CUDA* d_parents,
  int numParents)
{
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numParents; idx += gridDim.x * blockDim.x) {
    d_parents[idx].x = -1;
    d_parents[idx].y = -1;
    d_parents[idx].z = -1;
  }
}

__global__
void copyParents3D__kernel(
  NodeData3D* d_nodes,
  Point3D_CUDA* d_parents,
  int numNodes)
{
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numNodes; idx += gridDim.x * blockDim.x) {
    d_parents[idx].x = d_nodes[idx].golden_parent_x;
    d_parents[idx].y = d_nodes[idx].golden_parent_y;
    d_parents[idx].z = d_nodes[idx].golden_parent_z;
  }
}

__global__
void initBatchPin3D__kernel(
  NodeData3D* d_nodes,
  int* d_pins,
  int* d_netPtr,
  int* d_netBatchIdx,
  int netIdStart,
  int netIdEnd,
  int numGrids)
{  
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int numNets = netIdEnd - netIdStart;
  for (tid; tid < numNets; tid += stride) {
    int netId = netIdStart + tid;
    int batchId = d_netBatchIdx[netId];
    int baseNodeId = batchId * numGrids;
    int pinIdxStart = d_netPtr[netId];
    int pinIdxEnd = d_netPtr[netId + 1];
    int pinId = d_pins[pinIdxStart] + baseNodeId;
    d_nodes[pinId].flags.src_flag = true;
    for (int pinIter = pinIdxStart + 1; pinIter < pinIdxEnd; pinIter++) {
      pinId = d_pins[pinIter] + baseNodeId;
      d_nodes[pinId].flags.dst_flag = true;
    }
  }
}


float FlexGR::GPUAccelerated3DMazeRoute_update(
  std::vector<std::unique_ptr<FlexGRWorker> >& uworkers,
  std::vector<std::vector<grNet*> >& netBatches,
  std::vector<int>& validBatches,
  std::vector<Point3D_CUDA>& h_parents_3D,
  std::vector<uint64_t>& h_costMap_3D,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  std::vector<int>& h_zHeights,
  float relaxThreshold,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST,
  int maxChunkSize,
  int xDim, int yDim, int zDim)
{
  // Start overall timing.
  auto totalStart = std::chrono::high_resolution_clock::now();
  int numGrids = xDim * yDim * zDim;
  int numBatches = validBatches.size();  
  if (numBatches == 0) {
    return 0.0;
  }

  int maxHPWL = 0; // We will run the algorithm for maxHPWL * relaxThreshold iteratively
  int maxBatchSize = 0;
  int minBatchSize = std::numeric_limits<int>::max();

  // To do list:
  // To avoid frequent memory allocation, we will allocate the memory once  
  std::vector<int> netPtr;
  std::vector<int> netBatchIdxVec; 
  std::vector<Rect2D_CUDA> netBBoxVec;
  std::vector<int> pinIdxVec;
  std::vector<int> chunkNetPtr; // store the first netIdx of each chunk
 
  netPtr.push_back(0);
  chunkNetPtr.push_back(0);
    
  int batchChunkIdx = 0;
  for (int batchIdx = 0; batchIdx < numBatches; batchIdx++) {
    auto& batch = netBatches[validBatches[batchIdx]];
    for (auto& net : batch) {
      for (auto& idx : net->getPinGCellAbsIdxs()) {
        pinIdxVec.push_back(locToIdx_3D(idx.x(), idx.y(), idx.z(), xDim, yDim));
      }
      netBatchIdxVec.push_back(batchChunkIdx);
      netPtr.push_back(pinIdxVec.size());    
      auto netBBox = net->getRouteAbsBBox();
      netBBoxVec.push_back(Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
      // To be updated 
      maxHPWL = std::max(maxHPWL, static_cast<int>((netBBox.xMax() - netBBox.xMin()) * (netBBox.yMax() - netBBox.yMin())));
    }
   
    batchChunkIdx++;
    if (batchChunkIdx % maxChunkSize == 0) {
      chunkNetPtr.push_back(netPtr.size() - 1);
      batchChunkIdx = 0;
    }
    
    maxBatchSize = std::max(maxBatchSize, static_cast<int>(batch.size()));
    minBatchSize = std::min(minBatchSize, static_cast<int>(batch.size()));
  }
 
  if (batchChunkIdx != 0) {
    chunkNetPtr.push_back(netPtr.size() - 1);
  }

  int numNets = static_cast<int>(netBBoxVec.size());
  int numChunks = static_cast<int>(chunkNetPtr.size()) - 1;
   
  int chunkSize = std::min(maxChunkSize, numBatches);
  int maxIters = static_cast<int>(maxHPWL * relaxThreshold);
  // numNodes == parentSize
  int numNodes = numGrids * chunkSize;
  int parentSize = numNodes;

  allocateCUDAMem3D(
    h_costMap_3D,
    h_xCoords,
    h_yCoords,
    h_zHeights,
    h_parents_3D,
    pinIdxVec,
    netPtr,
    netBBoxVec,
    netBatchIdxVec,
    numGrids, 
    maxChunkSize,
    numNodes);
  
  cudaCheckError();


  for (int chunkId = 0; chunkId < numChunks; chunkId++) {
    int netStartId = chunkNetPtr[chunkId];
    int netEndId = chunkNetPtr[chunkId + 1];
    
    // Perform Global Initialization
    // Just use the part that we need
    int numThreads = 1024;
    int numBatchBlocks = (numNodes + numThreads - 1) / numThreads;
    int numParentBlocks = (parentSize + numThreads - 1) / numThreads;    
    initParent3D__kernel<<<numParentBlocks, numThreads>>>(d_parents_3D_, parentSize);
    initBatchNodeData3D__kernel<<<numBatchBlocks, numThreads>>>(d_nodes_3D_, numNodes);
    
    hipDeviceSynchronize();
    cudaCheckError();
  
    int numNets = netEndId - netStartId;
    int numNetBlocks = (numNets + numThreads - 1) / numThreads;
    initBatchPin3D__kernel<<<numNetBlocks, numThreads>>>(
      d_nodes_3D_,
      d_pinIdxVec_, 
      d_netPtr_,
      d_netBatchIdx_,
      netStartId,
      netEndId,
      numGrids);
    hipDeviceSynchronize();
    cudaCheckError();

    int numBlocks = numNets;
    biwaveBellmanFord3D__kernel<<<numBlocks, numThreads>>>(
      netStartId,
      netEndId,
      d_netBatchIdx_,
      d_netPtr_,
      d_netBBox_,
      d_pinIdxVec_,
      d_costMap_,
      d_nodes_3D_,
      d_dX_,
      d_dY_,
      d_dZ_,
      d_xCoords_,
      d_yCoords_,
      d_zHeights_,
      maxIters,
      xDim,
      yDim,
      zDim,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST,
      HISTCOST);
    hipDeviceSynchronize();
    cudaCheckError();
   
    // copy the back results to the d_parents
    copyParents3D__kernel<<<numParentBlocks, numThreads>>>(
      d_nodes_3D_, 
      d_parents_3D_, 
      numNodes);
    hipDeviceSynchronize();
    cudaCheckError();

    hipMemcpy(
      h_parents_3D.data(), 
      d_parents_3D_, 
      h_parents_3D.size() * sizeof(Point3D_CUDA), 
      hipMemcpyDeviceToHost);
    cudaCheckError();
  }  

  auto totalEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> totalTime = totalEnd - totalStart;
  return totalTime.count();
}



void FlexGR::allocateCUDAMem3D(
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  std::vector<int>& h_zHeights,
  std::vector<Point3D_CUDA>& h_parents,
  std::vector<int>& pinIdxVec,
  std::vector<int>& netPtr,
  std::vector<Rect2D_CUDA>& netBBoxVec,
  std::vector<int>& netBatchIdxVec,
  int numGrids,
  int maxChunkSize,
  int numNodes)
{  
  if (d_dZ_ == nullptr) {
    std::vector<int> h_dX = {1, 0, -1, 0, 0, 0};
    std::vector<int> h_dY = {0, 1, 0, -1, 0, 0};
    std::vector<int> h_dZ = {0, 0, 0, 0, 1, -1};
    hipMalloc(&d_dX_, 6 * sizeof(int));
    hipMalloc(&d_dY_, 6 * sizeof(int));
    hipMalloc(&d_dZ_, 6 * sizeof(int));
    hipMemcpy(d_dX_, h_dX.data(), 6 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dY_, h_dY.data(), 6 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dZ_, h_dZ.data(), 6 * sizeof(int), hipMemcpyHostToDevice);
  }
  
  if (h_xCoords.size() > h_xCoords_size_) {
    h_xCoords_size_ = h_xCoords.size();
    hipFree(d_xCoords_);
    hipMalloc(&d_xCoords_, h_xCoords.size() * sizeof(int));
    hipMemcpy(d_xCoords_, h_xCoords.data(), h_xCoords.size() * sizeof(int), hipMemcpyHostToDevice); 
  }

  if (h_yCoords.size() > h_yCoords_size_) {
    h_yCoords_size_ = h_yCoords.size();
    hipFree(d_yCoords_);
    hipMalloc(&d_yCoords_, h_yCoords.size() * sizeof(int));
    hipMemcpy(d_yCoords_, h_yCoords.data(), h_yCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  }

  if (h_zHeights.size() > h_zHeights_size_) {
    h_zHeights_size_ = h_zHeights.size();
    hipFree(d_zHeights_);
    hipMalloc(&d_zHeights_, h_zHeights.size() * sizeof(int));
    hipMemcpy(d_zHeights_, h_zHeights.data(), h_zHeights.size() * sizeof(int), hipMemcpyHostToDevice);
  }

  if (h_costMap.size() > h_costMap_size_) {
    h_costMap_size_ = h_costMap.size();
    hipFree(d_costMap_);
    hipMalloc(&d_costMap_, h_costMap.size() * sizeof(uint64_t));
  }
  hipMemcpy(d_costMap_, h_costMap.data(), h_costMap.size() * sizeof(uint64_t), hipMemcpyHostToDevice);


  if (pinIdxVec.size() > h_pinIdxVec_size_) {
    h_pinIdxVec_size_ = pinIdxVec.size();
    hipFree(d_pinIdxVec_);
    hipMalloc(&d_pinIdxVec_, pinIdxVec.size() * sizeof(int));
  }
  hipMemcpy(d_pinIdxVec_, pinIdxVec.data(), pinIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);

  if (netPtr.size() > h_netPtr_size_) {
    h_netPtr_size_ = netPtr.size();
    hipFree(d_netPtr_);
    hipMalloc(&d_netPtr_, netPtr.size() * sizeof(int));
  }
  hipMemcpy(d_netPtr_, netPtr.data(), netPtr.size() * sizeof(int), hipMemcpyHostToDevice);

  if (netBBoxVec.size() > h_netBBoxVec_size_) {
    h_netBBoxVec_size_ = netBBoxVec.size();
    hipFree(d_netBBox_);
    hipMalloc(&d_netBBox_, netBBoxVec.size() * sizeof(Rect2D_CUDA));
  }
  hipMemcpy(d_netBBox_, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);

  if (netBatchIdxVec.size() > h_netBatchIdxVec_size_) {
    h_netBatchIdxVec_size_ = netBatchIdxVec.size();
    hipFree(d_netBatchIdx_);
    hipMalloc(&d_netBatchIdx_, netBatchIdxVec.size() * sizeof(int));
  }
  hipMemcpy(d_netBatchIdx_, netBatchIdxVec.data(), netBatchIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);

  if (h_parents.size() > h_parents_size_3D_) {
    // We only allocate once
    int maxParentSize = numGrids * maxChunkSize;
    h_parents_size_3D_ = maxParentSize;
    hipFree(d_parents_3D_);
    hipMalloc(&d_parents_3D_, maxParentSize * sizeof(Point3D_CUDA));
  }

  if (numNodes > h_nodes_size_) {
    int maxNodeSize = numGrids * maxChunkSize;
    h_nodes_size_3D_ = maxNodeSize;
    hipFree(d_nodes_3D_);
    hipMalloc(&d_nodes_3D_, maxNodeSize * sizeof(NodeData3D));    
  }
  cudaCheckError();
}


void FlexGR::freeCUDAMem3D()
{
  hipFree(d_dX_);
  hipFree(d_dY_);
  hipFree(d_dZ_);
  hipFree(d_costMap_);
  hipFree(d_xCoords_);
  hipFree(d_yCoords_);
  hipFree(d_zHeights_);
  hipFree(d_nodes_3D_);
  hipFree(d_parents_3D_);
  hipFree(d_pinIdxVec_);
  hipFree(d_netPtr_);
  hipFree(d_netBBox_);
  hipFree(d_netBatchIdx_);

  d_dX_ = nullptr;
  d_dY_ = nullptr;
  d_dZ_ = nullptr;
  d_costMap_ = nullptr;
  d_xCoords_ = nullptr;
  d_yCoords_ = nullptr;
  d_zHeights_ = nullptr;
  d_nodes_3D_ = nullptr;
  d_parents_3D_ = nullptr;
  d_pinIdxVec_ = nullptr;
  d_netPtr_ = nullptr;
  d_netBBox_ = nullptr;
  d_netBatchIdx_ = nullptr;

  h_costMap_size_ = 0;
  h_xCoords_size_ = 0;
  h_yCoords_size_ = 0;
  h_zHeights_size_ = 0;
  h_nodes_size_3D_ = 0;
  h_parents_size_3D_ = 0;
  h_pinIdxVec_size_ = 0;
  h_netPtr_size_ = 0;
  h_netBBoxVec_size_ = 0;
  h_netBatchIdxVec_size_ = 0;

  cudaCheckError();
}










} // namespace drt
