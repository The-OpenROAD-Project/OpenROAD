#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


 #include <iostream>
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime.h>
 #include <thrust/device_vector.h>
 #include <thrust/functional.h>
 #include <thrust/host_vector.h>
 #include <thrust/transform_reduce.h>
 #include <string>
 #include <functional>
 
 #include "db/grObj/grShape.h"
 #include "db/grObj/grVia.h"
 #include "db/infra/frTime.h"
 #include "db/obj/frGuide.h"
 #include "odb/db.h"
 #include "utl/exception.h"
 #include "FlexGR_util.h"
 #include "stt/SteinerTreeBuilder.h"
 #include "gr/FlexGR_util_update.h"
 #include "gr/FlexGRCMap.h"
 
 
 
namespace drt {
 
 
__device__
int getIdx_device(int x, int y, int z, int x_dim, int y_dim, int z_dim)
{
  return x + y * x_dim + z * x_dim * y_dim;
}
 
__device__
bool getBit_device(uint64_t* cmap, unsigned idx, unsigned pos)
{
  return (cmap[idx] >> pos) & 1;
}
 
__device__
unsigned getBits_device(uint64_t* cmap, unsigned idx, unsigned pos, unsigned length)
{
  auto tmp = cmap[idx] & (((1ull << length) - 1) << pos);
  return tmp >> pos;
}
 
 
__device__
void setBits_device(uint64_t* cmap, unsigned idx, unsigned pos, unsigned length, unsigned val)
{
  cmap[idx] &= ~(((1ull << length) - 1) << pos);  // clear related bits to 0
  cmap[idx] |= ((uint64_t) val & ((1ull << length) - 1))
                 << pos;  // only get last length bits of val
}
 
 
__device__
void addToBits_device(uint64_t* cmap, unsigned idx, unsigned pos, unsigned length, unsigned val)
{
  auto tmp = getBits_device(cmap, idx, pos, length) + val;
  tmp = (tmp > (1u << length)) ? (1u << length) : tmp;
  setBits_device(cmap, idx, pos, length, tmp);
}
 
 
__device__
unsigned getRawDemand_device(uint64_t* cmap,
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir)
{
  unsigned demand = 0;
  auto idx = getIdx_device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      demand = getBits_device(cmap, idx, 48, CMAPDEMANDSIZE);
      break;
    case frDirEnum::N:
      demand = getBits_device(cmap, idx, 32, CMAPDEMANDSIZE);
      break;
    case frDirEnum::U:;
      break;
    default:;
  }
 
  return demand;
}
 
 
__device__
unsigned getRawSupply_device(uint64_t* cmap, 
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir)
{
  unsigned supply = 0;
  auto idx = getIdx_device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      supply = getBits_device(cmap, idx, 24, CMAPSUPPLYSIZE);
      break;
    case frDirEnum::N:
      supply = getBits_device(cmap, idx, 16, CMAPSUPPLYSIZE);
      break;
    case frDirEnum::U:;
      break;
    default:;
  }
 
  return supply << CMAPFRACSIZE;
}
 
 
__device__
bool hasBlock_device(uint64_t* cmap, 
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir)
{
  bool sol = false;
  auto idx = getIdx_device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      sol = getBit_device(cmap, idx, 3);
      break;
    case frDirEnum::N:
      sol = getBit_device(cmap, idx, 2);
      break;
    case frDirEnum::U:;
      break;
    default:;
  }
   
  return sol;
}
 
/*
__device__
void addRawDemand_device(
  uint64_t* cmap,
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir, unsigned delta)
{
  int idx = getIdx_device(x, y, z, xDim, yDim, zDim);
  switch (dir) {
    case frDirEnum::E:
      addToBits_device(cmap, idx, 48, CMAPDEMANDSIZE, delta);
      break;
    case frDirEnum::N:
      addToBits_device(cmap, idx, 32, CMAPDEMANDSIZE, delta);
      break;
    case frDirEnum::U:
      break;
    default:;
  }
}
*/


__device__
void atomicAddBits(uint64_t* bits, size_t idx, int bitOffset, int bitSize, unsigned delta) {
    uint64_t mask = ((1ULL << bitSize) - 1) << bitOffset; // Create a mask for the target bits

    unsigned long long oldVal, newVal; // Use atomicCAS for atomic updates
    do {
        oldVal = atomicCAS((unsigned long long*)&bits[idx], bits[idx], bits[idx]); // Read current value
        uint64_t currentBits = (oldVal & mask) >> bitOffset; // Extract relevant bits
        uint64_t updatedBits = min(
            static_cast<uint64_t>(currentBits + delta),
            static_cast<uint64_t>((1ULL << bitSize) - 1)
        ); // Add delta and saturate
        newVal = (oldVal & ~mask) | (updatedBits << bitOffset); // Update the relevant bits
    } while (atomicCAS((unsigned long long*)&bits[idx], oldVal, newVal) != oldVal); // Repeat until successful
}



__device__
void addRawDemand_device(
  uint64_t* cmap,
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z, frDirEnum dir, unsigned delta)
{
  int idx = getIdx_device(x, y, z, xDim, yDim, zDim);
  int shiftAmount = 0;

  switch (dir) {
    case frDirEnum::E:
      shiftAmount = 48;
      break;
    case frDirEnum::N:
      shiftAmount = 32;
      break;
    default:
      return;
  }

  // Use atomic operations to safely update the relevant bits
  atomicAddBits(cmap, idx, shiftAmount, GRDEMANDSIZE, delta);
}


__device__
unsigned getHistoryCost_device(
  uint64_t* cmap,
  int xDim, int yDim, int zDim,
  unsigned x, unsigned y, unsigned z)
{
  int idx = getIdx_device(x, y, z, xDim, yDim, zDim);
  return getBits_device(cmap, idx, 8, GRGRIDGRAPHHISTCOSTSIZE);
}



__device__
float getCongCost_device(unsigned supply, unsigned demand)
{
  //return demand * (1.0 + 8.0 / (1.0 + exp(supply - demand))) / (supply + 1);
  // Modify the cost function to avoid overflow
  // Clamping to avoid overflow in exp()
  float exp_val = exp(std::min(10.0f, static_cast<float>(supply) - demand));  
  // Calculate the factor safely
  float factor = 8.0f / (1.0f + exp_val); 
  // Compute congestion cost with safety limits
  float congCost = demand * (1.0f + factor) / (supply + 1.0f);
  return congCost;  
}


} // namespace drt 

