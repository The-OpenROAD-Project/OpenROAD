#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR.h"
#include <omp.h> 
#include <cmath>
#include <fstream>
#include <iostream>
#include <future>

#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <string>
#include <stdint.h> // For fixed-width integers
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <map>
#include <queue>
#include <fstream>
#include <iostream>
#include <iterator>
#include <memory>
#include <set>
#include <sstream>
#include <string>
#include <vector>
#include <numeric>
#include <algorithm>

// We always work on the entire grid 
// The grid system is always in terms of global coordinates
// So for each net, we need to translate the local index system into global system

namespace drt {

constexpr int GRGRIDGRAPHHISTCOSTSIZE = 8;
constexpr int GRSUPPLYSIZE = 8;
constexpr int GRDEMANDSIZE = 16;
constexpr int GRFRACSIZE = 1;
constexpr int VERBOSE = 0; 

namespace cg = cooperative_groups;

#define cudaCheckError()                                                   \
{                                                                          \
    hipError_t err = hipGetLastError();                                  \
    if (err != hipSuccess) {                                              \
        fprintf(stderr, "CUDA error at %s:%d: %s\n",                       \
                __FILE__, __LINE__, hipGetErrorString(err));              \
        exit(1);                                                           \
    }                                                                      \
}

// We treat 0xFFFF as "infinite" cost for 32-bit fields
__device__ __host__ __constant__ uint32_t INF32 = 0xFFFFFFFF;



__host__ __device__ 
void initNodeData2D(NodeData2D& nd) {
  nd.forward_h_cost = INF32;
  nd.forward_g_cost = INF32;
  nd.backward_h_cost = INF32;
  nd.backward_g_cost = INF32;
  nd.forward_h_cost_prev = INF32;
  nd.forward_g_cost_prev = INF32;
  nd.backward_h_cost_prev = INF32;
  nd.backward_g_cost_prev = INF32;
  nd.forward_direction = DIR_NONE;
  nd.backward_direction = DIR_NONE;
  nd.forward_direction_prev = DIR_NONE;
  nd.backward_direction_prev = DIR_NONE;
  nd.golden_parent_x = -1;
  nd.golden_parent_y = -1;
  nd.flags.src_flag = 0;
  nd.flags.dst_flag = 0;
  nd.flags.forward_update_flag = 0;
  nd.flags.backward_update_flag = 0;
  nd.flags.forward_visited_flag = 0;
  nd.flags.backward_visited_flag = 0;
  nd.flags.forward_visited_flag_prev = 0;
  nd.flags.backward_visited_flag_prev = 0;
}


__host__ __device__
void printNode2D(NodeData2D& nd) {
  printf(" forward_g_cost = %d, backward_g_cost = %d ", nd.forward_g_cost, nd.backward_g_cost);
  printf(" forward_g_cost_prev = %d, backward_g_cost_prev = %d ", nd.forward_g_cost_prev, nd.backward_g_cost_prev);
  printf(" forward_direction = %d, backward_direction = %d ", nd.forward_direction, nd.backward_direction);
  printf(" forward_visited_flag = %d, backward_visited_flag = %d ", nd.flags.forward_visited_flag, nd.flags.backward_visited_flag);
  printf(" forward_visited_flag_prev = %d, backward_visited_flag_prev = %d ", nd.flags.forward_visited_flag_prev, nd.flags.backward_visited_flag_prev);
  printf(" parent_x = %d, parent_y = %d ", nd.golden_parent_x, nd.golden_parent_y);
  printf(" src_flag = %d, dst_flag = %d\n", nd.flags.src_flag, nd.flags.dst_flag);
}



__device__  
uint8_t computeParentDirection2D(int d) {
  switch(d) {
    case 0: return DIR_NORTH;
    case 1: return DIR_RIGHT;
    case 2: return DIR_SOUTH;
    case 3: return DIR_LEFT;
    default: return DIR_NONE;
  }
}


// Invert direction for backtracking
__device__  
uint8_t invertDirection2D(uint8_t d) {
  switch(d) {
    case DIR_NORTH:    return DIR_SOUTH;
    case DIR_SOUTH:    return DIR_NORTH;
    case DIR_LEFT:     return DIR_RIGHT;
    case DIR_RIGHT:    return DIR_LEFT;
    default:           return DIR_NONE;
  }
}


// Define the idxToLoc_2D function
// Convert linear index -> (x,y)
__device__ __host__  
int2 idxToLoc_2D(int idx, int xDim) {
  int x = idx % xDim;
  int y = idx / xDim;
  return make_int2(x,y);
}


// Define the locToIdx_2D function
// Convert (x,y) -> linear index
__device__ __host__  
int locToIdx_2D(int x, int y, int xDim) {
  return y * xDim + x;
}


// Bit related functions
__host__ __device__ 
bool getBit(const uint64_t* cmap, unsigned idx, unsigned pos)
{
  return (cmap[idx] >> pos) & 1;
}
 
__host__ __device__ 
unsigned getBits(const uint64_t* cmap, unsigned idx, unsigned pos, unsigned length)
{
  auto tmp = cmap[idx] & (((1ull << length) - 1) << pos);
  return tmp >> pos;
}
 

__host__ __device__ 
unsigned getHistoryCost(const uint64_t* cmap, int idx)
{
  return getBits(cmap, idx, 8, GRGRIDGRAPHHISTCOSTSIZE);
}


/*
__host__ __device__ 
float getCongCost(unsigned demand, unsigned supply)
{
  float exp_val = exp(std::min(10.0f, static_cast<float>(supply) - demand));  
  float factor = 4.0f / (1.0f + exp_val); 
  float congCost = demand * (1.0f + factor) / (supply + 1.0f);
  return congCost;  
}
*/


// Please DO NOT TOUCH the following function
// The performance of the function is critical to the overall performance of the router.
__host__ __device__
double getCongCost(unsigned demand, unsigned supply)
{
  return (demand * (4 / (1.0 + exp(static_cast<double>(supply) - demand))) / (supply + 1));
}


__host__ __device__
unsigned getRawDemand2D(const uint64_t* cmap, int idx, Directions2D dir)
{
  unsigned demand = 0;
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      demand = getBits(cmap, idx, 48, GRDEMANDSIZE);
      break;
    case Directions2D::DIR_NORTH:
      demand = getBits(cmap, idx, 32, GRDEMANDSIZE);
      break;
    default:;
  }
  return demand;
}


__host__ __device__
unsigned getRawSupply2D(const uint64_t* cmap, int idx, Directions2D dir)
{
  unsigned supply = 0;
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      supply = getBits(cmap, idx, 24, GRSUPPLYSIZE);
      break;
    case Directions2D::DIR_NORTH:
      supply = getBits(cmap, idx, 16, GRSUPPLYSIZE);
      break;
    default:;
  }
  return supply << GRFRACSIZE;
}


__host__ __device__
bool hasBlock2D(const uint64_t* cmap, int idx, Directions2D dir)
{
  bool sol = false;
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      sol = getBit(cmap, idx, 3);
      break;
    case Directions2D::DIR_NORTH:
      sol = getBit(cmap, idx, 2);
      break;
    default:;
  }
  return sol;
}


__host__ __device__
bool hasEdge2D(const uint64_t* bits, int idx, Directions2D dir)
{
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      return getBit(bits, idx, 0);
    case Directions2D::DIR_NORTH:
      return getBit(bits, idx, 1);
    default:;
  }  
}


__host__ __device__
unsigned getEdgeLength2D(const int* xCoords, const int* yCoords, 
  int x, int y, Directions2D dir)
{
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      return xCoords[x + 1] - xCoords[x];
    case Directions2D::DIR_NORTH:
      return yCoords[y + 1] - yCoords[y];
    default:
      return 0;
  }
}



// We do not consider bending cost in this version
__host__ __device__
uint32_t getEdgeCost2D(
  const uint64_t* d_costMap,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST,
  int idx, int x, int y,
  Directions2D dir)
{
  bool blockCost = hasBlock2D(d_costMap, idx, dir);
  unsigned histCost = getHistoryCost(d_costMap, idx);
  unsigned rawDemand = getRawDemand2D(d_costMap, idx, dir);
  unsigned rawSupply = getRawSupply2D(d_costMap, idx, dir) * congThreshold;
  bool overflowCost = (rawDemand >= rawSupply);
  double congCost = getCongCost(rawDemand, rawSupply);
  unsigned edgeLength = getEdgeLength2D(d_xCoords, d_yCoords, x, y, dir);

  // cost 
  uint32_t edgeCost = edgeLength
    + edgeLength * congCost
    + (histCost ? HISTCOST * edgeLength * congCost * HISTCOST : 0)
    + (blockCost ? BLOCKCOST * edgeLength : 0)
    + (overflowCost ? OVERFLOWCOST * edgeLength : 0);

  return edgeCost;
}


// Define the device function for node initialization

__device__
void initNodeData2D__device(
  NodeData2D* d_nodes,
  int* d_pins, int pinIterStart, int pinIter,  // Pin related variables
  int LLX, int LLY, int URX, int URY, // Bounding box
  int xDim)
{ 
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
  int xDimTemp = URX - LLX + 1;
  for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    int2 xy = idxToLoc_2D(idx, xDim);
    int2 src = idxToLoc_2D(d_pins[pinIterStart + pinIter - 1], xDim);
    int2 dst = idxToLoc_2D(d_pins[pinIterStart + pinIter], xDim);
  
    // The experimental results show that the heuristic cost is not needed
    d_nodes[idx].forward_h_cost = 0;
    d_nodes[idx].backward_h_cost = 0;

    if (d_nodes[idx].flags.src_flag) {
      d_nodes[idx].forward_g_cost = 0;
      d_nodes[idx].forward_g_cost_prev = 0;
      d_nodes[idx].flags.forward_visited_flag = true;
      d_nodes[idx].flags.forward_visited_flag_prev = true;
    } else {
      d_nodes[idx].forward_g_cost = INF32;
      d_nodes[idx].forward_g_cost_prev = INF32;
      d_nodes[idx].flags.forward_visited_flag = false;
      d_nodes[idx].flags.forward_visited_flag_prev = false;
    }

    if (d_nodes[idx].flags.dst_flag) {
      d_nodes[idx].backward_g_cost = 0;
      d_nodes[idx].backward_g_cost_prev = 0;
      d_nodes[idx].flags.backward_visited_flag = true;
      d_nodes[idx].flags.backward_visited_flag_prev = true;
    } else {
      d_nodes[idx].backward_g_cost = INF32;
      d_nodes[idx].backward_g_cost_prev = INF32;
      d_nodes[idx].flags.backward_visited_flag = false;
      d_nodes[idx].flags.backward_visited_flag_prev = false;
    }

    d_nodes[idx].forward_direction = DIR_NONE;
    d_nodes[idx].backward_direction = DIR_NONE;
    d_nodes[idx].forward_direction_prev = DIR_NONE;
    d_nodes[idx].backward_direction_prev = DIR_NONE;
    d_nodes[idx].flags.forward_update_flag = false;
    d_nodes[idx].flags.backward_update_flag = false;
    d_nodes[idx].flags.forward_visited_flag = false;
    d_nodes[idx].flags.backward_visited_flag = false;
    d_nodes[idx].flags.forward_visited_flag_prev = false;
    d_nodes[idx].flags.backward_visited_flag_prev = false;
  } 
}


__device__ 
uint32_t getNeighorCost2D(
  const uint64_t* d_costMap,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST,
  int idx, int x, int y,
  int nbrIdx, int nx, int ny)
{
  uint32_t newG = 0;
  if (nx == x && ny == y - 1) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      nbrIdx, nx, ny, Directions2D::DIR_NORTH);
  } else if (nx == x && ny == y + 1) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      idx, x, y, Directions2D::DIR_NORTH);
  } else if (nx == x - 1 && ny == y) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      nbrIdx, nx, ny, Directions2D::DIR_RIGHT);
  } else if (nx == x + 1 && ny == y) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      idx, x, y, Directions2D::DIR_RIGHT);
  }

  return newG;
}



// Define the device function for the biwaveBellmanFord_2D_v3__device
__device__
void runBiBellmanFord2D_v3__device(
  NodeData2D* d_nodes,
  uint64_t* d_costMap, 
  const int* __restrict__ d_dX, 
  const int* __restrict__ d_dY,
  const int* __restrict__ d_xCoords,
  const int* __restrict__ d_yCoords,
  int LLX, int LLY, int URX, int URY,
  int xDim, 
  int maxIters,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST)
{
  // Each device function is handled by a single block
  int total = (URX - LLX + 1) * (URY - LLY + 1);
  int tid = threadIdx.x;
  int stride = blockDim.x; 
  int xDimTemp = URX - LLX + 1;
  
  
  // define the shared memory for d_dx and d_dy
  __shared__ int s_dX[4];
  __shared__ int s_dY[4];
  volatile __shared__ int s_doneFlag;
  volatile __shared__ int s_minCost;
  volatile __shared__ int s_meetId;
  __shared__ int tracebackError;   // 0: no error; 1: error detected


  // Load the d_dX and d_dY into shared memory
  if (tid < 4) {
    s_dX[tid] = d_dX[tid];
    s_dY[tid] = d_dY[tid];
    if (tid == 0) {
      s_doneFlag = 0;
      s_minCost = 0x7FFFFFFF;
      s_meetId = 0x7FFFFFFF;
      tracebackError = 0;
    }
  }

  __syncthreads();

  // We’ll do up to maxIters or until no changes / front-meet
  for (int iter = 0; iter < maxIters && (s_doneFlag == 0); iter++)
  {
    bool localFrontsMeet = false;
    ////////////////////////////////////////////////////////////////////////////
    // (1) Forward & backward relaxation phase
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      NodeData2D &nd = d_nodes[idx];

      // Forward relaxation
      // Typically: newCost = min over neighbors of (neighborCost + edgeWeight).
      // Be sure to skip if src_flag is set (source node may be pinned).
      if (!nd.flags.src_flag) {
        uint32_t bestCost = nd.forward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
        
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          // We want neighbor's cost plus the edge weight, e.g. 100
          uint32_t neighborCost = d_nodes[nbrIdx].forward_g_cost_prev;
          // If neighbor is effectively infinite, skip
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
        
          uint32_t newG = neighborCost +
            getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, nbrIdx, nx, ny);

          // Check if we found a better cost
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) { // We found an improvement
          nd.forward_g_cost = bestCost;
          nd.forward_direction = computeParentDirection2D(bestD);
          nd.flags.forward_update_flag = 1;
        }
      } // end forward

      // Backward relaxation
      // Typically: newCost = min over neighbors of (neighbor.backward_cost + edgeWeight)
      // Skip if dst_flag is set (destination node may be pinned).
      if (!nd.flags.dst_flag) {
        uint32_t bestCost = nd.backward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
        
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          uint32_t neighborCost = d_nodes[nbrIdx].backward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
        
          uint32_t newG = neighborCost +
          getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
            congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
            idx, x, y, nbrIdx, nx, ny);
        
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) {
          nd.backward_g_cost = bestCost;
          nd.backward_direction = computeParentDirection2D(bestD);
          nd.flags.backward_update_flag = 1;
        }
      } // end backward
    } // end “for each node” (forward + backward)

    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // (2) Commit updated costs (double-buffering technique)
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = d_nodes[idx];
      // If forward_update_flag is set, copy forward_g_cost -> forward_g_cost_prev
      if (nd.flags.forward_update_flag) {
        nd.flags.forward_update_flag = false;
        nd.forward_g_cost_prev = nd.forward_g_cost;
      }
      
      // If backward_update_flag is set, copy backward_g_cost -> backward_g_cost_prev
      if (nd.flags.backward_update_flag) {
        nd.flags.backward_update_flag = false;
        nd.backward_g_cost_prev = nd.backward_g_cost;
      }

      nd.flags.forward_visited_flag_prev = nd.flags.forward_visited_flag;
      nd.flags.backward_visited_flag_prev = nd.flags.backward_visited_flag;
    }

    __syncthreads();


    // Needs to be updated
    ////////////////////////////////////////////////////////////////////////////
    // (3) Check if forward and backward fronts meet
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      NodeData2D &nd = d_nodes[idx];

      // if (idx == 6007 && (nd.forward_g_cost == 91514 || nd.flags.forward_visited_flag_prev)) {
      //  printf("idx = %d, x = %d, y = %d, cost = %d, forward_visited_flag = %d\n", 
      //    idx, x, y, nd.forward_g_cost, nd.flags.forward_visited_flag_prev);
      //}

      // Check the forward visited flag
      if (!nd.flags.forward_visited_flag_prev) {
        bool localForwardMin = (nd.forward_g_cost_prev != 0xFFFFFFFF);
      

        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
         
                  
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          NodeData2D &nbr = d_nodes[nbrIdx];
          
          /*
          int neighborCost = nbr.forward_g_cost_prev;          
          uint32_t newG = neighborCost +
          getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
            congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
            idx, x, y, nbrIdx, nx, ny);

          if (idx == 6007 && nd.forward_g_cost_prev == 91514) {
            printf("nbrIdx = %d, neighborCost = %d, nd.cost = %d\n", nbrIdx, neighborCost, nd.forward_g_cost_prev);
          }*/

      
          // Check forward minimum
          if (!nbr.flags.forward_visited_flag_prev && 
              (nbr.forward_g_cost_prev + nbr.forward_h_cost < nd.forward_g_cost_prev + nd.forward_h_cost)) {
            localForwardMin = false;
          } 

        }

        if (localForwardMin) {
          nd.flags.forward_visited_flag = true;
        }
      }

      // Check the backward visited flag
      if (!nd.flags.backward_visited_flag_prev) {
        bool localBackwardMin = (nd.backward_g_cost_prev != 0xFFFFFFFF);
        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          NodeData2D &nbr = d_nodes[nbrIdx];
          

          /*
          int neighborCost = nbr.backward_g_cost_prev;          
          uint32_t newG = neighborCost +
          getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
            congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
            idx, x, y, nbrIdx, nx, ny);
          */
       
          
          // Check forward minimum
          if (!nbr.flags.backward_visited_flag_prev && 
              (nbr.backward_g_cost_prev + nbr.backward_h_cost < nd.backward_g_cost_prev + nd.backward_h_cost)) {
            localBackwardMin = false;
          } 
        }

        if (localBackwardMin) {
          nd.flags.backward_visited_flag = true;
        }
      }
    } // end “for each node”
    
  
    __syncthreads();

    // Check if any thread found a front-meet
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = d_nodes[idx];
      if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
        localFrontsMeet = true;
      }
    }

    __syncthreads();

    if (localFrontsMeet) {
      atomicExch((int*)&s_doneFlag, 1);
    }
    
    __syncthreads();

  } // end for (iter)

  __syncthreads();

  // Ensure all threads know the doneFlag
  bool converged = (s_doneFlag == 1);
  if (!converged) {
    if (tid == 0) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. doneFlag = false \n");
    }
    __syncthreads();
    return;
  }

  __syncthreads();  

  // identify the minimum cost
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    auto& nd = d_nodes[idx];
    if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
      int32_t cost = nd.forward_g_cost + nd.backward_g_cost;
      atomicMin((int*)&s_minCost, cost);
    }    
  }

  // identify the meetId
  __syncthreads();
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    auto& nd = d_nodes[idx];
    if ((nd.forward_g_cost != INF32 && nd.backward_g_cost != INF32) &&
        (nd.forward_g_cost + nd.backward_g_cost == s_minCost)) {
      atomicMin((int*)&s_meetId, idx);      
    }
  }

  // identify the parent
  __syncthreads();

  
  // Check if s_meetId is valid. (All threads in warp 0 & 1 perform the check.)
  if (s_meetId == 0x7FFFFFFF) {
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. meetId = 0x7FFFFFFF\n");
    }
    // Set the error flag (using atomicExch for safety in parallel regions)
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      atomicExch(&tracebackError, 1);
    }
  }
  __syncthreads();
  
  // Only threads 0 and 1 perform the traceback if no error occurred.
  if (tracebackError == 0) {
    // ----- Forward Traceback (Thread 0) -----
    if (threadIdx.x == 0) {
      printf("Start the traceback\n");
    
      int tempIter = 0;
      // Update the meetId accrodingly to remove reduant path
      while (d_nodes[s_meetId].forward_direction == d_nodes[s_meetId].backward_direction && tempIter < total) {
        if (d_nodes[s_meetId].forward_direction == DIR_NONE) {
          printf("Error: forward_direction == DIR_NONE\n");
          break;
        }
      
        printf("s_meetId = %d, forward_direction = %d, backward_direction = %d\n", s_meetId, d_nodes[s_meetId].forward_direction, d_nodes[s_meetId].backward_direction);
        int2 xy = idxToLoc_2D(s_meetId, xDim);
        auto direction = d_nodes[s_meetId].forward_direction;
        int nx = xy.x + s_dX[direction];
        int ny = xy.y + s_dY[direction];
        s_meetId = locToIdx_2D(nx, ny, xDim);
        tempIter++;
      }

      if (tempIter >= total) {
        printf("Error: Forward traceback exceeded maximum iterations.\n");
      }
  
      printf("Start the traceback\n");

      int forwardCurId = s_meetId;
      int forwardIteration = 0;
      while (!d_nodes[forwardCurId].flags.src_flag && forwardIteration < total) {
        uint8_t fwdDir = d_nodes[forwardCurId].forward_direction;
        int2 xy = idxToLoc_2D(forwardCurId, xDim);
        int nx = xy.x + s_dX[fwdDir];
        int ny = xy.y + s_dY[fwdDir];
        // Break if the next position is out of bounds.
        if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
          break;
        }
        // Record the golden parent before moving on.
        d_nodes[forwardCurId].golden_parent_x = nx;
        d_nodes[forwardCurId].golden_parent_y = ny;
        // Mark the node as processed.
        d_nodes[forwardCurId].flags.src_flag = 1;
        
        // Move to the next node.
        forwardCurId = locToIdx_2D(nx, ny, xDim);
        forwardIteration++;
      }
      if (forwardIteration >= total) {
        printf("Warning: Forward traceback exceeded maximum iterations.\n");
      }
    // }

    printf("Start the backward traceback\n");
      
    // ----- Backward Traceback (Thread 1) -----
    // if (threadIdx.x == 1) {
      int backwardCurId = s_meetId;
      int backwardIteration = 0;
      // Special handling if the starting node is already marked as destination.
      if (d_nodes[backwardCurId].flags.dst_flag == 1) {
        d_nodes[backwardCurId].flags.dst_flag = 0; // Reset dst flag.
        d_nodes[backwardCurId].flags.src_flag = 1;
      } else {
        while (!d_nodes[backwardCurId].flags.dst_flag && backwardIteration < total) {
          int2 xy = idxToLoc_2D(backwardCurId, xDim);
          uint8_t backwardDir = d_nodes[backwardCurId].backward_direction;
          int nx = xy.x + s_dX[backwardDir];
          int ny = xy.y + s_dY[backwardDir];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            break;
          }
          int nextId = locToIdx_2D(nx, ny, xDim);
          // Check if backward traceback collides with the forward traceback.
          if (d_nodes[nextId].golden_parent_x != -1 || d_nodes[nextId].golden_parent_y != -1) {
            printf("Error: Backward traceback meets forward traceback.\n");
            printf("meetId = %d\n", s_meetId); 
            for (int localIdx = 0; localIdx < total; localIdx++) {
              int local_x = localIdx % xDimTemp + LLX;
              int local_y = localIdx / xDimTemp + LLY;
              int idx = locToIdx_2D(local_x, local_y, xDim);
              printf("node id = %d, x = %d, y = %d ", idx, local_x, local_y);
              printNode2D(d_nodes[idx]);
            }
            
            printf("nextId = %d, x = %d, y = %d,  golden_parent_x = %d, golden_parent_y = %d, dst_flag = %d, src_flag = %d\n", 
              nextId, nx, ny, 
              d_nodes[nextId].golden_parent_x, d_nodes[nextId].golden_parent_y, 
              d_nodes[nextId].flags.dst_flag, d_nodes[nextId].flags.src_flag);
            atomicExch(&tracebackError, 1);
            break;      
          }
          
          // Update parent's information.
          d_nodes[nextId].flags.src_flag = 1;
          d_nodes[nextId].golden_parent_x = xy.x;
          d_nodes[nextId].golden_parent_y = xy.y;
          
          backwardCurId = nextId;
          backwardIteration++;
        }
        // Reset dst flag at the final node.
        d_nodes[backwardCurId].flags.dst_flag = 0;
        if (backwardIteration >= total) {
          printf("Warning: Backward traceback exceeded maximum iterations.\n");
        }
      }

      printf("End the traceback\n");
    }
  }
  __syncthreads();


  /*
  // We only need the first thread to update the parent
  if (tid == 0) {
    if (s_meetId == 0x7FFFFFFF) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. meetId = 0x7FFFFFFF \n");
      return;
    }

    int curId = s_meetId;
    int maxIterations = total;
    int iteration = 0;

    // Forward traceback
    while (d_nodes[curId].flags.src_flag == 0 && iteration < maxIterations) {
      // Ensure forward_direction is valid (e.g., 0 <= forward_direction < 4)
      uint8_t fwdDir = d_nodes[curId].forward_direction;
      // Record the golden parent BEFORE moving on.
      int2 xy = idxToLoc_2D(curId, xDim);
      int nx = xy.x + s_dX[fwdDir];
      int ny = xy.y + s_dY[fwdDir];
      if (nx < LLX || nx > URX || ny < LLY || ny > URY) { break; }
      
      d_nodes[curId].golden_parent_x = nx;
      d_nodes[curId].golden_parent_y = ny;
      
      // Mark this node as processed.
      d_nodes[curId].flags.src_flag = 1;
      
      // Move to the next node.
      curId = locToIdx_2D(nx, ny, xDim);
      iteration++;
    }
   
    if (iteration >= maxIterations) {
      printf("Warning: Forward traceback exceeded maximum iterations.\n");
    }
  } else if (tid == 1) {   
    // Backward traceback
    if (s_meetId == 0x7FFFFFFF) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. meetId = 0x7FFFFFFF \n");
      return;
    }
    
    int maxIterations = total;
    int curId = s_meetId;
    int iteration = 0;
    
    if (d_nodes[curId].flags.dst_flag == 1) { 
      d_nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
      d_nodes[curId].flags.src_flag = 1;
      return;
    }
  
    while (d_nodes[curId].flags.dst_flag == 0 && iteration < maxIterations) {
      int2 xy = idxToLoc_2D(curId, xDim);
      uint8_t backwardDirection = d_nodes[curId].backward_direction;
      int nx = xy.x + s_dX[backwardDirection];
      int ny = xy.y + s_dY[backwardDirection];
      if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
        break;
      }  
     
      int nextId = locToIdx_2D(nx, ny, xDim);
      if (d_nodes[nextId].golden_parent_x != -1) {
        printf("Error: Backward traceback meets forward traceback.\n");
      }    
      
      d_nodes[nextId].flags.src_flag = 1;
      d_nodes[nextId].golden_parent_x = xy.x;
      d_nodes[nextId].golden_parent_y = xy.y;
      
      curId = nextId;
      iteration++;
    }
    
    d_nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
    if (iteration >= maxIterations) {
      printf("Warning: Backward traceback exceeded maximum iterations.\n");
    }
  }

  __syncthreads();
  */
}



// Define the device function for the biwaveBellmanFord_2D_v4__device
__device__
void runBiBellmanFord2D_v4__device(
  NodeData2D* d_nodes,
  uint64_t* d_costMap, 
  const int* __restrict__ d_dX, 
  const int* __restrict__ d_dY,
  const int* __restrict__ d_xCoords,
  const int* __restrict__ d_yCoords,
  int LLX, int LLY, int URX, int URY,
  int xDim, 
  int maxIters,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST)
{
  // Each device function is handled by a single block
  int total = (URX - LLX + 1) * (URY - LLY + 1);
  int tid = threadIdx.x;
  int stride = blockDim.x; 
  int xDimTemp = URX - LLX + 1;
  
  // Define the shared memory for d_dx and d_dy
  __shared__ int s_dX[4];
  __shared__ int s_dY[4];
  volatile __shared__ int s_doneFlag;
  volatile __shared__ int s_minCost;
  volatile __shared__ int s_meetId;
  __shared__ int tracebackError;   // 0: no error; 1: error detected

  // Load the d_dX and d_dY into shared memory
  if (tid < 4) {
    s_dX[tid] = d_dX[tid];
    s_dY[tid] = d_dY[tid];
    if (tid == 0) {
      s_doneFlag = 0;
      s_minCost = 0x7FFFFFFF;
      s_meetId = 0x7FFFFFFF;
      tracebackError = 0;
    }
  }
  __syncthreads();

  // We'll do up to maxIters or until no changes / front-meet
  for (int iter = 0; iter < maxIters && (s_doneFlag == 0); iter++)
  {
    bool localFrontsMeet = false;
    ////////////////////////////////////////////////////////////////////////////
    // (1) Forward & backward relaxation phase
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      NodeData2D &nd = d_nodes[idx];

      // Forward relaxation:
      // newCost = min over neighbors of (neighborCost + edgeWeight).
      // Skip if src_flag is set.
      if (!nd.flags.src_flag) {
        uint32_t bestCost = nd.forward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          uint32_t neighborCost = d_nodes[nbrIdx].forward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          uint32_t newG = neighborCost +
            getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, nbrIdx, nx, ny);
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) { // Found an improvement
          nd.forward_g_cost = bestCost;
          nd.forward_direction = computeParentDirection2D(bestD);
          nd.flags.forward_update_flag = 1;
        }
      } // end forward

      // Backward relaxation:
      // newCost = min over neighbors of (neighbor.backward_cost + edgeWeight).
      // Skip if dst_flag is set.
      if (!nd.flags.dst_flag) {
        uint32_t bestCost = nd.backward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          uint32_t neighborCost = d_nodes[nbrIdx].backward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          uint32_t newG = neighborCost +
            getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, nbrIdx, nx, ny);
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) {
          nd.backward_g_cost = bestCost;
          nd.backward_direction = computeParentDirection2D(bestD);
          nd.flags.backward_update_flag = 1;
        }
      } // end backward
    } // end for each node (relaxation)
    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // (2) Commit updated costs (double-buffering technique)
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = d_nodes[idx];
      if (nd.flags.forward_update_flag) {
        nd.flags.forward_update_flag = false;
        nd.forward_g_cost_prev = nd.forward_g_cost;
      }
      if (nd.flags.backward_update_flag) {
        nd.flags.backward_update_flag = false;
        nd.backward_g_cost_prev = nd.backward_g_cost;
      }
      // Save previous visited flags for later comparisons.
      nd.flags.forward_visited_flag_prev = nd.flags.forward_visited_flag;
      nd.flags.backward_visited_flag_prev = nd.flags.backward_visited_flag;
    }
    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // (3) Mark nodes using parallel reduction to select the minimum cost nodes.
    //     For the forward side, we find the node with the minimum forward_g_cost
    //     (among nodes not already visited) and mark its forward_visited_flag.
    //     Similarly for the backward side.
    ////////////////////////////////////////////////////////////////////////////
    __shared__ int s_minForwardCost;
    __shared__ int s_minBackwardCost;
  
    if (threadIdx.x == 0) {
      s_minForwardCost = 0x7FFFFFFF;
      s_minBackwardCost = 0x7FFFFFFF;
    }

    __syncthreads();

    // Each thread examines a subset of nodes.
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      NodeData2D &nd = d_nodes[idx];

      // For forward: only consider nodes not yet visited with a valid cost.
      if (!nd.flags.forward_visited_flag_prev && nd.forward_g_cost_prev != 0xFFFFFFFF) {
        atomicMin(&s_minForwardCost, nd.forward_g_cost_prev);
      }
      
      // For backward: only consider nodes not yet visited with a valid cost.
      if (!nd.flags.backward_visited_flag_prev && nd.backward_g_cost_prev != 0xFFFFFFFF) {
        atomicMin(&s_minBackwardCost, nd.backward_g_cost_prev);
      }
    }
    __syncthreads();
    
    // Alternatively, one could scan all nodes to check for any node that has both flags true.
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = d_nodes[idx];
      if (nd.forward_g_cost_prev == s_minForwardCost) {
        nd.flags.forward_visited_flag = true;
      }

      if (nd.backward_g_cost_prev == s_minBackwardCost) {
        nd.flags.backward_visited_flag = true;
      }

      // Check if the forward and backward fronts meet.
      if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
        localFrontsMeet = true;
      }
    }

    __syncthreads();

    if (localFrontsMeet) {
      atomicExch((int*)&s_doneFlag, 1);
    }
    __syncthreads();

  } // end for (iter)

  __syncthreads();

  // Ensure all threads know the doneFlag
  bool converged = (s_doneFlag == 1);
  if (!converged) {
    if (tid == 0) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. doneFlag = false \n");
    }
    __syncthreads();
    return;
  } 
  __syncthreads();  

  // Identify the minimum total cost (forward + backward) among meeting nodes.
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    auto& nd = d_nodes[idx];
    if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
      int32_t cost = nd.forward_g_cost + nd.backward_g_cost;
      atomicMin((int*)&s_minCost, cost);
    }    
  }

  // Identify the meetId corresponding to the minimum cost.
  __syncthreads();
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    auto& nd = d_nodes[idx];
    if ((nd.forward_g_cost != INF32 && nd.backward_g_cost != INF32) &&
        (nd.forward_g_cost + nd.backward_g_cost == s_minCost)) {
      atomicMin((int*)&s_meetId, idx);      
    }
  }
  __syncthreads();

  // Check if s_meetId is valid.
  if (s_meetId == 0x7FFFFFFF) {
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. meetId = 0x7FFFFFFF\n");
    }
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      atomicExch(&tracebackError, 1);
    }
  }
  __syncthreads();
  
  // Only threads 0 and 1 perform the traceback if no error occurred.
  if (tracebackError == 0) {
    // ----- Forward Traceback (Thread 0) -----
    if (threadIdx.x == 0) {
      // printf("Start the traceback\n");
      int tempIter = 0;      
      // Update the meetId accordingly to remove redundant path
      while (d_nodes[s_meetId].forward_direction == d_nodes[s_meetId].backward_direction && tempIter < total) {
        if (d_nodes[s_meetId].forward_direction == DIR_NONE) {
          printf("Warning: forward_direction == DIR_NONE\n");
          break;
        }
        
        int2 xy = idxToLoc_2D(s_meetId, xDim);
        auto direction = d_nodes[s_meetId].forward_direction;
        int nx = xy.x + s_dX[direction];
        int ny = xy.y + s_dY[direction];
        s_meetId = locToIdx_2D(nx, ny, xDim);
        tempIter++;
      }

      if (tempIter >= total) {
        printf("Warning: reduce iteration exceeded maximum iterations.\n");
      }
            
      // printf("Start the forward traceback\n");

      int forwardCurId = s_meetId;
      int forwardIteration = 0;
      while (!d_nodes[forwardCurId].flags.src_flag && forwardIteration < total) {
        uint8_t fwdDir = d_nodes[forwardCurId].forward_direction;
        int2 xy = idxToLoc_2D(forwardCurId, xDim);
        int nx = xy.x + s_dX[fwdDir];
        int ny = xy.y + s_dY[fwdDir];
        if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
          break;
        }
        d_nodes[forwardCurId].golden_parent_x = nx;
        d_nodes[forwardCurId].golden_parent_y = ny;
        d_nodes[forwardCurId].flags.src_flag = 1;
        forwardCurId = locToIdx_2D(nx, ny, xDim);
        forwardIteration++;
      }
      if (forwardIteration >= total) {
        printf("Warning: Forward traceback exceeded maximum iterations.\n");
      }
    // }
    

     //  printf("Start the backward traceback\n");
    // ----- Backward Traceback (Thread 1) -----
    // if (threadIdx.x == 1) {
      int backwardCurId = s_meetId;
      int backwardIteration = 0;
      if (d_nodes[backwardCurId].flags.dst_flag == 1) {
        d_nodes[backwardCurId].flags.dst_flag = 0; // Reset dst flag.
        d_nodes[backwardCurId].flags.src_flag = 1;
      } else {
        while (!d_nodes[backwardCurId].flags.dst_flag && backwardIteration < total) {
          int2 xy = idxToLoc_2D(backwardCurId, xDim);
          uint8_t backwardDir = d_nodes[backwardCurId].backward_direction;
          int nx = xy.x + s_dX[backwardDir];
          int ny = xy.y + s_dY[backwardDir];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            break;
          }
          int nextId = locToIdx_2D(nx, ny, xDim);
          
          /*
          if (d_nodes[nextId].golden_parent_x != -1 || d_nodes[nextId].golden_parent_y != -1) {
            printf("Error: Backward traceback meets forward traceback.\n");
            printf("meetId = %d\n", s_meetId); 
            for (int localIdx = 0; localIdx < total; localIdx++) {
              int local_x = localIdx % xDimTemp + LLX;
              int local_y = localIdx / xDimTemp + LLY;
              int idx = locToIdx_2D(local_x, local_y, xDim);
              printf("node id = %d, x = %d, y = %d ", idx, local_x, local_y);
              printNode2D(d_nodes[idx]);
            }
            printf("nextId = %d, x = %d, y = %d,  golden_parent_x = %d, golden_parent_y = %d, dst_flag = %d, src_flag = %d\n", 
              nextId, nx, ny, 
              d_nodes[nextId].golden_parent_x, d_nodes[nextId].golden_parent_y, 
              d_nodes[nextId].flags.dst_flag, d_nodes[nextId].flags.src_flag);
            atomicExch(&tracebackError, 1);
            break;      
          } */

          d_nodes[nextId].flags.src_flag = 1;
          d_nodes[nextId].golden_parent_x = xy.x;
          d_nodes[nextId].golden_parent_y = xy.y;
          backwardCurId = nextId;
          backwardIteration++;
        }
        d_nodes[backwardCurId].flags.dst_flag = 0;
        if (backwardIteration >= total) {
          printf("Warning: Backward traceback exceeded maximum iterations.\n");
        }
      }
      // printf("End the traceback\n");

    }
  }
  __syncthreads();
}






// Define the device function for the biwaveBellmanFord_2D_v4__device
__device__
void runBiBellmanFord2D_v5__device(
  int netId,
  NodeData2D* d_nodes,
  uint64_t* d_costMap, 
  const int* __restrict__ d_dX, 
  const int* __restrict__ d_dY,
  const int* __restrict__ d_xCoords,
  const int* __restrict__ d_yCoords,
  int LLX, int LLY, int URX, int URY,
  int xDim, 
  int maxIters,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST)
{
  // Each device function is handled by a single block
  int total = (URX - LLX + 1) * (URY - LLY + 1);
  int tid = threadIdx.x;
  int stride = blockDim.x; 
  int xDimTemp = URX - LLX + 1;
  
  // Define the shared memory for d_dx and d_dy
  __shared__ int s_dX[4];
  __shared__ int s_dY[4];
  __shared__ volatile int s_doneFlag;
  __shared__ volatile int s_minCost;
  //__shared__ volatile int s_meetId;
  __shared__ unsigned long long s_meet;
  __shared__ volatile int tracebackError;   // 0: no error; 1: error detected
  __shared__ int s_minForwardCost;
  __shared__ int s_minBackwardCost;
  
  // Load the d_dX and d_dY into shared memory
  if (tid < 4) {
    s_dX[tid] = d_dX[tid];
    s_dY[tid] = d_dY[tid];
    if (tid == 0) {
      s_doneFlag = 0;
      s_minCost = 0x7FFFFFFF;
      //s_meetId = 0x7FFFFFFF;
      s_meet = 0xFFFFFFFFFFFFFFFFULL;
      tracebackError = 0;
      s_minForwardCost = 0x7FFFFFFF;
      s_minBackwardCost = 0x7FFFFFFF;
    }
  }
  __syncthreads();

  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int x = localIdx % xDimTemp + LLX;
    int y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(x, y, xDim);
    if (d_nodes[idx].flags.dst_flag) {
      atomicExch((int*)&s_doneFlag, 1);
      break; // Exit early if any destination node is found.
    }
  }
  __syncthreads();

  if (!s_doneFlag) {
    return;
  } 
  
  if (tid == 0) {
    s_doneFlag = 0;
  }

  __syncthreads();

  // We'll do up to maxIters or until no changes / front-meet
  for (int iter = 0; iter < maxIters && (s_doneFlag == 0); iter++)
  {
    bool localFrontsMeet = false;

    ////////////////////////////////////////////////////////////////////////////
    // (1) Forward & backward relaxation phase
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      NodeData2D &nd = d_nodes[idx];

      // Forward relaxation:
      // Skip if src_flag is set.
      if (!nd.flags.src_flag) {
        uint32_t bestCost = nd.forward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          uint32_t neighborCost = d_nodes[nbrIdx].forward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          uint32_t newG = neighborCost +
            getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, nbrIdx, nx, ny);
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) { // Found an improvement
          nd.forward_g_cost = bestCost;
          nd.forward_direction = computeParentDirection2D(bestD);
          nd.flags.forward_update_flag = 1;
        }
      } // end forward

      // Backward relaxation:
      // newCost = min over neighbors of (neighbor.backward_cost + edgeWeight).
      // Skip if dst_flag is set.
      if (!nd.flags.dst_flag) {
        uint32_t bestCost = nd.backward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + s_dX[d];
          int ny = y + s_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          uint32_t neighborCost = d_nodes[nbrIdx].backward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          uint32_t newG = neighborCost +
            getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, nbrIdx, nx, ny);
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) {
          nd.backward_g_cost = bestCost;
          nd.backward_direction = computeParentDirection2D(bestD);
          nd.flags.backward_update_flag = 1;
        }
      } // end backward
    } // end for each node (relaxation)
    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // (2) Commit updated costs (double-buffering technique)
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = d_nodes[idx];
      if (nd.flags.forward_update_flag) {
        nd.flags.forward_update_flag = false;
        nd.forward_g_cost_prev = nd.forward_g_cost;
        atomicMin(&s_minForwardCost, nd.forward_g_cost);
      }
      if (nd.flags.backward_update_flag) {
        nd.flags.backward_update_flag = false;
        nd.backward_g_cost_prev = nd.backward_g_cost;
        atomicMin(&s_minBackwardCost, nd.backward_g_cost);
      }
      // Save previous visited flags for later comparisons.
      nd.flags.forward_visited_flag_prev = nd.flags.forward_visited_flag;
      nd.flags.backward_visited_flag_prev = nd.flags.backward_visited_flag;
    }
    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // (3) Mark nodes using parallel reduction to select the minimum cost nodes.
    //     For the forward side, we find the node with the minimum forward_g_cost
    //     (among nodes not already visited) and mark its forward_visited_flag.
    //     Similarly for the backward side.    
    // Alternatively, one could scan all nodes to check for any node that has both flags true.
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = d_nodes[idx];
      if (nd.forward_g_cost_prev <= s_minForwardCost) {
        nd.flags.forward_visited_flag = true;
      }

      if (nd.backward_g_cost_prev <= s_minBackwardCost) {
        nd.flags.backward_visited_flag = true;
      }

      // Check if the forward and backward fronts meet.
      if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
        localFrontsMeet = true;
      }
    }

    __syncthreads();

    if (localFrontsMeet) {
      atomicExch((int*)&s_doneFlag, 1);
    }
    
    if (tid == 0) {
      s_minForwardCost = 0x7FFFFFFF;
      s_minBackwardCost = 0x7FFFFFFF;
    }
    
    __syncthreads();

  } // end for (iter)

  __syncthreads();

  // Ensure all threads know the doneFlag
  bool converged = (s_doneFlag == 1);
  if (!converged) {
    if (tid == 0) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. doneFlag = false netId = %d\n", netId);
      for (int localIdx = 0; localIdx < total; localIdx++) {
        int local_x = localIdx % xDimTemp + LLX;
        int local_y = localIdx / xDimTemp + LLY;
        int idx = locToIdx_2D(local_x, local_y, xDim);
        printf("node id = %d, x = %d, y = %d ", idx, local_x, local_y);
        printNode2D(d_nodes[idx]);
      }
    }
    __syncthreads();
    return;
  } 
  __syncthreads();  


  /*
  // Identify the meetId corresponding to the minimum cost.
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    auto& nd = d_nodes[idx];
    if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
      atomicMin((int*)&s_meetId, idx);      
    }
  }
  __syncthreads();
  */


  // Iterate over your domain. Assume tid and stride are defined appropriately.
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    auto& nd = d_nodes[idx];
  
    // Only consider nodes visited from both directions.
    if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
      // Assume each node has an integer cost value.
      int cost = nd.forward_g_cost + nd.backward_g_cost;
      // Pack the cost and idx into a 64-bit value.
      // Lower cost (in high bits) gives a lower overall value.
      unsigned long long candidate = (((unsigned long long)cost) << 32) | ((unsigned int)idx);
      atomicMin(&s_meet, candidate);
    }
  }
  __syncthreads();


  // After the loop, the meetId is stored in the lower 32 bits of s_meet.
  int s_meetId = (int)(s_meet & 0xFFFFFFFF);


  // Check if s_meetId is valid.
  if (s_meetId == 0x7FFFFFFF) {
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      printf("Error! biwaveBellmanFord_2D_v3__device did not converge. meetId = 0x7FFFFFFF, netId = %d\n", netId);
    }
    if (threadIdx.x == 0 || threadIdx.x == 1) {
      atomicExch((int*)&tracebackError, 1);
    }
  }
  __syncthreads();
  
  // Only threads 0 and 1 perform the traceback if no error occurred.
  if (tracebackError == 0) {
    // ----- Forward Traceback (Thread 0) -----
    if (threadIdx.x == 0) {
      
      /*
      if (netId == 537) {
        printf("*****************************************************************\n");
        printf("meetId = %d, netId = %d\n", s_meetId, netId);
        for (int idx = 0; idx < total; idx++) {
          int local_x = idx % xDimTemp + LLX;
          int local_y = idx / xDimTemp + LLY;
          int id = locToIdx_2D(local_x, local_y, xDim);
          printf("node id = %d, x = %d, y = %d ", id, local_x, local_y);
          printNode2D(d_nodes[id]);
        }
        printf("\n");
      }*/
      
      
      
      // printf("Start the traceback\n");
      int tempIter = 0;      
      // Update the meetId accordingly to remove redundant path
      while (d_nodes[s_meetId].forward_direction == d_nodes[s_meetId].backward_direction && tempIter < total) {
        if (d_nodes[s_meetId].forward_direction == DIR_NONE) {
          printf("Warning: forward_direction == DIR_NONE netId = %d s_meetId = %d\n", netId, s_meetId);
          for (int idx = 0; idx < total; idx++) {
            int local_x = idx % xDimTemp + LLX;
            int local_y = idx / xDimTemp + LLY;
            int id = locToIdx_2D(local_x, local_y, xDim);
            printf("node id = %d, x = %d, y = %d ", id, local_x, local_y);
            printNode2D(d_nodes[id]);
          }
          break;
        }
        
        int2 xy = idxToLoc_2D(s_meetId, xDim);
        auto direction = d_nodes[s_meetId].forward_direction;
        int nx = xy.x + s_dX[direction];
        int ny = xy.y + s_dY[direction];
        s_meetId = locToIdx_2D(nx, ny, xDim);
        tempIter++;
      }

      if (tempIter >= total) {
        printf("Warning: reduce iteration exceeded maximum iterations. netId = %d\n", netId);
      }
            
      // printf("Start the forward traceback\n");

      int forwardCurId = s_meetId;
      int forwardIteration = 0;
      while (!d_nodes[forwardCurId].flags.src_flag && forwardIteration < total) {
        uint8_t fwdDir = d_nodes[forwardCurId].forward_direction;
        int2 xy = idxToLoc_2D(forwardCurId, xDim);
        int nx = xy.x + s_dX[fwdDir];
        int ny = xy.y + s_dY[fwdDir];
        if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
          break;
        }
        d_nodes[forwardCurId].golden_parent_x = nx;
        d_nodes[forwardCurId].golden_parent_y = ny;
        d_nodes[forwardCurId].flags.src_flag = 1;
        forwardCurId = locToIdx_2D(nx, ny, xDim);
        forwardIteration++;
      }
      if (forwardIteration >= total) {
        printf("Warning: Forward traceback exceeded maximum iterations. netId = %d\n", netId);
      }
    // }
    

     //  printf("Start the backward traceback\n");
    // ----- Backward Traceback (Thread 1) -----
    // if (threadIdx.x == 1) {
      int backwardCurId = s_meetId;
      int backwardIteration = 0;
      if (d_nodes[backwardCurId].flags.dst_flag == 1) {
        d_nodes[backwardCurId].flags.dst_flag = 0; // Reset dst flag.
        d_nodes[backwardCurId].flags.src_flag = 1;
      } else {
        while (!d_nodes[backwardCurId].flags.dst_flag && backwardIteration < total) {
          int2 xy = idxToLoc_2D(backwardCurId, xDim);
          uint8_t backwardDir = d_nodes[backwardCurId].backward_direction;
          int nx = xy.x + s_dX[backwardDir];
          int ny = xy.y + s_dY[backwardDir];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            break;
          }
          int nextId = locToIdx_2D(nx, ny, xDim);
          
          /*
          if (d_nodes[nextId].golden_parent_x != -1 || d_nodes[nextId].golden_parent_y != -1) {
            printf("Error: Backward traceback meets forward traceback.\n");
            printf("meetId = %d\n", s_meetId); 
            for (int localIdx = 0; localIdx < total; localIdx++) {
              int local_x = localIdx % xDimTemp + LLX;
              int local_y = localIdx / xDimTemp + LLY;
              int idx = locToIdx_2D(local_x, local_y, xDim);
              printf("node id = %d, x = %d, y = %d ", idx, local_x, local_y);
              printNode2D(d_nodes[idx]);
            }
            printf("nextId = %d, x = %d, y = %d,  golden_parent_x = %d, golden_parent_y = %d, dst_flag = %d, src_flag = %d\n", 
              nextId, nx, ny, 
              d_nodes[nextId].golden_parent_x, d_nodes[nextId].golden_parent_y, 
              d_nodes[nextId].flags.dst_flag, d_nodes[nextId].flags.src_flag);
            atomicExch(&tracebackError, 1);
            break;      
          } */
           
          d_nodes[nextId].flags.src_flag = 1;
          d_nodes[nextId].golden_parent_x = xy.x;
          d_nodes[nextId].golden_parent_y = xy.y;
          backwardCurId = nextId;
          backwardIteration++;
        }
        d_nodes[backwardCurId].flags.dst_flag = 0;
        if (backwardIteration >= total) {
          printf("Warning: Backward traceback exceeded maximum iterations. netId = %d\n", netId);
        }
      }
      // printf("End the traceback\n");

    }
  }
  __syncthreads();
}




__device__
void initNodeData2D_v3__device(
  NodeData2D* d_nodes,
  int* d_pins, int pinIterStart, int pinIter,  // Pin related variables
  int LLX, int LLY, int URX, int URY, // Bounding box
  int xDim)
{ 
  int total = (URX - LLX + 1) * (URY - LLY + 1);
  int xDimTemp = URX - LLX + 1;
  int tid = threadIdx.x;
  int stride = blockDim.x;
  
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    // int2 xy = idxToLoc_2D(idx, xDim);
    // int2 src = idxToLoc_2D(d_pins[pinIterStart + pinIter - 1], xDim);
    // int2 dst = idxToLoc_2D(d_pins[pinIterStart + pinIter], xDim);
  
    // The experimental results show that the heuristic cost is not needed
    d_nodes[idx].forward_h_cost = 0;
    d_nodes[idx].backward_h_cost = 0;

    if (d_nodes[idx].flags.src_flag) {
      d_nodes[idx].forward_g_cost = 0;
      d_nodes[idx].forward_g_cost_prev = 0;
      d_nodes[idx].flags.forward_visited_flag = true;
      d_nodes[idx].flags.forward_visited_flag_prev = true;
    } else {
      d_nodes[idx].forward_g_cost = INF32;
      d_nodes[idx].forward_g_cost_prev = INF32;
      d_nodes[idx].flags.forward_visited_flag = false;
      d_nodes[idx].flags.forward_visited_flag_prev = false;
    }

    if (d_nodes[idx].flags.dst_flag) {
      d_nodes[idx].backward_g_cost = 0;
      d_nodes[idx].backward_g_cost_prev = 0;
      d_nodes[idx].flags.backward_visited_flag = true;
      d_nodes[idx].flags.backward_visited_flag_prev = true;
    } else {
      d_nodes[idx].backward_g_cost = INF32;
      d_nodes[idx].backward_g_cost_prev = INF32;
      d_nodes[idx].flags.backward_visited_flag = false;
      d_nodes[idx].flags.backward_visited_flag_prev = false;
    }

    d_nodes[idx].forward_direction = DIR_NONE;
    d_nodes[idx].backward_direction = DIR_NONE;
    d_nodes[idx].forward_direction_prev = DIR_NONE;
    d_nodes[idx].backward_direction_prev = DIR_NONE;
    d_nodes[idx].flags.forward_update_flag = false;
    d_nodes[idx].flags.backward_update_flag = false;
    d_nodes[idx].flags.forward_visited_flag = false;
    d_nodes[idx].flags.backward_visited_flag = false;
    d_nodes[idx].flags.forward_visited_flag_prev = false;
    d_nodes[idx].flags.backward_visited_flag_prev = false;
  } 
}


__device__
void checkConnectivity__device(
  NodeData2D* d_nodes,
  int LLX, int LLY, int URX, int URY, // Bounding box
  int xDim)
{
  int total = (URX - LLX + 1) * (URY - LLY + 1);
  int xDimTemp = URX - LLX + 1;
  int tid = threadIdx.x;
  int stride = blockDim.x;
  
  for (int localIdx = tid; localIdx < total; localIdx += stride) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    if (d_nodes[idx].flags.dst_flag) {
      printf("Error: dst_flag is set. idx = %d, x = %d, y = %d\n", idx, local_x, local_y);
    }

    if (d_nodes[idx].golden_parent_x == -1 && d_nodes[idx].golden_parent_y != d_nodes[idx].golden_parent_x) {
      printf("Error: golden_parent_x = %d, golden_parent_y = %d, idx = %d, x = %d, y = %d\n", 
        d_nodes[idx].golden_parent_x, d_nodes[idx].golden_parent_y, idx, local_x, local_y);
    }
  }
}




__device__ 
void biwaveBellmanFord2D_v3__device(
  int netId,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  const int* d_xCoords,
  const int* d_yCoords,
  int maxIters,
  int xDim,
  int yDim,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST, 
  int HISTCOST)
{
  // for this net
  int pinIdxStart = d_netPtr[netId];
  int pinIdxEnd = d_netPtr[netId + 1];
  int numPins = pinIdxEnd - pinIdxStart;
  Rect2D_CUDA netBBox = d_netBBoxVec[netId];
  int LLX = netBBox.xMin;
  int LLY = netBBox.yMin;
  int URX = netBBox.xMax;
  int URY = netBBox.yMax;

  for (int pinIter = 1; pinIter < numPins; pinIter++) {
    // Initialize the node data
    initNodeData2D_v3__device(
      d_nodes,
      d_pins, pinIdxStart, pinIter, 
      LLX, LLY, URX, URY, 
      xDim);

    __syncthreads(); // Synchronize all threads in the block

    // Run the Bellman Ford algorithm
    runBiBellmanFord2D_v5__device(
      netId, 
      d_nodes, d_costMap, d_dX, d_dY, d_xCoords, d_yCoords,
      LLX, LLY, URX, URY, xDim, maxIters,
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST);  

    __syncthreads(); // Synchronize all threads in the block
  }

  // Check the connectivity
  checkConnectivity__device(
    d_nodes,
    LLX, LLY, URX, URY, xDim);
}



__global__ 
void biwaveBellmanFord2D_v3__kernel(
  int netStartId,
  int netEndId,
  int* d_netBatchIdx,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  const int* d_xCoords,
  const int* d_yCoords,
  int maxIters,
  int xDim,
  int yDim,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST, 
  int HISTCOST)
{
  // Each net is handled by a single block
  for (int netId = netStartId + blockIdx.x; netId < netEndId; netId += gridDim.x) {
    biwaveBellmanFord2D_v3__device(
      netId,
      d_netPtr,
      d_netBBoxVec,
      d_pins,
      d_costMap,
      d_nodes + d_netBatchIdx[netId] * xDim * yDim,
      d_dX,
      d_dY,
      d_xCoords,
      d_yCoords,
      maxIters,
      xDim,
      yDim,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST, 
      HISTCOST);
  }
}



__global__ 
void initBatchNodeData2D_v3__kernel(
  NodeData2D* d_nodes,
  int numNodes)
{
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numNodes; idx += gridDim.x * blockDim.x) {
    initNodeData2D(d_nodes[idx]);
  }
}


__global__
void initParent2D__kernel(
  Point2D_CUDA* d_parents,
  int numParents)
{
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numParents; idx += gridDim.x * blockDim.x) {
    d_parents[idx].x = -1;
    d_parents[idx].y = -1;
  }
}


__global__
void copyParents2D__kernel(
  NodeData2D* d_nodes,
  Point2D_CUDA* d_parents,
  int numNodes)
{
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numNodes; idx += gridDim.x * blockDim.x) {
    d_parents[idx].x = d_nodes[idx].golden_parent_x;
    d_parents[idx].y = d_nodes[idx].golden_parent_y;
  }
}


__global__
void initBatchPin2D_v3__kernel(
  NodeData2D* d_nodes,
  int* d_pins,
  int* d_netPtr,
  int* d_netBatchIdx,
  int netIdStart,
  int netIdEnd,
  int numGrids)
{  
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int numNets = netIdEnd - netIdStart;
  for (tid; tid < numNets; tid += stride) {
    int netId = netIdStart + tid;
    int batchId = d_netBatchIdx[netId];
    int baseNodeId = batchId * numGrids;
    int pinIdxStart = d_netPtr[netId];
    int pinIdxEnd = d_netPtr[netId + 1];
    if (batchId < 0 || batchId >= 200) {
      printf("Error: Invalid batchId = %d\n", batchId);
    }

    int pinId = d_pins[pinIdxStart] + baseNodeId;
    if (pinId > numGrids * 200) {
      printf("Error: Invalid pinId = %d\n", pinId);
    }

    d_nodes[pinId].flags.src_flag = true;

    for (int pinIter = pinIdxStart + 1; pinIter < pinIdxEnd; pinIter++) {
      pinId = d_pins[pinIter] + baseNodeId;
      if (pinId > numGrids * 200) {
        printf("Error: Invalid pinId = %d\n", pinId);
      }
      d_nodes[pinId].flags.dst_flag = true;
    }
  }
}



float ChunkPathSyncUp(
  std::vector<std::unique_ptr<FlexGRWorker>>& uworkers,
  std::vector<grNet*>& nets,
  std::vector<Rect2D_CUDA>& netBBoxVec,
  std::vector<Point2D_CUDA>& h_parents,
  int netIdStart,
  int netIdEnd,
  int xDim)
{ 
  auto syncupTimeStart = std::chrono::high_resolution_clock::now();
  for (int netId = netIdStart; netId < netIdEnd; netId++) {
    auto& net = nets[netId];
    auto& uworker = uworkers[net->getWorkerId()];
    auto& gridGraph = uworker->getGridGraph();
    auto workerLL = uworkers[net->getWorkerId()]->getRouteGCellIdxLL();
    int workerLX = workerLL.x();
    int workerLY = workerLL.y();  
    auto& netBBox = netBBoxVec[netId];
    int LLX = netBBox.xMin;
    int LLY = netBBox.yMin;
    int URX = netBBox.xMax;
    int URY = netBBox.yMax;
    int xDimTemp = URX - LLX + 1;
    int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
    for (int localIdx = 0; localIdx < numNodes; localIdx++) {
      int localX = localIdx % xDimTemp;
      int localY = localIdx / xDimTemp;

      int x = localX + LLX;
      int y = localY + LLY;      
      int idx = locToIdx_2D(x, y, xDim);

      x -= workerLX;
      y -= workerLY;

      int parentX = h_parents[idx].x - workerLX;
      int parentY = h_parents[idx].y - workerLY;

      //int parentX = nodes[idx].golden_parent_x - workerLX;
      //int parentY = nodes[idx].golden_parent_y - workerLY;

      gridGraph.setGoldenParent2D(x, y, parentX, parentY);      
      /*
      if (nodes[idx].golden_parent_x != -1 || nodes[idx].golden_parent_y != -1) {
        std::cout << "Net " << netId << " x = " << x << " y = " << y << " "
                  << "Parent " << parentX << "  " << parentY << std::endl;
      }
      */
    }    
  }

  auto syncupTimeEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> syncupTime = syncupTimeEnd - syncupTimeStart;
  return syncupTime.count();
}


void FlexGR::allocateCUDAMem(
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  std::vector<Point2D_CUDA>& h_parents,
  std::vector<int>& pinIdxVec,
  std::vector<int>& netPtr,
  std::vector<Rect2D_CUDA>& netBBoxVec,
  std::vector<int>& netBatchIdxVec,
  int numGrids,
  int numNodes)
{  
  // We have defined the following variables
  // h_costMap_size_
  // h_xCoords_size_
  // h_yCoords_size_
  // h_parents_size_
  // h_pinIdxVec_size_
  // h_netPtr_size_
  // h_netBBoxVec_size_
  // h_netBatchIdxVec_size_ 
  // h_nodes_size_

  if (d_dX_ == nullptr) {
    std::vector<int> h_dX = {0, 1, 0, -1};
    std::vector<int> h_dY = {1, 0, -1, 0};
    hipMalloc(&d_dX_, 4 * sizeof(int));
    hipMalloc(&d_dY_, 4 * sizeof(int));
    hipMemcpy(d_dX_, h_dX.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dY_, h_dY.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  }

  if (h_xCoords.size() > h_costMap_size_) {
    h_xCoords_size_ = h_xCoords.size();
    hipFree(d_xCoords_);
    hipMalloc(&d_xCoords_, h_xCoords.size() * sizeof(int));
    hipMemcpy(d_xCoords_, h_xCoords.data(), h_xCoords.size() * sizeof(int), hipMemcpyHostToDevice); 
  }

  if (h_yCoords.size() > h_costMap_size_) {
    h_yCoords_size_ = h_yCoords.size();
    hipFree(d_yCoords_);
    hipMalloc(&d_yCoords_, h_yCoords.size() * sizeof(int));
    hipMemcpy(d_yCoords_, h_yCoords.data(), h_yCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  }

  if (h_costMap.size() > h_costMap_size_) {
    h_costMap_size_ = h_costMap.size();
    hipFree(d_costMap_);
    hipMalloc(&d_costMap_, h_costMap.size() * sizeof(uint64_t));
  }
  hipMemcpy(d_costMap_, h_costMap.data(), h_costMap.size() * sizeof(uint64_t), hipMemcpyHostToDevice);

  if (h_parents.size() > h_parents_size_) {
    h_parents_size_ = h_parents.size();
    hipFree(d_parents_);
    hipMalloc(&d_parents_, h_parents.size() * sizeof(Point2D_CUDA));
  }
  //hipMemcpy(d_parents_, h_parents.data(), h_parents.size() * sizeof(Point2D_CUDA), hipMemcpyHostToDevice);

  if (pinIdxVec.size() > h_pinIdxVec_size_) {
    h_pinIdxVec_size_ = pinIdxVec.size();
    hipFree(d_pinIdxVec_);
    hipMalloc(&d_pinIdxVec_, pinIdxVec.size() * sizeof(int));
  }
  hipMemcpy(d_pinIdxVec_, pinIdxVec.data(), pinIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);

  if (netPtr.size() > h_netPtr_size_) {
    h_netPtr_size_ = netPtr.size();
    hipFree(d_netPtr_);
    hipMalloc(&d_netPtr_, netPtr.size() * sizeof(int));
  }
  hipMemcpy(d_netPtr_, netPtr.data(), netPtr.size() * sizeof(int), hipMemcpyHostToDevice);

  if (netBBoxVec.size() > h_netBBoxVec_size_) {
    h_netBBoxVec_size_ = netBBoxVec.size();
    hipFree(d_netBBox_);
    hipMalloc(&d_netBBox_, netBBoxVec.size() * sizeof(Rect2D_CUDA));
  }
  hipMemcpy(d_netBBox_, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);

  if (netBatchIdxVec.size() > h_netBatchIdxVec_size_) {
    h_netBatchIdxVec_size_ = netBatchIdxVec.size();
    hipFree(d_netBatchIdx_);
    hipMalloc(&d_netBatchIdx_, netBatchIdxVec.size() * sizeof(int));
  }
  hipMemcpy(d_netBatchIdx_, netBatchIdxVec.data(), netBatchIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);

  if (numNodes > h_nodes_size_) {
    h_nodes_size_ = numNodes;
    hipFree(d_nodes_);
    hipMalloc(&d_nodes_, numNodes * sizeof(NodeData2D));
  }
  cudaCheckError();
}



void FlexGR::freeCUDAMem()
{
  hipFree(d_dX_);
  hipFree(d_dY_);
  hipFree(d_costMap_);
  hipFree(d_xCoords_);
  hipFree(d_yCoords_);
  hipFree(d_nodes_);
  hipFree(d_parents_);
  hipFree(d_pinIdxVec_);
  hipFree(d_netPtr_);
  hipFree(d_netBBox_);
  hipFree(d_netBatchIdx_);

  d_dX_ = nullptr;
  d_dY_ = nullptr;
  d_costMap_ = nullptr;
  d_xCoords_ = nullptr;
  d_yCoords_ = nullptr;
  d_nodes_ = nullptr;
  d_parents_ = nullptr;
  d_pinIdxVec_ = nullptr;
  d_netPtr_ = nullptr;
  d_netBBox_ = nullptr;
  d_netBatchIdx_ = nullptr;

  h_costMap_size_ = 0;
  h_xCoords_size_ = 0;
  h_yCoords_size_ = 0;
  h_parents_size_ = 0;
  h_pinIdxVec_size_ = 0;
  h_netPtr_size_ = 0;
  h_netBBoxVec_size_ = 0;
  h_netBatchIdxVec_size_ = 0;
  h_nodes_size_ = 0;

  cudaCheckError();
}


float FlexGR::GPUAccelerated2DMazeRoute_update_v3(
  std::vector<std::unique_ptr<FlexGRWorker> >& uworkers,
  std::vector<std::vector<grNet*> >& netBatches,
  std::vector<int>& validBatches,
  std::vector<Point2D_CUDA>& h_parents,
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  RouterConfiguration* router_cfg,
  float relaxThreshold,
  float congThreshold,
  int xDim, int yDim)
{
  // Start overall timing.
  auto totalStart = std::chrono::high_resolution_clock::now();
  int numGrids = xDim * yDim;
  int numBatches = validBatches.size();  
  
  if (VERBOSE > 0) {
    std::cout << "[INFO] Number of batches: " << numBatches << std::endl;
  }
  
  if (numBatches == 0) {
    return 0.0;
  }

  // Set the GPU device to 1.
  hipSetDevice(1);

  std::vector<Point2D_CUDA> netVec;
  std::vector<int> netPtr;
  std::vector<int> netBatchIdxVec; 
  std::vector<Rect2D_CUDA> netBBoxVec;
  std::vector<int> pinIdxVec;
  int maxHPWL = 0; // We will run the algorithm for maxHPWL * relaxThreshold iteratively
  
  // We need to divide the batches into multiple chunks due to the memory limitation
  int maxChunkSize = 200;  // Basically we allows 200 batches to be processed in one chunk
  // For testing 
  // int maxChunkSize = 1;
  
  std::vector<int> chunkNetPtr; // store the first netIdx of each chunk
  
  if (VERBOSE > 0) {
    std::cout << "[INFO] Max chunk size: " << maxChunkSize << std::endl;
  }

  netPtr.push_back(0);
  chunkNetPtr.push_back(0);

  int maxBatchSize = 0;
  int minBatchSize = std::numeric_limits<int>::max();

  int batchChunkIdx = 0;
  for (int batchIdx = 0; batchIdx < numBatches; batchIdx++) {
    auto& batch = netBatches[validBatches[batchIdx]];
    for (auto& net : batch) {
      for (auto& idx : net->getPinGCellAbsIdxs()) {
        netVec.push_back(Point2D_CUDA(idx.x(), idx.y()));
        pinIdxVec.push_back(locToIdx_2D(idx.x(), idx.y(), xDim));
      }
      netBatchIdxVec.push_back(batchChunkIdx);
      netPtr.push_back(netVec.size());
      auto netBBox = net->getRouteAbsBBox();
      netBBoxVec.push_back(
        Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
      // maxHPWL = std::max(maxHPWL, net->getHPWL());
      maxHPWL = std::max(maxHPWL, static_cast<int>((netBBox.xMax() - netBBox.xMin()) * (netBBox.yMax() - netBBox.yMin())));
    }
   
    batchChunkIdx++;

    if (batchChunkIdx % maxChunkSize == 0) {
      chunkNetPtr.push_back(netPtr.size() - 1);
      batchChunkIdx = 0;
    }
    
    maxBatchSize = std::max(maxBatchSize, static_cast<int>(batch.size()));
    minBatchSize = std::min(minBatchSize, static_cast<int>(batch.size()));
  }
 
  if (batchChunkIdx != 0) {
    chunkNetPtr.push_back(netPtr.size() - 1);
  }

  int numNets = static_cast<int>(netBBoxVec.size());
  int numChunks = static_cast<int>(chunkNetPtr.size()) - 1;
  
  if (numChunks > numBatches) {
    std::cout << "[ERROR] Number of chunks is larger than the number of batches." << std::endl;
    exit(1);
    return 0.0;
  }  
  
  int chunkSize = std::min(maxChunkSize, numBatches);
  int maxIters = static_cast<int>(maxHPWL * relaxThreshold);
  int numNodes = numGrids * chunkSize;

  if (VERBOSE > 0) {
    std::cout << "[INFO] Number of nets: " << numNets << std::endl;
    std::cout << "[INFO] Number of chunks: " << numChunks << std::endl;
    std::cout << "[INFO] Chunk size: " << chunkSize << std::endl; 
    std::cout << "[INFO] Max batch size: " << maxBatchSize << std::endl;
    std::cout << "[INFO] Min batch size: " << minBatchSize << std::endl;
    std::cout << "[INFO] Max HPWL: " << maxHPWL << std::endl;
    std::cout << "[INFO] Max iterations: " << maxIters << std::endl;
    std::cout << "[INFO] Number of nodes: " << numNodes << std::endl;
    std::cout << "[INFO] Number of grids: " << numGrids << std::endl;
  }

  for (int i = 0; i < netBatchIdxVec.size(); i++) {
    if (netBatchIdxVec[i] < 0 || netBatchIdxVec[i] >= chunkSize) {
      std::cout << "[ERROR] Net " << i << " is in batch " << netBatchIdxVec[i] << std::endl;
    }
  }


  allocateCUDAMem(
    h_costMap,
    h_xCoords,
    h_yCoords,
    h_parents,
    pinIdxVec,
    netPtr,
    netBBoxVec,
    netBatchIdxVec,
    numGrids, 
    numNodes);


  if (VERBOSE > 0) {
    std::cout << "[INFO] Device memory allocation is done." << std::endl;
  }

  // According to the original code
  unsigned BLOCKCOST = router_cfg->BLOCKCOST * 100;
  unsigned OVERFLOWCOST = 128;
  unsigned HISTCOST = 4;
 
  for (int chunkId = 0; chunkId < numChunks; chunkId++) {
    int netStartId = chunkNetPtr[chunkId];
    int netEndId = chunkNetPtr[chunkId + 1];
    
    // Perform Global Initialization
    int numThreads = 1024;
    int numBatchBlocks = (numNodes + numThreads - 1) / numThreads;
    
    int numParentsBlocks = (h_parents_size_ + numThreads - 1) / numThreads;
    initParent2D__kernel<<<numParentsBlocks, numThreads>>>(d_parents_, h_parents.size());

    initBatchNodeData2D_v3__kernel<<<numBatchBlocks, numThreads>>>(
      d_nodes_, 
      numNodes);
    hipDeviceSynchronize();

    cudaCheckError();

  
    int numNets = netEndId - netStartId;
    if (VERBOSE > 0) {
      std::cout << "[INFO] Chunk " << chunkId << " has " << numNets << " nets." << std::endl;
    }
  
    int numNetBlocks = (numNets + numThreads - 1) / numThreads;
    initBatchPin2D_v3__kernel<<<numNetBlocks, numThreads>>>(
      d_nodes_,
      d_pinIdxVec_, 
      d_netPtr_,
      d_netBatchIdx_,
      netStartId,
      netEndId,
      numGrids);
    hipDeviceSynchronize();
  

    cudaCheckError();
    // std::cout << "[INFO] Initialization is done." << std::endl;

    //int numThreads = 1024;
    auto netRouteStart = std::chrono::high_resolution_clock::now();
    int numBlocks = numNets;
    biwaveBellmanFord2D_v3__kernel<<<numBlocks, numThreads>>>(
      netStartId,
      netEndId,
      d_netBatchIdx_,
      d_netPtr_,
      d_netBBox_,
      d_pinIdxVec_,
      d_costMap_,
      d_nodes_,
      d_dX_,
      d_dY_,
      d_xCoords_,
      d_yCoords_,
      maxIters,
      xDim,
      yDim,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST,
      HISTCOST);
    
    std::cout << "Congestion threshold: " << congThreshold << std::endl;

    cudaCheckError();
    
    hipDeviceSynchronize();
    auto netRouteEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> netRouteTime = netRouteEnd - netRouteStart;
   
    // copy the back results to the d_parents
    copyParents2D__kernel<<<numBatchBlocks, numThreads>>>(
      d_nodes_, 
      d_parents_, 
      numNodes);

    hipDeviceSynchronize();
    cudaCheckError();


    hipMemcpy(h_parents.data(), d_parents_, h_parents.size() * sizeof(Point2D_CUDA), hipMemcpyDeviceToHost);

    // Check the parents
    cudaCheckError();
  }  
 
  for (auto& parent : h_parents) {
    if (parent.x < 0 || parent.y < 0) {
      if (parent.x != -1 || parent.y != -1) {
        std::cout << "[ERROR] Invalid parent: " << parent.x << " " << parent.y << std::endl;
      } 
    }
  }  

  cudaCheckError();
  
  if (VERBOSE > 0) {
    std::cout << "[INFO] Kernel execution is done." << std::endl;
  }

  auto totalEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> totalTime = totalEnd - totalStart;
  return totalTime.count();
}



// In the V3 version, we will use node-level parallelism directly
// Basically, we do not do the batch-level synchronization anymore
// Just let the tool run for maxHPWL * value iteratively
// And see how it works
float FlexGR::GPUAccelerated2DMazeRoute_update_v3_old(
  std::vector<std::unique_ptr<FlexGRWorker> >& uworkers,
  std::vector<std::vector<grNet*> >& netBatches,
  std::vector<int>& validBatches,
  std::vector<Point2D_CUDA>& h_parents,
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  RouterConfiguration* router_cfg,
  float relaxThreshold,
  float congThreshold,
  int xDim, int yDim)
{
  // Start overall timing.
  auto totalStart = std::chrono::high_resolution_clock::now();
  int numGrids = xDim * yDim;
  int numBatches = validBatches.size();  
  
  if (VERBOSE > 0) {
    std::cout << "[INFO] Number of batches: " << numBatches << std::endl;
  }
  
  if (numBatches == 0) {
    return 0.0;
  }

  // Set the GPU device to 1.
  hipSetDevice(1);

  std::vector<Point2D_CUDA> netVec;
  std::vector<int> netPtr;
  std::vector<int> netBatchIdxVec; 
  std::vector<Rect2D_CUDA> netBBoxVec;
  std::vector<int> pinIdxVec;
  int maxHPWL = 0; // We will run the algorithm for maxHPWL * relaxThreshold iteratively
  
  // We need to divide the batches into multiple chunks due to the memory limitation
  int maxChunkSize = 200;  // Basically we allows 200 batches to be processed in one chunk
  // For testing 
  // int maxChunkSize = 1;
  
  std::vector<int> chunkNetPtr; // store the first netIdx of each chunk
  
  if (VERBOSE > 0) {
    std::cout << "[INFO] Max chunk size: " << maxChunkSize << std::endl;
  }

  netPtr.push_back(0);
  chunkNetPtr.push_back(0);

  int maxBatchSize = 0;
  int minBatchSize = std::numeric_limits<int>::max();

  int batchChunkIdx = 0;
  for (int batchIdx = 0; batchIdx < numBatches; batchIdx++) {
    auto& batch = netBatches[validBatches[batchIdx]];
    for (auto& net : batch) {
      for (auto& idx : net->getPinGCellAbsIdxs()) {
        netVec.push_back(Point2D_CUDA(idx.x(), idx.y()));
        pinIdxVec.push_back(locToIdx_2D(idx.x(), idx.y(), xDim));
      }
      netBatchIdxVec.push_back(batchChunkIdx);
      netPtr.push_back(netVec.size());
      auto netBBox = net->getRouteAbsBBox();
      netBBoxVec.push_back(
        Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
      // maxHPWL = std::max(maxHPWL, net->getHPWL());
      maxHPWL = std::max(maxHPWL, static_cast<int>((netBBox.xMax() - netBBox.xMin()) * (netBBox.yMax() - netBBox.yMin())));
    }
   
    batchChunkIdx++;

    if (batchChunkIdx % maxChunkSize == 0) {
      chunkNetPtr.push_back(netPtr.size() - 1);
      batchChunkIdx = 0;
    }
    
    maxBatchSize = std::max(maxBatchSize, static_cast<int>(batch.size()));
    minBatchSize = std::min(minBatchSize, static_cast<int>(batch.size()));
  }
 
  if (batchChunkIdx != 0) {
    chunkNetPtr.push_back(netPtr.size() - 1);
  }

  int numNets = static_cast<int>(netBBoxVec.size());
  int numChunks = static_cast<int>(chunkNetPtr.size()) - 1;
  
  if (numChunks > numBatches) {
    std::cout << "[ERROR] Number of chunks is larger than the number of batches." << std::endl;
    exit(1);
    return 0.0;
  }
  
  
  int chunkSize = std::min(maxChunkSize, numBatches);
  int maxIters = static_cast<int>(maxHPWL * relaxThreshold);
  int numNodes = numGrids * chunkSize;


  // if (VERBOSE > 0) {
  if (1) {
    std::cout << "[INFO] Number of nets: " << numNets << std::endl;
    std::cout << "[INFO] Number of chunks: " << numChunks << std::endl;
    std::cout << "[INFO] Chunk size: " << chunkSize << std::endl; 
    std::cout << "[INFO] Max batch size: " << maxBatchSize << std::endl;
    std::cout << "[INFO] Min batch size: " << minBatchSize << std::endl;
    std::cout << "[INFO] Max HPWL: " << maxHPWL << std::endl;
    std::cout << "[INFO] Max iterations: " << maxIters << std::endl;
    std::cout << "[INFO] Number of nodes: " << numNodes << std::endl;
    std::cout << "[INFO] Number of grids: " << numGrids << std::endl;
  }

  for (int i = 0; i < netBatchIdxVec.size(); i++) {
    if (netBatchIdxVec[i] < 0 || netBatchIdxVec[i] >= chunkSize) {
      std::cout << "[ERROR] Net " << i << " is in batch " << netBatchIdxVec[i] << std::endl;
    }
  }

  // We need to define the needed utility variables
  std::vector<int> h_dX = {0, 1, 0, -1};
  std::vector<int> h_dY = {1, 0, -1, 0};
 
  /*
  int* d_dX = nullptr;
  int* d_dY = nullptr;

  // For the design specific variables (numGrids)
  uint64_t* d_costMap = nullptr;
  int* d_xCoords = nullptr;
  int* d_yCoords = nullptr;  
  // For the chunk specific variables
  NodeData2D* d_nodes = nullptr; // (numGrids * chunkSize);
  
  // Point2D_CUDA* d_parents = nullptr;
  int* d_pinIdxVec = nullptr;
  int* d_netPtr = nullptr;
  Rect2D_CUDA* d_netBBox = nullptr;
  int* d_netBatchIdx = nullptr;
  Point2D_CUDA* d_parents = nullptr;
  */

 


  
  cudaCheckError();

  /*
  allocateCUDAMem(
    d_dX_,
    d_dY_,
    d_costMap_,
    d_xCoords_,
    d_yCoords_,
    d_nodes_,
    d_parents_,
    d_pinIdxVec_,
    d_netPtr_,
    d_netBBox_,
    d_netBatchIdx_,
    h_costMap,
    h_xCoords,
    h_yCoords,
    h_parents,
    pinIdxVec,
    netPtr,
    netBBoxVec,
    netBatchIdxVec,
    numGrids, 
    numNodes);
  */

  int* d_dX = d_dX_;
  int* d_dY = d_dY_;
  uint64_t* d_costMap = d_costMap_;
  int* d_xCoords = d_xCoords_;
  int* d_yCoords = d_yCoords_;
  Point2D_CUDA* d_parents = d_parents_;
  int* d_pinIdxVec = d_pinIdxVec_;
  int* d_netPtr = d_netPtr_;
  Rect2D_CUDA* d_netBBox = d_netBBox_;
  int* d_netBatchIdx = d_netBatchIdx_;
  NodeData2D* d_nodes = d_nodes_;
   
  cudaCheckError();

  std::cout << "Finish Memory Allocation" << std::endl;




  /*  
  allocateCUDAMem(
    d_dX_,
    d_dY_,
    d_costMap_,
    d_xCoords_,
    d_yCoords_,
    d_nodes_,
    d_parents_,
    d_pinIdxVec_,
    d_netPtr_,
    d_netBBox_,
    d_netBatchIdx_,
    h_costMap,
    h_xCoords,
    h_yCoords,
    h_parents,
    pinIdxVec,
    netPtr,
    netBBoxVec,
    netBatchIdxVec,
    numNodes);

  int* d_dX = d_dX_;
  int* d_dY = d_dY_;

  uint64_t* d_costMap = d_costMap_;
  int* d_xCoords = d_xCoords_;
  int* d_yCoords = d_yCoords_;
  Point2D_CUDA* d_parents = d_parents_;
  int* d_pinIdxVec = d_pinIdxVec_;
  int* d_netPtr = d_netPtr_;
  Rect2D_CUDA* d_netBBox = d_netBBox_;
  int* d_netBatchIdx = d_netBatchIdx_;
  NodeData2D* d_nodes = d_nodes_;
 
    
  if (d_dX == nullptr) {
    std::vector<int> h_dX = {0, 1, 0, -1};
    std::vector<int> h_dY = {1, 0, -1, 0};
    int* d_dX = nullptr;
    int* d_dY = nullptr;
    hipMalloc(&d_dX, 4 * sizeof(int));
    hipMalloc(&d_dY, 4 * sizeof(int));
    hipMemcpy(d_dX, h_dX.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dY, h_dY.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();
  }

  if (h_xCoords.size() > h_xCoords_size_) {
    h_xCoords_size_ = h_xCoords.size();
    hipFree(d_xCoords);
    hipMalloc(&d_xCoords, h_xCoords.size() * sizeof(int));
    hipMemcpy(d_xCoords, h_xCoords.data(), h_xCoords.size() * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();
  }

  if (h_yCoords.size() > h_yCoords_size_) {
    h_yCoords_size_ = h_yCoords.size();
    hipFree(d_yCoords);
    hipMalloc(&d_yCoords, h_yCoords.size() * sizeof(int));
    hipMemcpy(d_yCoords, h_yCoords.data(), h_yCoords.size() * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();
  }

  if (h_costMap.size() > h_costMap_size_) {
    h_costMap_size_ = h_costMap.size();
    hipFree(d_costMap);
    hipMalloc(&d_costMap, h_costMap.size() * sizeof(uint64_t));
  }
  hipMemcpy(d_costMap, h_costMap.data(), h_costMap.size() * sizeof(uint64_t), hipMemcpyHostToDevice);

  if (h_parents.size() > h_parents_size_) {
    h_parents_size_ = h_parents.size();
    hipFree(d_parents);
    hipMalloc(&d_parents, h_parents.size() * sizeof(Point2D_CUDA));
  }
  hipMemcpy(d_parents, h_parents.data(), h_parents.size() * sizeof(Point2D_CUDA), hipMemcpyHostToDevice);

  if (pinIdxVec.size() > h_pinIdxVec_size_) {
    h_pinIdxVec_size_ = pinIdxVec.size();
    hipFree(d_pinIdxVec);
    hipMalloc(&d_pinIdxVec, pinIdxVec.size() * sizeof(int));
  }
  hipMemcpy(d_pinIdxVec, pinIdxVec.data(), pinIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);

  if (netPtr.size() > h_netPtr_size_) {
    h_netPtr_size_ = netPtr.size();
    hipFree(d_netPtr);
    hipMalloc(&d_netPtr, netPtr.size() * sizeof(int));
  }
  hipMemcpy(d_netPtr, netPtr.data(), netPtr.size() * sizeof(int), hipMemcpyHostToDevice);
  

  if (netBBoxVec.size() > h_netBBoxVec_size_) {
    h_netBBoxVec_size_ = netBBoxVec.size();
    hipFree(d_netBBox);
    hipMalloc(&d_netBBox, netBBoxVec.size() * sizeof(Rect2D_CUDA));
  }
  hipMemcpy(d_netBBox, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);

  if (netBatchIdxVec.size() > h_netBatchIdxVec_size_) {
    h_netBatchIdxVec_size_ = netBatchIdxVec.size();
    hipFree(d_netBatchIdx);
    hipMalloc(&d_netBatchIdx, netBatchIdxVec.size() * sizeof(int));
  }
  hipMemcpy(d_netBatchIdx, netBatchIdxVec.data(), netBatchIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);

  
  if (numNodes > h_nodes_size_) {
    h_nodes_size_ = numNodes;
    std::cout << "h_nodes_size_ = " << h_nodes_size_ << std::endl;
    hipFree(d_nodes);
    hipMalloc(&d_nodes, numNodes * sizeof(NodeData2D));
  }

  cudaCheckError();

  if (h_nodes_size_ != h_parents_size_) {
    std::cout << "Error: h_nodes_size_ != h_parents_size_" << std::endl;
    exit(1);
  }

  std::cout << "[INFO] Number of nodes = " << numNodes << std::endl;
  std::cout << "[INFO] Number of parents = " << h_parents.size() << std::endl;
  std::cout << "[INFO] h_nodes_size_ = " << h_nodes_size_ << std::endl;
  std::cout << "[INFO] h_parents_size_ = " << h_parents_size_ << std::endl;
  */



  if (VERBOSE > 0) {
    std::cout << "[INFO] Device memory allocation is done." << std::endl;
  }

  // According to the original code
  unsigned BLOCKCOST = router_cfg->BLOCKCOST * 100;
  unsigned OVERFLOWCOST = 128;
  unsigned HISTCOST = 4;
 
  for (int chunkId = 0; chunkId < numChunks; chunkId++) {
    int netStartId = chunkNetPtr[chunkId];
    int netEndId = chunkNetPtr[chunkId + 1];
    
    // Perform Global Initialization
    int numThreads = 1024;
    int numBatchBlocks = (numNodes + numThreads - 1) / numThreads;
    
    initBatchNodeData2D_v3__kernel<<<numBatchBlocks, numThreads>>>(
      d_nodes, 
      numNodes);
    hipDeviceSynchronize();

    cudaCheckError();

    std::cout << "Test a " << std::endl;

  
    int numNets = netEndId - netStartId;
    if (VERBOSE > 0) {
      std::cout << "[INFO] Chunk " << chunkId << " has " << numNets << " nets." << std::endl;
    }
  
    int numNetBlocks = (numNets + numThreads - 1) / numThreads;
    initBatchPin2D_v3__kernel<<<numNetBlocks, numThreads>>>(
      d_nodes,
      d_pinIdxVec, 
      d_netPtr,
      d_netBatchIdx,
      netStartId,
      netEndId,
      numGrids);
    hipDeviceSynchronize();
  

    cudaCheckError();
    // std::cout << "[INFO] Initialization is done." << std::endl;

    std::cout << "Test b " << std::endl;

    //int numThreads = 1024;
    auto netRouteStart = std::chrono::high_resolution_clock::now();
    int numBlocks = numNets;
    biwaveBellmanFord2D_v3__kernel<<<numBlocks, numThreads>>>(
      netStartId,
      netEndId,
      d_netBatchIdx,
      d_netPtr,
      d_netBBox,
      d_pinIdxVec,
      d_costMap,
      d_nodes,
      d_dX,
      d_dY,
      d_xCoords,
      d_yCoords,
      maxIters,
      xDim,
      yDim,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST,
      HISTCOST);
    
    cudaCheckError();
    
    hipDeviceSynchronize();
    auto netRouteEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> netRouteTime = netRouteEnd - netRouteStart;

    std::cout << "Test c" << std::endl;
   
    /*
    int numPins = netPtr[netStartId + 1] - netPtr[netStartId];
    int HPWL = (netBBoxVec[netStartId].xMax - netBBoxVec[netStartId].xMin) + 
               (netBBoxVec[netStartId].yMax - netBBoxVec[netStartId].yMin);

    std::cout << "[INFO] Net: HPWL = " << HPWL << "  "
              << "numPins = " << numPins << " "
              << "netRouteTime = " << netRouteTime.count() << " ms" << std::endl;
    */

    // copy the back results to the d_parents
    copyParents2D__kernel<<<numBatchBlocks, numThreads>>>(
      d_nodes, 
      d_parents, 
      numNodes);

    hipDeviceSynchronize();
    cudaCheckError();

    std::cout << "Test d" << std::endl;

    hipMemcpy(h_parents.data(), d_parents, h_parents.size() * sizeof(Point2D_CUDA), hipMemcpyDeviceToHost);

    // Check the parents
    cudaCheckError();

    /*
    ChunkPathSyncUp(
      uworkers,
      netBatches[validBatches[0]],
      netBBoxVec,
      h_parents,
      netStartId,
      netEndId,
      xDim);
    */
  }  
  
  cudaCheckError();
  
  if (VERBOSE > 0) {
    std::cout << "[INFO] Kernel execution is done." << std::endl;
  }

  /*
  // Clear the memory
  hipFree(d_dX);
  hipFree(d_dY);
  hipFree(d_costMap);
  hipFree(d_xCoords);
  hipFree(d_yCoords);
  hipFree(d_nodes);
  hipFree(d_parents);
  hipFree(d_pinIdxVec);
  hipFree(d_netPtr);
  hipFree(d_netBBox);
  hipFree(d_netBatchIdx);
  cudaCheckError();
  */

  auto totalEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> totalTime = totalEnd - totalStart;
  return totalTime.count();
}



// ------------------------------------------------------------------------------------



// Define the device function for the biwaveBellmanFord2D
__device__
void runBiBellmanFord_2D__device(
  cooperative_groups::grid_group& g,   // grid-level cooperative group
  NodeData2D* nodes,
  uint64_t* d_costMap, 
  int* d_dX, int* d_dY,
  int* d_doneFlag,
  int LLX, int LLY, int URX, int URY,
  int xDim, int maxIters,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST)
{
  // A typical 1D decomposition over the entire 2D domain
  int total = (URX - LLX + 1) * (URY - LLY + 1);
  int tid     = blockDim.x * blockIdx.x + threadIdx.x;
  int stride  = blockDim.x * gridDim.x;
  int xDimTemp = URX - LLX + 1;
  maxIters = total;

  /*
  if (tid == 0) 
  { 
    printf("total = %d\n", total);
    for (int id = 0; id < total; id++) {
      int local_x = id % xDimTemp + LLX;
      int local_y = id / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = nodes[idx];
      if (nd.flags.src_flag == 1) {
        printf("device src_flag : id = %d, x = %d, y = %d cost = %d\n", idx, local_x, local_y, nd.forward_g_cost_prev);
      }
      
      if (nd.flags.dst_flag == 1) {
        printf("device dst_flag : id = %d, x = %d, y = %d, cost = %d\n", 
          idx, local_x, local_y, nd.backward_g_cost_prev);
      }
    }
  }
  */

  g.sync();

  bool globalDone = false;

  // We’ll do up to maxIters or until no changes / front-meet
  for (int iter = 0; iter < maxIters && !globalDone; iter++)
  {
    bool localFrontsMeet = false;
    ////////////////////////////////////////////////////////////////////////////
    // (1) Forward & backward relaxation phase
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      NodeData2D &nd = nodes[idx];

      // Forward relaxation
      // Typically: newCost = min over neighbors of (neighborCost + edgeWeight).
      // Be sure to skip if src_flag is set (source node may be pinned).
      if (!nd.flags.src_flag) {
        uint32_t bestCost = nd.forward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + d_dX[d];
          int ny = y + d_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          // We want neighbor's cost plus the edge weight, e.g. 100
          uint32_t neighborCost = nodes[nbrIdx].forward_g_cost_prev;
          // If neighbor is effectively infinite, skip
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          
          uint32_t newG = neighborCost +
            getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, nbrIdx, nx, ny);

          //printf("id = %d, x = %d, y = %d, d = %d, newG = %d, bestCost = %d, bestD = %d\n", 
          //  idx, x, y, d, newG, bestCost, bestD);

          // Check if we found a better cost
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) { // We found an improvement
          nd.forward_g_cost = bestCost;
          nd.forward_direction = computeParentDirection2D(bestD);
          nd.flags.forward_update_flag = 1;
          // printf("id = %d, x = %d, y = %d, forward_cost = %d, forward_update_flag = %d\n", 
          //  idx, x, y, nd.forward_g_cost, nd.flags.forward_update_flag);
        }
      } // end forward

      // Backward relaxation
      // Typically: newCost = min over neighbors of (neighbor.backward_cost + edgeWeight)
      // Skip if dst_flag is set (destination node may be pinned).
      if (!nd.flags.dst_flag) {
        uint32_t bestCost = nd.backward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + d_dX[d];
          int ny = y + d_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          uint32_t neighborCost = nodes[nbrIdx].backward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          
          uint32_t newG = neighborCost +
          getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
            congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
            idx, x, y, nbrIdx, nx, ny);
          
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) {
          nd.backward_g_cost = bestCost;
          nd.backward_direction = computeParentDirection2D(bestD);
          nd.flags.backward_update_flag = 1;
          //printf("id = %d, x = %d, y = %d, backward_cost = %d, backward_update_flag = %d\n", 
          //  idx, x, y, nd.backward_g_cost, nd.flags.backward_update_flag);
        }
      } // end backward
    } // end “for each node” (forward + backward)

    g.sync();


    /*
    if (tid == 0) {
      printf("iter = %d, maxIters = %d\n", iter, maxIters);
      for (int id = 0; id < total; id++) {
        int local_x = id % xDimTemp + LLX;
        int local_y = id / xDimTemp + LLY;
        int idx = locToIdx_2D(local_x, local_y, xDim);
        NodeData2D &nd = nodes[idx];
        if (nd.flags.backward_update_flag == 1) {
          printf("summary id = %d, x = %d, y = %d,  backward_cost = %d\n", idx, local_x, local_y,  nd.backward_g_cost);
        }

        if (nd.flags.forward_update_flag == 1) {
          printf("summary id = %d, x = %d, y = %d,  forward_cost = %d\n", idx, local_x, local_y, nd.forward_g_cost);
        }
      }
    }
    */
 
    ////////////////////////////////////////////////////////////////////////////
    // (2) Commit updated costs (double-buffering technique)
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = nodes[idx];
      // If forward_update_flag is set, copy forward_g_cost -> forward_g_cost_prev
      if (nd.flags.forward_update_flag) {
        nd.flags.forward_update_flag = false;
        nd.forward_g_cost_prev = nd.forward_g_cost;
      }
      
      // If backward_update_flag is set, copy backward_g_cost -> backward_g_cost_prev
      if (nd.flags.backward_update_flag) {
        nd.flags.backward_update_flag = false;
        nd.backward_g_cost_prev = nd.backward_g_cost;
      }

      nd.flags.forward_visited_flag_prev = nd.flags.forward_visited_flag;
      nd.flags.backward_visited_flag_prev = nd.flags.backward_visited_flag;
    }

    // Another full grid sync before the “stop” checks:
    g.sync();


    // Needs to be updated
    ////////////////////////////////////////////////////////////////////////////
    // (3) Check if forward and backward fronts meet
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      NodeData2D &nd = nodes[idx];
      /*
      // If either side is "unreached," skip
      if (nd.forward_g_cost_prev == 0xFFFFFFFF || 
          nd.backward_g_cost_prev == 0xFFFFFFFF)
      {
        continue;
      }
      */

      // Check the visited flag
      // bool localForwardMin = true;
      // bool localBackwardMin = true;
      
      
      if (!nd.flags.forward_visited_flag_prev) {
        bool localForwardMin = (nd.forward_g_cost_prev != 0xFFFFFFFF);
        for (int d = 0; d < 4; d++) {
          int nx = x + d_dX[d];
          int ny = y + d_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          NodeData2D &nbr = nodes[nbrIdx];
          
          // Check forward minimum
          if (!nbr.flags.forward_visited_flag_prev && 
              (nbr.forward_g_cost_prev + nbr.forward_h_cost < nd.forward_g_cost_prev + nd.forward_h_cost)) {
              localForwardMin = false;
          }
        }
  
        if (localForwardMin) {
          nd.flags.forward_visited_flag = true;
        }
      }


      if (!nd.flags.backward_visited_flag_prev) {
        bool localBackwardMin = (nd.backward_g_cost_prev != 0xFFFFFFFF);
        for (int d = 0; d < 4; d++) {
          int nx = x + d_dX[d];
          int ny = y + d_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          NodeData2D &nbr = nodes[nbrIdx];
          
          // Check forward minimum
          if (!nbr.flags.backward_visited_flag_prev && 
              (nbr.backward_g_cost_prev + nbr.backward_h_cost < nd.backward_g_cost_prev + nd.backward_h_cost)) {
              localBackwardMin = false;
          }
        }
  
        if (localBackwardMin) {
          nd.flags.backward_visited_flag = true;
        }
      }
    
    
      /*
      bool localForwardMin = (nd.forward_g_cost_prev != 0xFFFFFFFF);
      bool localBackwardMin = (nd.backward_g_cost_prev != 0xFFFFFFFF);
      for (int d = 0; d < 4; d++) {
        int nx = x + d_dX[d];
        int ny = y + d_dY[d];
        if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
          continue;  // out of bounds
        }
        
        int nbrIdx = locToIdx_2D(nx, ny, xDim);
        NodeData2D &nbr = nodes[nbrIdx];
        
        // Check forward minimum
        if (!nbr.flags.forward_visited_flag_prev && 
            (nbr.forward_g_cost_prev + nbr.forward_h_cost < nd.forward_g_cost_prev + nd.forward_h_cost)) {
            localForwardMin = false;
        }
    
        // Check backward minimum
        if (!nbr.flags.backward_visited_flag_prev &&
            (nbr.backward_g_cost_prev + nbr.backward_h_cost < nd.backward_g_cost_prev + nd.backward_h_cost)) {
            localBackwardMin = false;
        }
      }
      */

      /*
      if (localForwardMin) {
        nd.flags.forward_visited_flag = true;
      }

      if (localBackwardMin) {
        nd.flags.backward_visited_flag = true;  
      } */

      /*
      if (localForwardMin && localBackwardMin) {
        nd.flags.forward_visited_flag = true;
        nd.flags.backward_visited_flag = true;
      }*/
      
      /*
      for (int d = 0; d < 4; d++) {
        int nx = x + d_dX[d];
        int ny = y + d_dY[d];
        if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
          continue;
        }

        int nbrIdx = locToIdx_2D(nx, ny, xDim);
        if (nodes[nbrIdx].forward_g_cost_prev + nodes[nbrIdx].forward_h_cost >= nd.forward_g_cost_prev + nd.forward_h_cost_prev &&
            nodes[nbrIdx].backward_g_cost_prev + nodes[nbrIdx].backward_h_cost >= nd.backward_g_cost_prev + nd.backward_h_cost_prev) {
          nd.flags.forward_visited_flag = true;
          nd.flags.backward_visited_flag = true;
        }
      }
      */

    } // end “for each node”
    
    g.sync();

    // Check if any thread found a front-meet
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = nodes[idx];
      if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
        localFrontsMeet = true;
        // printf("localFrontsMeet = %d", localFrontsMeet);
      }


      /*
      // Check neighbors for meeting fronts.
      if (nd.flags.forward_visited_flag || nd.flags.backward_visited_flag) {
        for (int d = 0; d < 4; d++) {
          int nx = local_x + d_dX[d];
          int ny = local_y + d_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) { continue; }
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          NodeData2D &nbr = nodes[nbrIdx];
          if (nd.flags.forward_visited_flag && nbr.flags.backward_visited_flag) { localFrontsMeet = true; }
          if (nd.flags.backward_visited_flag && nbr.flags.forward_visited_flag) { localFrontsMeet = true; }
        }
      } */
    }
    
    g.sync();

    /*
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
      printf("iter = %d, maxIters = %d\n", iter, maxIters);
      for (int id = 0; id < total; id++) {
        int local_x = id % xDimTemp + LLX;
        int local_y = id / xDimTemp + LLY;
        int idx = locToIdx_2D(local_x, local_y, xDim);
        NodeData2D &nd = nodes[idx];
        if (nd.flags.backward_visited_flag == true) {
          printf("backward_visited_flag id = %d, backward_cost = %d", idx, nd.backward_g_cost);
        }

        if (nd.flags.forward_visited_flag == true) {
          printf("forward_visited_flag id = %d, forward_cost = %d", idx, nd.forward_g_cost);
        }
      }
    } */
 
    if (localFrontsMeet) {
      atomicExch(d_doneFlag, 1);
    }
    
    g.sync();

    if (*d_doneFlag == 1) {
      globalDone = true;
    }

    g.sync();

    //if (*d_doneFlag == 1) {
    //  *d_doneFlag = 0x7FFFFFFF;
    //  return;
    //}
  } // end for (iter)

  if (tid == 0) {
    if (*d_doneFlag == 1) {
      *d_doneFlag = 0x7FFFFFFF;
    }
  }
  g.sync();
  return;
}


// Define the device function for the meetId check
__device__
void findMeetIdAndTraceBackCost2D__device(
  NodeData2D* nodes,
  int* d_doneFlag, 
  int LLX, int LLY, int URX, int URY,
  int xDim)
{ 
  if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
    if (*d_doneFlag == 0) { printf("Error ! d_doneFlag = 0\n"); }
  }

  int xDimTemp = URX - LLX + 1;
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
  for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    if (nodes[idx].forward_g_cost != INF32 && nodes[idx].backward_g_cost != INF32) {
      int32_t cost = nodes[idx].forward_g_cost + nodes[idx].backward_g_cost;
      atomicMin(d_doneFlag, cost);      
    }
    /*
    if (nodes[idx].flags.forward_visited_flag && nodes[idx].flags.backward_visited_flag) {
      int32_t cost = nodes[idx].forward_g_cost + nodes[idx].backward_g_cost;
      atomicMin(d_doneFlag, cost);      
    } */
  }

  //if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
  //  printf("Cost2D MinCost = %d\n", *d_doneFlag);
  //}
}

__device__
void findMeetIdAndTraceBackId2D__device(
  NodeData2D* nodes,
  int* d_doneFlag, 
  int* d_meetId,
  int LLX, int LLY, int URX, int URY,
  int xDim)
{ 
  int xDimTemp = URX - LLX + 1;
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
  /*
  for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    if (nodes[idx].flags.forward_visited_flag && nodes[idx].flags.backward_visited_flag && 
        (nodes[idx].forward_g_cost + nodes[idx].backward_g_cost == *d_doneFlag)) {
      atomicMin(d_meetId, idx);      
    }
  }

  if (*d_meetId == 0x7FFFFFFF) {
    for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      if (nodes[idx].flags.forward_visited_flag && nodes[idx].flags.dst_flag) {
        atomicMin(d_meetId, idx);      
      }
    }
  }*/
  
  for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    if ((nodes[idx].forward_g_cost != INF32 && nodes[idx].backward_g_cost != INF32) &&
        (nodes[idx].forward_g_cost + nodes[idx].backward_g_cost == *d_doneFlag)) {
      atomicMin(d_meetId, idx);      
    }
  }
}

__device__
void forwardTraceBack2D__single_thread__device(
  NodeData2D* nodes, 
  int* d_meetId, 
  int* d_dX, int* d_dY,
  int LLX, int LLY, int URX, int URY,
  int xDim)
{
  if (*d_meetId == 0x7FFFFFFF) {
    return; // No meetId found
  }
  
  int curId = *d_meetId;
  int maxIterations = (URX - LLX + 1) * (URY - LLY + 1);
  int iteration = 0;
  
  /*
  while (nodes[curId].flags.src_flag == 0 && iteration < maxIterations) {
    uint8_t forwardDirection = nodes[curId].forward_direction;
    nodes[curId].flags.src_flag = 1;
    int2 xy = idxToLoc_2D(curId, xDim);
    int nx = xy.x + d_dX[forwardDirection];
    int ny = xy.y + d_dY[forwardDirection];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
      break;
    }  
    
    nodes[curId].golden_parent_x = nx;
    nodes[curId].golden_parent_y = ny;
    curId = locToIdx_2D(nx, ny, xDim);
    iteration++;
  }*/

  while (nodes[curId].flags.src_flag == 0 && iteration < maxIterations) {
    // Ensure forward_direction is valid (e.g., 0 <= forward_direction < 4)
    uint8_t fwdDir = nodes[curId].forward_direction;
    
    // Record the golden parent BEFORE moving on.
    int2 xy = idxToLoc_2D(curId, xDim);
    int nx = xy.x + d_dX[fwdDir];
    int ny = xy.y + d_dY[fwdDir];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) break;
    
    nodes[curId].golden_parent_x = nx;
    nodes[curId].golden_parent_y = ny;
    
    // Mark this node as processed.
    nodes[curId].flags.src_flag = 1;
    
    // Move to the next node.
    curId = locToIdx_2D(nx, ny, xDim);
    iteration++;
  }


  if (iteration >= maxIterations) {
    printf("Warning: Forward traceback exceeded maximum iterations.\n");
  }
}

__device__
void backwardTraceBack2D__single__thread__device(
  NodeData2D* nodes, 
  int* d_meetId, 
  int* d_dX, int* d_dY,
  int LLX, int LLY, int URX, int URY,
  int xDim)
{  
  if (*d_meetId == 0x7FFFFFFF) {
    return; // No meetId found
  }
  
  int curId = *d_meetId;
  if (nodes[curId].flags.dst_flag == 1) { 
    nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
    nodes[curId].flags.src_flag = 1;
    return;
  }
  
  int maxIterations = (URX - LLX + 1) * (URY - LLY + 1);
  int iteration = 0;

  while (nodes[curId].flags.dst_flag == 0 && iteration < maxIterations) {
    int2 xy = idxToLoc_2D(curId, xDim);
    uint8_t backwardDirection = nodes[curId].backward_direction;
    int nx = xy.x + d_dX[backwardDirection];
    int ny = xy.y + d_dY[backwardDirection];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
      break;
    }  
   
    int nextId = locToIdx_2D(nx, ny, xDim);
    if (nodes[nextId].golden_parent_x != -1) {
      printf("Error: Backward traceback meets forward traceback.\n");
    }    
    
    nodes[nextId].flags.src_flag = 1;
    nodes[nextId].golden_parent_x = xy.x;
    nodes[nextId].golden_parent_y = xy.y;
    
    curId = nextId;
    iteration++;
  }
  
  nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
  /*
  while (iteration < maxIterations) {
    // Record current position.
    int2 xy = idxToLoc_2D(curId, xDim);
    uint8_t bwdDir = nodes[curId].backward_direction;
    int nx = xy.x + d_dX[bwdDir];
    int ny = xy.y + d_dY[bwdDir];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) break;
    
    // Set the golden parent pointer for the current node before moving.
    nodes[curId].golden_parent_x = xy.x;
    nodes[curId].golden_parent_y = xy.y;
    // Mark the node as processed.
    nodes[curId].flags.src_flag = 1;
    
    // Check if moving to the next node would conflict with an already set pointer.
    int nextId = locToIdx_2D(nx, ny, xDim);
    if (nodes[nextId].golden_parent_x != -1) {
      break;
    }
    
    curId = nextId;
    
    // If we reach a node flagged as destination, clear the flag and stop.
    if (nodes[curId].flags.dst_flag == 1) {
      nodes[curId].flags.dst_flag = 0;
      break;
    }
    iteration++;
  } */

  if (iteration >= maxIterations) {
    printf("Warning: Backward traceback exceeded maximum iterations.\n");
  }
}



// Fused cooperative kernel that processes a single net.
__device__ 
void biwaveBellmanFord2D__device(
  cooperative_groups::grid_group& grid,   // grid-level cooperative group		
  int netId,
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlags,
  int* d_meetIds,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST, 
  int HISTCOST)
{
  // for this net
  int pinIdxStart = d_netPtr[netId];
  int pinIdxEnd = d_netPtr[netId + 1];
  int numPins = pinIdxEnd - pinIdxStart;
  int maxIters = d_netHPWL[netId];
  Rect2D_CUDA netBBox = d_netBBoxVec[netId];
  int LLX = netBBox.xMin;
  int LLY = netBBox.yMin;
  int URX = netBBox.xMax;
  int URY = netBBox.yMax;

  int* d_doneFlag = d_doneFlags + netId;
  int* d_meetId = d_meetIds + netId;

  // Connect the pin one by one
  //for (int pinIter = 1; pinIter < 2; pinIter++) {
  for (int pinIter = 1; pinIter < numPins; pinIter++) {
    // Initilization
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
      *d_doneFlag = 0;
      *d_meetId = 0x7FFFFFFF;

      /*
      for (int i = 0; i < xDim * yDim; i++) {
        int2 xy = idxToLoc_2D(i, xDim);
        if (d_nodes[i].flags.src_flag == 1) {
          printf("pinIter = %d, src_flag : id = %d, x = %d, y = %d\n", pinIter, i, xy.x, xy.y);
        }

        if (d_nodes[i].flags.dst_flag == 1) {
          printf("pinIter = %d, dst_flag : id = %d, x = %d, y = %d\n", pinIter, i, xy.x, xy.y);
        }
      }
      */

      // Check the temp
      // printf("pinIter = %d, netId = %d, numPins = %d, LLX = %d, LLY = %d, URX = %d, URY = %d\n", pinIter, netId, numPins, LLX, LLY, URX, URY);
      
      /*
      int total = (URX - LLX + 1) * (URY - LLY + 1);
      int xDimTemp = URX - LLX + 1;
      for (int id = 0; id < total; id++) {
        int local_x = id % xDimTemp + LLX;
        int local_y = id / xDimTemp + LLY;
        int idx = locToIdx_2D(local_x, local_y, xDim);
        NodeData2D &nd = d_nodes[idx];
        if (nd.flags.src_flag == true) {
          printf("local src_flag : id = %d, x = %d, y = %d\n", idx, local_x, local_y);
        }
          
        if (nd.flags.dst_flag == true) {
          printf("local dst_flag : id = %d, x = %d, y = %d\n", idx, local_x, local_y);
        }
      }
      */
    }


    initNodeData2D__device(
      d_nodes,
      d_pins, pinIdxStart, pinIter, 
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    // Run the Bellman Ford algorithm
    runBiBellmanFord_2D__device(
      grid, d_nodes, d_costMap, d_dX, d_dY, 
      d_doneFlag, LLX, LLY, URX, URY, xDim, maxIters,
      d_xCoords, d_yCoords, congThreshold,
      BLOCKCOST, OVERFLOWCOST, HISTCOST);  

    grid.sync();


    // Find the d_meetId
    findMeetIdAndTraceBackCost2D__device(
      d_nodes, d_doneFlag, 
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    findMeetIdAndTraceBackId2D__device(
      d_nodes, d_doneFlag, d_meetId,
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    // Traceback
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
      // printf("d_doneFlag = %d, d_meetId = %d\n",  *d_doneFlag,  *d_meetId);
      
      // trace back
      forwardTraceBack2D__single_thread__device(
        d_nodes, d_meetId, d_dX, d_dY, 
        LLX, LLY, URX, URY, xDim);

      // printf("finish forward traceback\n");    

      backwardTraceBack2D__single__thread__device(
        d_nodes, d_meetId, d_dX, d_dY, 
        LLX, LLY, URX, URY, xDim);

      // printf("finish backward traceback\n");
    }

    grid.sync(); // Synchronize all threads in the grid
  }
  //grid.sync();
}




// Fused cooperative kernel that processes a single net.
__global__
void biwaveBellmanFord2D__kernel(
  int netId,
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlags,
  int* d_meetIds,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST,
  int HISTCOST)
{
  // Obtain a handle to the entire cooperative grid.
  cg::grid_group grid = cg::this_grid();
  biwaveBellmanFord2D__device(
    grid,
    netId,
    d_netHPWL,
    d_netPtr,
    d_netBBoxVec,
    d_pins,
    d_costMap,
    d_nodes,
    d_dX,
    d_dY,
    d_doneFlags,
    d_meetIds,
    xDim, yDim,
    d_xCoords,
    d_yCoords,
    congThreshold,
    BLOCKCOST,
    OVERFLOWCOST,
    HISTCOST);    
  grid.sync(); // Synchronize all threads in the grid
}


// Just a wrapper function to call the kernel
void launchMazeRouteStream(
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBox,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlag,
  int* d_meetId,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int CONGCOST,
  int HISTCOST,
  int netId,
  int totalThreads,
  hipStream_t stream)
{
  void* kernelArgs[] = {
    &netId,
    &d_netHPWL,
    &d_netPtr,
    &d_netBBox,
    &d_pins,
    &d_costMap,
    &d_nodes,
    &d_dX,
    &d_dY,
    &d_doneFlag,
    &d_meetId,
    &xDim, 
    &yDim,
    &d_xCoords,
    &d_yCoords,
    &congThreshold,
    &BLOCKCOST, 
    &CONGCOST, 
    &HISTCOST
  };


  /*
  // Calculate the maximum number of blocks that can run cooperatively
  int deviceId = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceId);
  if (deviceProp.cooperativeLaunch == 0) {
    printf("Device does not support cooperative grid launch.\n");
    exit(1);
  } 

  if (VERBOSE > 0) {
    printf("Device supports cooperative grid launch.\n");
    printf("Cooperative launch: %d\n", deviceProp.cooperativeLaunch);
    printf("Multi-device coop: %d\n", deviceProp.cooperativeMultiDeviceLaunch);
    printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
  }

  int threadsPerBlock = 1024;
  int numBlocksPerSm = 0;
  hipError_t occErr = hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, biwaveBellmanFord2D__kernel, threadsPerBlock, 0);
  if (occErr != hipSuccess) {
    printf("Occupancy calculation error: %s\n", hipGetErrorString(occErr));
  } 

  if (numBlocksPerSm == 0) {
    numBlocksPerSm = 1;
    printf("Reset numBlocksPerSm to 1\n");
  }

  int numSms = deviceProp.multiProcessorCount;
  int numBlocks = numBlocksPerSm * numSms;
  //  printf("numBlocksPerSm = %d, numSms = %d, numBlocks = %d\n", numBlocksPerSm, numSms, numBlocks);

  // Ensure the grid size does not exceed the maximum allowed for cooperative launch
  int maxBlocksPerGrid = 0;
  hipDeviceGetAttribute(&maxBlocksPerGrid, hipDeviceAttributeMaxGridDimX, deviceId);
  // printf("maxBlocksPerGrid = %d\n", maxBlocksPerGrid);

  // For cooperative kernels that use grid-wide sync, you must launch the full grid,
  // even if totalThreads would allow a smaller grid. (Inside the kernel, extra blocks
  // should simply exit if their blockIdx.x exceeds the work limit.)
  numBlocks = min(numBlocks, maxBlocksPerGrid);
  // printf("Launching kernel with %d blocks\n", numBlocks);
  */

  int threadsPerBlock = 1024;
  int numBlocks = 108;

  hipError_t err = hipLaunchCooperativeKernel(
    (void*)biwaveBellmanFord2D__kernel,
    numBlocks, threadsPerBlock,
    kernelArgs,
    0,       // additional dynamic shared memory (if needed)
    stream); // launch on the given stream

  if (err != hipSuccess) {
    printf("Kernel launch error (net %d): %s\n", netId, hipGetErrorString(err));
  }
}


// We need to restore the connected path from the golden parent
float batchPathSyncUp(
  std::vector<std::unique_ptr<FlexGRWorker>>& uworkers,
  std::vector<grNet*>& nets,
  std::vector<Rect2D_CUDA>& netBBoxVec,
  std::vector<NodeData2D>& nodes,
  int xDim)
{ 
  auto syncupTimeStart = std::chrono::high_resolution_clock::now();
  
  for (int netId = 0; netId < nets.size(); netId++) {
    auto& net = nets[netId];
    auto& uworker = uworkers[net->getWorkerId()];
    auto& gridGraph = uworker->getGridGraph();
    auto workerLL = uworkers[net->getWorkerId()]->getRouteGCellIdxLL();
    int workerLX = workerLL.x();
    int workerLY = workerLL.y();  
    auto& netBBox = netBBoxVec[netId];
    int LLX = netBBox.xMin;
    int LLY = netBBox.yMin;
    int URX = netBBox.xMax;
    int URY = netBBox.yMax;
    int xDimTemp = URX - LLX + 1;
    int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
    for (int localIdx = 0; localIdx < numNodes; localIdx++) {
      int localX = localIdx % xDimTemp;
      int localY = localIdx / xDimTemp;

      int x = localX + LLX;
      int y = localY + LLY;      
      int idx = locToIdx_2D(x, y, xDim);

      x -= workerLX;
      y -= workerLY;

      int parentX = nodes[idx].golden_parent_x - workerLX;
      int parentY = nodes[idx].golden_parent_y - workerLY;

      gridGraph.setGoldenParent2D(x, y, parentX, parentY);      
      /*
      if (nodes[idx].golden_parent_x != -1 || nodes[idx].golden_parent_y != -1) {
        std::cout << "Net " << netId << " x = " << x << " y = " << y << " "
                  << "Parent " << parentX << "  " << parentY << std::endl;
      }
      */
    }    
  }

  auto syncupTimeEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> syncupTime = syncupTimeEnd - syncupTimeStart;
  return syncupTime.count();
}




__global__ 
void initBatchNodeData2D__kernel(
  NodeData2D* d_nodes,
  int numNodes)
{
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numNodes; idx += gridDim.x * blockDim.x) {
    initNodeData2D(d_nodes[idx]);
  }
}



__global__
void initBatchPin2D__kernel(
  NodeData2D* d_nodes,
  int* d_pins,
  int* d_netPtr,
  int netIdStart,
  int numNets)
{  
  int netId = blockIdx.x * blockDim.x + threadIdx.x;
  if (netId >= numNets) {
    return;
  }

  netId += netIdStart;

  int pinIdxStart = d_netPtr[netId];
  int pinIdxEnd = d_netPtr[netId + 1];
  d_nodes[d_pins[pinIdxStart]].flags.src_flag = true;
  for (int pinIter = pinIdxStart + 1; pinIter < pinIdxEnd; pinIter++) {
    d_nodes[d_pins[pinIter]].flags.dst_flag = true;
  }
}



__global__
void initBatchPin2D_v2__kernel(
  NodeData2D* d_nodes,
  int* d_pins,
  int* d_netPtr,
  int* d_netBatchIdx,
  int numNets,
  int numGrids)
{  
  for (int netId = blockIdx.x * blockDim.x + threadIdx.x; netId < numNets; netId += gridDim.x * blockDim.x) {
    int batchId = d_netBatchIdx[netId];
    int baseNodeId = batchId * numGrids;
    int pinIdxStart = d_netPtr[netId];
    int pinIdxEnd = d_netPtr[netId + 1];
    d_nodes[d_pins[pinIdxStart] + baseNodeId].flags.src_flag = true;
    for (int pinIter = pinIdxStart + 1; pinIter < pinIdxEnd; pinIter++) {
      d_nodes[d_pins[pinIter] + baseNodeId].flags.dst_flag = true;
    }
  }
}






// Just a wrapper function to call the kernel
void launchMazeRouteStream_update_v2(
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBox,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlag,
  int* d_meetId,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  Point2D_CUDA* d_parents,
  float congThreshold,
  int BLOCKCOST,
  int CONGCOST,
  int HISTCOST,
  int netId,
  int netIdx,
  int batchIdx,
  int totalThreads,
  int nodeBaseIdx,
  hipStream_t stream)
{
  void* kernelArgs[] = {
    &netId,
    &netIdx,
    &batchIdx,
    &nodeBaseIdx,
    &d_parents,
    &d_netHPWL,
    &d_netPtr,
    &d_netBBox,
    &d_pins,
    &d_costMap,
    &d_nodes,
    &d_dX,
    &d_dY,
    &d_doneFlag,
    &d_meetId,
    &xDim, 
    &yDim,
    &d_xCoords,
    &d_yCoords,
    &congThreshold,
    &BLOCKCOST, 
    &CONGCOST, 
    &HISTCOST
  };

  int threadsPerBlock = 1024;
  int numBlocks = 108;

  /*
  hipError_t err = hipLaunchCooperativeKernel(
    (void*)biwaveBellmanFord2D_update_v2__kernel,
    numBlocks, threadsPerBlock,
    kernelArgs,
    0,       // additional dynamic shared memory (if needed)
    stream); // launch on the given stream


  if (err != hipSuccess) {
    printf("Kernel launch error (net %d): %s\n", netId, hipGetErrorString(err));
  }*/
}































float FlexGR::GPUAccelerated2DMazeRoute_update_v2(
  std::vector<std::unique_ptr<FlexGRWorker> >& uworkers,
  std::vector<std::vector<grNet*> >& netBatches,
  std::vector<int>& validBatches,
  std::vector<Point2D_CUDA>& h_parents,
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  RouterConfiguration* router_cfg,
  float congThreshold,
  int xDim, int yDim)
{
  // Start overall timing.
  auto totalStart = std::chrono::high_resolution_clock::now();
  int numGrids = xDim * yDim;
  
  std::vector<Point2D_CUDA> netVec;
  std::vector<int> netPtr;
  std::vector<int> netBatchIdxVec; 
  std::vector<int> netHWPL;
  std::vector<Rect2D_CUDA> netBBoxVec;
  std::vector<int> pinIdxVec;
  std::vector<int> batchPtr;
  
  netPtr.push_back(0);
  batchPtr.push_back(0);  
  int maxBatchSize = 0;
  int minBatchSize = std::numeric_limits<int>::max();
  int batchIdx = 0;
  for (auto& batchId : validBatches) {
    auto& batch = netBatches[batchId];
    for (auto& net : batch) {
      for (auto& idx : net->getPinGCellAbsIdxs()) {
        netVec.push_back(Point2D_CUDA(idx.x(), idx.y()));
        pinIdxVec.push_back(locToIdx_2D(idx.x(), idx.y(), xDim));
      }
      netBatchIdxVec.push_back(batchIdx);
      netPtr.push_back(netVec.size());
      auto netBBox = net->getRouteAbsBBox();
      netBBoxVec.push_back(
        Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
      netHWPL.push_back(net->getHPWL());
    }
    batchPtr.push_back(netHWPL.size());
    batchIdx++;
    maxBatchSize = std::max(maxBatchSize, static_cast<int>(batch.size()));
    minBatchSize = std::min(minBatchSize, static_cast<int>(batch.size()));
  }

  int numBatches = validBatches.size();
  int numNets = netHWPL.size();
  
  // std::cout << "[INFO] Number of batches: " << numBatches << std::endl;
  std::cout << "[INFO] Max batch size: " << maxBatchSize << std::endl;
  std::cout << "[INFO] Min batch size: " << minBatchSize << std::endl;
  // std::vector<Point2D_CUDA> h_parents(numGrids * numBatches, Point2D_CUDA(-1, -1));

  // Allocate and copy device memory
  // We need to define the needed utility variables
  std::vector<int> h_dX = {0, 1, 0, -1};
  std::vector<int> h_dY = {1, 0, -1, 0};
  
  int* d_dX = nullptr;
  int* d_dY = nullptr;
  

  int* d_doneFlag = nullptr; // This is allocated for each net seperately (maxBatchSize)
  int* d_meetId = nullptr; // This is allocated for each net seperately (maxBatchSize)
  
  // For the design specific variables (numGrids)
  uint64_t* d_costMap = nullptr;
  int* d_xCoords = nullptr;
  int* d_yCoords = nullptr;
  NodeData2D* d_nodes = nullptr;
  Point2D_CUDA* d_parents = nullptr;
  
  int* d_pinIdxVec = nullptr;
  int* d_netHPWL = nullptr;
  int* d_netPtr = nullptr;
  Rect2D_CUDA* d_netBBox = nullptr;
  int* d_netBatchIdx = nullptr;

  int numBatchesParallel = numBatches;
  //int numBatchesParallel = 500;  
  int numNodes = numGrids * numBatchesParallel;

  // Allocate the device memory for the d_dX and d_dY
  hipMalloc(&d_dX, 4 * sizeof(int));
  hipMalloc(&d_dY, 4 * sizeof(int));
  
  hipMalloc(&d_doneFlag, numNets * sizeof(int));
  hipMalloc(&d_meetId, numNets * sizeof(int));
  
  hipMalloc(&d_costMap, numNodes * sizeof(uint64_t));
  hipMalloc(&d_xCoords, h_xCoords.size() * sizeof(int));
  hipMalloc(&d_yCoords, h_yCoords.size() * sizeof(int));
  hipMalloc(&d_nodes, numNodes * sizeof(NodeData2D));
  hipMalloc(&d_parents, numNodes * sizeof(Point2D_CUDA));

  hipMalloc(&d_pinIdxVec, pinIdxVec.size() * sizeof(int));
  hipMalloc(&d_netHPWL, netHWPL.size() * sizeof(int));
  hipMalloc(&d_netPtr, netPtr.size() * sizeof(int));
  hipMalloc(&d_netBBox, netBBoxVec.size() * sizeof(Rect2D_CUDA));
  hipMalloc(&d_netBatchIdx, netBatchIdxVec.size() * sizeof(int));

  hipMemcpy(d_dX, h_dX.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dY, h_dY.data(), 4 * sizeof(int), hipMemcpyHostToDevice);


  hipMemcpy(d_costMap, h_costMap.data(), numNodes * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_xCoords, h_xCoords.data(), h_xCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_yCoords, h_yCoords.data(), h_yCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_parents, h_parents.data(), numNodes * sizeof(Point2D_CUDA), hipMemcpyHostToDevice);

  hipMemcpy(d_pinIdxVec, pinIdxVec.data(), pinIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netHPWL, netHWPL.data(), netHWPL.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netPtr, netPtr.data(), netPtr.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netBBox, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);
  hipMemcpy(d_netBatchIdx, netBatchIdxVec.data(), netBatchIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);  
  
  cudaCheckError();


  std::vector<hipStream_t> netStreams(maxBatchSize);
  for (auto& stream : netStreams) {
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  }
 
  int numStreams = static_cast<int>(netStreams.size());


  // According to the original code
  unsigned BLOCKCOST = router_cfg->BLOCKCOST * 100;
  unsigned OVERFLOWCOST = 128;
  unsigned HISTCOST = 4;
 
  // Perform Global Initialization
  int numThreads = 1024;
  int numBatchBlocks = (numNodes + numThreads - 1) / numThreads;
  initBatchNodeData2D__kernel<<<numBatchBlocks, numThreads>>>(
    d_nodes, numNodes);

  hipDeviceSynchronize();

  int numNetBlocks = (numNets + numThreads - 1) / numThreads;
  initBatchPin2D_v2__kernel<<<numNetBlocks, numThreads>>>(
    d_nodes,
    d_pinIdxVec, 
    d_netPtr,
    d_netBatchIdx,
    numNets,
    numGrids);
  hipDeviceSynchronize();

  for (int netId = 0; netId < numNets; netId++) {
    // pick which stream to use
    int streamIdx = netId % numStreams;
    hipStream_t stream = netStreams[streamIdx];
    auto& netBBox = netBBoxVec[netId];
    int localGridSize = (netBBox.xMax - netBBox.xMin + 1) * (netBBox.yMax - netBBox.yMin + 1);
    int nodeBaseIdx = netBatchIdxVec[netId] * numGrids;

    launchMazeRouteStream_update_v2(
      d_netHPWL, d_netPtr, d_netBBox, d_pinIdxVec, 
      d_costMap, d_nodes, 
      d_dX, d_dY, d_doneFlag, d_meetId,
      xDim, yDim, 
      d_xCoords,
      d_yCoords,
      d_parents,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST,
      HISTCOST,
      netId,
      netId,
      netBatchIdxVec[netId], 
      localGridSize,
      nodeBaseIdx,
      stream);
  }

  
  // Sync up the golden parents
  cudaCheckError();

  for (int i = 0; i < maxBatchSize; i++) {
    hipStreamSynchronize(netStreams[i]);
  }

  hipMemcpy(h_parents.data(), d_parents, numNodes * sizeof(Point2D_CUDA), hipMemcpyDeviceToHost);
  cudaCheckError();
  
  for (int i = 0; i < maxBatchSize; i++) {
    hipStreamDestroy(netStreams[i]);
  }


  // Clear the memory
  hipFree(d_dX);
  hipFree(d_dY);
  hipFree(d_doneFlag);
  hipFree(d_meetId);
  hipFree(d_costMap);
  hipFree(d_xCoords);
  hipFree(d_yCoords);
  hipFree(d_nodes);
  hipFree(d_parents);
  
  hipFree(d_pinIdxVec);
  hipFree(d_netHPWL);
  hipFree(d_netPtr);
  hipFree(d_netBBox);

  cudaCheckError();

  auto totalEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> totalTime = totalEnd - totalStart;
  return totalTime.count();
}























// Fused cooperative kernel that processes a single net.
__device__ 
void biwaveBellmanFord2D_update__device(
  cooperative_groups::grid_group& grid,   // grid-level cooperative group		
  int netId,
  int netIdx,
  int batchIdx,
  Point2D_CUDA* d_parents,
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlags,
  int* d_meetIds,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST, 
  int HISTCOST)
{
  // for this net
  int pinIdxStart = d_netPtr[netId];
  int pinIdxEnd = d_netPtr[netId + 1];
  int numPins = pinIdxEnd - pinIdxStart;
  int maxIters = d_netHPWL[netId];
  Rect2D_CUDA netBBox = d_netBBoxVec[netId];
  int LLX = netBBox.xMin;
  int LLY = netBBox.yMin;
  int URX = netBBox.xMax;
  int URY = netBBox.yMax;

  int* d_doneFlag = d_doneFlags + netIdx;
  int* d_meetId = d_meetIds + netIdx;

  // if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
  //  printf("netId = %d, netIdx = %d, batchIdx = %d, pinIdxStart = %d, pinIdxEnd = %d, numPins = %d, maxIters = %d\n", 
  //          netId, netIdx, batchIdx, pinIdxStart, pinIdxEnd, numPins, maxIters);
  //}


  // Connect the pin one by one
  //for (int pinIter = 1; pinIter < 2; pinIter++) {
  for (int pinIter = 1; pinIter < numPins; pinIter++) {
    // Initilization
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
      *d_doneFlag = 0;
      *d_meetId = 0x7FFFFFFF;
    }

    initNodeData2D__device(
      d_nodes,
      d_pins, pinIdxStart, pinIter, 
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    // Run the Bellman Ford algorithm
    runBiBellmanFord_2D__device(
      grid, d_nodes, d_costMap, d_dX, d_dY, 
      d_doneFlag, LLX, LLY, URX, URY, xDim, maxIters,
      d_xCoords, d_yCoords, congThreshold,
      BLOCKCOST, OVERFLOWCOST, HISTCOST);  

    grid.sync();

    // Find the d_meetId
    findMeetIdAndTraceBackCost2D__device(
      d_nodes, d_doneFlag, 
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    findMeetIdAndTraceBackId2D__device(
      d_nodes, d_doneFlag, d_meetId,
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    // Traceback
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
      // printf("d_doneFlag = %d, d_meetId = %d\n",  *d_doneFlag,  *d_meetId);
      
      // trace back
      forwardTraceBack2D__single_thread__device(
        d_nodes, d_meetId, d_dX, d_dY, 
        LLX, LLY, URX, URY, xDim);

      // printf("finish forward traceback\n");    

      backwardTraceBack2D__single__thread__device(
        d_nodes, d_meetId, d_dX, d_dY, 
        LLX, LLY, URX, URY, xDim);

      // printf("finish backward traceback\n");
    }

    grid.sync(); // Synchronize all threads in the grid
  }

  //grid.sync();
}


// Fused cooperative kernel that processes a single net.
__global__
void biwaveBellmanFord2D_update__kernel(
  int netId,
  int netIdx,
  int batchIdx,
  Point2D_CUDA* d_parents,
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlags,
  int* d_meetIds,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST,
  int HISTCOST)
{
  // Obtain a handle to the entire cooperative grid.
  cg::grid_group grid = cg::this_grid();
  biwaveBellmanFord2D_update__device(
    grid,
    netId,
    netIdx,
    batchIdx,
    d_parents,
    d_netHPWL,
    d_netPtr,
    d_netBBoxVec,
    d_pins,
    d_costMap,
    d_nodes,
    d_dX,
    d_dY,
    d_doneFlags,
    d_meetIds,
    xDim, yDim,
    d_xCoords,
    d_yCoords,
    congThreshold,
    BLOCKCOST,
    OVERFLOWCOST,
    HISTCOST);    
  grid.sync(); // Synchronize all threads in the grid
}


// Just a wrapper function to call the kernel
void launchMazeRouteStream_update(
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBox,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlag,
  int* d_meetId,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  Point2D_CUDA* d_parents,
  float congThreshold,
  int BLOCKCOST,
  int CONGCOST,
  int HISTCOST,
  int netIdx,
  int netId,
  int batchIdx,
  int totalThreads,
  hipStream_t stream)
{
  void* kernelArgs[] = {
    &netId,
    &netIdx,
    &batchIdx,
    &d_parents,
    &d_netHPWL,
    &d_netPtr,
    &d_netBBox,
    &d_pins,
    &d_costMap,
    &d_nodes,
    &d_dX,
    &d_dY,
    &d_doneFlag,
    &d_meetId,
    &xDim, 
    &yDim,
    &d_xCoords,
    &d_yCoords,
    &congThreshold,
    &BLOCKCOST, 
    &CONGCOST, 
    &HISTCOST
  };

  int threadsPerBlock = 1024;
  int numBlocks = 108;

  hipError_t err = hipLaunchCooperativeKernel(
    (void*)biwaveBellmanFord2D_update__kernel,
    numBlocks, threadsPerBlock,
    kernelArgs,
    0,       // additional dynamic shared memory (if needed)
    stream); // launch on the given stream

  if (err != hipSuccess) {
    printf("Kernel launch error (net %d): %s\n", netId, hipGetErrorString(err));
  }
}




// Fused cooperative kernel that processes a single net.
__global__
void biwaveBellmanFord2D_update_v2__kernel(
  int netId,
  int netIdx,
  int batchIdx,
  int nodeBaseIdx,
  Point2D_CUDA* d_parents,
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlags,
  int* d_meetIds,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST,
  int HISTCOST)
{
  // Obtain a handle to the entire cooperative grid.
  cg::grid_group grid = cg::this_grid();
  biwaveBellmanFord2D_update__device(
    grid,
    netId,
    netIdx,
    batchIdx,
    d_parents,
    d_netHPWL,
    d_netPtr,
    d_netBBoxVec,
    d_pins,
    d_costMap,
    d_nodes + nodeBaseIdx,
    d_dX,
    d_dY,
    d_doneFlags,
    d_meetIds,
    xDim, yDim,
    d_xCoords,
    d_yCoords,
    congThreshold,
    BLOCKCOST,
    OVERFLOWCOST,
    HISTCOST);    
  grid.sync(); // Synchronize all threads in the grid
}





float FlexGR::GPUAccelerated2DMazeRoute_update(
  std::vector<std::unique_ptr<FlexGRWorker> >& uworkers,
  std::vector<std::vector<grNet*> >& netBatches,
  std::vector<int>& validBatches,
  std::vector<Point2D_CUDA>& h_parents,
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  RouterConfiguration* router_cfg,
  float congThreshold,
  int xDim, int yDim)
{
  // Start overall timing.
  auto totalStart = std::chrono::high_resolution_clock::now();
  int numGrids = xDim * yDim;
  
  std::vector<Point2D_CUDA> netVec;
  std::vector<int> netPtr; 
  std::vector<int> netHWPL;
  std::vector<Rect2D_CUDA> netBBoxVec;
  std::vector<int> pinIdxVec;
  std::vector<int> batchPtr;
  
  netPtr.push_back(0);
  batchPtr.push_back(0);  
  int maxBatchSize = 0;
  int minBatchSize = std::numeric_limits<int>::max();
  for (auto& batchId : validBatches) {
    auto& batch = netBatches[batchId];
    for (auto& net : batch) {
      for (auto& idx : net->getPinGCellAbsIdxs()) {
        netVec.push_back(Point2D_CUDA(idx.x(), idx.y()));
        pinIdxVec.push_back(locToIdx_2D(idx.x(), idx.y(), xDim));
      }
      netPtr.push_back(netVec.size());
      auto netBBox = net->getRouteAbsBBox();
      netBBoxVec.push_back(
        Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
      netHWPL.push_back(net->getHPWL());
    }
    batchPtr.push_back(netHWPL.size());
    maxBatchSize = std::max(maxBatchSize, static_cast<int>(batch.size()));
    minBatchSize = std::min(minBatchSize, static_cast<int>(batch.size()));
  }

  int numBatches = validBatches.size();
  // std::cout << "[INFO] Number of batches: " << numBatches << std::endl;
  std::cout << "[INFO] Max batch size: " << maxBatchSize << std::endl;
  std::cout << "[INFO] Min batch size: " << minBatchSize << std::endl;
  // std::vector<Point2D_CUDA> h_parents(numGrids * numBatches, Point2D_CUDA(-1, -1));

  // Allocate and copy device memory
  // We need to define the needed utility variables
  std::vector<int> h_dX = {0, 1, 0, -1};
  std::vector<int> h_dY = {1, 0, -1, 0};
  
  int* d_dX = nullptr;
  int* d_dY = nullptr;
  

  int* d_doneFlag = nullptr; // This is allocated for each net seperately (maxBatchSize)
  int* d_meetId = nullptr; // This is allocated for each net seperately (maxBatchSize)
  
  // For the design specific variables (numGrids)
  uint64_t* d_costMap = nullptr;
  int* d_xCoords = nullptr;
  int* d_yCoords = nullptr;
  NodeData2D* d_nodes = nullptr;
  Point2D_CUDA* d_parents = nullptr;
  
  int* d_pinIdxVec = nullptr;
  int* d_netHPWL = nullptr;
  int* d_netPtr = nullptr;
  Rect2D_CUDA* d_netBBox = nullptr;

  // Allocate the device memory for the d_dX and d_dY
  hipMalloc(&d_dX, 4 * sizeof(int));
  hipMalloc(&d_dY, 4 * sizeof(int));
  
  hipMalloc(&d_doneFlag, maxBatchSize * sizeof(int));
  hipMalloc(&d_meetId, maxBatchSize * sizeof(int));
  
  hipMalloc(&d_costMap, numGrids * sizeof(uint64_t));
  hipMalloc(&d_xCoords, h_xCoords.size() * sizeof(int));
  hipMalloc(&d_yCoords, h_yCoords.size() * sizeof(int));
  hipMalloc(&d_nodes, numGrids * sizeof(NodeData2D));
  hipMalloc(&d_parents, numGrids * numBatches * sizeof(Point2D_CUDA));

  hipMalloc(&d_pinIdxVec, pinIdxVec.size() * sizeof(int));
  hipMalloc(&d_netHPWL, netHWPL.size() * sizeof(int));
  hipMalloc(&d_netPtr, netPtr.size() * sizeof(int));
  hipMalloc(&d_netBBox, netBBoxVec.size() * sizeof(Rect2D_CUDA));


  hipMemcpy(d_dX, h_dX.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dY, h_dY.data(), 4 * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(d_costMap, h_costMap.data(), numGrids * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_xCoords, h_xCoords.data(), h_xCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_yCoords, h_yCoords.data(), h_yCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_parents, h_parents.data(), numGrids * numBatches * sizeof(Point2D_CUDA), hipMemcpyHostToDevice);

  hipMemcpy(d_pinIdxVec, pinIdxVec.data(), pinIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netHPWL, netHWPL.data(), netHWPL.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netPtr, netPtr.data(), netPtr.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netBBox, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);
  cudaCheckError();



  std::vector<hipStream_t> netStreams(maxBatchSize);
  for (auto& stream : netStreams) {
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  }
  

  // According to the original code
  unsigned BLOCKCOST = router_cfg->BLOCKCOST * 100;
  unsigned OVERFLOWCOST = 128;
  unsigned HISTCOST = 4;

  for (int batchIdx = 0; batchIdx < numBatches; batchIdx++) {
    int netBatchStart = batchPtr[batchIdx];
    int netBatchEnd = batchPtr[batchIdx + 1];
    int numNets = netBatchEnd - netBatchStart;

    // Initialize the nodes accordingly
    int numThreads = 1024;
    int numBatchBlocks = (numGrids + numThreads - 1) / numThreads;
    initBatchNodeData2D__kernel<<<numBatchBlocks, numThreads>>>(
      d_nodes, numGrids);
    cudaCheckError();
    hipDeviceSynchronize();

    int numNetBlocks = (numNets + numThreads - 1) / numThreads;
    initBatchPin2D__kernel<<<numNetBlocks, numThreads>>>(
      d_nodes,
      d_pinIdxVec, 
      d_netPtr,
      netBatchStart,
      numNets);
    cudaCheckError();
    hipDeviceSynchronize();
      
    std::vector<NodeData2D> h_nodes(numGrids);
    hipMemcpy(h_nodes.data(), d_nodes, numGrids * sizeof(NodeData2D), hipMemcpyDeviceToHost);
    cudaCheckError();

    /*
    std::cout << "numGrids = " << numGrids << std::endl;
    for (int nodeId = 0; nodeId < numGrids; nodeId++) {
      if (h_nodes[nodeId].flags.src_flag == true) {
        std::cout << "src_flag : id = " << nodeId << std::endl;
      }

      if (h_nodes[nodeId].flags.dst_flag == true) {
        std::cout << "dst_flag : id = " << nodeId << std::endl;
      }
    } 
    */

    // Perform the routing here
    for (int netIdx = 0; netIdx < numNets; netIdx++) {
      int netId = netBatchStart + netIdx;
      auto& netBBox = netBBoxVec[netId];
      int localGridSize = (netBBox.xMax - netBBox.xMin + 1) * (netBBox.yMax - netBBox.yMin + 1);
    
      launchMazeRouteStream_update(
        d_netHPWL, d_netPtr, d_netBBox, d_pinIdxVec, 
        d_costMap, d_nodes, 
        d_dX, d_dY, d_doneFlag, d_meetId,
        xDim, yDim, 
        d_xCoords,
        d_yCoords,
        d_parents,
        congThreshold,
        BLOCKCOST,
        OVERFLOWCOST,
        HISTCOST,
        netIdx, 
        netId,
        batchIdx,  
        (netBBox.xMax - netBBox.xMin + 1) * (netBBox.yMax - netBBox.yMin + 1),
        netStreams[netIdx]);
    }

    // Wait for all streams to finish
    for (int i = 0; i < numNets; i++) {
      hipStreamSynchronize(netStreams[i]);
    }

    //std::cout << "numNets = " << numNets << std::endl;
  }

  // Sync up the golden parents
  cudaCheckError();

  hipMemcpy(h_parents.data(), d_parents, numGrids * numBatches * sizeof(Point2D_CUDA), hipMemcpyDeviceToHost);
  cudaCheckError();

  
  for (int i = 0; i < maxBatchSize; i++) {
    hipStreamDestroy(netStreams[i]);
  }


  // Clear the memory
  hipFree(d_dX);
  hipFree(d_dY);
  hipFree(d_doneFlag);
  hipFree(d_meetId);
  hipFree(d_costMap);
  hipFree(d_xCoords);
  hipFree(d_yCoords);
  hipFree(d_nodes);
  hipFree(d_parents);
  
  hipFree(d_pinIdxVec);
  hipFree(d_netHPWL);
  hipFree(d_netPtr);
  hipFree(d_netBBox);

  cudaCheckError();

  auto totalEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> totalTime = totalEnd - totalStart;
  return totalTime.count();
}








float FlexGR::GPUAccelerated2DMazeRoute(
  std::vector<std::unique_ptr<FlexGRWorker>>& uworkers,
  std::vector<grNet*>& nets,
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  RouterConfiguration* router_cfg,
  float congThreshold,
  int xDim, int yDim)
{
  // Start overall timing.
  auto totalStart = std::chrono::high_resolution_clock::now();

  if (VERBOSE > 0) {
    std::cout << "[INFO] GPU accelerated 2D Maze Routing" << std::endl;
    std::cout << "[INFO] Number of nets: " << nets.size() << std::endl;
  }

  int numGrids = xDim * yDim;
  int numNets = nets.size();
  
  std::vector<Point2D_CUDA> netVec;
  std::vector<int> netPtr; 
  std::vector<int> netHWPL;
  std::vector<Rect2D_CUDA> netBBoxVec;
  std::vector<int> pinIdxVec;
 

  netPtr.push_back(0);
  for (auto& net : nets) {
    for (auto& idx : net->getPinGCellAbsIdxs()) {
      netVec.push_back(Point2D_CUDA(idx.x(), idx.y()));
      pinIdxVec.push_back(locToIdx_2D(idx.x(), idx.y(), xDim));
      if (VERBOSE > 0) {
        std::cout << "Pin x = " << idx.x() << " y = " << idx.y() << " idx = " << locToIdx_2D(idx.x(), idx.y(), xDim) << std::endl;
      }
    }
    netPtr.push_back(netVec.size());
    auto netBBox = net->getRouteAbsBBox();
    netBBoxVec.push_back(
      Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
    netHWPL.push_back(net->getHPWL());
  }


  //===========================================================================
  NodeData2D* h_nodesPinned = nullptr;
  hipHostAlloc(&h_nodesPinned, numGrids * sizeof(NodeData2D), hipHostMallocDefault);
  for (int i = 0; i < numGrids; i++) {
    initNodeData2D(h_nodesPinned[i]);
  }

  // Mark the source and destination nodes for each net.
  for (int netId = 0; netId < numNets; netId++) {
    int pinIdxStart = netPtr[netId];
    int pinIdxEnd   = netPtr[netId + 1];
    h_nodesPinned[pinIdxVec[pinIdxStart]].flags.src_flag = 1;
    for (int idx = pinIdxStart + 1; idx < pinIdxEnd; idx++) {
      h_nodesPinned[pinIdxVec[idx]].flags.dst_flag = 1;
    }
  }



  // Allocate and copy device memory
  // We need to define the needed utility variables
  std::vector<int> h_dX = {0, 1, 0, -1};
  std::vector<int> h_dY = {1, 0, -1, 0};
  
  int* d_dX = nullptr;
  int* d_dY = nullptr;
  int* d_doneFlag = nullptr; // This is allocated for each net seperately
  int* d_meetId = nullptr; // This is allocated for each net seperately
  // For the design specific variables
  uint64_t* d_costMap = nullptr;
  int* d_xCoords = nullptr;
  int* d_yCoords = nullptr;
  int* d_pinIdxVec = nullptr;
  NodeData2D* d_nodes = nullptr;
  int* d_netHPWL = nullptr;
  int* d_netPtr = nullptr;
  Rect2D_CUDA* d_netBBox = nullptr;


  // Allocate the device memory for the d_dX and d_dY
  hipMalloc(&d_dX, 4 * sizeof(int));
  hipMalloc(&d_dY, 4 * sizeof(int));
  hipMalloc(&d_doneFlag, nets.size() * sizeof(int));
  hipMalloc(&d_meetId, nets.size() * sizeof(int));
  hipMalloc(&d_costMap, numGrids * sizeof(uint64_t));
  hipMalloc(&d_xCoords, h_xCoords.size() * sizeof(int));
  hipMalloc(&d_yCoords, h_yCoords.size() * sizeof(int));
  hipMalloc(&d_pinIdxVec, pinIdxVec.size() * sizeof(int));
  hipMalloc(&d_nodes, numGrids * sizeof(NodeData2D));
  hipMalloc(&d_netHPWL, netHWPL.size() * sizeof(int));
  hipMalloc(&d_netPtr, netPtr.size() * sizeof(int));
  hipMalloc(&d_netBBox, netBBoxVec.size() * sizeof(Rect2D_CUDA));


  hipMemcpy(d_dX, h_dX.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dY, h_dY.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_costMap, h_costMap.data(), numGrids * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_xCoords, h_xCoords.data(), h_xCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_yCoords, h_yCoords.data(), h_yCoords.size() * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(d_pinIdxVec, pinIdxVec.data(), pinIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);
  // Initialize d_nodes from h_nodesPinned.
  hipMemcpy(d_nodes, h_nodesPinned, numGrids * sizeof(NodeData2D), hipMemcpyHostToDevice);
  hipMemcpy(d_netHPWL, netHWPL.data(), netHWPL.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netPtr, netPtr.data(), netPtr.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netBBox, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);

  cudaCheckError();

  // Unfortunately, the cooperative kernel launch is not supported
  // by the dynamic parallelism.
  // We have to lunch the kernel in the host side
  // So we need to use the cudaStream for each net

  // Create a stream per net
  hipStream_t* netStreams = new hipStream_t[numNets];
  for (int i = 0; i < numNets; i++) {
    hipStreamCreate(&netStreams[i]);
  }

  // According to the original code
  unsigned BLOCKCOST = router_cfg->BLOCKCOST * 100;
  unsigned OVERFLOWCOST = 128;
  unsigned HISTCOST = 4;

  // launch one cooperative kernel per net concurrently using different streams
  // the d_netHPWL is used to determine the maximum iterations
  for (int netId = 0; netId < numNets; netId++) {
    auto& netBBox = netBBoxVec[netId];
    launchMazeRouteStream(
      d_netHPWL, d_netPtr, d_netBBox,
      d_pinIdxVec, d_costMap, d_nodes, 
      d_dX, d_dY, d_doneFlag, d_meetId,
      xDim, yDim, 
      d_xCoords,
      d_yCoords,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST,
      HISTCOST,
      netId, 
      (netBBox.xMax - netBBox.xMin + 1) * (netBBox.yMax - netBBox.yMin + 1),
      netStreams[netId]);

    // Determine the bounding box region for this net.
    int xMin   = netBBox.xMin;
    int yMin   = netBBox.yMin;
    int width  = netBBox.xMax - netBBox.xMin + 1;
    int height = netBBox.yMax - netBBox.yMin + 1;

    // Source pointer in device memory and destination pointer in pinned host memory.
    NodeData2D* srcPtr = d_nodes + yMin * xDim + xMin;
    NodeData2D* dstPtr = h_nodesPinned + yMin * xDim + xMin;
    size_t pitch = xDim * sizeof(NodeData2D);

    // Enqueue an asynchronous 2D memory copy in the same stream.
    hipMemcpy2DAsync(dstPtr, pitch, srcPtr, pitch,
      width * sizeof(NodeData2D), height,
      hipMemcpyDeviceToHost,
      netStreams[netId]);
  }

  if (VERBOSE > 0) {
    std::cout << "Finish launchMazeRouteStream" << std::endl;
  }

  cudaCheckError();


  //===========================================================================
  // Launch asynchronous CPU tasks (one per net) to process the restored paths.
  //     Each task waits on its corresponding stream (i.e. for its GPU kernel and async copy
  //     to finish) and then processes the net's bounding-box region.
  std::vector<std::future<void> > cpuFutures;
  cpuFutures.reserve(numNets);
  for (int netId = 0; netId < numNets; netId++) {
    cpuFutures.push_back(std::async(std::launch::async, [&, netId]() {
      // Wait for the GPU work (kernel + async memcopy) in this stream to complete.
      hipStreamSynchronize(netStreams[netId]);

      // Process the CPU-side path sync for this net.
      auto& net    = nets[netId];
      auto& uworker= uworkers[net->getWorkerId()];
      auto& gridGraph = uworker->getGridGraph();
      auto workerLL  = uworker->getRouteGCellIdxLL();
      int workerLX = workerLL.x();
      int workerLY = workerLL.y();
      auto& netBBox = netBBoxVec[netId];
      int LLX = netBBox.xMin;
      int LLY = netBBox.yMin;
      int URX = netBBox.xMax;
      int URY = netBBox.yMax;
      int xDimTemp = URX - LLX + 1;
      int numNodesLocal = xDimTemp * (URY - LLY + 1);
      
      for (int localIdx = 0; localIdx < numNodesLocal; localIdx++) {
        int localX = localIdx % xDimTemp;
        int localY = localIdx / xDimTemp;
        int x = localX + LLX;
        int y = localY + LLY;
        int idx = locToIdx_2D(x, y, xDim);

        int xRel = x - workerLX;
        int yRel = y - workerLY;
        int parentX = h_nodesPinned[idx].golden_parent_x - workerLX;
        int parentY = h_nodesPinned[idx].golden_parent_y - workerLY;
        gridGraph.setGoldenParent2D(xRel, yRel, parentX, parentY);
      }
    }));
  }

  // Wait for all CPU tasks to complete.
  for (auto& f : cpuFutures) {
    f.get();
  }


  /*  
  // hipDeviceSynchronize();
  // Wait for all nets to finish
  for (int i = 0; i < numNets; i++) {
    hipStreamSynchronize(netStreams[i]);
  }
  
  // We need to trace back the routing path on the CPU side
  hipMemcpy(nodes.data(), d_nodes, numGrids * sizeof(NodeData2D), hipMemcpyDeviceToHost);
  cudaCheckError();
  */

  /*
  int LX = netBBoxVec[0].xMin;
  int LY = netBBoxVec[0].yMin;
  int UX = netBBoxVec[0].xMax;
  int UY = netBBoxVec[0].yMax;
  if (LX == 74 && LY == 57 && UX == 83 && UY == 72) {
    for (int id = 0; id < nodes.size(); id++) {
      int2 xy = idxToLoc_2D(id, xDim);
      if (xy.x < LX || xy.x > UX || xy.y < LY || xy.y > UY) {
        continue;
      }
      std::cout << "id = " << id << " "
                << "x = " << xy.x << " y = " << xy.y << " "
                << "isSrc = " << (nodes[id].flags.src_flag == 1) << " "
                << "isDst = " << (nodes[id].flags.dst_flag == 1) << " "
                << "forward_visited_flag = " << (nodes[id].flags.forward_visited_flag == 1) << " "
                << "backward_visited_flag = " << (nodes[id].flags.backward_visited_flag == 1) << " "
                << "forward_g_cost = " << nodes[id].forward_g_cost << " "
                << "backward_g_cost = " << nodes[id].backward_g_cost << " "
                << "golden_parent_x = " << nodes[id].golden_parent_x << " "
                << "golden_parent_y = " << nodes[id].golden_parent_y << " "
                << std::endl;
    }
  }
  */

  if (VERBOSE > 0) {
    std::cout << "Finish the GPU routing" << std::endl;
  }

  // float syncupTime = batchPathSyncUp(uworkers, nets, netBBoxVec, nodes, xDim);
  // Reconstruct the nets similar to the CPU version
  // batchPathSyncUp(uworkers, nets, netBBoxVec, nodes, xDim);

  for (int i = 0; i < numNets; i++) {
    hipStreamDestroy(netStreams[i]);
  }

  delete[] netStreams;

  // Clear the memory
  hipFree(d_dX);
  hipFree(d_dY);
  hipFree(d_doneFlag);
  hipFree(d_meetId);
  hipFree(d_costMap);
  hipFree(d_xCoords);
  hipFree(d_yCoords);
  hipFree(d_pinIdxVec);
  hipFree(d_nodes);
  hipFree(d_netHPWL);
  hipFree(d_netPtr);
  hipFree(d_netBBox);

  hipHostFree(h_nodesPinned);

  cudaCheckError();

  auto totalEnd = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> totalTime = totalEnd - totalStart;
  return totalTime.count();
}

} // namespace drt




