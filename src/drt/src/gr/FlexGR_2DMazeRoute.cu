#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR.h"
#include <omp.h> 
#include <cmath>
#include <fstream>
#include <iostream>
 
#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <string>
#include <stdint.h> // For fixed-width integers
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <map>
#include <queue>
#include <fstream>
#include <iostream>
#include <iterator>
#include <memory>
#include <set>
#include <sstream>
#include <string>
#include <vector>
#include <numeric>
#include <algorithm>

// We always work on the entire grid 
// The grid system is always in terms of global coordinates
// So for each net, we need to translate the local index system into global system

namespace drt {

constexpr int GRGRIDGRAPHHISTCOSTSIZE = 8;
constexpr int GRSUPPLYSIZE = 8;
constexpr int GRDEMANDSIZE = 16;
constexpr int GRFRACSIZE = 1;
  

namespace cg = cooperative_groups;

#define cudaCheckError()                                                   \
{                                                                          \
    hipError_t err = hipGetLastError();                                  \
    if (err != hipSuccess) {                                              \
        fprintf(stderr, "CUDA error at %s:%d: %s\n",                       \
                __FILE__, __LINE__, hipGetErrorString(err));              \
        exit(1);                                                           \
    }                                                                      \
}

// We treat 0xFFFF as "infinite" cost for 32-bit fields
__device__ __host__ __constant__ uint32_t INF32 = 0xFFFFFFFF;


struct Point2D_CUDA {
  int x;
  int y;

  Point2D_CUDA(int x, int y) : x(x), y(y) {}
};

struct Rect2D_CUDA {
  int xMin;
  int yMin;
  int xMax;
  int yMax;

  Rect2D_CUDA(int xMin, int yMin, int xMax, int yMax) : xMin(xMin), yMin(yMin), xMax(xMax), yMax(yMax) {}
};


enum Directions2D {
  DIR_NORTH    = 0,
  DIR_RIGHT = 1,
  DIR_SOUTH  = 2,
  DIR_LEFT  = 3,
  DIR_NONE  = 255
};


struct NodeData2D {
  // forward and backward propagation (heuristic and real cost) (32 bits each)
  uint32_t forward_h_cost; // heuristic cost
  uint32_t forward_g_cost; // real cost
  uint32_t backward_h_cost; // heuristic cost
  uint32_t backward_g_cost; // real cost
  uint32_t forward_h_cost_prev; 
  uint32_t forward_g_cost_prev;
  uint32_t backward_h_cost_prev;
  uint32_t backward_g_cost_prev;
  
  // Store the direction (for turning point cost and path reconstruction)
  uint8_t forward_direction;
  uint8_t backward_direction;
  uint8_t forward_direction_prev;
  uint8_t backward_direction_prev;
  int golden_parent_x;
  int golden_parent_y;

  // Flags (1 bit each, packed into a single 8-bit field)
  struct Flags {
    uint8_t src_flag : 1; // 1 if this node is the source
    uint8_t dst_flag : 1; // 1 if this node is the destination
    uint8_t forward_update_flag: 1; // 1 if the forward cost is updated
    uint8_t backward_update_flag: 1; // 1 if the backward cost is updated
    uint8_t forward_visited_flag: 1; // 1 if the forward node is visited
    uint8_t backward_visited_flag: 1; // 1 if the backward node is visited
    uint8_t forward_visited_flag_prev: 1; // 1 if the forward node is visited
    uint8_t backward_visited_flag_prev: 1; // 1 if the backward node is visited
  }  flags;
};



__host__ __device__ 
void initNodeData2D(NodeData2D& nd) {
  nd.forward_h_cost = INF32;
  nd.forward_g_cost = INF32;
  nd.backward_h_cost = INF32;
  nd.backward_g_cost = INF32;
  nd.forward_h_cost_prev = INF32;
  nd.forward_g_cost_prev = INF32;
  nd.backward_h_cost_prev = INF32;
  nd.backward_g_cost_prev = INF32;
  nd.forward_direction = DIR_NONE;
  nd.backward_direction = DIR_NONE;
  nd.forward_direction_prev = DIR_NONE;
  nd.backward_direction_prev = DIR_NONE;
  nd.golden_parent_x = -1;
  nd.golden_parent_y = -1;
  nd.flags.src_flag = 0;
  nd.flags.dst_flag = 0;
  nd.flags.forward_update_flag = 0;
  nd.flags.backward_update_flag = 0;
  nd.flags.forward_visited_flag = 0;
  nd.flags.backward_visited_flag = 0;
  nd.flags.forward_visited_flag_prev = 0;
  nd.flags.backward_visited_flag_prev = 0;
}



__device__ __forceinline__ 
uint8_t computeParentDirection2D(int d) {
  switch(d) {
    case 0: return DIR_NORTH;
    case 1: return DIR_RIGHT;
    case 2: return DIR_SOUTH;
    case 3: return DIR_LEFT;
    default: return DIR_NONE;
  }
}


// Invert direction for backtracking
__device__ __forceinline__ 
uint8_t invertDirection2D(uint8_t d) {
  switch(d) {
    case DIR_NORTH:    return DIR_SOUTH;
    case DIR_SOUTH:    return DIR_NORTH;
    case DIR_LEFT:     return DIR_RIGHT;
    case DIR_RIGHT:    return DIR_LEFT;
    default:           return DIR_NONE;
  }
}


// Define the idxToLoc_2D function
// Convert linear index -> (x,y)
__device__ __host__ __forceinline__ 
int2 idxToLoc_2D(int idx, int xDim) {
  int x = idx % xDim;
  int y = idx / xDim;
  return make_int2(x,y);
}


// Define the locToIdx_2D function
// Convert (x,y) -> linear index
__device__ __host__ __forceinline__ 
int locToIdx_2D(int x, int y, int xDim) {
  return y * xDim + x;
}


// Bit related functions
__host__ __device__ __forceinline__
bool getBit(const uint64_t* cmap, unsigned idx, unsigned pos)
{
  return (cmap[idx] >> pos) & 1;
}
 
__host__ __device__ __forceinline__
unsigned getBits(const uint64_t* cmap, unsigned idx, unsigned pos, unsigned length)
{
  auto tmp = cmap[idx] & (((1ull << length) - 1) << pos);
  return tmp >> pos;
}
 

__host__ __device__ __forceinline__
unsigned getHistoryCost(const uint64_t* cmap, int idx)
{
  return getBits(cmap, idx, 8, GRGRIDGRAPHHISTCOSTSIZE);
}


__host__ __device__ __forceinline__
float getCongCost(unsigned supply, unsigned demand)
{
  float exp_val = exp(std::min(10.0f, static_cast<float>(supply) - demand));  
  float factor = 4.0f / (1.0f + exp_val); 
  float congCost = demand * (1.0f + factor) / (supply + 1.0f);
  return congCost;  
}



__host__ __device__
unsigned getRawDemand2D(const uint64_t* cmap, int idx, Directions2D dir)
{
  unsigned demand = 0;
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      demand = getBits(cmap, idx, 48, CMAPDEMANDSIZE);
      break;
    case Directions2D::DIR_NORTH:
      demand = getBits(cmap, idx, 32, CMAPDEMANDSIZE);
      break;
    default:;
  }
  return demand;
}


__host__ __device__
unsigned getRawSupply2D(const uint64_t* cmap, int idx, Directions2D dir)
{
  unsigned supply = 0;
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      supply = getBits(cmap, idx, 24, CMAPSUPPLYSIZE);
      break;
    case Directions2D::DIR_NORTH:
      supply = getBits(cmap, idx, 16, CMAPSUPPLYSIZE);
      break;
    default:;
  }
  return supply << CMAPFRACSIZE;
}


__host__ __device__
bool hasBlock2D(const uint64_t* cmap, int idx, Directions2D dir)
{
  bool sol = false;
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      sol = getBit(cmap, idx, 3);
      break;
    case Directions2D::DIR_NORTH:
      sol = getBit(cmap, idx, 2);
      break;
    default:;
  }
  return sol;
}


__host__ __device__
bool hasEdge2D(const uint64_t* bits, int idx, Directions2D dir)
{
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      return getBit(bits, idx, 0);
    case Directions2D::DIR_NORTH:
      return getBit(bits, idx, 1);
    default:;
  }  
}


__host__ __device__
unsigned getEdgeLength2D(const int* xCoords, const int* yCoords, 
  int x, int y, Directions2D dir)
{
  switch (dir) {
    case Directions2D::DIR_RIGHT:
      return xCoords[x + 1] - xCoords[x];
    case Directions2D::DIR_NORTH:
      return yCoords[y + 1] - yCoords[y];
    default:
      return 0;
  }
}



// We do not consider bending cost in this version
__host__ __device__
uint32_t getEdgeCost2D(
  const uint64_t* d_costMap,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST,
  int idx, int x, int y,
  Directions2D dir)
{
  bool blockCost = hasBlock2D(d_costMap, idx, dir);
  unsigned histCost = getHistoryCost(d_costMap, idx);
  unsigned rawDemand = getRawDemand2D(d_costMap, idx, dir) * congThreshold;
  unsigned rawSupply = getRawSupply2D(d_costMap, idx, dir);
  bool overflowCost = (rawDemand >= rawSupply);
  float congCost = getCongCost(rawSupply, rawDemand);
  unsigned edgeLength = getEdgeLength2D(d_xCoords, d_yCoords, x, y, dir);

  // cost 
  uint32_t edgeCost = edgeLength
    + edgeLength * congCost
    + (histCost ? HISTCOST * edgeLength * congCost * HISTCOST : 0)
    + (blockCost ? BLOCKCOST * edgeLength : 0)
    + (overflowCost ? OVERFLOWCOST * edgeLength : 0);

  return edgeCost;
}


// Define the device function for node initialization

__device__
void initNodeData2D__device(
  NodeData2D* d_nodes,
  int* d_pins, int pinIterStart, int pinIter,  // Pin related variables
  int LLX, int LLY, int URX, int URY, // Bounding box
  int xDim)
{ 
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
  int xDimTemp = URX - LLX + 1;
  for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    int2 xy = idxToLoc_2D(idx, xDim);
    int2 src = idxToLoc_2D(d_pins[pinIterStart + pinIter - 1], xDim);
    int2 dst = idxToLoc_2D(d_pins[pinIterStart + pinIter], xDim);
  
    // The experimental results show that the heuristic cost is not needed
    d_nodes[idx].forward_h_cost = 0;
    d_nodes[idx].backward_h_cost = 0;

    if (d_nodes[idx].flags.src_flag) {
      d_nodes[idx].forward_g_cost = 0;
      d_nodes[idx].forward_g_cost_prev = 0;
      d_nodes[idx].flags.forward_visited_flag = true;
      d_nodes[idx].flags.forward_visited_flag_prev = true;
    } else {
      d_nodes[idx].forward_g_cost = INF32;
      d_nodes[idx].forward_g_cost_prev = INF32;
      d_nodes[idx].flags.forward_visited_flag = false;
      d_nodes[idx].flags.forward_visited_flag_prev = false;
    }

    if (d_nodes[idx].flags.dst_flag) {
      d_nodes[idx].backward_g_cost = 0;
      d_nodes[idx].backward_g_cost_prev = 0;
      d_nodes[idx].flags.backward_visited_flag = true;
      d_nodes[idx].flags.backward_visited_flag_prev = true;
    } else {
      d_nodes[idx].backward_g_cost = INF32;
      d_nodes[idx].backward_g_cost_prev = INF32;
      d_nodes[idx].flags.backward_visited_flag = false;
      d_nodes[idx].flags.backward_visited_flag_prev = false;
    }

    d_nodes[idx].forward_direction = DIR_NONE;
    d_nodes[idx].backward_direction = DIR_NONE;
    d_nodes[idx].forward_direction_prev = DIR_NONE;
    d_nodes[idx].backward_direction_prev = DIR_NONE;
    d_nodes[idx].flags.forward_update_flag = false;
    d_nodes[idx].flags.backward_update_flag = false;
    d_nodes[idx].flags.forward_visited_flag = false;
    d_nodes[idx].flags.backward_visited_flag = false;
    d_nodes[idx].flags.forward_visited_flag_prev = false;
    d_nodes[idx].flags.backward_visited_flag_prev = false;
  }  
}


__device__ __forceinline__
uint32_t getNeighorCost2D(
  const uint64_t* d_costMap,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST,
  int idx, int x, int y,
  int nbrIdx, int nx, int ny)
{
  uint32_t newG = 0;
  if (nx == x && ny == y - 1) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      nbrIdx, nx, ny, Directions2D::DIR_NORTH);
  } else if (nx == x && ny == y + 1) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      idx, x, y, Directions2D::DIR_NORTH);
  } else if (nx == x - 1 && ny == y) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      nbrIdx, nx, ny, Directions2D::DIR_RIGHT);
  } else if (nx == x + 1 && ny == y) {
    newG += getEdgeCost2D(d_costMap, d_xCoords, d_yCoords, 
      congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
      idx, x, y, Directions2D::DIR_RIGHT);
  }

  return newG;
}


// Define the device function for the biwaveBellmanFord2D
__device__
void runBiBellmanFord_2D__device(
  cooperative_groups::grid_group& g,   // grid-level cooperative group
  NodeData2D* nodes,
  uint64_t* d_costMap, 
  int* d_dX, int* d_dY,
  int& d_doneFlag,
  int LLX, int LLY, int URX, int URY,
  int xDim, int maxIters,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  unsigned BLOCKCOST,
  unsigned OVERFLOWCOST,
  unsigned HISTCOST)
{
  // A typical 1D decomposition over the entire 2D domain
  int total = (URX - LLX + 1) * (URY - LLY + 1);
  int tid     = blockDim.x * blockIdx.x + threadIdx.x;
  int stride  = blockDim.x * gridDim.x;
  int xDimTemp = URX - LLX + 1;

  // We’ll do up to maxIters or until no changes / front-meet
  for (int iter = 0; iter < maxIters; iter++)
  {
    bool localFrontsMeet = false;
    ////////////////////////////////////////////////////////////////////////////
    // (1) Forward & backward relaxation phase
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = nodes[idx];
      int2 xy = idxToLoc_2D(idx, xDim);
      int  x  = xy.x;
      int  y  = xy.y;

      // Forward relaxation
      // Typically: newCost = min over neighbors of (neighborCost + edgeWeight).
      // Be sure to skip if src_flag is set (source node may be pinned).
      if (!nd.flags.src_flag) {
        uint32_t bestCost = nd.forward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + d_dX[d];
          int ny = y + d_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          // We want neighbor's cost plus the edge weight, e.g. 100
          uint32_t neighborCost = nodes[nbrIdx].forward_g_cost_prev;
          // If neighbor is effectively infinite, skip
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          
          uint32_t newG = neighborCost +
            getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
              congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
              idx, x, y, nbrIdx, nx, ny);

          // Check if we found a better cost
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) { // We found an improvement
          nd.forward_g_cost = bestCost;
          nd.forward_direction = computeParentDirection2D(bestD);
          nd.flags.forward_update_flag = true;
        }
      } // end forward

      // Backward relaxation
      // Typically: newCost = min over neighbors of (neighbor.backward_cost + edgeWeight)
      // Skip if dst_flag is set (destination node may be pinned).
      if (!nd.flags.dst_flag) {
        uint32_t bestCost = nd.backward_g_cost_prev;
        int      bestD    = -1;
        for (int d = 0; d < 4; d++) {
          int nx = x + d_dX[d];
          int ny = y + d_dY[d];
          if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
            continue;  // out of bounds
          }
          
          int nbrIdx = locToIdx_2D(nx, ny, xDim);
          uint32_t neighborCost = nodes[nbrIdx].backward_g_cost_prev;
          if (neighborCost == 0xFFFFFFFF) {
            continue;
          }
          
          uint32_t newG = neighborCost +
          getNeighorCost2D(d_costMap, d_xCoords, d_yCoords,
            congThreshold, BLOCKCOST, OVERFLOWCOST, HISTCOST,
            idx, x, y, nbrIdx, nx, ny);
          
          if (newG < bestCost) {
            bestCost = newG;
            bestD    = d;
          }
        } // end neighbor loop

        if (bestD != -1) {
          nd.backward_g_cost = bestCost;
          nd.backward_direction = computeParentDirection2D(bestD);
          nd.flags.backward_update_flag = true;
        }
      } // end backward
    } // end “for each node” (forward + backward)

    g.sync();

    ////////////////////////////////////////////////////////////////////////////
    // (2) Commit updated costs (double-buffering technique)
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = nodes[idx];
      // If forward_update_flag is set, copy forward_g_cost -> forward_g_cost_prev
      if (nd.flags.forward_update_flag) {
        nd.flags.forward_update_flag = false;
        nd.forward_g_cost_prev = nd.forward_g_cost;
      }
      
      // If backward_update_flag is set, copy backward_g_cost -> backward_g_cost_prev
      if (nd.flags.backward_update_flag) {
        nd.flags.backward_update_flag = false;
        nd.backward_g_cost_prev = nd.backward_g_cost;
      }

      nd.flags.forward_visited_flag_prev = nd.flags.forward_visited_flag;
      nd.flags.backward_visited_flag_prev = nd.flags.backward_visited_flag;
    }

    // Another full grid sync before the “stop” checks:
    g.sync();


    // Needs to be updated
    ////////////////////////////////////////////////////////////////////////////
    // (3) Check if forward and backward fronts meet
    ////////////////////////////////////////////////////////////////////////////
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = nodes[idx];
      // If either side is "unreached," skip
      if (nd.forward_g_cost_prev == 0xFFFFFFFF || 
          nd.backward_g_cost_prev == 0xFFFFFFFF)
      {
        continue;
      }

      // Check the visited flag
      bool localForwardMin = true;
      bool localBackwardMin = true;

      int2 xy = idxToLoc_2D(idx, xDim);
      int  x  = xy.x;
      int  y  = xy.y;

      for (int d = 0; d < 4; d++) {
        int nx = x + d_dX[d];
        int ny = y + d_dY[d];
        if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
          continue;  // out of bounds
        }
        
        int nbrIdx = locToIdx_2D(nx, ny, xDim);
        // check forward case
        if ((nodes[nbrIdx].flags.forward_visited_flag_prev == false) && 
            (nodes[nbrIdx].forward_g_cost_prev + nodes[nbrIdx].forward_h_cost < nd.forward_g_cost_prev + nd.forward_h_cost_prev)) {
          localForwardMin = false;
        } 

        if ((nodes[nbrIdx].flags.backward_visited_flag_prev == false) &&
            (nodes[nbrIdx].backward_g_cost_prev + nodes[nbrIdx].backward_h_cost >= nd.backward_g_cost_prev + nd.backward_h_cost_prev)) {
          localBackwardMin = false;
        }      
      }

      if (localForwardMin == true) {
        nd.flags.forward_visited_flag = true;
      }

      if (localBackwardMin == true) {
        nd.flags.backward_visited_flag = true;
      }
    } // end “for each node”
    
    g.sync();

    // Check if any thread found a front-meet
    for (int localIdx = tid; localIdx < total; localIdx += stride) {
      int local_x = localIdx % xDimTemp + LLX;
      int local_y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(local_x, local_y, xDim);
      NodeData2D &nd = nodes[idx];
      if (nd.flags.forward_visited_flag && nd.flags.backward_visited_flag) {
        localFrontsMeet = true;
      }
    }
    
    g.sync();

    bool localDone = localFrontsMeet;
    if (localDone) {
      atomicExch(&d_doneFlag, 1);
    }
    
    g.sync();

    if (d_doneFlag == 1) {
      d_doneFlag = INF32;
      return;
    }
  } // end for (iter)
}


// Define the device function for the meetId check
__device__
void findMeetIdAndTraceBackCost2D__device(
  NodeData2D* nodes,
  int& d_doneFlag, 
  int LLX, int LLY, int URX, int URY,
  int xDim)
{ 
  int xDimTemp = URX - LLX + 1;
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
  for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    if (nodes[idx].flags.forward_visited_flag && nodes[idx].flags.backward_visited_flag) {
      int32_t cost = nodes[idx].forward_g_cost + nodes[idx].backward_g_cost;
      atomicMin(&d_doneFlag, cost);      
    }
  }
}

__device__
void findMeetIdAndTraceBackId2D__device(
  NodeData2D* nodes,
  int& d_doneFlag, 
  int& d_meetId,
  int LLX, int LLY, int URX, int URY,
  int xDim)
{ 
  int xDimTemp = URX - LLX + 1;
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
  for (int localIdx = threadIdx.x; localIdx < numNodes; localIdx += blockDim.x) {
    int local_x = localIdx % xDimTemp + LLX;
    int local_y = localIdx / xDimTemp + LLY;
    int idx = locToIdx_2D(local_x, local_y, xDim);
    if (nodes[idx].flags.forward_visited_flag && nodes[idx].flags.backward_visited_flag && 
        (nodes[idx].forward_g_cost + nodes[idx].backward_g_cost == d_doneFlag)) {
      atomicMin(&d_meetId, idx);      
    }
  }
}

__device__
void forwardTraceBack2D__single_thread__device(
  NodeData2D* nodes, 
  int& d_meetId, 
  int* d_dX, int* d_dY,
  int LLX, int LLY, int URX, int URY,
  int xDim)
{
  if (d_meetId == 0x7FFFFFFF) {
    return; // No meetId found
  }
  
  int curId = d_meetId;
  int maxIterations = (URX - LLX + 1) * (URY - LLY + 1);
  int iteration = 0;
  while (nodes[curId].flags.src_flag == 0 && iteration < maxIterations) {
    uint8_t forwardDirection = nodes[curId].forward_direction;
    nodes[curId].flags.src_flag = 1;
    int2 xy = idxToLoc_2D(curId, xDim);
    int nx = xy.x + d_dX[forwardDirection];
    int ny = xy.y + d_dY[forwardDirection];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
      break;
    }  
    
    nodes[curId].golden_parent_x = nx;
    nodes[curId].golden_parent_y = ny;
    curId = locToIdx_2D(nx, ny, xDim);
    iteration++;
  }

  if (iteration >= maxIterations) {
    printf("Warning: Forward traceback exceeded maximum iterations.\n");
  }
}

__device__
void backwardTraceBack2D__single__thread__device(
  NodeData2D* nodes, 
  int& d_meetId, 
  int* d_dX, int* d_dY,
  int LLX, int LLY, int URX, int URY,
  int xDim)
{  
  if (d_meetId == 0x7FFFFFFF) {
    return; // No meetId found
  }
  
  int curId = d_meetId;
  if (nodes[curId].flags.dst_flag == 1) { 
    nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
    nodes[curId].flags.src_flag = 1;
    return;
  }
  
  int maxIterations = (URX - LLX + 1) * (URY - LLY + 1);
  int iteration = 0;

  while (iteration < maxIterations) {
    int2 xy = idxToLoc_2D(curId, xDim);
    uint8_t backwardDirection = nodes[curId].backward_direction;
    int nx = xy.x + d_dX[backwardDirection];
    int ny = xy.y + d_dY[backwardDirection];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
      break;
    }  
    
    curId = locToIdx_2D(nx, ny, xDim);
    nodes[curId].flags.src_flag = 1;
    nodes[curId].golden_parent_x = xy.x;
    nodes[curId].golden_parent_y = xy.y;
    if (nodes[curId].flags.dst_flag == 1) {
      nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
      break;
    }
    iteration++;
  }

  if (iteration >= maxIterations) {
    printf("Warning: Backward traceback exceeded maximum iterations.\n");
  }
}



// Fused cooperative kernel that processes a single net.
__global__ 
void biwaveBellmanFord2D__kernel(
  int netId,
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBoxVec,
  int* d_pins,
  uint64_t* d_costMap,
  NodeData2D* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlags,
  int* d_meetIds,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int OVERFLOWCOST, 
  int HISTCOST)
{
  // Obtain a handle to the entire cooperative grid.
  cg::grid_group grid = cg::this_grid();

  // for this net
  int pinIdxStart = d_netPtr[netId];
  int pinIdxEnd = d_netPtr[netId + 1];
  int numPins = pinIdxEnd - pinIdxStart;
  int maxIters = d_netHPWL[netId];
  Rect2D_CUDA netBBox = d_netBBoxVec[netId];
  int LLX = netBBox.xMin;
  int LLY = netBBox.yMin;
  int URX = netBBox.xMax;
  int URY = netBBox.yMax;

  int& d_doneFlag = d_doneFlags[netId];
  int& d_meetId = d_meetIds[netId];

  // Connect the pin one by one
  for (int pinIter = 1; pinIter < numPins; pinIter++) {
    // Initilization
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
      d_doneFlag = 0;
      d_meetId = 0x7FFFFFFF;
    }

    initNodeData2D__device(
      d_nodes,
      d_pins, pinIdxStart, pinIter, 
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    // Run the Bellman Ford algorithm
    runBiBellmanFord_2D__device(
      grid, d_nodes, d_costMap, d_dX, d_dY, 
      d_doneFlag, LLX, LLY, URX, URY, xDim, maxIters,
      d_xCoords, d_yCoords, congThreshold,
      BLOCKCOST, OVERFLOWCOST, HISTCOST);  

    grid.sync();

    // Find the d_meetId
    findMeetIdAndTraceBackCost2D__device(
      d_nodes, d_doneFlag, 
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    findMeetIdAndTraceBackId2D__device(
      d_nodes, d_doneFlag, d_meetId,
      LLX, LLY, URX, URY, 
      xDim);

    grid.sync(); // Synchronize all threads in the grid

    // Traceback
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
      // trace back
      forwardTraceBack2D__single_thread__device(
        d_nodes, d_meetId, d_dX, d_dY, 
        LLX, LLY, URX, URY, xDim);
      
      backwardTraceBack2D__single__thread__device(
        d_nodes, d_meetId, d_dX, d_dY, 
        LLX, LLY, URX, URY, xDim);
    }

    grid.sync(); // Synchronize all threads in the grid
  }
}


// Just a wrapper function to call the kernel
void launchMazeRouteStream(
  int* d_netHPWL,
  int* d_netPtr,
  Rect2D_CUDA* d_netBBox,
  int* d_pins,
  uint64_t* d_costMap,
  int* d_nodes,
  int* d_dX,
  int* d_dY,
  int* d_doneFlag,
  int* d_meetId,
  int xDim, int yDim,
  const int* d_xCoords,
  const int* d_yCoords,
  float congThreshold,
  int BLOCKCOST,
  int CONGCOST,
  int HISTCOST,
  int netId,
  int totalThreads,
  hipStream_t stream)
{
  void* kernelArgs[] = {
    &netId,
    &d_netHPWL,
    &d_netPtr,
    &d_netBBox,
    &d_pins,
    &d_costMap,
    &d_nodes,
    &d_dX,
    &d_dY,
    &d_doneFlag,
    &d_meetId,
    &xDim, 
    &yDim,
    &d_xCoords,
    &d_yCoords,
    &congThreshold,
    &BLOCKCOST, 
    &CONGCOST, 
    &HISTCOST
  };

  // Calculate the maximum number of blocks that can run cooperatively
  int deviceId = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceId);

  int threadsPerBlock = 1024;
  int numBlocksPerSm = 0;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, biwaveBellmanFord2D__kernel, threadsPerBlock, 0);
  int numSms = deviceProp.multiProcessorCount;
  int numBlocks = numBlocksPerSm * numSms;

  // Ensure the grid size does not exceed the maximum allowed for cooperative launch
  int maxBlocksPerGrid = 0;
  hipDeviceGetAttribute(&maxBlocksPerGrid, hipDeviceAttributeMaxGridDimX, deviceId);
  numBlocks = min(numBlocks, maxBlocksPerGrid);
  numBlocks = min(numBlocks, (totalThreads + threadsPerBlock - 1) / threadsPerBlock);
  numBlocks = max(numBlocks, 1);

  printf("Launching kernel with %d blocks\n", numBlocks);

  hipError_t err = hipLaunchCooperativeKernel(
    (void*)biwaveBellmanFord2D__kernel,
    numBlocks, threadsPerBlock,
    kernelArgs,
    0,       // additional dynamic shared memory (if needed)
    stream); // launch on the given stream

  if (err != hipSuccess) {
    printf("Kernel launch error (net %d): %s\n", netId, hipGetErrorString(err));
  }
}


// We need to restore the connected path from the golden parent
void batchPathSyncUp(
  std::vector<std::unique_ptr<FlexGRWorker>>& uworkers,
  std::vector<grNet*>& nets,
  std::vector<Rect2D_CUDA>& netBBoxVec,
  std::vector<NodeData2D>& nodes,
  int xDim)
{
  for (int netId = 0; netId < nets.size(); netId++) {
    auto& net = nets[netId];
    auto& uworker = uworkers[netId];
    auto& gridGraph = uworker->getGridGraph();
    auto& netBBox = netBBoxVec[netId];
    int LLX = netBBox.xMin;
    int LLY = netBBox.yMin;
    int URX = netBBox.xMax;
    int URY = netBBox.yMax;
    int xDimTemp = URX - LLX + 1;
    int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
    for (int localIdx = 0; localIdx < numNodes; localIdx++) {
      int x = localIdx % xDimTemp + LLX;
      int y = localIdx / xDimTemp + LLY;
      int idx = locToIdx_2D(x, y, xDim);
      gridGraph.setGoldenParent2D(x, y, nodes[idx].golden_parent_x, nodes[idx].golden_parent_y);
    }
  }
}

void FlexGR::GPUAccelerated2DMazeRoute(
  std::vector<std::unique_ptr<FlexGRWorker>>& uworkers,
  std::vector<grNet*>& nets,
  std::vector<uint64_t>& h_costMap,
  std::vector<int>& h_xCoords,
  std::vector<int>& h_yCoords,
  RouterConfiguration* router_cfg,
  float congThreshold,
  int xDim, int yDim)
{
  std::cout << "[INFO] GPU accelerated 2D Maze Routing" << std::endl;
  std::cout << "[INFO] Number of nets: " << nets.size() << std::endl;
  
  int numGrids = xDim * yDim;
  int numNets = nets.size();
  
  std::vector<Point2D_CUDA> netVec;
  std::vector<int> netPtr; 
  std::vector<int> netHWPL;
  std::vector<Rect2D_CUDA> netBBoxVec;
  std::vector<int> pinIdxVec;
  std::vector<NodeData2D> nodes;
  for (auto& node : nodes) {
    initNodeData2D(node);
  }

  netPtr.push_back(0);
  for (auto& net : nets) {
    for (auto& idx : net->getPinGCellAbsIdxs()) {
      netVec.push_back(Point2D_CUDA(idx.x(), idx.y()));
      pinIdxVec.push_back(locToIdx_2D(idx.x(), idx.y(), xDim));
    }
    netPtr.push_back(netVec.size());
    auto netBBox = net->getRouteAbsBBox();
    netBBoxVec.push_back(
      Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
    netHWPL.push_back(net->getHPWL());
  }

  // Perform the initialization
  for (int netId = 0; netId < numNets; netId++) {
    // Mark the first pin of the net as src
    // and the remaining pins as dst
    int pinIdxStart = netPtr[netId];
    int pinIdxEnd = netPtr[netId + 1];
    nodes[pinIdxVec[pinIdxStart]].flags.src_flag = 1;    
    for (int idx = pinIdxStart + 1; idx < pinIdxEnd; idx++) {
      nodes[pinIdxVec[idx]].flags.dst_flag = 1;
    }
  }

  // We need to define the needed utility variables
  std::vector<int> h_dX = {0, 1, 0, -1};
  std::vector<int> h_dY = {1, 0, -1, 0};
  
  int* d_dX = nullptr;
  int* d_dY = nullptr;
  int* d_doneFlag = nullptr; // This is allocated for each net seperately
  int* d_meetId = nullptr; // This is allocated for each net seperately
  
  // For the design specific variables
  uint64_t* d_costMap = nullptr;
  int* d_xCoords = nullptr;
  int* d_yCoords = nullptr;
  int* d_pinIdxVec = nullptr;
  int* d_nodes = nullptr;
  int* d_netHPWL = nullptr;
  int* d_netPtr = nullptr;
  Rect2D_CUDA* d_netBBox = nullptr;


  // Allocate the device memory for the d_dX and d_dY
  hipMalloc(&d_dX, 4 * sizeof(int));
  hipMalloc(&d_dY, 4 * sizeof(int));
  hipMalloc(&d_doneFlag, nets.size() * sizeof(int));
  hipMalloc(&d_meetId, nets.size() * sizeof(int));
  hipMalloc(&d_costMap, numGrids * sizeof(uint64_t));
  hipMalloc(&d_xCoords, h_xCoords.size() * sizeof(int));
  hipMalloc(&d_yCoords, h_yCoords.size() * sizeof(int));
  hipMalloc(&d_pinIdxVec, pinIdxVec.size() * sizeof(int));

  hipMemcpy(d_dX, h_dX.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dY, h_dY.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_costMap, h_costMap.data(), numGrids * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_xCoords, h_xCoords.data(), h_xCoords.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_yCoords, h_yCoords.data(), h_yCoords.size() * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(d_pinIdxVec, pinIdxVec.data(), pinIdxVec.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nodes, nodes.data(), numGrids * sizeof(NodeData2D), hipMemcpyHostToDevice);
  hipMemcpy(d_netHPWL, netHWPL.data(), netHWPL.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netPtr, netPtr.data(), netPtr.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_netBBox, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);

  cudaCheckError();

  // Unfortunately, the cooperative kernel launch is not supported
  // by the dynamic parallelism.
  // We have to lunch the kernel in the host side
  // So we need to use the cudaStream for each net

  // Create a stream per net
  hipStream_t* netStreams = new hipStream_t[numNets];
  for (int i = 0; i < numNets; i++) {
    hipStreamCreate(&netStreams[i]);
  }

  // According to the original code
  unsigned BLOCKCOST = router_cfg->BLOCKCOST * 100;
  unsigned OVERFLOWCOST = 128;
  unsigned HISTCOST = 4;

  // launch one cooperative kernel per net concurrently using different streams
  // the d_netHPWL is used to determine the maximum iterations
  for (int netId = 0; netId < numNets; netId++) {
    auto& netBBox = netBBoxVec[netId];
    launchMazeRouteStream(
      d_netHPWL, d_netPtr, d_netBBox,
      d_pinIdxVec, d_costMap, d_nodes, 
      d_dX, d_dY, d_doneFlag, d_meetId,
      xDim, yDim, 
      d_xCoords,
      d_yCoords,
      congThreshold,
      BLOCKCOST,
      OVERFLOWCOST,
      HISTCOST,
      netId, 
      (netBBox.xMax - netBBox.xMin + 1) * (netBBox.yMax - netBBox.yMin + 1),
      netStreams[netId]);
  }

  // hipDeviceSynchronize();
  // Wait for all nets to finish
  for (int i = 0; i < numNets; i++) {
    hipStreamSynchronize(netStreams[i]);
  }
  
  // We need to trace back the routing path on the CPU side
  hipMemcpy(nodes.data(), d_nodes, numGrids * sizeof(NodeData2D), hipMemcpyDeviceToHost);
  cudaCheckError();

  // Reconstruct the nets similar to the CPU version
  batchPathSyncUp(uworkers, nets, netBBoxVec, nodes, xDim);

  for (int i = 0; i < numNets; i++) {
    hipStreamDestroy(netStreams[i]);
  }

  delete[] netStreams;

  // Clear the memory
  hipFree(d_dX);
  hipFree(d_dY);
  hipFree(d_doneFlag);
  hipFree(d_meetId);
  hipFree(d_costMap);
  hipFree(d_xCoords);
  hipFree(d_yCoords);
  hipFree(d_pinIdxVec);
  hipFree(d_nodes);
  hipFree(d_netHPWL);
  hipFree(d_netPtr);
  hipFree(d_netBBox);
}

} // namespace drt




