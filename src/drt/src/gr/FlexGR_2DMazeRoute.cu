#include "hip/hip_runtime.h"
/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR.h"
#include <omp.h> 
#include <cmath>
#include <fstream>
#include <iostream>
 
#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <string>
#include <stdint.h> // For fixed-width integers
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <map>
#include <queue>
#include <fstream>
#include <iostream>
#include <iterator>
#include <memory>
#include <set>
#include <sstream>
#include <string>
#include <vector>
#include <numeric>
#include <algorithm>

// We always work on the entire grid 
// The grid system is always in terms of global coordinates
// So for each net, we need to translate the local index system into global system

namespace drt {

namespace cg = cooperative_groups;

#define cudaCheckError()                                                   \
{                                                                          \
    hipError_t err = hipGetLastError();                                  \
    if (err != hipSuccess) {                                              \
        fprintf(stderr, "CUDA error at %s:%d: %s\n",                       \
                __FILE__, __LINE__, hipGetErrorString(err));              \
        exit(1);                                                           \
    }                                                                      \
}

// We treat 0xFFFF as "infinite" cost for 32-bit fields
__device__ __host__ __constant__ uint32_t INF32 = 0xFFFFFFFF;


struct Point2D_CUDA {
  int x;
  int y;

  Point2D_CUDA(int x, int y) : x(x), y(y) {}
};

struct Rect2D_CUDA {
  int xMin;
  int yMin;
  int xMax;
  int yMax;

  Rect2D_CUDA(int xMin, int yMin, int xMax, int yMax) : xMin(xMin), yMin(yMin), xMax(xMax), yMax(yMax) {}
};


enum Directions2D {
  DIR_NORTH    = 0,
  DIR_RIGHT = 1,
  DIR_SOUTH  = 2,
  DIR_LEFT  = 3,
  DIR_NONE  = 255
};


struct NodeData2D {
  // forward and backward propagation (heuristic and real cost) (32 bits each)
  uint32_t forward_h_cost; // heuristic cost
  uint32_t forward_g_cost; // real cost
  uint32_t backward_h_cost; // heuristic cost
  uint32_t backward_g_cost; // real cost
  uint32_t forward_h_cost_prev; 
  uint32_t forward_g_cost_prev;
  uint32_t backward_h_cost_prev;
  uint32_t backward_g_cost_prev;
  
  // Store the direction (for turning point cost and path reconstruction)
  uint8_t forward_direction;
  uint8_t backward_direction;
  uint8_t forward_direction_prev;
  uint8_t backward_direction_prev;
  int golden_parent_x;
  int golden_parent_y;

  // Flags (1 bit each, packed into a single 8-bit field)
  struct Flags {
    uint8_t src_flag : 1; // 1 if this node is the source
    uint8_t dst_flag : 1; // 1 if this node is the destination
    uint8_t forward_update_flag: 1; // 1 if the forward cost is updated
    uint8_t backward_update_flag: 1; // 1 if the backward cost is updated
    uint8_t forward_visited_flag: 1; // 1 if the forward node is visited
    uint8_t backward_visited_flag: 1; // 1 if the backward node is visited
    uint8_t not_used: 2; // 2 bits not used
  }  flags;
};


__host__ __device__ 
void initNodeData2D(NodeData2D& nd) {
  nd.forward_h_cost = INF32;
  nd.forward_g_cost = INF32;
  nd.backward_h_cost = INF32;
  nd.backward_g_cost = INF32;
  nd.forward_h_cost_prev = INF32;
  nd.forward_g_cost_prev = INF32;
  nd.backward_h_cost_prev = INF32;
  nd.backward_g_cost_prev = INF32;
  nd.forward_direction = DIR_NONE;
  nd.backward_direction = DIR_NONE;
  nd.forward_direction_prev = DIR_NONE;
  nd.backward_direction_prev = DIR_NONE;
  nd.golden_parent_x = -1;
  nd.golden_parent_y = -1;
  nd.flags.src_flag = 0;
  nd.flags.dst_flag = 0;
  nd.flags.forward_update_flag = 0;
  nd.flags.backward_update_flag = 0;
}


__device__ __forceinline__ 
uint8_t computeParentDirection2D(int d) {
  switch(d) {
    case 0: return DIR_NORTH;
    case 1: return DIR_RIGHT;
    case 2: return DIR_SOUTH;
    case 3: return DIR_LEFT;
    default: return DIR_NONE;
  }
}


// Invert direction for backtracking
__device__ __forceinline__ 
uint8_t invertDirection2D(uint8_t d) {
  switch(d) {
    case DIR_NORTH:    return DIR_SOUTH;
    case DIR_SOUTH:    return DIR_NORTH;
    case DIR_LEFT:     return DIR_RIGHT;
    case DIR_RIGHT:    return DIR_LEFT;
    default:           return DIR_NONE;
  }
}


// Define the idxToLoc_2D function
// Convert linear index -> (x,y)
__device__ __host__ __forceinline__ 
int2 idxToLoc_2D(int idx, int xDim) {
  int x = idx % xDim;
  int y = idx / xDim;
  return make_int2(x,y);
}


// Define the locToIdx_2D function
// Convert (x,y) -> linear index
__device__ __host__ __forceinline__ 
int locToIdx_2D(int x, int y, int xDim) {
  return y * xDim + x;
}



__global__
void initNodeData__kernel(
  NodeData2D* d_nodes,
  int* d_doneFlag, int* d_meetId,  int netId,  // Net related variables
  int* d_pins, int pinIterStart, int numPins,  // Pin related variables
  int LLX, int LLY, int URX, int URY, // Bounding box
  int xDim, int pinIter, int avgCost)
{ 
  int local_idx = blockDim.x * blockIdx.x + threadIdx.x;
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);
  if (local_idx > numNodes) {
    return;
  }

  int xDimTemp = URX - LLX + 1;
  int x = local_idx % xDimTemp + LLX;
  int y = local_idx / xDimTemp + LLY;
  int idx = locToIdx_2D(x, y, xDim);

  if (idx == 0) {
    d_doneFlag[netId] = 0;
    d_meetId[netId] = 0x7FFFFFFF;
  }

  int2 xy = idxToLoc_2D(idx, xDim);
  int2 src = idxToLoc_2D(d_pins[pinIterStart + pinIter - 1], xDim);
  int2 dst = idxToLoc_2D(d_pins[pinIterStart + pinIter], xDim);
  
  d_nodes[idx].forward_h_cost = (abs(xy.x - dst.x) + abs(xy.y - dst.y)) * avgCost;
  d_nodes[idx].backward_h_cost = (abs(xy.x - src.x) + abs(xy.y - src.y)) * avgCost;
  //d_nodes[idx].forward_h_cost = 0;
  //d_nodes[idx].backward_h_cost = 0;

  if (d_nodes[idx].flags.src_flag) {
    d_nodes[idx].forward_g_cost = 0;
    d_nodes[idx].forward_g_cost_prev = 0;
  } else {
    d_nodes[idx].forward_g_cost = INF32;
    d_nodes[idx].forward_g_cost_prev = INF32;
  }

  if (d_nodes[idx].flags.dst_flag) {
    d_nodes[idx].backward_g_cost = 0;
    d_nodes[idx].backward_g_cost_prev = 0;
  } else {
    d_nodes[idx].backward_g_cost = INF32;
    d_nodes[idx].backward_g_cost_prev = INF32;
  }

  d_nodes[idx].forward_direction = DIR_NONE;
  d_nodes[idx].backward_direction = DIR_NONE;
  d_nodes[idx].forward_direction_prev = DIR_NONE;
  d_nodes[idx].backward_direction_prev = DIR_NONE;
  d_nodes[idx].flags.forward_update_flag = false;
  d_nodes[idx].flags.backward_update_flag = false;
  d_nodes[idx].flags.forward_visited_flag = false;
  d_nodes[idx].flags.backward_visited_flag = false;
}



////////////////////////////////////////////////////////////////////////////////
// Kernel or device function that uses a grid-wide cooperative group:
////////////////////////////////////////////////////////////////////////////////
__device__
void runBiBellmanFord2D__device(
  cooperative_groups::grid_group& g,   // grid-level cooperative group
  NodeData2D* nodes,
  int* costMap, 
  int* d_dX,
  int* d_dY,
  int* d_doneFlag,
  int LLX, int LLY, int URX, int URY,
  int netId,
  int xDim, int yDim,
  int TURNCOST,
  int maxIters)
{
  // This need to be updated later 
}



// Kernel to find the meetId and launch traceback kernels
// We always work on the entire grid
__global__
void findMeetIdAndTraceBack__kernel(
  NodeData2D* nodes,
  int* d_meetId, int netId, 
  int xDim,
  int LLX, int LLY, int URX, int URY)
{ 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int numNodes = (URX - LLX + 1) * (URY - LLY + 1);  
  int xDimTemp = URX - LLX + 1;
  auto& meetId = d_meetId[netId];
  if (idx < numNodes) {
    int x = idx % xDimTemp + LLX;
    int y = idx / xDimTemp + LLY;
    int nodeIdx = locToIdx_2D(x, y, xDim);    
    if (nodes[idx].flags.forward_visited_flag && nodes[idx].flags.backward_visited_flag) {
      atomicMin(&meetId, idx); // Ensure only the smallest meetId is stored
    }
  }

  // Synchronize threads to ensure meetId is updated
  __syncthreads();
}


// Kernel for forward traceback  
// Please note that this is only called by single thread
__global__
void forwardTraceBack__single_thread__kernel(
  NodeData2D* nodes, 
  int* d_meetId, int netId, 
  int* d_dX, int* d_dY,
  int xDim,
  int LLX, int LLY, int URX, int URY)
{
  auto& meetId = d_meetId[netId];  
  if (meetId == 0x7FFFFFFF) {
    return; // No meetId found
  }
  
  int curId = meetId;
  int maxIterations = (URX - LLX + 1) * (URY - LLY + 1);
  int iteration = 0;
  while (nodes[curId].flags.src_flag == 0 && iteration < maxIterations) {
    uint8_t forwardDirection = nodes[curId].forward_direction;
    nodes[curId].flags.src_flag = 1;
    int2 xy = idxToLoc_2D(curId, xDim);
    int nx = xy.x + d_dX[forwardDirection];
    int ny = xy.y + d_dY[forwardDirection];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
      break;
    }  
    
    nodes[curId].golden_parent_x = nx;
    nodes[curId].golden_parent_y = ny;
    curId = locToIdx_2D(nx, ny, xDim);
    iteration++;
  }

  if (iteration >= maxIterations) {
    printf("Warning: Forward traceback exceeded maximum iterations.\n");
  }
}


// Kernel for backward traceback
__global__
void backwardTraceBack__single__thread__kernel(
  NodeData2D* nodes, 
  int* d_meetId, int netId, 
  int* d_dX, int* d_dY,
  int xDim,
  int LLX, int LLY, int URX, int URY)
{  
  auto& meetId = d_meetId[netId];
  if (meetId == 0x7FFFFFFF) {
    return; // No meetId found
  }
  
  int curId = meetId;
  if (nodes[curId].flags.dst_flag == 1) { 
    nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
    nodes[curId].flags.src_flag = 1;
    return;
  }
  
  int maxIterations = (URX - LLX + 1) * (URY - LLY + 1);
  int iteration = 0;

  while (iteration < maxIterations) {
    int2 xy = idxToLoc_2D(curId, xDim);
    uint8_t backwardDirection = nodes[curId].backward_direction;
    int nx = xy.x + d_dX[backwardDirection];
    int ny = xy.y + d_dY[backwardDirection];
    if (nx < LLX || nx > URX || ny < LLY || ny > URY) {
      break;
    }  
    
    curId = locToIdx_2D(nx, ny, xDim);
    nodes[curId].flags.src_flag = 1;
    nodes[curId].golden_parent_x = xy.x;
    nodes[curId].golden_parent_y = xy.y;
    if (nodes[curId].flags.dst_flag == 1) {
      nodes[curId].flags.dst_flag = 0; // change the dst flag to 0
      break;
    }
    iteration++;
  }

  if (iteration >= maxIterations) {
    printf("Warning: Backward traceback exceeded maximum iterations.\n");
  }
}


void FlexGR::GPUAccelerated2DMazeRoute(
  std::vector<grNet*>& nets,
  std::vector<uint64_t>& h_costMap,
  int xDim, int yDim)
{
  std::cout << "[INFO] GPU accelerated 2D Maze Routing" << std::endl;
  std::cout << "[INFO] Number of nets: " << nets.size() << std::endl;
  
  int numGrids = xDim * yDim;
  int numNets = nets.size();
  
  std::vector<Point2D_CUDA> netVec;
  std::vector<int> netPtr; 
  std::vector<Rect2D_CUDA> netBBoxVec;

  netPtr.push_back(0);
  for (auto& net : nets) {
    for (auto& idx : net->getPinGCellAbsIdxs()) {
      netVec.push_back(Point2D_CUDA(idx.x(), idx.y()));
    }
    netPtr.push_back(netVec.size());
    auto netBBox = net->getRouteAbsBBox();
    netBBoxVec.push_back(
      Rect2D_CUDA(netBBox.xMin(), netBBox.yMin(), netBBox.xMax(), netBBox.yMax()));
  }

  // We need to define the needed utility variables
  std::vector<int> h_dX = {0, 1, 0, -1};
  std::vector<int> h_dY = {1, 0, -1, 0};
  
  int* d_dX;
  int* d_dY;
  int* d_doneFlag; // This is allocated for each net seperately
  int* d_meetId; // This is allocated for each net seperately

  // For the design specific variables
  uint64_t* d_costMap;
  Point2D_CUDA* d_netVec;
  Rect2D_CUDA* d_netBBoxVec;

  // Allocate the device memory for the d_dX and d_dY
  hipMalloc(&d_dX, 4 * sizeof(int));
  hipMalloc(&d_dY, 4 * sizeof(int));
  hipMalloc(&d_doneFlag, nets.size() * sizeof(int));
  hipMalloc(&d_meetId, nets.size() * sizeof(int));

  hipMalloc(&d_costMap, numGrids * sizeof(uint64_t));
  hipMalloc(&d_netVec, netVec.size() * sizeof(Point2D_CUDA));
  hipMalloc(&d_netBBoxVec, netBBoxVec.size() * sizeof(Rect2D_CUDA));

  hipMemcpy(d_dX, h_dX.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dY, h_dY.data(), 4 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_costMap, h_costMap.data(), numGrids * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_netVec, netVec.data(), netVec.size() * sizeof(Point2D_CUDA), hipMemcpyHostToDevice);
  hipMemcpy(d_netBBoxVec, netBBoxVec.data(), netBBoxVec.size() * sizeof(Rect2D_CUDA), hipMemcpyHostToDevice);

  cudaCheckError();

  // Clear the memory
  hipFree(d_dX);
  hipFree(d_dY);
  hipFree(d_doneFlag);
  hipFree(d_meetId);
  hipFree(d_costMap);
  hipFree(d_netVec);
  hipFree(d_netBBoxVec);
}

} // namespace drt




