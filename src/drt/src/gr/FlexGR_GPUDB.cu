/* Authors: Zhiang Wang */
/*
 * Copyright (c) 2025, The Regents of the University of California
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the University nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE REGENTS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FlexGR_GPUDB.h"

#include <omp.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <string>

#include "db/grObj/grShape.h"
#include "db/grObj/grVia.h"
#include "db/infra/frTime.h"
#include "db/obj/frGuide.h"
#include "odb/db.h"
#include "utl/exception.h"

namespace drt {

void FlexGRGPUDB::init(FlexGRCMap* cmap, FlexGRCMap* cmap2D)
{
  cmap->getDim(xDim, yDim, zDim);
  auto& cmap_bits = cmap->getBits();
  auto& cmap2D_bits = cmap2D->getBits();

  cmap_bits_3D_size = cmap_bits.size();
  cmap_bits_2D_size = cmap2D_bits.size();
  
  // Allocate memory on the GPU side
  hipMalloc((void**)&cmap_bits_3D, cmap_bits_3D_size * sizeof(uint64_t));
  hipMalloc((void**)&cmap_bits_2D, cmap_bits_2D_size * sizeof(uint64_t));
  cudaCheckError();
  
  // Copy the data from the host to the device
  hipMemcpy(cmap_bits_3D, cmap_bits.data(), cmap_bits_3D_size * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(cmap_bits_2D, cmap2D_bits.data(), cmap_bits_2D_size * sizeof(uint64_t), hipMemcpyHostToDevice);
  cudaCheckError();


  std::string msg = std::string("[INFO] ")
                  + std::string("FlexGRGPUDB initialized with dimensions: \n")
                  + "\t xDim: " + std::to_string(xDim) + ", "
                  + "yDim: " + std::to_string(yDim) + ", "
                  + "zDim: " + std::to_string(zDim) + "\n"
                  + "\t cmap_bits_2D_size: " + std::to_string(cmap_bits_2D_size) + ", "
                  + "cmap_bits_3D_size: " + std::to_string(cmap_bits_3D_size);

  logger_->report(msg);
}

void FlexGRGPUDB::freeCUDAMem()
{
  if (cmap_bits_3D) {
    hipFree(cmap_bits_3D);
    cmap_bits_3D = nullptr;
  }
  
  if (cmap_bits_2D) {
    hipFree(cmap_bits_2D);
    cmap_bits_2D = nullptr;
  }
  
  cudaCheckError();
  
  logger_->report("FlexGRGPUDB CUDA memory freed ....");
}

}  // namespace drt
